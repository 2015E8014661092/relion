#include "hip/hip_runtime.h"
#undef ALTCPU
#include <sys/time.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <ctime>
#include <vector>
#include <iostream>
#include "src/ml_optimiser.h"
#include <hip/hip_runtime.h>
#include "src/acc/acc_ptr.h"
#include "src/acc/acc_projector.h"
#include "src/acc/acc_backprojector.h"
#include "src/acc/acc_projector_plan.h"
#include "src/acc/cuda/cuda_benchmark_utils.h"
#include "src/acc/cuda/cuda_kernels/helper.cuh"
#include "src/acc/cuda/cuda_kernels/diff2.cuh"
#include "src/acc/cuda/cuda_kernels/wavg.cuh"
#include "src/acc/cuda/cuda_mem_utils.h"
#include "src/acc/cuda/cuda_fft.h"
#include "src/acc/data_types.h"
#include "src/complex.h"
#include "src/helix.h"
#include "src/error.h"
#include <fstream>
#include "src/parallel.h"
#include <signal.h>
#include <map>

#ifdef CUDA_FORCESTL
#include "src/acc/cuda/cuda_utils_stl.cuh"
#else
#include "src/acc/cuda/cuda_utils_cub.cuh"
#endif

#include "src/acc/utilities.h"
#include "src/acc/utilities_impl.h"

#include "src/acc/acc_ml_optimiser.h"
#include "src/acc/cuda/cuda_ml_optimiser.h"
#include "src/acc/acc_helper_functions.h"
#include "src/acc/acc_ml_optimiser_impl.h"

// -------------------------------  Some explicit template instantiations
template __global__ void CudaKernels::cuda_kernel_translate2D<XFLOAT>(XFLOAT *,
    XFLOAT*, int, int, int, int, int);

template __global__ void CudaKernels::cuda_kernel_translate3D<XFLOAT>(XFLOAT *,
    XFLOAT *, int, int, int, int, int, int, int);

template __global__ void cuda_kernel_multi<XFLOAT>( XFLOAT *,
	XFLOAT *, XFLOAT, int);

template __global__ void CudaKernels::cuda_kernel_multi<XFLOAT>( XFLOAT *,
	XFLOAT, int);

template __global__ void cuda_kernel_multi<XFLOAT>( XFLOAT *, XFLOAT *,
	XFLOAT *, XFLOAT, int);

// ----------------------------------------------------------------------

// High-level CUDA objects

size_t MlDeviceBundle::checkFixedSizedObjects(int shares)
{
	int devCount;
	size_t BoxLimit;
	HANDLE_ERROR(hipGetDeviceCount(&devCount));
	if(device_id >= devCount)
		CRITICAL(ERR_GPUID);

	HANDLE_ERROR(hipSetDevice(device_id));

	size_t free(0), total(0);
	DEBUG_HANDLE_ERROR(hipMemGetInfo( &free, &total ));
	float margin(1.05);
	BoxLimit = pow(free/(margin*2.5*sizeof(XFLOAT)*((float)shares)),(1/3.0)) / (2.0);
	size_t BytesNeeded = ((float)shares)*margin*2.5*sizeof(XFLOAT)*pow((baseMLO->mymodel.ori_size*2),3);

	return(BoxLimit);
}
void MlDeviceBundle::setupFixedSizedObjects()
{
	unsigned nr_classes = baseMLO->mymodel.nr_classes;

	int devCount;
	HANDLE_ERROR(hipGetDeviceCount(&devCount));
	if(device_id >= devCount)
	{
		//std::cerr << " using device_id=" << device_id << " (device no. " << device_id+1 << ") which is higher than the available number of devices=" << devCount << std::endl;
		CRITICAL(ERR_GPUID);
	}
	else
		HANDLE_ERROR(hipSetDevice(device_id));

	//Can we pre-generate projector plan and corresponding euler matrices for all particles
	if (baseMLO->do_skip_align || baseMLO->do_skip_rotate || baseMLO->do_auto_refine || baseMLO->mymodel.orientational_prior_mode != NOPRIOR)
		generateProjectionPlanOnTheFly = true;
	else
		generateProjectionPlanOnTheFly = false;

	// clear() called on std::vector appears to set size=0, even if we have an explicit
	// destructor for each member, so we need to set the size to what is was before
	projectors.resize(nr_classes);
	backprojectors.resize(nr_classes);

	/*======================================================
	              PROJECTOR AND BACKPROJECTOR
	======================================================*/

	//Loop over classes
	for (int iclass = 0; iclass < nr_classes; iclass++)
	{
		projectors[iclass].setMdlDim(
				baseMLO->mymodel.PPref[iclass].data.xdim,
				baseMLO->mymodel.PPref[iclass].data.ydim,
				baseMLO->mymodel.PPref[iclass].data.zdim,
				baseMLO->mymodel.PPref[iclass].data.yinit,
				baseMLO->mymodel.PPref[iclass].data.zinit,
				baseMLO->mymodel.PPref[iclass].r_max,
				baseMLO->mymodel.PPref[iclass].padding_factor);

		projectors[iclass].initMdl(baseMLO->mymodel.PPref[iclass].data.data);

		backprojectors[iclass].setMdlDim(
				baseMLO->wsum_model.BPref[iclass].data.xdim,
				baseMLO->wsum_model.BPref[iclass].data.ydim,
				baseMLO->wsum_model.BPref[iclass].data.zdim,
				baseMLO->wsum_model.BPref[iclass].data.yinit,
				baseMLO->wsum_model.BPref[iclass].data.zinit,
				baseMLO->wsum_model.BPref[iclass].r_max,
				baseMLO->wsum_model.BPref[iclass].padding_factor);

		backprojectors[iclass].initMdl();
	}

	/*======================================================
	                    CUSTOM ALLOCATOR
	======================================================*/

	int memAlignmentSize;
	hipDeviceGetAttribute ( &memAlignmentSize, hipDeviceAttributeTextureAlignment, device_id );
	allocator = new CudaCustomAllocator(0, memAlignmentSize);
}

void MlDeviceBundle::setupTunableSizedObjects(size_t allocationSize)
{
	unsigned nr_classes = baseMLO->mymodel.nr_classes;
	int devCount;
	HANDLE_ERROR(hipGetDeviceCount(&devCount));
	if(device_id >= devCount)
	{
		//std::cerr << " using device_id=" << device_id << " (device no. " << device_id+1 << ") which is higher than the available number of devices=" << devCount << std::endl;
		CRITICAL(ERR_GPUID);
	}
	else
		HANDLE_ERROR(hipSetDevice(device_id));

	/*======================================================
	                    CUSTOM ALLOCATOR
	======================================================*/
#ifdef DEBUG_CUDA
	printf("DEBUG: Total GPU allocation size set to %zu MB on device id %d.\n", allocationSize / (1000*1000), device_id);
#endif
#ifndef CUDA_NO_CUSTOM_ALLOCATION
	allocator->resize(allocationSize);
#endif


	/*======================================================
	                    PROJECTION PLAN
	======================================================*/

	coarseProjectionPlans.resize(nr_classes, allocator);

	for (int iclass = 0; iclass < nr_classes; iclass++)
	{
		//If doing predefined projector plan at all and is this class significant
		if (!generateProjectionPlanOnTheFly && baseMLO->mymodel.pdf_class[iclass] > 0.)
		{
			std::vector<int> exp_pointer_dir_nonzeroprior;
			std::vector<int> exp_pointer_psi_nonzeroprior;
			std::vector<RFLOAT> exp_directions_prior;
			std::vector<RFLOAT> exp_psi_prior;

			long unsigned itrans_max = baseMLO->sampling.NrTranslationalSamplings() - 1;
			long unsigned nr_idir = baseMLO->sampling.NrDirections(0, &exp_pointer_dir_nonzeroprior);
			long unsigned nr_ipsi = baseMLO->sampling.NrPsiSamplings(0, &exp_pointer_psi_nonzeroprior );

			coarseProjectionPlans[iclass].setup(
					baseMLO->sampling,
					exp_directions_prior,
					exp_psi_prior,
					exp_pointer_dir_nonzeroprior,
					exp_pointer_psi_nonzeroprior,
					NULL, //Mcoarse_significant
					baseMLO->mymodel.pdf_class,
					baseMLO->mymodel.pdf_direction,
					nr_idir,
					nr_ipsi,
					0, //idir_min
					nr_idir - 1, //idir_max
					0, //ipsi_min
					nr_ipsi - 1, //ipsi_max
					0, //itrans_min
					itrans_max,
					0, //current_oversampling
					1, //nr_oversampled_rot
					iclass,
					true, //coarse
					!IS_NOT_INV,
					baseMLO->do_skip_align,
					baseMLO->do_skip_rotate,
					baseMLO->mymodel.orientational_prior_mode
					);
		}
	}
};

void MlOptimiserCuda::resetData()
{
	int devCount;
	HANDLE_ERROR(hipGetDeviceCount(&devCount));
	if(device_id >= devCount)
	{
		//std::cerr << " using device_id=" << device_id << " (device no. " << device_id+1 << ") which is higher than the available number of devices=" << devCount << std::endl;
		CRITICAL(ERR_GPUID);
	}
	else
		HANDLE_ERROR(hipSetDevice(device_id));

	unsigned nr_classes = baseMLO->mymodel.nr_classes;

	classStreams.resize(nr_classes, 0);
	for (int i = 0; i < nr_classes; i++)
		HANDLE_ERROR(hipStreamCreate(&classStreams[i])); //HANDLE_ERROR(hipStreamCreateWithFlags(&classStreams[i],hipStreamNonBlocking));

	transformer1.clear();
	transformer2.clear();
};

void MlOptimiserCuda::doThreadExpectationSomeParticles(int thread_id)
{
#ifdef TIMING
	// Only time one thread
	if (thread_id == 0)
		baseMLO->timer.tic(baseMLO->TIMING_ESP_THR);
#endif
//	CTOC(cudaMLO->timer,"interParticle");

	int devCount;
	HANDLE_ERROR(hipGetDeviceCount(&devCount));
	if(device_id >= devCount)
	{
		//std::cerr << " using device_id=" << device_id << " (device no. " << device_id+1 << ") which is higher than the available number of devices=" << devCount << std::endl;
		CRITICAL(ERR_GPUID);
	}
	else
		DEBUG_HANDLE_ERROR(hipSetDevice(device_id));
	//std::cerr << " calling on device " << device_id << std::endl;
	//put mweight allocation here
	size_t first_ipart = 0, last_ipart = 0;

	while (baseMLO->exp_ipart_ThreadTaskDistributor->getTasks(first_ipart, last_ipart))
	{
		CTIC(timer,"oneTask");
		for (long unsigned ipart = first_ipart; ipart <= last_ipart; ipart++)
		{
#ifdef TIMING
	// Only time one thread
	if (thread_id == 0)
		baseMLO->timer.tic(baseMLO->TIMING_ESP_DIFF2_A);
#endif
			unsigned my_ori_particle = baseMLO->exp_my_first_ori_particle + ipart;

			AccPtrFactory ptrFactory(allocator, hipStreamPerThread);
            accDoExpectationOneParticle<MlOptimiserCuda>(this, my_ori_particle, thread_id, ptrFactory);

		}
		CTOC(timer,"oneTask");
	}

//	CTIC(cudaMLO->timer,"interParticle");
//	exit(0);

#ifdef TIMING
	// Only time one thread
	if (thread_id == 0)
		baseMLO->timer.toc(baseMLO->TIMING_ESP_THR);
#endif
}

