#include "hip/hip_runtime.h"
#include "src/gpu_utils/cuda_kernels/proj_diff.cuh"
#include <vector>
#include <iostream>

__global__ void cuda_kernel_diff2_fine(
		FLOAT *g_eulers,
		FLOAT *g_imgs_real,
		FLOAT *g_imgs_imag,
		Cuda3DProjectorKernel projector,
		FLOAT *g_Minvsigma2,
		FLOAT *g_diff2s,
		unsigned image_size,
		FLOAT sum_init,
		unsigned long orientation_num,
		unsigned long translation_num,
		unsigned long todo_blocks,
		unsigned long *d_rot_idx,
		unsigned long *d_trans_idx,
		unsigned long *d_job_idx,
		unsigned long *d_job_num
		)
{
	int bid = blockIdx.y * gridDim.x + blockIdx.x;
	int tid = threadIdx.x;

//    // Specialize BlockReduce for a 1D block of 128 threads on type FLOAT
//    typedef hipcub::BlockReduce<FLOAT, 128> BlockReduce;
//    // Allocate shared memory for BlockReduce
//    __shared__ typename BlockReduce::TempStorage temp_storage;

	int pixel;
	FLOAT ref_real;
	FLOAT ref_imag;

	__shared__ FLOAT s[BLOCK_SIZE*PROJDIFF_CHUNK_SIZE]; //We MAY have to do up to PROJDIFF_CHUNK_SIZE translations in each block
	__shared__ FLOAT s_outs[PROJDIFF_CHUNK_SIZE];
	// inside the padded 2D orientation gri
	if( bid < todo_blocks ) // we only need to make
	{
		unsigned trans_num   = d_job_num[bid]; //how many transes we have for this rot
		for (int itrans=0; itrans<trans_num; itrans++)
		{
			s[itrans*BLOCK_SIZE+tid] = 0.0f;
		}
		// index of comparison
		unsigned long int ix = d_rot_idx[d_job_idx[bid]];
		unsigned long int iy;
		unsigned pass_num(ceilf(   ((float)image_size) / (float)BLOCK_SIZE  ));

		for (unsigned pass = 0; pass < pass_num; pass++) // finish an entire ref image each block
		{
			pixel = (pass * BLOCK_SIZE) + tid;

			if(pixel < image_size)
			{
				projector.project(
						pixel,
						__ldg(&g_eulers[ix*9  ]), __ldg(&g_eulers[ix*9+1]),
						__ldg(&g_eulers[ix*9+3]), __ldg(&g_eulers[ix*9+4]),
						__ldg(&g_eulers[ix*9+6]), __ldg(&g_eulers[ix*9+7]),
						ref_real, ref_imag);

				FLOAT diff_real;
				FLOAT diff_imag;
				for (int itrans=0; itrans<trans_num; itrans++) // finish all translations in each partial pass
				{
					iy=d_trans_idx[d_job_idx[bid]]+itrans;
					unsigned long img_start(iy * image_size);
					unsigned long img_pixel_idx = img_start + pixel;
					diff_real =  ref_real - __ldg(&g_imgs_real[img_pixel_idx]); // TODO  Put g_img_* in texture (in such a way that fetching of next image might hit in cache)
					diff_imag =  ref_imag - __ldg(&g_imgs_imag[img_pixel_idx]);
					s[itrans*BLOCK_SIZE + tid] += (diff_real * diff_real + diff_imag * diff_imag) * 0.5f * __ldg(&g_Minvsigma2[pixel]);
				}
				__syncthreads();
			}
		}
		for(int j=(BLOCK_SIZE/2); j>0; j/=2)
		{
			if(tid<j)
			{
				for (int itrans=0; itrans<trans_num; itrans++) // finish all translations in each partial pass
				{
					s[itrans*BLOCK_SIZE+tid] += s[itrans*BLOCK_SIZE+tid+j];
				}
			}
			__syncthreads();
		}
		if (tid < trans_num)
		{
			s_outs[tid]=s[tid*BLOCK_SIZE]+sum_init;
		}
		if (tid < trans_num)
		{
			iy=d_job_idx[bid]+tid;
			g_diff2s[iy] = s_outs[tid];
		}
	}
}
