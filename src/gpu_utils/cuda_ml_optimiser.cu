#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <ctime>
#include <iostream>
#include "src/gpu_utils/cuda_ml_optimiser.h"
#include "src/complex.h"

class CudaComplex
{
public:
	double real, imag;

	inline
	__device__ __host__ CudaComplex(): real(), imag() {};
	inline
	__device__ __host__ CudaComplex(double real, double imag): real(real), imag(imag) {};
};

class CudaImages
{
public:
	long unsigned x,y,xy,num,max_num;
	CudaComplex* start;

	inline
	__device__ __host__ CudaImages(long unsigned x, long unsigned y, long unsigned max_num):
			x(x), y(y), num(0), max_num(max_num), xy(x*y), start(new CudaComplex[xy*max_num])
	{};

	inline
	__device__ __host__ CudaComplex* current() { return start + (num*xy); };

	inline
	__device__ __host__ void increment() { num++; };

	inline
	__device__ __host__ CudaComplex* operator [](long unsigned i) { return start + (i*xy); };

	inline
	__device__ __host__ long unsigned alloc_size() { return num*xy; };

	inline
	__device__ __host__ ~CudaImages() { delete[] start; }
};

__global__ void kernel_diff2(CudaImages *ref, CudaImages *img, CudaComplex *Minvsigma2, double *partial_sums)
{
	//Dummy for now
}

void MlOptimiserCUDA::getAllSquaredDifferences(
		long int my_ori_particle, int exp_current_image_size,
		int exp_ipass, int exp_current_oversampling, int metadata_offset,
		int exp_idir_min, int exp_idir_max, int exp_ipsi_min, int exp_ipsi_max,
		int exp_itrans_min, int exp_itrans_max, int exp_iclass_min, int exp_iclass_max,
		std::vector<double> &exp_min_diff2,
		std::vector<double> &exp_highres_Xi2_imgs,
		std::vector<MultidimArray<Complex > > &exp_Fimgs,
		std::vector<MultidimArray<double> > &exp_Fctfs,
		MultidimArray<double> &exp_Mweight,
		MultidimArray<bool> &exp_Mcoarse_significant,
		std::vector<int> &exp_pointer_dir_nonzeroprior, std::vector<int> &exp_pointer_psi_nonzeroprior,
		std::vector<double> &exp_directions_prior, std::vector<double> &exp_psi_prior,
		std::vector<MultidimArray<Complex> > &exp_local_Fimgs_shifted,
		std::vector<MultidimArray<double> > &exp_local_Minvsigma2s,
		std::vector<MultidimArray<double> > &exp_local_Fctfs,
		std::vector<double> &exp_local_sqrtXi2)
{
	// Initialise min_diff and exp_Mweight for this pass
	int exp_nr_particles = mydata.ori_particles[my_ori_particle].particles_id.size();
	long int exp_nr_dir = (do_skip_align || do_skip_rotate) ? 1 : sampling.NrDirections(0, &exp_pointer_dir_nonzeroprior);
	long int exp_nr_psi = (do_skip_align || do_skip_rotate) ? 1 : sampling.NrPsiSamplings(0, &exp_pointer_psi_nonzeroprior);
	long int exp_nr_trans = (do_skip_align) ? 1 : sampling.NrTranslationalSamplings();
	long int exp_nr_oversampled_rot = sampling.oversamplingFactorOrientations(exp_current_oversampling);
	long int exp_nr_oversampled_trans = sampling.oversamplingFactorTranslations(exp_current_oversampling);

	exp_Mweight.resize(exp_nr_particles, mymodel.nr_classes * exp_nr_dir * exp_nr_psi * exp_nr_trans * exp_nr_oversampled_rot * exp_nr_oversampled_trans);
	exp_Mweight.initConstant(-999.);
	if (exp_ipass==0)
		exp_Mcoarse_significant.clear();

	exp_min_diff2.clear();
	exp_min_diff2.resize(exp_nr_particles, 99.e99);

	std::vector<MultidimArray<Complex > > dummy;
	precalculateShiftedImagesCtfsAndInvSigma2s(false, my_ori_particle, exp_current_image_size, exp_current_oversampling,
			exp_itrans_min, exp_itrans_max, exp_Fimgs, dummy, exp_Fctfs, exp_local_Fimgs_shifted, dummy,
			exp_local_Fctfs, exp_local_sqrtXi2, exp_local_Minvsigma2s);

	// Loop only from exp_iclass_min to exp_iclass_max to deal with seed generation in first iteration
	for (int exp_iclass = exp_iclass_min; exp_iclass <= exp_iclass_max; exp_iclass++)
	{
		if (mymodel.pdf_class[exp_iclass] > 0.)
		{
			// Local variables
			std::vector< double > oversampled_rot, oversampled_tilt, oversampled_psi;
			std::vector< double > oversampled_translations_x, oversampled_translations_y, oversampled_translations_z;
			MultidimArray<Complex > Fimg, Fref, Frefctf, Fimg_otfshift;
			double *Minvsigma2;
			Matrix2D<double> A;

			CudaImages Frefs(exp_local_Minvsigma2s[0].xdim, exp_local_Minvsigma2s[0].ydim,
					(exp_idir_max - exp_idir_min + 1) * (exp_ipsi_max - exp_ipsi_min + 1) * exp_nr_oversampled_rot);

			// Mapping index look-up table
			std::vector< long unsigned > iorientclasses(Frefs.max_num), iover_rots(Frefs.max_num);
			long unsigned orientation_num(0);

			/*=======================================================================================
			                           Generate Reference Projections
			=========================================================================================*/

			Fref.resize(exp_local_Minvsigma2s[0]);
			Frefctf.resize(exp_local_Minvsigma2s[0]);
			if (do_shifts_onthefly)
				Fimg_otfshift.resize(Frefctf);

			std::cerr << "This is calculation of the reference projections." << std::endl;

			for (long int idir = exp_idir_min, iorient = 0; idir <= exp_idir_max; idir++)
			{
				for (long int ipsi = exp_ipsi_min; ipsi <= exp_ipsi_max; ipsi++, iorient++)
				{
					long int iorientclass = exp_iclass * exp_nr_dir * exp_nr_psi + iorient;

					// Get prior for this direction and skip calculation if prior==0
					double pdf_orientation;
					if (do_skip_align || do_skip_rotate)
					{
						pdf_orientation = mymodel.pdf_class[exp_iclass];
					}
					else if (mymodel.orientational_prior_mode == NOPRIOR)
					{
						pdf_orientation = DIRECT_MULTIDIM_ELEM(mymodel.pdf_direction[exp_iclass], idir);
					}
					else
					{
						pdf_orientation = exp_directions_prior[idir] * exp_psi_prior[ipsi];
					}
					// In the first pass, always proceed
					// In the second pass, check whether one of the translations for this orientation of any of the particles had a significant weight in the first pass
					// if so, proceed with projecting the reference in that direction
					bool do_proceed = (exp_ipass==0) ? true :
						isSignificantAnyParticleAnyTranslation(iorientclass, exp_itrans_min, exp_itrans_max, exp_Mcoarse_significant);
					if (do_proceed && pdf_orientation > 0.)
					{
						// Now get the oversampled (rot, tilt, psi) triplets
						// This will be only the original (rot,tilt,psi) triplet in the first pass (exp_current_oversampling==0)
						sampling.getOrientations(idir, ipsi, exp_current_oversampling, oversampled_rot, oversampled_tilt, oversampled_psi,
								exp_pointer_dir_nonzeroprior, exp_directions_prior, exp_pointer_psi_nonzeroprior, exp_psi_prior);
						// Loop over all oversampled orientations (only a single one in the first pass)
						for (long int iover_rot = 0; iover_rot < exp_nr_oversampled_rot; iover_rot++)
						{
							// Get the Euler matrix
							Euler_angles2matrix(oversampled_rot[iover_rot],
												oversampled_tilt[iover_rot],
												oversampled_psi[iover_rot], A);

							Fref.data = (Complex*) Frefs.current();

							// Project the reference map (into Fref)
							(mymodel.PPref[exp_iclass]).get2DFourierTransform(Fref, A, IS_NOT_INV);

							//TODO REMOVE ONCE YOU KNOW THIS IS ALLWAYS TRUE
							if (Frefs.x != Fref.xdim || Frefs.y != Fref.ydim)
								std::cerr << "!!!!!!! BAD Fref size x:" << Fref.xdim << ":" << Frefs.x << " y:" << Fref.ydim << ":" << Frefs.y << std::endl;

							Frefs.increment();
							iorientclasses.push_back(iorientclass);
							iover_rots.push_back(iover_rot);
							orientation_num ++;
						}
					}
				}
			}

			CudaImages *d_Frefs;

			hipMalloc( (void**) &d_Frefs, Frefs.alloc_size());
			hipMemcpy( d_Frefs, Frefs.start, Frefs.alloc_size(), hipMemcpyHostToDevice);

			/*=======================================================================================
			                                  	  Particle Iteration
			=========================================================================================*/

			for (long int ipart = 0; ipart < mydata.ori_particles[my_ori_particle].particles_id.size(); ipart++)
			{
				/*====================================
				        Generate Translations
				======================================*/

				CudaImages Fimgs(Frefs.x, Frefs.y,
						orientation_num * ( exp_itrans_max - exp_itrans_min + 1) * exp_nr_oversampled_trans);

				long unsigned translation_num(0);

				for (long int itrans = exp_itrans_min; itrans <= exp_itrans_max; itrans++)
				{
					sampling.getTranslations(itrans, exp_current_oversampling,
							oversampled_translations_x, oversampled_translations_y, oversampled_translations_z );

					for (long int iover_trans = 0; iover_trans < exp_nr_oversampled_trans; iover_trans++)
					{
						/// Now get the shifted image
						// Use a pointer to avoid copying the entire array again in this highly expensive loop
						Complex *myAB;
						if (exp_current_oversampling == 0)
						{
							myAB = (Frefs.y == coarse_size) ? global_fftshifts_ab_coarse[itrans].data
									: global_fftshifts_ab_current[itrans].data;
						}
						else
						{
							int iitrans = itrans * exp_nr_oversampled_trans +  iover_trans;
							myAB = (strict_highres_exp > 0.) ? global_fftshifts_ab2_coarse[iitrans].data
									: global_fftshifts_ab2_current[iitrans].data;
						}
						FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(exp_local_Fimgs_shifted[ipart])
						{
							double real = (*(myAB + n)).real * (DIRECT_MULTIDIM_ELEM(exp_local_Fimgs_shifted[ipart], n)).real
									- (*(myAB + n)).imag *(DIRECT_MULTIDIM_ELEM(exp_local_Fimgs_shifted[ipart], n)).imag;
							double imag = (*(myAB + n)).real * (DIRECT_MULTIDIM_ELEM(exp_local_Fimgs_shifted[ipart], n)).imag
									+ (*(myAB + n)).imag *(DIRECT_MULTIDIM_ELEM(exp_local_Fimgs_shifted[ipart], n)).real;

							*(Fimgs.current() + n) = CudaComplex(real, imag);
						}
						Fimgs.increment();
						translation_num ++;
					}
				}

				/*====================================
				   Initiate Particle Related On GPU
				======================================*/

				Minvsigma2 = exp_local_Minvsigma2s[ipart].data;

				CudaImages *d_Fimgs;
				double *d_Minvsigma2;

				hipMalloc( (void**) &d_Fimgs, Fimgs.alloc_size());
				hipMalloc( (void**) &d_Minvsigma2, Fimgs.xy);
				hipMemcpy( d_Fimgs, Fimgs.start, Fimgs.alloc_size(), hipMemcpyHostToDevice);
				hipMemcpy( d_Minvsigma2, Minvsigma2, Fimgs.xy, hipMemcpyHostToDevice);

				CudaImages *d_diff2s;
				hipMalloc( (void**) &d_diff2s, orientation_num*translation_num);
				hipMemset( (void**) &d_diff2s, 0, orientation_num*translation_num); //Initiate diff2 values with zeros

				/*====================================
				    		Kernel Calls
				======================================*/

				for (unsigned iorient = 0; iorient < orientation_num; iorient ++)
				{
					for (unsigned itrans = 0; itrans < translation_num; itrans ++)
					{

					}
				}

				/*====================================
				    	   Retrieve Results
				======================================*/

				double* diff2s = new double[orientation_num*translation_num];
				hipMemcpy( diff2s, d_diff2s, orientation_num*translation_num, hipMemcpyDeviceToHost );

				/*====================================
				    	Write To Destination TODO
				======================================*/

				/*
				for (long int i = 0; i < ihidden_overs.size(); i++)
				{
					DIRECT_A2D_ELEM(exp_Mweight, ipart, ihidden_overs[i]) = diff2s[i];

					// Keep track of minimum of all diff2, only for the last image in this series
					if (diff2s[i] < exp_min_diff2[ipart])
						exp_min_diff2[ipart] = diff2s[i];
				}
				*/

				hipFree(d_Fimgs);

			} // end loop ipart

			hipFree(d_Frefs);

		} // end if class significant
	} // end loop iclass
}

//void MlOptimiserCUDA::precalculateModelProjectionsCtfsAndInvSigma2s(bool do_also_unmasked,
//	    int exp_current_image_size, int exp_current_oversampling,
//		std::vector<MultidimArray<Complex > > &model,
//		std::vector<MultidimArray<Complex > > &exp_Fimgs_nomask,
//		std::vector<MultidimArray<double> > &exp_Fctfs,
//		std::vector<MultidimArray<Complex > > &exp_local_Fimgs_shifted,
//		std::vector<MultidimArray<Complex > > &exp_local_Fimgs_shifted_nomask,
//		std::vector<MultidimArray<double> > &exp_local_Fctfs,
//		std::vector<double> &exp_local_sqrtXi2,
//		std::vector<MultidimArray<double> > &exp_local_Minvsigma2s)
//{
//
//}

void MlOptimiserCUDA::precalculateShiftedImagesCtfsAndInvSigma2s(bool do_also_unmasked,
		long int my_ori_particle, int exp_current_image_size, int exp_current_oversampling,
		int exp_itrans_min, int exp_itrans_max,
		std::vector<MultidimArray<Complex > > &exp_Fimgs,
		std::vector<MultidimArray<Complex > > &exp_Fimgs_nomask,
		std::vector<MultidimArray<double> > &exp_Fctfs,
		std::vector<MultidimArray<Complex > > &exp_local_Fimgs_shifted,
		std::vector<MultidimArray<Complex > > &exp_local_Fimgs_shifted_nomask,
		std::vector<MultidimArray<double> > &exp_local_Fctfs,
		std::vector<double> &exp_local_sqrtXi2,
		std::vector<MultidimArray<double> > &exp_local_Minvsigma2s)
{

	int exp_nr_particles = mydata.ori_particles[my_ori_particle].particles_id.size();
	int nr_shifts = (do_shifts_onthefly || do_skip_align) ? exp_nr_particles : exp_nr_particles * sampling.NrTranslationalSamplings(exp_current_oversampling);
	// Don't re-do if nothing has changed....
	bool do_ctf_invsig = (exp_local_Fctfs.size() > 0) ? YSIZE(exp_local_Fctfs[0])  != exp_current_image_size : true; // size has changed
	bool do_masked_shifts = (do_ctf_invsig || nr_shifts != exp_local_Fimgs_shifted.size()); // size or nr_shifts has changed

	// Use pre-sized vectors instead of push_backs!!
	exp_local_Fimgs_shifted.resize(nr_shifts);
	if (do_also_unmasked)
		exp_local_Fimgs_shifted_nomask.resize(nr_shifts);
	exp_local_Minvsigma2s.resize(exp_nr_particles);
	exp_local_Fctfs.resize(exp_nr_particles);
	exp_local_sqrtXi2.resize(exp_nr_particles);

	MultidimArray<Complex > Fimg, Fimg_nomask;
	for (int ipart = 0, my_trans_image = 0; ipart < mydata.ori_particles[my_ori_particle].particles_id.size(); ipart++)
	{
		long int part_id = mydata.ori_particles[my_ori_particle].particles_id[ipart];
		int group_id = mydata.getGroupId(part_id);

		if (do_masked_shifts)
			windowFourierTransform(exp_Fimgs[ipart], Fimg, exp_current_image_size);
		if (do_also_unmasked)
			windowFourierTransform(exp_Fimgs_nomask[ipart], Fimg_nomask, exp_current_image_size);

		if (do_ctf_invsig)
		{
			// Also precalculate the sqrt of the sum of all Xi2
			// Could exp_current_image_size ever be different from mymodel.current_size?
			// Probably therefore do it here rather than in getFourierTransforms
			if ((iter == 1 && do_firstiter_cc) || do_always_cc)
			{
				double sumxi2 = 0.;
				FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(Fimg)
				{
					sumxi2 += norm(DIRECT_MULTIDIM_ELEM(Fimg, n));
				}
				// Normalised cross-correlation coefficient: divide by power of reference (power of image is a constant)
				exp_local_sqrtXi2[ipart] = sqrt(sumxi2);
			}

			// Also store downsized Fctfs
			// In the second pass of the adaptive approach this will have no effect,
			// since then exp_current_image_size will be the same as the size of exp_Fctfs
			windowFourierTransform(exp_Fctfs[ipart], exp_local_Fctfs[ipart], exp_current_image_size);

			// Also prepare Minvsigma2
			if (mymodel.data_dim == 3)
				exp_local_Minvsigma2s[ipart].initZeros(ZSIZE(Fimg), YSIZE(Fimg), XSIZE(Fimg));
			else
				exp_local_Minvsigma2s[ipart].initZeros(YSIZE(Fimg), XSIZE(Fimg));

			int *myMresol = (YSIZE(Fimg) == coarse_size) ? Mresol_coarse.data : Mresol_fine.data;
			// With group_id and relevant size of Fimg, calculate inverse of sigma^2 for relevant parts of Mresol
			FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(exp_local_Minvsigma2s[ipart])
			{
				int ires = *(myMresol + n);
				// Exclude origin (ires==0) from the Probability-calculation
				// This way we are invariant to additive factors
				if (ires > 0)
					DIRECT_MULTIDIM_ELEM(exp_local_Minvsigma2s[ipart], n) = 1. / (sigma2_fudge * DIRECT_A1D_ELEM(mymodel.sigma2_noise[group_id], ires));
			}

		}

		if (do_shifts_onthefly)
		{
			// Store a single, down-sized version of exp_Fimgs[ipart] in exp_local_Fimgs_shifted
			if (do_masked_shifts)
				exp_local_Fimgs_shifted[ipart] = Fimg;
			if (do_also_unmasked)
				exp_local_Fimgs_shifted_nomask[ipart] = Fimg_nomask;
		}
		else
		{
			// Store all translated variants of Fimg
			for (long int itrans = exp_itrans_min; itrans <= exp_itrans_max; itrans++)
			{
				// First get the non-oversampled translations as defined by the sampling object
				std::vector<double > oversampled_translations_x, oversampled_translations_y, oversampled_translations_z;
				sampling.getTranslations(itrans, exp_current_oversampling, oversampled_translations_x,
						oversampled_translations_y, oversampled_translations_z);
				// Then loop over all its oversampled relatives
				for (long int iover_trans = 0; iover_trans < oversampled_translations_x.size(); iover_trans++, my_trans_image++)
				{
					// Shift through phase-shifts in the Fourier transform
					// Note that the shift search range is centered around (exp_old_xoff, exp_old_yoff)
					if (do_masked_shifts)
					{
						exp_local_Fimgs_shifted[my_trans_image].resize(Fimg);
						if (mymodel.data_dim ==2)
							shiftImageInFourierTransform(Fimg, exp_local_Fimgs_shifted[my_trans_image],
									tab_sin, tab_cos, (double)mymodel.ori_size,
									oversampled_translations_x[iover_trans],
									oversampled_translations_y[iover_trans]);
						else
							shiftImageInFourierTransform(Fimg, exp_local_Fimgs_shifted[my_trans_image],
									tab_sin, tab_cos, (double)mymodel.ori_size,
									oversampled_translations_x[iover_trans],
									oversampled_translations_y[iover_trans],
									oversampled_translations_z[iover_trans]);
					}
					if (do_also_unmasked)
					{
						exp_local_Fimgs_shifted_nomask[my_trans_image].resize(Fimg_nomask);
						if (mymodel.data_dim ==2)
							shiftImageInFourierTransform(Fimg_nomask, exp_local_Fimgs_shifted_nomask[my_trans_image],
								tab_sin, tab_cos, (double)mymodel.ori_size,
								oversampled_translations_x[iover_trans],
								oversampled_translations_y[iover_trans]);
						else
							shiftImageInFourierTransform(Fimg_nomask, exp_local_Fimgs_shifted_nomask[my_trans_image],
								tab_sin, tab_cos, (double)mymodel.ori_size,
								oversampled_translations_x[iover_trans],
								oversampled_translations_y[iover_trans],
								oversampled_translations_z[iover_trans]);
					}
				}
			}
		}
	}
}


bool MlOptimiserCUDA::isSignificantAnyParticleAnyTranslation(long int iorient, int exp_itrans_min, int exp_itrans_max, MultidimArray<bool> &exp_Mcoarse_significant)
{

	long int exp_nr_trans = exp_itrans_max - exp_itrans_min + 1;
	for (long int ipart = 0; ipart < YSIZE(exp_Mcoarse_significant); ipart++)
	{
		long int ihidden = iorient * exp_nr_trans;
		for (long int itrans = exp_itrans_min; itrans <= exp_itrans_max; itrans++, ihidden++)
		{
			if (DIRECT_A2D_ELEM(exp_Mcoarse_significant, ipart, ihidden))
				return true;
		}
	}
	return false;

}
