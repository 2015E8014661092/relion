#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <ctime>
#include <iostream>
#include "src/gpu_utils/cuda_ml_optimiser.h"
#include "src/gpu_utils/cuda_img_operations.h"
#include "src/gpu_utils/cuda_utils.cuh"
#include "src/complex.h"
#include <fstream>
#include <hip/hip_runtime.h>

#define MAX_RESOL_SHARED_MEM 32
#define BLOCK_SIZE 128         // This is optimally set as big as possible without its ceil:ed multiple exceeding imagesize by too much.
#define NR_CLASS_MUTEXES 5

static pthread_mutex_t global_mutex2[NR_CLASS_MUTEXES] = { PTHREAD_MUTEX_INITIALIZER };
static pthread_mutex_t global_mutex = PTHREAD_MUTEX_INITIALIZER;

__global__ void cuda_kernel_diff2(	CudaComplex *g_refs, CudaComplex *g_imgs,
									FLOAT *g_Minvsigma2, FLOAT *g_diff2s,
									unsigned img_size, FLOAT sum_init,
									unsigned long significant_num,
									unsigned long translation_num,
									unsigned long *d_rotidx,
									unsigned long *d_transidx)
{
	// blockid
	int ex = blockIdx.y * gridDim.x + blockIdx.x;

	// inside the padded 2D orientation grid
	if( ex < significant_num )
	{
		// index of comparison
		unsigned long int ix=d_rotidx[ex];
		unsigned long int iy=d_transidx[ex];

		__shared__ FLOAT s[BLOCK_SIZE];
		s[threadIdx.x] = 0;

		unsigned pass_num(ceilf((float)img_size/(float)BLOCK_SIZE)), pixel;

		unsigned long ref_start(ix * img_size);
		unsigned long img_start(iy * img_size);
		unsigned long ref_pixel_idx;
		unsigned long img_pixel_idx;

		for (unsigned pass = 0; pass < pass_num; pass ++)
		{
			pixel = pass * BLOCK_SIZE + threadIdx.x;

			if (pixel < img_size) //Is inside image
			{
				ref_pixel_idx = ref_start + pixel;
				img_pixel_idx = img_start + pixel;

				FLOAT diff_real = g_refs[ref_pixel_idx].real - g_imgs[img_pixel_idx].real;
				FLOAT diff_imag = g_refs[ref_pixel_idx].imag - g_imgs[img_pixel_idx].imag;

				s[threadIdx.x] += (diff_real * diff_real + diff_imag * diff_imag) * 0.5 * g_Minvsigma2[pixel];
			}
		}

		// This version should run in             BLOCK_SIZE                  cycles
		// -------------------------------------------------------------------------
	//		if (threadIdx.x == 0)
	//		{
	//			double sum(sum_init);
	//			for (unsigned i = 0; i < BLOCK_SIZE; i++)
	//				sum += s[i];
	//
	//			g_diff2s[ex * translation_num + ey] = sum;
	//		}
		// -------------------------------------------------------------------------

		// This version should run in     BLOCK_SIZE/trads + log2(trads)      cycles
		// ( Runs ~2x as fast as the above one for BLOCK_SIZE=32 )
		// -------------------------------------------------------------------------
		__syncthreads();
		int trads = 32;
		int itr = BLOCK_SIZE/trads;
		if(threadIdx.x<trads)
		{
			for(int i=1; i<itr; i++)
			{
				s[threadIdx.x] += s[i*trads + threadIdx.x];
				//__syncthreads();
			}
		}

		for(int j=(trads/2); j>0; j/=2)
		{
			if(threadIdx.x<j)
			{
				s[threadIdx.x] += s[threadIdx.x+j];
			}
		}
		__syncthreads();
//		if (threadIdx.x*ex == 0)
		{
			g_diff2s[ix * translation_num + iy] = s[0]+sum_init;
		}
		// -------------------------------------------------------------------------
	}
}

__global__ void cuda_kernel_cc_diff2(	CudaComplex *g_refs, CudaComplex *g_imgs,
										FLOAT *g_Minvsigma2, FLOAT *g_diff2s,
										unsigned img_size, FLOAT exp_local_sqrtXi2,
										unsigned long significant_num,
										unsigned long translation_num,
										unsigned long *d_rotidx,
										unsigned long *d_transidx)
{
	// blockid
	int ex = blockIdx.y * gridDim.x + blockIdx.x;
	// inside the padded 2D orientation grid
	if( ex < significant_num )
	{
		// index of comparison
		unsigned long int ix=d_rotidx[ex];
		unsigned long int iy=d_transidx[ex];
		__shared__ double    s[BLOCK_SIZE];
		__shared__ double norm[BLOCK_SIZE];
		s[threadIdx.x] = 0;
		unsigned pass_num(ceilf((float)img_size/(float)BLOCK_SIZE));
		unsigned long pixel,
		ref_start(ix * img_size),
		img_start(iy * img_size);
		unsigned long ref_pixel_idx;
		unsigned long img_pixel_idx;
		for (unsigned pass = 0; pass < pass_num; pass ++)
		{
			pixel = pass * BLOCK_SIZE + threadIdx.x;

			if (pixel < img_size) //Is inside image
			{
				ref_pixel_idx = ref_start + pixel;
				img_pixel_idx = img_start + pixel;

				double diff_real = g_refs[ref_pixel_idx].real * g_imgs[img_pixel_idx].real;
				double diff_imag = g_refs[ref_pixel_idx].imag * g_imgs[img_pixel_idx].imag;

				double nR = g_refs[ref_pixel_idx].real*g_refs[ref_pixel_idx].real;
				double nI = g_refs[ref_pixel_idx].imag*g_refs[ref_pixel_idx].imag;

				s[threadIdx.x] -= (diff_real + diff_imag);
				norm[threadIdx.x] += nR+nI;
			}
		}
		// -------------------------------------------------------------------------
		__syncthreads();
		int trads = 32;
		int itr = BLOCK_SIZE/trads;
		if(threadIdx.x<trads)
		{
			for(int i=1; i<itr; i++)
			{
				s[threadIdx.x] += s[i*trads + threadIdx.x];
				norm[threadIdx.x] += norm[i*trads + threadIdx.x];
			}
		}
		for(int j=(trads/2); j>0; j/=2)
		{
			if(threadIdx.x<j)
			{
				s[threadIdx.x] += s[threadIdx.x+j];
				norm[threadIdx.x] += norm[threadIdx.x+j];
			}
		}
		__syncthreads();
		// -------------------------------------------------------------------------
		g_diff2s[ix * translation_num + iy] = s[0]/(sqrt(norm[0])*exp_local_sqrtXi2);
	}
}


//  Takes a boolean N-by-M matrix and returns pointer pairs to coordinates in two corresponding objects
//__global__ void cuda_kernel_boolToPointers(	bool *matrix,
//												int yLength,
//												int** yPoints)
//{
//	//save the current index of the partial array to a shared location
//	__shared__  long int  length[blockDim.x*BLOCK_SIZE];
//	length[threadIdx.x]=0;
//
//	unsigned yiter(ceilf((float)yLength/(float)BLOCK_SIZE));
//
//	for(i=0; i<yiter; i++)
//	{
//		int pos = ylength*blockIdx.x + i*BLOCK_SIZE + threadIdx.x
//		if(matrix[pos]==1)
//		{
//			yPoints[blockIdx.x][length[blockIdx.x*BLOCK_SIZE+threadidx.x]]=blockIdx.x;
//			length[blockIdx.x*BLOCK_SIZE+threadidx.x]+=1;
//		}
//	}
//
//}

void MlOptimiserCUDA::getAllSquaredDifferences(
		long int my_ori_particle, int exp_current_image_size,
		int exp_ipass, int exp_current_oversampling, int metadata_offset,
		int exp_idir_min, int exp_idir_max, int exp_ipsi_min, int exp_ipsi_max,
		int exp_itrans_min, int exp_itrans_max, int exp_iclass_min, int exp_iclass_max,
		std::vector<double> &exp_min_diff2,
		std::vector<double> &exp_highres_Xi2_imgs,
		std::vector<MultidimArray<Complex > > &exp_Fimgs,
		std::vector<MultidimArray<double> > &exp_Fctfs,
		MultidimArray<double> &exp_Mweight,
		MultidimArray<bool> &exp_Mcoarse_significant,
		std::vector<int> &exp_pointer_dir_nonzeroprior, std::vector<int> &exp_pointer_psi_nonzeroprior,
		std::vector<double> &exp_directions_prior, std::vector<double> &exp_psi_prior,
		std::vector<MultidimArray<Complex> > &exp_local_Fimgs_shifted,
		std::vector<MultidimArray<double> > &exp_local_Minvsigma2s,
		std::vector<MultidimArray<double> > &exp_local_Fctfs,
		std::vector<double> &exp_local_sqrtXi2)
{

	CUDA_CPU_TIC("diff_pre_gpu");

	// Initialise min_diff and exp_Mweight for this pass
	int exp_nr_particles = mydata.ori_particles[my_ori_particle].particles_id.size();
	long int exp_nr_dir = (do_skip_align || do_skip_rotate) ? 1 : sampling.NrDirections(0, &exp_pointer_dir_nonzeroprior);
	long int exp_nr_psi = (do_skip_align || do_skip_rotate) ? 1 : sampling.NrPsiSamplings(0, &exp_pointer_psi_nonzeroprior);
	long int exp_nr_trans = (do_skip_align) ? 1 : sampling.NrTranslationalSamplings();
	long int exp_nr_oversampled_rot = sampling.oversamplingFactorOrientations(exp_current_oversampling);
	long int exp_nr_oversampled_trans = sampling.oversamplingFactorTranslations(exp_current_oversampling);

	//for scale_correction
	int group_id;

	//printf("exp_nr_oversampled_rot=%d\n", (unsigned)exp_nr_oversampled_rot);

	exp_Mweight.resize(exp_nr_particles, mymodel.nr_classes * exp_nr_dir * exp_nr_psi * exp_nr_trans * exp_nr_oversampled_rot * exp_nr_oversampled_trans);
	exp_Mweight.initConstant(-999.);
	if (exp_ipass==0)
		exp_Mcoarse_significant.clear();

	exp_min_diff2.clear();
	exp_min_diff2.resize(exp_nr_particles, 99.e99);

	std::vector<MultidimArray<Complex > > dummy;
	precalculateShiftedImagesCtfsAndInvSigma2s(false, my_ori_particle, exp_current_image_size, exp_current_oversampling,
			exp_itrans_min, exp_itrans_max, exp_Fimgs, dummy, exp_Fctfs, exp_local_Fimgs_shifted, dummy,
			exp_local_Fctfs, exp_local_sqrtXi2, exp_local_Minvsigma2s);

	MultidimArray<Complex > Fref;
	Fref.resize(exp_local_Minvsigma2s[0]);

	unsigned image_size = exp_local_Minvsigma2s[0].nzyxdim;

	CUDA_CPU_TOC("diff_pre_gpu");

	// Loop only from exp_iclass_min to exp_iclass_max to deal with seed generation in first iteration
	for (int exp_iclass = exp_iclass_min; exp_iclass <= exp_iclass_max; exp_iclass++)
	{
		if (mymodel.pdf_class[exp_iclass] > 0.)
		{
			// Local variables
			std::vector< double > oversampled_rot, oversampled_tilt, oversampled_psi;
			std::vector< double > oversampled_translations_x, oversampled_translations_y, oversampled_translations_z;
			CudaGlobalPtr<FLOAT> gpuMinvsigma2(image_size);
			gpuMinvsigma2.device_alloc();

			Matrix2D<double> A;

			CudaGlobalPtr<CudaComplex> Frefs(image_size * exp_nr_dir * exp_nr_psi * exp_nr_oversampled_rot);

			// Mapping index look-up table
			std::vector< long unsigned > iorientclasses, iover_rots;
			long unsigned orientation_num(0);

			/*=======================================================================================
			                           Generate Reference Projections
			=========================================================================================*/

			CUDA_CPU_TIC("projection_1");

			for (long int idir = exp_idir_min, iorient = 0; idir <= exp_idir_max; idir++)
			{
				for (long int ipsi = exp_ipsi_min; ipsi <= exp_ipsi_max; ipsi++, iorient++)
				{
					long int iorientclass = exp_iclass * exp_nr_dir * exp_nr_psi + iorient;

					// Get prior for this direction and skip calculation if prior==0
					double pdf_orientation;
					if (do_skip_align || do_skip_rotate)
					{
						pdf_orientation = mymodel.pdf_class[exp_iclass];
					}
					else if (mymodel.orientational_prior_mode == NOPRIOR)
					{
						pdf_orientation = DIRECT_MULTIDIM_ELEM(mymodel.pdf_direction[exp_iclass], idir);
					}
					else
					{
						pdf_orientation = exp_directions_prior[idir] * exp_psi_prior[ipsi];
					}
					// In the first pass, always proceed
					// In the second pass, check whether one of the translations for this orientation of any of the particles had a significant weight in the first pass
					// if so, proceed with projecting the reference in that direction
					bool do_proceed = (exp_ipass==0) ? true :
						isSignificantAnyParticleAnyTranslation(iorientclass, exp_itrans_min, exp_itrans_max, exp_Mcoarse_significant);
					if (do_proceed && pdf_orientation > 0.)
					{
						// Now get the oversampled (rot, tilt, psi) triplets
						// This will be only the original (rot,tilt,psi) triplet in the first pass (exp_current_oversampling==0)
						sampling.getOrientations(idir, ipsi, exp_current_oversampling, oversampled_rot, oversampled_tilt, oversampled_psi,
								exp_pointer_dir_nonzeroprior, exp_directions_prior, exp_pointer_psi_nonzeroprior, exp_psi_prior);

						// Loop over all oversampled orientations (only a single one in the first pass)
						for (long int iover_rot = 0; iover_rot < exp_nr_oversampled_rot; iover_rot++)
						{
							// Get the Euler matrix
							Euler_angles2matrix(oversampled_rot[iover_rot],
												oversampled_tilt[iover_rot],
												oversampled_psi[iover_rot], A);

							//Fref.data = &Frefs[image_size * orientation_num];
							(mymodel.PPref[exp_iclass]).get2DFourierTransform(Fref, A, IS_NOT_INV);

							for (unsigned i = 0; i < image_size; i++)
							{
								Frefs[image_size * orientation_num + i].real = Fref.data[i].real;
								Frefs[image_size * orientation_num + i].imag = Fref.data[i].imag;
							}

							orientation_num ++;
							iorientclasses.push_back(iorientclass);
							iover_rots.push_back(iover_rot);
						}
					}
				}
			}

			Frefs.size = orientation_num * image_size;
			Frefs.device_alloc();
			Frefs.cp_to_device();
			Frefs.free_host();

			CUDA_CPU_TOC("projection_1");

			/*=======================================================================================
			                                  	  Particle Iteration
			=========================================================================================*/

			for (long int ipart = 0; ipart < mydata.ori_particles[my_ori_particle].particles_id.size(); ipart++)
			{
				/*====================================
				        Generate Translations
				======================================*/

				CUDA_CPU_TIC("translation_1");

				CudaGlobalPtr<CudaComplex> Fimgs(image_size * exp_nr_trans * exp_nr_oversampled_trans);

				long int part_id = mydata.ori_particles[my_ori_particle].particles_id[ipart];
				long unsigned translation_num(0), ihidden(0);
				std::vector< long unsigned > iover_transes, itranses, ihiddens;

				for (long int itrans = exp_itrans_min; itrans <= exp_itrans_max; itrans++, ihidden++)
				{
					sampling.getTranslations(itrans, exp_current_oversampling,
							oversampled_translations_x, oversampled_translations_y, oversampled_translations_z );

					for (long int iover_trans = 0; iover_trans < exp_nr_oversampled_trans; iover_trans++)
					{
						/// Now get the shifted image
						// Use a pointer to avoid copying the entire array again in this highly expensive loop
						Complex *myAB;
						if (exp_current_oversampling == 0)
						{
							myAB = (Fref.ydim == coarse_size) ? global_fftshifts_ab_coarse[itrans].data
									: global_fftshifts_ab_current[itrans].data;
						}
						else
						{
							int iitrans = itrans * exp_nr_oversampled_trans +  iover_trans;
							myAB = (strict_highres_exp > 0.) ? global_fftshifts_ab2_coarse[iitrans].data
									: global_fftshifts_ab2_current[iitrans].data;
						}


						FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(exp_local_Fimgs_shifted[ipart])
						{
							FLOAT real = (*(myAB + n)).real * (DIRECT_MULTIDIM_ELEM(exp_local_Fimgs_shifted[ipart], n)).real
									- (*(myAB + n)).imag *(DIRECT_MULTIDIM_ELEM(exp_local_Fimgs_shifted[ipart], n)).imag;
							FLOAT imag = (*(myAB + n)).real * (DIRECT_MULTIDIM_ELEM(exp_local_Fimgs_shifted[ipart], n)).imag
									+ (*(myAB + n)).imag *(DIRECT_MULTIDIM_ELEM(exp_local_Fimgs_shifted[ipart], n)).real;

							//When on gpu, it makes more sense to ctf-correct translated images, rather than anti-ctf-correct ref-projections
							if (do_scale_correction)
							{
								//group_id = mydata.getGroupId(part_id);
								FLOAT myscale = mymodel.scale_correction[group_id];
								real /= myscale;
								imag /= myscale;
							}
							if (do_ctf_correction && refs_are_ctf_corrected)
							{
								real /= DIRECT_MULTIDIM_ELEM(exp_local_Fctfs[ipart], n);
								imag /= DIRECT_MULTIDIM_ELEM(exp_local_Fctfs[ipart], n);
							}
							Fimgs[translation_num * image_size + n].real = real;
							Fimgs[translation_num * image_size + n].imag = imag;
						}
						translation_num ++;

						ihiddens.push_back(ihidden);
						itranses.push_back(itrans);
						iover_transes.push_back(iover_trans);
					}
				}

				CUDA_CPU_TOC("translation_1");

				/*===========================================
				   Determine significant comparison indices
				=============================================*/
				//      This section is annoying to test because
				//		it can't complete on first pass, since
				//		the significance has never been set


				CUDA_CPU_TIC("pair_list_1");

				CudaGlobalPtr<long unsigned> transidx(orientation_num*translation_num), rotidx(orientation_num*translation_num);

				long unsigned coarse_num = exp_nr_dir*exp_nr_psi*exp_nr_trans;
				long unsigned significant_num(0);

				if (exp_ipass == 0)
				{
					exp_Mcoarse_significant.resize(coarse_num, 1);
					for (long unsigned i = 0; i < orientation_num; i++)
					{
						for (long unsigned j = 0; j < translation_num; j++)
						{
							rotidx[significant_num] = i;
							transidx[significant_num] = j;
							significant_num++;
//							DIRECT_A2D_ELEM(exp_Mcoarse_significant, ipart, i)=1;
//							std::cerr << "exp_Mcoarse_significant("<< i <<") = " <<    DIRECT_A2D_ELEM(exp_Mcoarse_significant, ipart, i) << std::endl;
//							std::cerr << "exp_Mcoarse_significant("<< i <<") = " << *(&DIRECT_A2D_ELEM(exp_Mcoarse_significant, ipart, 0)+i*sizeof(bool)) << std::endl;
						}
					}
				}
				else
				{
					for (long unsigned i = 0; i < orientation_num; i++)
					{
						long int iover_rot = iover_rots[i];
//						long int iover_rot = i % exp_nr_oversampled_rot
						long int coarse_rot = floor(i/exp_nr_oversampled_rot);
						for (long unsigned j = 0; j < translation_num; j++)
						{
							long int iover_trans = iover_transes[j];
//							long int iover_trans = j % exp_nr_oversampled_trans
							long int coarse_trans = floor(j/exp_nr_oversampled_trans);
							long int ihidden = iorientclasses[i] * exp_nr_trans + ihiddens[j];
							if(DIRECT_A2D_ELEM(exp_Mcoarse_significant, ipart, ihidden)==1)
							{
								 long int ihidden_over = sampling.getPositionOversampledSamplingPoint(ihidden,
										                  exp_current_oversampling, iover_rot, iover_trans);

								rotidx[significant_num] = i;
								transidx[significant_num] = j;
								significant_num++;
							}
						}
					}
				}

				CUDA_CPU_TOC("pair_list_1");

//				std::cerr << "orientation_num "<< orientation_num << std::endl;
//				std::cerr << "translation_num "<< translation_num << std::endl;
//				std::cerr << "my_nr_significant_coarse_samples "<< DIRECT_A2D_ELEM(exp_metadata, metadata_offset + ipart, METADATA_NR_SIGN) << std::endl;
//				std::cerr << "significant_num "<< significant_num << std::endl;

				/*====================================
				   Initiate Particle Related On GPU
				======================================*/


				CUDA_CPU_TIC("kernel_init_1");

				// Since we hijack Minvsigma to carry a bit more info into the GPU-kernel
				// we need to make a modified copy, since the global object shouldn't be
				// changed


				FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(exp_local_Fimgs_shifted[ipart])
				{
					gpuMinvsigma2[n] = *(exp_local_Minvsigma2s[ipart].data + n );
					//std::cerr <<  *(exp_local_Minvsigma2s[ipart].data + n )<< " ";
				}

				if (do_ctf_correction && refs_are_ctf_corrected)
				{
					FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(exp_local_Fimgs_shifted[ipart])
					{
						gpuMinvsigma2[n] *= (DIRECT_MULTIDIM_ELEM(exp_local_Fctfs[ipart], n)*DIRECT_MULTIDIM_ELEM(exp_local_Fctfs[ipart], n));
					}
				}
				// TODO :    + Assure accuracy with the implemented GPU-based ctf-scaling
				//           + Make setting of myscale robust between here and above.
				//  (scale_correction turns off by default with only one group: ml_optimiser-line 1067,
				//   meaning small-scale test will probably not catch this malfunctioning when/if it breaks.)
				if (do_scale_correction)
				{
					FLOAT myscale = mymodel.scale_correction[group_id];
					FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(exp_local_Fimgs_shifted[ipart])
					{
						gpuMinvsigma2[n] *= (myscale*myscale);
					}
				}

				Fimgs.size = translation_num * image_size;
				Fimgs.device_alloc();
				Fimgs.cp_to_device();

				gpuMinvsigma2.cp_to_device();

				CudaGlobalPtr<FLOAT> diff2s(orientation_num*translation_num);
				diff2s.device_alloc();

				rotidx.size = significant_num;
				rotidx.device_alloc();
				rotidx.cp_to_device();

				transidx.size = significant_num;
				transidx.device_alloc();
				transidx.cp_to_device();

				/*====================================
				    		Kernel Calls
				======================================*/
				unsigned orient1, orient2;

				if(significant_num>65535)
				{
					orient1 = ceil(sqrt(significant_num));
					orient2 = orient1;
				}
				else
				{
					orient1 = significant_num;
					orient2 = 1;
				}
				dim3 block_dim(orient1,orient2);

				CUDA_CPU_TOC("kernel_init_1");

				CUDA_GPU_TIC("cuda_kernel_diff2");

				if ((iter == 1 && do_firstiter_cc) || do_always_cc) // do cross-correlation instead of diff
				{
					cuda_kernel_cc_diff2<<<block_dim,BLOCK_SIZE>>>(~Frefs, ~Fimgs, ~gpuMinvsigma2,  ~diff2s,
																	image_size, exp_highres_Xi2_imgs[ipart],
																	significant_num,
																	translation_num,
																	~rotidx,
																	~transidx);
				}
				else
				{
					cuda_kernel_diff2<<<block_dim,BLOCK_SIZE>>>(~Frefs, ~Fimgs, ~gpuMinvsigma2, ~diff2s,
																image_size, exp_highres_Xi2_imgs[ipart] / 2.,
																significant_num,
																translation_num,
																~rotidx,
																~transidx);
				}

				CUDA_GPU_TAC("cuda_kernel_diff2");

				/*====================================
				    	   Retrieve Results
				======================================*/

				HANDLE_ERROR(hipDeviceSynchronize()); //TODO Apparently this is not required here

				CUDA_GPU_TOC("cuda_kernel_diff2");

				diff2s.cp_to_host();

				if (exp_ipass == 0)
				{
					exp_Mcoarse_significant.clear();
				}

				/*====================================
				    	Write To Destination
				======================================*/


				CUDA_CPU_TIC("collect_data_1");

				for (long unsigned k = 0; k < significant_num; k ++)
				{
					long unsigned i = rotidx[k];
					long unsigned j = transidx[k];
					long int iover_rot = iover_rots[i];

					long int ihidden = iorientclasses[i] * exp_nr_trans + ihiddens[j];
					long int iover_trans = iover_transes[j];

					long int ihidden_over = sampling.getPositionOversampledSamplingPoint(ihidden, exp_current_oversampling,
																						iover_rot, iover_trans);

					double diff2 = diff2s[i * translation_num + j];

					DIRECT_A2D_ELEM(exp_Mweight, ipart, ihidden_over) = diff2;

					// Keep track of minimum of all diff2, only for the last image in this series
					if (diff2 < exp_min_diff2[ipart])
						exp_min_diff2[ipart] = diff2;
				}

				CUDA_CPU_TOC("collect_data_1");

			} // end loop ipart

		} // end if class significant
	} // end loop iclass
}

__global__ void cuda_kernel_wavg(	CudaComplex *g_refs, CudaComplex *g_imgs, CudaComplex *g_imgs_nomask,
									FLOAT* g_weights, FLOAT* g_ctfs, FLOAT* g_Minvsigma2s,
									FLOAT *g_wdiff2s_parts, CudaComplex *g_wavgs, FLOAT* g_Fweights,
									unsigned long translation_num, FLOAT weight_norm,
									FLOAT significant_weight, unsigned image_size,
									bool refs_are_ctf_corrected)
{
	unsigned long iorient = blockIdx.y*gridDim.x + blockIdx.x;
	unsigned tid = threadIdx.x;

	//TODO Consider Mresol_fine to speed up this kernel

	unsigned pass_num(ceilf((float)image_size/(float)BLOCK_SIZE)),pixel;
	FLOAT Fweight, wavgs_real, wavgs_imag, wdiff2s_parts;

	for (unsigned pass = 0; pass < pass_num; pass ++)
	{
		wavgs_real = 0;
		wavgs_imag = 0;
		wdiff2s_parts = 0;
		Fweight = 0;

		pixel = pass * BLOCK_SIZE + tid;

		if (pixel < image_size)
		{
			unsigned long orientation_pixel = iorient * image_size + pixel;

			for (unsigned long itrans = 0; itrans < translation_num; itrans++)
			{
				FLOAT weight = g_weights[iorient * translation_num + itrans];

				if (weight >= significant_weight)
				{
					weight /= weight_norm;

					unsigned long img_pixel_idx = itrans * image_size + pixel;

					FLOAT myctf = g_ctfs[pixel];
					CudaComplex ref = g_refs[orientation_pixel];
					if (refs_are_ctf_corrected) //FIXME Create two kernels for the different cases
					{
						ref.real *= myctf;
						ref.imag *= myctf;
					}
					FLOAT diff_real = ref.real - g_imgs[img_pixel_idx].real;
					FLOAT diff_imag = ref.imag - g_imgs[img_pixel_idx].imag;
					wdiff2s_parts += weight * (diff_real*diff_real + diff_imag*diff_imag);
					FLOAT weightxinvsigma2 = weight * myctf * g_Minvsigma2s[pixel];
					wavgs_real += g_imgs_nomask[img_pixel_idx].real * weightxinvsigma2;
					wavgs_imag += g_imgs_nomask[img_pixel_idx].imag * weightxinvsigma2;
					Fweight += weightxinvsigma2 * myctf;
				}
			}

			g_wavgs[orientation_pixel].real = wavgs_real; //TODO should be buffered into shared
			g_wavgs[orientation_pixel].imag = wavgs_imag; //TODO should be buffered into shared
			g_wdiff2s_parts[orientation_pixel] = wdiff2s_parts; //TODO this could be further reduced in here
			g_Fweights[orientation_pixel] = Fweight; //TODO should be buffered into shared
		}
	}
}



void MlOptimiserCUDA::storeWeightedSums(long int my_ori_particle, int exp_current_image_size,
		int exp_current_oversampling, int metadata_offset,
		int exp_idir_min, int exp_idir_max, int exp_ipsi_min, int exp_ipsi_max,
		int exp_itrans_min, int exp_itrans_max, int exp_iclass_min, int exp_iclass_max,
		std::vector<double> &exp_min_diff2,
		std::vector<double> &exp_highres_Xi2_imgs,
		std::vector<MultidimArray<Complex > > &exp_Fimgs,
		std::vector<MultidimArray<Complex > > &exp_Fimgs_nomask,
		std::vector<MultidimArray<double> > &exp_Fctfs,
		std::vector<MultidimArray<double> > &exp_power_imgs,
		std::vector<Matrix1D<double> > &exp_old_offset,
		std::vector<Matrix1D<double> > &exp_prior,
		MultidimArray<double> &exp_Mweight,
		MultidimArray<bool> &exp_Mcoarse_significant,
		std::vector<double> &exp_significant_weight,
		std::vector<double> &exp_sum_weight,
		std::vector<double> &exp_max_weight,
		std::vector<int> &exp_pointer_dir_nonzeroprior, std::vector<int> &exp_pointer_psi_nonzeroprior,
		std::vector<double> &exp_directions_prior, std::vector<double> &exp_psi_prior,
		std::vector<MultidimArray<Complex > > &exp_local_Fimgs_shifted,
		std::vector<MultidimArray<Complex > > &exp_local_Fimgs_shifted_nomask,
		std::vector<MultidimArray<double> > &exp_local_Minvsigma2s,
		std::vector<MultidimArray<double> > &exp_local_Fctfs,
		std::vector<double> &exp_local_sqrtXi2)
{
	CUDA_CPU_TIC("store_pre_gpu");

	int exp_nr_particles = mydata.ori_particles[my_ori_particle].particles_id.size();
	long int exp_nr_dir = (do_skip_align || do_skip_rotate) ? 1 : sampling.NrDirections(0, &exp_pointer_dir_nonzeroprior);
	long int exp_nr_psi = (do_skip_align || do_skip_rotate) ? 1 : sampling.NrPsiSamplings(0, &exp_pointer_psi_nonzeroprior);
	long int exp_nr_trans = (do_skip_align) ? 1 : sampling.NrTranslationalSamplings();
	long int exp_nr_oversampled_rot = sampling.oversamplingFactorOrientations(exp_current_oversampling);
	long int exp_nr_oversampled_trans = sampling.oversamplingFactorTranslations(exp_current_oversampling);

	// Re-do below because now also want unmasked images AND if (stricht_highres_exp >0.) then may need to resize
	precalculateShiftedImagesCtfsAndInvSigma2s(true, my_ori_particle, exp_current_image_size, exp_current_oversampling,
			exp_itrans_min, exp_itrans_max, exp_Fimgs, exp_Fimgs_nomask, exp_Fctfs, exp_local_Fimgs_shifted, exp_local_Fimgs_shifted_nomask,
			exp_local_Fctfs, exp_local_sqrtXi2, exp_local_Minvsigma2s);

	// In doThreadPrecalculateShiftedImagesCtfsAndInvSigma2s() the origin of the exp_local_Minvsigma2s was omitted.
	// Set those back here
	for (long int ipart = 0; ipart < mydata.ori_particles[my_ori_particle].particles_id.size(); ipart++)
	{
		long int part_id = mydata.ori_particles[my_ori_particle].particles_id[ipart];
		int group_id = mydata.getGroupId(part_id);
		DIRECT_MULTIDIM_ELEM(exp_local_Minvsigma2s[ipart], 0) = 1. / (sigma2_fudge * DIRECT_A1D_ELEM(mymodel.sigma2_noise[group_id], 0));
	}

	// Initialise the maximum of all weights to a negative value
	exp_max_weight.clear();
	exp_max_weight.resize(exp_nr_particles, -1.);

	// For norm_correction and scale_correction of all particles of this ori_particle
	std::vector<double> exp_wsum_norm_correction;
	std::vector<MultidimArray<double> > exp_wsum_scale_correction_XA, exp_wsum_scale_correction_AA;
	std::vector<MultidimArray<double> > thr_wsum_signal_product_spectra, thr_wsum_reference_power_spectra;
	exp_wsum_norm_correction.resize(exp_nr_particles, 0.);

	// For scale_correction
	if (do_scale_correction)
	{
		MultidimArray<double> aux;
		aux.initZeros(mymodel.ori_size/2 + 1);
		exp_wsum_scale_correction_XA.resize(exp_nr_particles, aux);
		exp_wsum_scale_correction_AA.resize(exp_nr_particles, aux);
		thr_wsum_signal_product_spectra.resize(mymodel.nr_groups, aux);
		thr_wsum_reference_power_spectra.resize(mymodel.nr_groups, aux);
	}

	std::vector< double> oversampled_rot, oversampled_tilt, oversampled_psi;
	std::vector<double> oversampled_translations_x, oversampled_translations_y, oversampled_translations_z;
	Matrix2D<double> A;
	MultidimArray<Complex > Fimg, Fimg_otfshift_nomask;
	MultidimArray<double> Fweight, Minvsigma2, Mctf;
	bool have_warned_small_scale = false;

	Fimg.resize(exp_Fimgs[0]);
	Fweight.resize(exp_Fimgs[0]);

	// Initialise Mctf to all-1 for if !do_ctf_corection
	Mctf.resize(exp_Fimgs[0]);
	Mctf.initConstant(1.);
	// Initialise Minvsigma2 to all-1 for if !do_map
	Minvsigma2.resize(exp_Fimgs[0]);
	Minvsigma2.initConstant(1.);

	// Make local copies of weighted sums (except BPrefs, which are too big)
	// so that there are not too many mutex locks below
	std::vector<MultidimArray<double> > thr_wsum_sigma2_noise, thr_wsum_pdf_direction;
	std::vector<double> thr_wsum_norm_correction, thr_sumw_group, thr_wsum_pdf_class, thr_wsum_prior_offsetx_class, thr_wsum_prior_offsety_class;
	double thr_wsum_sigma2_offset;
	MultidimArray<double> thr_metadata, zeroArray;
	// Wsum_sigma_noise2 is a 1D-spectrum for each group
	zeroArray.initZeros(mymodel.ori_size/2 + 1);
	thr_wsum_sigma2_noise.resize(mymodel.nr_groups, zeroArray);
	// wsum_pdf_direction is a 1D-array (of length sampling.NrDirections()) for each class
	zeroArray.initZeros(sampling.NrDirections());
	thr_wsum_pdf_direction.resize(mymodel.nr_classes, zeroArray);
	// sumw_group is a double for each group
	thr_sumw_group.resize(mymodel.nr_groups, 0.);
	// wsum_pdf_class is a double for each class
	thr_wsum_pdf_class.resize(mymodel.nr_classes, 0.);
	if (mymodel.ref_dim == 2)
	{
		thr_wsum_prior_offsetx_class.resize(mymodel.nr_classes, 0.);
		thr_wsum_prior_offsety_class.resize(mymodel.nr_classes, 0.);
	}
	// wsum_sigma2_offset is just a double
	thr_wsum_sigma2_offset = 0.;

	unsigned image_size = exp_local_Minvsigma2s[0].xdim*exp_local_Minvsigma2s[0].ydim;

	CUDA_CPU_TOC("store_pre_gpu");

	// Loop from iclass_min to iclass_max to deal with seed generation in first iteration
	for (int exp_iclass = exp_iclass_min; exp_iclass <= exp_iclass_max; exp_iclass++)
	{

		/*=======================================================================================
		                            REFERENCE PROJECTION GENERATION
		=======================================================================================*/
		CUDA_CPU_TIC("projection_2");

		CudaGlobalPtr<CudaComplex> Frefs(image_size * exp_nr_dir * exp_nr_psi * exp_nr_oversampled_rot);

		std::vector< long unsigned > iorientclasses, idirs, iover_rots;
		std::vector< double > rots, tilts, psis;
		long unsigned orientation_num(0);

		for (long int idir = exp_idir_min, iorient = 0; idir <= exp_idir_max; idir++)
		{
			for (long int ipsi = exp_ipsi_min; ipsi <= exp_ipsi_max; ipsi++, iorient++)
			{
				long int iorientclass = exp_iclass * exp_nr_dir * exp_nr_psi + iorient;

				sampling.getOrientations(idir, ipsi, adaptive_oversampling, oversampled_rot, oversampled_tilt, oversampled_psi,
						exp_pointer_dir_nonzeroprior, exp_directions_prior, exp_pointer_psi_nonzeroprior, exp_psi_prior);

				if (isSignificantAnyParticleAnyTranslation(iorientclass, exp_itrans_min, exp_itrans_max, exp_Mcoarse_significant))
				{
					for (long unsigned iover_rot = 0; iover_rot < exp_nr_oversampled_rot; iover_rot++)
					{
						double rot = oversampled_rot[iover_rot];
						double tilt = oversampled_tilt[iover_rot];
						double psi = oversampled_psi[iover_rot];
						// Get the Euler matrix
						Euler_angles2matrix(rot, tilt, psi, A);

						rots.push_back(rot);
						tilts.push_back(tilt);
						psis.push_back(psi);

						//Fref.data = &Frefs[image_size * orientation_num];
						(mymodel.PPref[exp_iclass]).get2DFourierTransform(Fimg, A, IS_NOT_INV);

						for (unsigned i = 0; i < image_size; i++)
						{
							Frefs[image_size * orientation_num + i].real = Fimg.data[i].real;
							Frefs[image_size * orientation_num + i].imag = Fimg.data[i].imag;
						}

						orientation_num ++;
						idirs.push_back(idir);
						iorientclasses.push_back(iorientclass);
						iover_rots.push_back(iover_rot);
					}
				}
			}
		}

		Frefs.size = orientation_num * image_size;
		Frefs.device_alloc();
		Frefs.cp_to_device();
		Frefs.free_host();

		CudaGlobalPtr<CudaComplex> wavgs(orientation_num * image_size);
		wavgs.device_alloc();
		//wavgs.device_init(0);

		CudaGlobalPtr<FLOAT> Fweights(orientation_num * image_size);
		Fweights.device_alloc();
		//Fweights.device_init(0);

		CUDA_CPU_TOC("projection_2");

		/*=======================================================================================
										  PARTICLE ITERATION
		=======================================================================================*/

		/// Now that reference projection has been made loop over all particles inside this ori_particle
		for (long int ipart = 0; ipart < mydata.ori_particles[my_ori_particle].particles_id.size(); ipart++)
		{
			long int part_id = mydata.ori_particles[my_ori_particle].particles_id[ipart];
			int group_id = mydata.getGroupId(part_id);

			double myprior_x, myprior_y, myprior_z;
			double old_offset_x = XX(exp_old_offset[ipart]);
			double old_offset_y = YY(exp_old_offset[ipart]);
			double old_offset_z;

			if (mymodel.ref_dim == 2)
			{
				myprior_x = XX(mymodel.prior_offset_class[exp_iclass]);
				myprior_y = YY(mymodel.prior_offset_class[exp_iclass]);
			}
			else
			{
				myprior_x = XX(exp_prior[ipart]);
				myprior_y = YY(exp_prior[ipart]);
				if (mymodel.data_dim == 3)
				{
					myprior_z = ZZ(exp_prior[ipart]);
					old_offset_z = ZZ(exp_old_offset[ipart]);
				}
			}


			/*======================================================
								 TRANSLATIONS
			======================================================*/

			CUDA_CPU_TIC("translation_2");

			CudaGlobalPtr<CudaComplex> Fimgs(image_size * exp_nr_trans * exp_nr_oversampled_trans);
			CudaGlobalPtr<CudaComplex> Fimgs_nomask(Fimgs.size);

			long unsigned translation_num(0), ihidden(0);
			std::vector< long unsigned > iover_transes, itranses, ihiddens;

			for (long int itrans = exp_itrans_min, iitrans = 0; itrans <= exp_itrans_max; itrans++, ihidden++)
			{
				sampling.getTranslations(itrans, adaptive_oversampling,
						oversampled_translations_x, oversampled_translations_y, oversampled_translations_z);
				for (long int iover_trans = 0; iover_trans < exp_nr_oversampled_trans; iover_trans++, iitrans++)
				{
					/// Now get the shifted image
					// Use a pointer to avoid copying the entire array again in this highly expensive loop
					Complex* myAB;
					myAB = (adaptive_oversampling == 0 ) ? global_fftshifts_ab_current[iitrans].data : global_fftshifts_ab2_current[iitrans].data;
					FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(exp_local_Fimgs_shifted[ipart])
					{
						FLOAT a = (*(myAB + n)).real;
						FLOAT b = (*(myAB + n)).imag;

						// Fimg_shift
						FLOAT real = a * (DIRECT_MULTIDIM_ELEM(exp_local_Fimgs_shifted[ipart], n)).real
								- b *(DIRECT_MULTIDIM_ELEM(exp_local_Fimgs_shifted[ipart], n)).imag;
						FLOAT imag = a * (DIRECT_MULTIDIM_ELEM(exp_local_Fimgs_shifted[ipart], n)).imag
								+ b *(DIRECT_MULTIDIM_ELEM(exp_local_Fimgs_shifted[ipart], n)).real;
						Fimgs[translation_num * image_size + n].real = real;
						Fimgs[translation_num * image_size + n].imag = imag;

						// Fimg_shift_nomask
						real = a * (DIRECT_MULTIDIM_ELEM(exp_local_Fimgs_shifted_nomask[ipart], n)).real
								- b *(DIRECT_MULTIDIM_ELEM(exp_local_Fimgs_shifted_nomask[ipart], n)).imag;
						imag = a * (DIRECT_MULTIDIM_ELEM(exp_local_Fimgs_shifted_nomask[ipart], n)).imag
								+ b *(DIRECT_MULTIDIM_ELEM(exp_local_Fimgs_shifted_nomask[ipart], n)).real;
						Fimgs_nomask[translation_num * image_size + n].real = real;
						Fimgs_nomask[translation_num * image_size + n].imag = imag;
					}

					translation_num ++;

					ihiddens.push_back(ihidden);
					itranses.push_back(itrans);
					iover_transes.push_back(iover_trans);
				}
			}

			Fimgs.size = translation_num * image_size;
			Fimgs.device_alloc();
			Fimgs.cp_to_device();

			Fimgs_nomask.size = translation_num * image_size;
			Fimgs_nomask.device_alloc();
			Fimgs_nomask.cp_to_device();

			CUDA_CPU_TOC("translation_2");

			/*======================================================
					            MAP WEIGHTS
			======================================================*/

			CudaGlobalPtr<FLOAT> sorted_weights(orientation_num * translation_num);

			for (long unsigned i = 0; i < orientation_num; i++)
			{
				long unsigned iover_rot = iover_rots[i];
				for (long unsigned j = 0; j < translation_num; j++)
				{
					long unsigned iover_trans = iover_transes[j];
					long unsigned ihidden = iorientclasses[i] * exp_nr_trans + ihiddens[j];
					long unsigned ihidden_over = sampling.getPositionOversampledSamplingPoint(ihidden,
											  exp_current_oversampling, iover_rot, iover_trans);
					sorted_weights[(long unsigned) i * translation_num + j] =
							DIRECT_A2D_ELEM(exp_Mweight, ipart, ihidden_over);
				}
			}


			/*======================================================
					            KERNEL CALL
			======================================================*/
#ifdef DEBUG_CUDA_MEM
		printf("Before Cpy to Device: ");
		cudaPrintMemInfo();
#endif

			sorted_weights.device_alloc();
			sorted_weights.cp_to_device();
			sorted_weights.free_host();

			CudaGlobalPtr<FLOAT> ctfs(image_size); //TODO Almost same size for all iparts, should be allocated once
			ctfs.device_alloc();

			if (do_ctf_correction)
			{
				for (unsigned i = 0; i < image_size; i++)
					ctfs[i] = (FLOAT) exp_local_Fctfs[ipart].data[i];
				ctfs.cp_to_device();
			}
			else
				ctfs.device_init(1.);

			CudaGlobalPtr<FLOAT> Minvsigma2s(image_size); //TODO Almost same size for all iparts, should be allocated once
			for (unsigned i = 0; i < image_size; i++)
				Minvsigma2s[i] = exp_local_Minvsigma2s[ipart].data[i];

			Minvsigma2s.device_alloc();
			Minvsigma2s.cp_to_device();

			CudaGlobalPtr<FLOAT> wdiff2s_parts(orientation_num * image_size); //TODO Almost same size for all iparts, should be allocated once
			wdiff2s_parts.device_alloc();

			unsigned orient1, orient2;
			if(orientation_num>65535)
			{
				orient1 = ceil(sqrt(orientation_num));
				orient2 = orient1;
			}
			else
			{
				orient1 = orientation_num;
				orient2 = 1;
			}
			dim3 block_dim(orient1,orient2);

			CUDA_GPU_TIC("cuda_kernel_wavg");

			cuda_kernel_wavg<<<block_dim,BLOCK_SIZE>>>(
												~Frefs, ~Fimgs, ~Fimgs_nomask,
												~sorted_weights, ~ctfs, ~Minvsigma2s,
												~wdiff2s_parts, ~wavgs, ~Fweights,
												translation_num,
												(FLOAT) exp_sum_weight[ipart],
												(FLOAT) exp_significant_weight[ipart],
												image_size,
												refs_are_ctf_corrected
												);

			CUDA_GPU_TAC("cuda_kernel_wavg");

			HANDLE_ERROR(hipDeviceSynchronize()); //TODO Apparently this is not required here

			CUDA_GPU_TOC("cuda_kernel_wavg");

			Fimgs.free_device();
			Fimgs_nomask.free_device();

			sorted_weights.free_device();
			ctfs.free_device();
			Minvsigma2s.free_device();

			/*======================================================
								COLLECT DATA
			======================================================*/

			CUDA_CPU_TIC("reduce_wdiff2s");

			//TODO Following reduction should be done on the GPU
			wdiff2s_parts.cp_to_host();
			wdiff2s_parts.free_device();

			for (long int j = 0; j < image_size; j++)
			{
				int ires = DIRECT_MULTIDIM_ELEM(Mresol_fine, j);
				if (ires > -1)
				{
					double sum = 0;
					for (long int i = 0; i < orientation_num; i++)
						sum += (double) wdiff2s_parts[i * image_size + j];
					thr_wsum_sigma2_noise[group_id].data[ires] += sum;
					exp_wsum_norm_correction[ipart] += sum;
				}
			}

			wdiff2s_parts.free_host();

			CUDA_CPU_TOC("reduce_wdiff2s");

			CUDA_CPU_TIC("collect_data_2");
			//TODO some in the following double loop can be GPU accelerated
			//TODO should be replaced with loop over pairs of projections and translations (like in the getAllSquaredDifferences-function)

			// exp_nr_dir * exp_nr_psi * exp_nr_oversampled_rot * exp_nr_trans * exp_nr_oversampled_trans
			for (int exp_iclass = exp_iclass_min; exp_iclass <= exp_iclass_max; exp_iclass++)
			{
				for (long int idir = exp_idir_min, iorient = 0; idir <= exp_idir_max; idir++)
				{
					for (long int ipsi = exp_ipsi_min; ipsi <= exp_ipsi_max; ipsi++, iorient++)
					{
						long int iorientclass = exp_iclass * exp_nr_dir * exp_nr_psi + iorient;

						// Only proceed if any of the particles had any significant coarsely sampled translation
						if (isSignificantAnyParticleAnyTranslation(iorientclass, exp_itrans_min, exp_itrans_max, exp_Mcoarse_significant))
						{
							// Now get the oversampled (rot, tilt, psi) triplets
							// This will be only the original (rot,tilt,psi) triplet if (adaptive_oversampling==0)
							sampling.getOrientations(idir, ipsi, adaptive_oversampling, oversampled_rot, oversampled_tilt, oversampled_psi,
									exp_pointer_dir_nonzeroprior, exp_directions_prior, exp_pointer_psi_nonzeroprior, exp_psi_prior);
							// Loop over all oversampled orientations (only a single one in the first pass)
							for (long int iover_rot = 0; iover_rot < exp_nr_oversampled_rot; iover_rot++)
							{
								double rot = oversampled_rot[iover_rot];
								double tilt = oversampled_tilt[iover_rot];
								double psi = oversampled_psi[iover_rot];

								// Get the Euler matrix
								Euler_angles2matrix(rot, tilt, psi, A);


								long int ihidden = iorientclass * exp_nr_trans;
								for (long int itrans = exp_itrans_min, iitrans = 0; itrans <= exp_itrans_max; itrans++, ihidden++)
								{
									sampling.getTranslations(itrans, adaptive_oversampling,
											oversampled_translations_x, oversampled_translations_y, oversampled_translations_z);
									for (long int iover_trans = 0; iover_trans < exp_nr_oversampled_trans; iover_trans++, iitrans++)
									{
										// Only deal with this sampling point if its weight was significant
										long int ihidden_over = ihidden * exp_nr_oversampled_trans * exp_nr_oversampled_rot +
												iover_rot * exp_nr_oversampled_trans + iover_trans;

										double weight = DIRECT_A2D_ELEM(exp_Mweight, ipart, ihidden_over);
										if (weight >= exp_significant_weight[ipart])
										{
											weight /= exp_sum_weight[ipart];

											// Store sum of weights for this group
											thr_sumw_group[group_id] += weight;
											// Store weights for this class and orientation
											thr_wsum_pdf_class[exp_iclass] += weight;

											// The following goes MUCH faster than the original lines below....
											if (mymodel.ref_dim == 2)
											{
												thr_wsum_prior_offsetx_class[exp_iclass] += weight * (old_offset_x + oversampled_translations_x[iover_trans]);
												thr_wsum_prior_offsety_class[exp_iclass] += weight * (old_offset_y + oversampled_translations_y[iover_trans]);
											}
											double diffx = myprior_x - old_offset_x - oversampled_translations_x[iover_trans];
											double diffy = myprior_y - old_offset_y - oversampled_translations_y[iover_trans];
											if (mymodel.data_dim == 3)
											{
												double diffz  = myprior_z - old_offset_z - oversampled_translations_z[iover_trans];
												thr_wsum_sigma2_offset += weight * (diffx*diffx + diffy*diffy + diffz*diffz);
											}
											else
											{
												thr_wsum_sigma2_offset += weight * (diffx*diffx + diffy*diffy);
											}

											// Store weight for this direction of this class
											if (do_skip_align || do_skip_rotate )
											{
												//ignore pdf_direction
											}
											else if (mymodel.orientational_prior_mode == NOPRIOR)
											{
												DIRECT_MULTIDIM_ELEM(thr_wsum_pdf_direction[exp_iclass], idir) += weight;
											}
											else
											{
												// In the case of orientational priors, get the original number of the direction back
												long int mydir = exp_pointer_dir_nonzeroprior[idir];
												DIRECT_MULTIDIM_ELEM(thr_wsum_pdf_direction[exp_iclass], mydir) += weight;
											}

											if (weight > exp_max_weight[ipart])
											{
												// Store optimal image parameters
												exp_max_weight[ipart] = weight;

												A = A.inv();
												A = A.inv();
												Euler_matrix2angles(A, rot, tilt, psi);

												DIRECT_A2D_ELEM(exp_metadata, metadata_offset + ipart, METADATA_ROT) = rot;
												DIRECT_A2D_ELEM(exp_metadata, metadata_offset + ipart, METADATA_TILT) = tilt;
												DIRECT_A2D_ELEM(exp_metadata, metadata_offset + ipart, METADATA_PSI) = psi;
												DIRECT_A2D_ELEM(exp_metadata, metadata_offset + ipart, METADATA_XOFF) = XX(exp_old_offset[ipart]) + oversampled_translations_x[iover_trans];
												DIRECT_A2D_ELEM(exp_metadata, metadata_offset + ipart, METADATA_YOFF) = YY(exp_old_offset[ipart]) + oversampled_translations_y[iover_trans];
												if (mymodel.data_dim == 3)
													DIRECT_A2D_ELEM(exp_metadata, metadata_offset + ipart, METADATA_ZOFF) = ZZ(exp_old_offset[ipart]) + oversampled_translations_z[iover_trans];
												DIRECT_A2D_ELEM(exp_metadata, metadata_offset + ipart, METADATA_CLASS) = (double)exp_iclass + 1;
												DIRECT_A2D_ELEM(exp_metadata, metadata_offset + ipart, METADATA_PMAX) = exp_max_weight[ipart];
											}
										}
									}
								}
							}
						}
					}
				}
			}

			CUDA_CPU_TOC("collect_data_2");

#ifdef DEBUG_CUDA_MEM
		printf("After Freeing Device Mem: ");
		cudaPrintMemInfo();
#endif

		} // end loop ipart

		Frefs.free_device();

		/*=======================================================================================
										   BACKPROJECTION
		=======================================================================================*/

		CUDA_CPU_TIC("backprojection");

		wavgs.cp_to_host();
		wavgs.free_device();

		Fweights.cp_to_host();
		Fweights.free_device();

#ifdef RELION_TESTING
		std::string fnm = std::string("gpu_out_exp_wsum_norm_correction.txt");
		char *text = &fnm[0];
		freopen(text,"w",stdout);
		for (long int ipart = 0; ipart < mydata.ori_particles[my_ori_particle].particles_id.size(); ipart++)
		{
			printf("%4.8f \n",exp_wsum_norm_correction[ipart]);
		}
		fclose(stdout);
		//----------
		fnm = std::string("gpu_out_thr_wsum_sigma2_noise.txt");
		text = &fnm[0];
		freopen(text,"w",stdout);
		FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(Mresol_fine)
		{
			printf("%4.8f \n",thr_wsum_sigma2_noise[0].data[n]);
		}
		fclose(stdout);
		//----------
		fnm = std::string("gpu_out_Fweights.txt");
		text = &fnm[0];
		freopen(text,"w",stdout);
		for(int n = 0; n < 1000; n++)
		{
			printf("%4.8f \n",Fweights[n*60+50]);
		}
		fclose(stdout);
#endif

		for (long int i = 0; i < orientation_num; i++)
		{
			Euler_angles2matrix(rots[i], tilts[i], psis[i], A);

			for (unsigned j = 0; j < image_size; j++)
			{
				Fimg.data[j].real = wavgs[i * image_size + j].real;
				Fimg.data[j].imag = wavgs[i * image_size + j].imag;
				Fweight.data[j] = Fweights[i * image_size + j];
			}

			int my_mutex = exp_iclass % NR_CLASS_MUTEXES;
			pthread_mutex_lock(&global_mutex2[my_mutex]);
			(wsum_model.BPref[exp_iclass]).set2DFourierTransform(Fimg, A, IS_NOT_INV, &Fweight);
			pthread_mutex_unlock(&global_mutex2[my_mutex]);
		}

		CUDA_CPU_TOC("backprojection");

	} // end loop iclass

	CUDA_CPU_TIC("store_post_gpu");

	// Extend norm_correction and sigma2_noise estimation to higher resolutions for all particles
	// Also calculate dLL for each particle and store in metadata
	// loop over all particles inside this ori_particle
	double thr_avg_norm_correction = 0.;
	double thr_sum_dLL = 0., thr_sum_Pmax = 0.;
	for (long int ipart = 0; ipart < mydata.ori_particles[my_ori_particle].particles_id.size(); ipart++)
	{
		long int part_id = mydata.ori_particles[my_ori_particle].particles_id[ipart];
		int group_id = mydata.getGroupId(part_id);

		// If the current images were smaller than the original size, fill the rest of wsum_model.sigma2_noise with the power_class spectrum of the images
		for (int ires = mymodel.current_size/2 + 1; ires < mymodel.ori_size/2 + 1; ires++)
		{
			DIRECT_A1D_ELEM(thr_wsum_sigma2_noise[group_id], ires) += DIRECT_A1D_ELEM(exp_power_imgs[ipart], ires);
			// Also extend the weighted sum of the norm_correction
			exp_wsum_norm_correction[ipart] += DIRECT_A1D_ELEM(exp_power_imgs[ipart], ires);
		}

		// Store norm_correction
		// Multiply by old value because the old norm_correction term was already applied to the image
		if (do_norm_correction)
		{
			double old_norm_correction = DIRECT_A2D_ELEM(exp_metadata, metadata_offset + ipart, METADATA_NORM);
			old_norm_correction /= mymodel.avg_norm_correction;
			// The factor two below is because exp_wsum_norm_correctiom is similar to sigma2_noise, which is the variance for the real/imag components
			// The variance of the total image (on which one normalizes) is twice this value!
			double normcorr = old_norm_correction * sqrt(exp_wsum_norm_correction[ipart] * 2.);
			thr_avg_norm_correction += normcorr;
			// Now set the new norm_correction in the relevant position of exp_metadata
			DIRECT_A2D_ELEM(exp_metadata, metadata_offset + ipart, METADATA_NORM) = normcorr;


			// Print warning for strange norm-correction values
			if (!(iter == 1 && do_firstiter_cc) && DIRECT_A2D_ELEM(exp_metadata, metadata_offset + ipart, METADATA_NORM) > 10.)
			{
				std::cout << " WARNING: norm_correction= "<< DIRECT_A2D_ELEM(exp_metadata, metadata_offset + ipart, METADATA_NORM) << " for particle " << part_id << " in group " << group_id + 1 << "; Are your groups large enough?" << std::endl;
			}

		}

		// Store weighted sums for scale_correction
		if (do_scale_correction)
		{
			// Divide XA by the old scale_correction and AA by the square of that, because was incorporated into Fctf
			exp_wsum_scale_correction_XA[ipart] /= mymodel.scale_correction[group_id];
			exp_wsum_scale_correction_AA[ipart] /= mymodel.scale_correction[group_id] * mymodel.scale_correction[group_id];

			thr_wsum_signal_product_spectra[group_id] += exp_wsum_scale_correction_XA[ipart];
			thr_wsum_reference_power_spectra[group_id] += exp_wsum_scale_correction_AA[ipart];
		}

		// Calculate DLL for each particle
		double logsigma2 = 0.;
		FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(Mresol_fine)
		{
			int ires = DIRECT_MULTIDIM_ELEM(Mresol_fine, n);
			// Note there is no sqrt in the normalisation term because of the 2-dimensionality of the complex-plane
			// Also exclude origin from logsigma2, as this will not be considered in the P-calculations
			if (ires > 0)
				logsigma2 += log( 2. * PI * DIRECT_A1D_ELEM(mymodel.sigma2_noise[group_id], ires));
		}
		if (exp_sum_weight[ipart]==0)
		{
			std::cerr << " part_id= " << part_id << std::endl;
			std::cerr << " ipart= " << ipart << std::endl;
			std::cerr << " exp_min_diff2[ipart]= " << exp_min_diff2[ipart] << std::endl;
			std::cerr << " logsigma2= " << logsigma2 << std::endl;
			int group_id = mydata.getGroupId(part_id);
			std::cerr << " group_id= " << group_id << std::endl;
			std::cerr << " ml_model.scale_correction[group_id]= " << mymodel.scale_correction[group_id] << std::endl;
			std::cerr << " exp_significant_weight[ipart]= " << exp_significant_weight[ipart] << std::endl;
			std::cerr << " exp_max_weight[ipart]= " << exp_max_weight[ipart] << std::endl;
			std::cerr << " ml_model.sigma2_noise[group_id]= " << mymodel.sigma2_noise[group_id] << std::endl;
			REPORT_ERROR("ERROR: exp_sum_weight[ipart]==0");
		}
		double dLL;
		if ((iter==1 && do_firstiter_cc) || do_always_cc)
			dLL = -exp_min_diff2[ipart];
		else
			dLL = log(exp_sum_weight[ipart]) - exp_min_diff2[ipart] - logsigma2;

		// Store dLL of each image in the output array, and keep track of total sum
		DIRECT_A2D_ELEM(exp_metadata, metadata_offset + ipart, METADATA_DLL) = dLL;
		thr_sum_dLL += dLL;

		// Also store sum of Pmax
		thr_sum_Pmax += DIRECT_A2D_ELEM(exp_metadata, metadata_offset + ipart, METADATA_PMAX);

	}

	// Now, inside a global_mutex, update the other weighted sums among all threads
	if (!do_skip_maximization)
	{
		pthread_mutex_lock(&global_mutex);
		for (int n = 0; n < mymodel.nr_groups; n++)
		{
			wsum_model.sigma2_noise[n] += thr_wsum_sigma2_noise[n];
			wsum_model.sumw_group[n] += thr_sumw_group[n];
			if (do_scale_correction)
			{
				wsum_model.wsum_signal_product_spectra[n] += thr_wsum_signal_product_spectra[n];
				wsum_model.wsum_reference_power_spectra[n] += thr_wsum_reference_power_spectra[n];
			}
		}
		for (int n = 0; n < mymodel.nr_classes; n++)
		{
			wsum_model.pdf_class[n] += thr_wsum_pdf_class[n];
			if (mymodel.ref_dim == 2)
			{
				XX(wsum_model.prior_offset_class[n]) += thr_wsum_prior_offsetx_class[n];
				YY(wsum_model.prior_offset_class[n]) += thr_wsum_prior_offsety_class[n];
			}

			if (!(do_skip_align || do_skip_rotate) )
				wsum_model.pdf_direction[n] += thr_wsum_pdf_direction[n];
		}
		wsum_model.sigma2_offset += thr_wsum_sigma2_offset;
		if (do_norm_correction)
			wsum_model.avg_norm_correction += thr_avg_norm_correction;
		wsum_model.LL += thr_sum_dLL;
		wsum_model.ave_Pmax += thr_sum_Pmax;
		pthread_mutex_unlock(&global_mutex);
	} // end if !do_skip_maximization

	CUDA_CPU_TOC("store_post_gpu");
}

//void MlOptimiserCUDA::precalculateModelProjectionsCtfsAndInvSigma2s(bool do_also_unmasked,
//	    int exp_current_image_size, int exp_current_oversampling,
//		std::vector<MultidimArray<Complex > > &model,
//		std::vector<MultidimArray<Complex > > &exp_Fimgs_nomask,
//		std::vector<MultidimArray<double> > &exp_Fctfs,
//		std::vector<MultidimArray<Complex > > &exp_local_Fimgs_shifted,
//		std::vector<MultidimArray<Complex > > &exp_local_Fimgs_shifted_nomask,
//		std::vector<MultidimArray<double> > &exp_local_Fctfs,
//		std::vector<double> &exp_local_sqrtXi2,
//		std::vector<MultidimArray<double> > &exp_local_Minvsigma2s)
//{
//
//}

void MlOptimiserCUDA::precalculateShiftedImagesCtfsAndInvSigma2s(bool do_also_unmasked,
		long int my_ori_particle, int exp_current_image_size, int exp_current_oversampling,
		int exp_itrans_min, int exp_itrans_max,
		std::vector<MultidimArray<Complex > > &exp_Fimgs,
		std::vector<MultidimArray<Complex > > &exp_Fimgs_nomask,
		std::vector<MultidimArray<double> > &exp_Fctfs,
		std::vector<MultidimArray<Complex > > &exp_local_Fimgs_shifted,
		std::vector<MultidimArray<Complex > > &exp_local_Fimgs_shifted_nomask,
		std::vector<MultidimArray<double> > &exp_local_Fctfs,
		std::vector<double> &exp_local_sqrtXi2,
		std::vector<MultidimArray<double> > &exp_local_Minvsigma2s)
{

	int exp_nr_particles = mydata.ori_particles[my_ori_particle].particles_id.size();
	int nr_shifts = (do_shifts_onthefly || do_skip_align) ? exp_nr_particles : exp_nr_particles * sampling.NrTranslationalSamplings(exp_current_oversampling);
	// Don't re-do if nothing has changed....
	bool do_ctf_invsig = (exp_local_Fctfs.size() > 0) ? YSIZE(exp_local_Fctfs[0])  != exp_current_image_size : true; // size has changed
	bool do_masked_shifts = (do_ctf_invsig || nr_shifts != exp_local_Fimgs_shifted.size()); // size or nr_shifts has changed

	// Use pre-sized vectors instead of push_backs!!
	exp_local_Fimgs_shifted.resize(nr_shifts);
	if (do_also_unmasked)
		exp_local_Fimgs_shifted_nomask.resize(nr_shifts);
	exp_local_Minvsigma2s.resize(exp_nr_particles);
	exp_local_Fctfs.resize(exp_nr_particles);
	exp_local_sqrtXi2.resize(exp_nr_particles);

	MultidimArray<Complex > Fimg, Fimg_nomask;
	for (int ipart = 0, my_trans_image = 0; ipart < mydata.ori_particles[my_ori_particle].particles_id.size(); ipart++)
	{
		long int part_id = mydata.ori_particles[my_ori_particle].particles_id[ipart];
		int group_id = mydata.getGroupId(part_id);

		if (do_masked_shifts)
			windowFourierTransform(exp_Fimgs[ipart], Fimg, exp_current_image_size);
		if (do_also_unmasked)
			windowFourierTransform(exp_Fimgs_nomask[ipart], Fimg_nomask, exp_current_image_size);

		if (do_ctf_invsig)
		{
			// Also precalculate the sqrt of the sum of all Xi2
			// Could exp_current_image_size ever be different from mymodel.current_size?
			// Probably therefore do it here rather than in getFourierTransforms
			if ((iter == 1 && do_firstiter_cc) || do_always_cc)
			{
				double sumxi2 = 0.;
				FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(Fimg)
				{
					sumxi2 += norm(DIRECT_MULTIDIM_ELEM(Fimg, n));
				}
				// Normalised cross-correlation coefficient: divide by power of reference (power of image is a constant)
				exp_local_sqrtXi2[ipart] = sqrt(sumxi2);
			}

			// Also store downsized Fctfs
			// In the second pass of the adaptive approach this will have no effect,
			// since then exp_current_image_size will be the same as the size of exp_Fctfs
			windowFourierTransform(exp_Fctfs[ipart], exp_local_Fctfs[ipart], exp_current_image_size);

			// Also prepare Minvsigma2
			if (mymodel.data_dim == 3)
				exp_local_Minvsigma2s[ipart].initZeros(ZSIZE(Fimg), YSIZE(Fimg), XSIZE(Fimg));
			else
				exp_local_Minvsigma2s[ipart].initZeros(YSIZE(Fimg), XSIZE(Fimg));

			int *myMresol = (YSIZE(Fimg) == coarse_size) ? Mresol_coarse.data : Mresol_fine.data;
			// With group_id and relevant size of Fimg, calculate inverse of sigma^2 for relevant parts of Mresol
			FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(exp_local_Minvsigma2s[ipart])
			{
				int ires = *(myMresol + n);
				// Exclude origin (ires==0) from the Probability-calculation
				// This way we are invariant to additive factors
				if (ires > 0)
					DIRECT_MULTIDIM_ELEM(exp_local_Minvsigma2s[ipart], n) = 1. / (sigma2_fudge * DIRECT_A1D_ELEM(mymodel.sigma2_noise[group_id], ires));
			}

		}

		if (do_shifts_onthefly)
		{
			// Store a single, down-sized version of exp_Fimgs[ipart] in exp_local_Fimgs_shifted
			if (do_masked_shifts)
				exp_local_Fimgs_shifted[ipart] = Fimg;
			if (do_also_unmasked)
				exp_local_Fimgs_shifted_nomask[ipart] = Fimg_nomask;
		}
		else
		{
			// Store all translated variants of Fimg
			for (long int itrans = exp_itrans_min; itrans <= exp_itrans_max; itrans++)
			{
				// First get the non-oversampled translations as defined by the sampling object
				std::vector<double > oversampled_translations_x, oversampled_translations_y, oversampled_translations_z;
				sampling.getTranslations(itrans, exp_current_oversampling, oversampled_translations_x,
						oversampled_translations_y, oversampled_translations_z);
				// Then loop over all its oversampled relatives
				for (long int iover_trans = 0; iover_trans < oversampled_translations_x.size(); iover_trans++, my_trans_image++)
				{
					// Shift through phase-shifts in the Fourier transform
					// Note that the shift search range is centered around (exp_old_xoff, exp_old_yoff)
					if (do_masked_shifts)
					{
						exp_local_Fimgs_shifted[my_trans_image].resize(Fimg);
						if (mymodel.data_dim ==2)
							shiftImageInFourierTransform(Fimg, exp_local_Fimgs_shifted[my_trans_image],
									tab_sin, tab_cos, (double)mymodel.ori_size,
									oversampled_translations_x[iover_trans],
									oversampled_translations_y[iover_trans]);
						else
							shiftImageInFourierTransform(Fimg, exp_local_Fimgs_shifted[my_trans_image],
									tab_sin, tab_cos, (double)mymodel.ori_size,
									oversampled_translations_x[iover_trans],
									oversampled_translations_y[iover_trans],
									oversampled_translations_z[iover_trans]);
					}
					if (do_also_unmasked)
					{
						exp_local_Fimgs_shifted_nomask[my_trans_image].resize(Fimg_nomask);
						if (mymodel.data_dim ==2)
							shiftImageInFourierTransform(Fimg_nomask, exp_local_Fimgs_shifted_nomask[my_trans_image],
								tab_sin, tab_cos, (double)mymodel.ori_size,
								oversampled_translations_x[iover_trans],
								oversampled_translations_y[iover_trans]);
						else
							shiftImageInFourierTransform(Fimg_nomask, exp_local_Fimgs_shifted_nomask[my_trans_image],
								tab_sin, tab_cos, (double)mymodel.ori_size,
								oversampled_translations_x[iover_trans],
								oversampled_translations_y[iover_trans],
								oversampled_translations_z[iover_trans]);
					}
				}
			}
		}
	}
}

bool MlOptimiserCUDA::isSignificantAnyParticleAnyTranslation(long int iorient, int exp_itrans_min, int exp_itrans_max, MultidimArray<bool> &exp_Mcoarse_significant)
{

	long int exp_nr_trans = exp_itrans_max - exp_itrans_min + 1;
	for (long int ipart = 0; ipart < YSIZE(exp_Mcoarse_significant); ipart++)
	{
		long int ihidden = iorient * exp_nr_trans;
		for (long int itrans = exp_itrans_min; itrans <= exp_itrans_max; itrans++, ihidden++)
		{
			if (DIRECT_A2D_ELEM(exp_Mcoarse_significant, ipart, ihidden))
				return true;
		}
	}
	return false;

}
