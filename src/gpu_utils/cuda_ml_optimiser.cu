#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <ctime>
#include <iostream>
#include "src/gpu_utils/cuda_ml_optimiser.h"
#include "src/gpu_utils/cuda_utils.cuh"
#include "src/complex.h"
#include <fstream>
#include <hip/hip_runtime.h>
#include "src/parallel.h"
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/extrema.h>

#define MAX_RESOL_SHARED_MEM 32
#define BLOCK_SIZE  128         	// -- Number of threads in a block --
									// This is optimally set as big as possible without
									// its ceil:ed multiple exceeding imagesize by too much.
#define SUM_BLOCK_SIZE 32

#define REF_GROUP_SIZE 3			// -- Number of references to be treated per block --
									// This applies to wavg and reduces global memory
									// accesses roughly proportionally, but scales shared
									// memory usage by allocating
									// ( 6*REF_GROUP_SIZE + 4 ) * BLOCK_SIZE
									// FLOATS.

#define NR_CLASS_MUTEXES 5

texture<FLOAT,hipTextureType3D,hipReadModeElementType> texModel_real;
texture<FLOAT,hipTextureType3D,hipReadModeElementType> texModel_imag;

static pthread_mutex_t global_mutex2[NR_CLASS_MUTEXES] = { PTHREAD_MUTEX_INITIALIZER };
static pthread_mutex_t global_mutex = PTHREAD_MUTEX_INITIALIZER;

/*
 * Maps weights to a decoupled indexing of translations and orientations
 */
inline
void mapWeights(CudaGlobalPtr<FLOAT> &mapped_weights, unsigned orientation_num, unsigned translation_num,
		HealpixSampling &sampling, long int ipart,
		std::vector< long unsigned > &iover_transes, std::vector< long unsigned > &ihiddens,
		std::vector< long unsigned > &iorientclasses, std::vector< long unsigned > &iover_rots,
		MultidimArray<FLOAT> &Mweight, unsigned long current_oversampling, unsigned long nr_trans)
{

	for (long unsigned i = 0; i < orientation_num; i++)
	{
		long unsigned iover_rot = iover_rots[i];
		for (long unsigned j = 0; j < translation_num; j++)
		{
			long unsigned iover_trans = iover_transes[j];
			long unsigned ihidden = iorientclasses[i] * nr_trans + ihiddens[j];
			long unsigned ihidden_over = sampling.getPositionOversampledSamplingPoint(ihidden,
									  current_oversampling, iover_rot, iover_trans);
			mapped_weights[(long unsigned) i * translation_num + j] =
					DIRECT_A2D_ELEM(Mweight, ipart, ihidden_over);
			//Mweight[(i)*(v).xdim+(j)]
		}
	}
}


inline
long unsigned imageTranslation(
		CudaGlobalPtr<FLOAT> &Fimgs_real, CudaGlobalPtr<FLOAT> &Fimgs_imag,
		CudaGlobalPtr<FLOAT> &Fimgs_nomask_real, CudaGlobalPtr<FLOAT> &Fimgs_nomask_imag,
		long int itrans_min, long int itrans_max, int adaptive_oversampling , HealpixSampling &sampling,
		std::vector<double> &oversampled_translations_x, std::vector<double> &oversampled_translations_y, std::vector<double> &oversampled_translations_z,
		unsigned long nr_oversampled_trans, std::vector<MultidimArray<Complex> > &global_fftshifts_ab_current, std::vector<MultidimArray<Complex> > &global_fftshifts_ab2_current,
		MultidimArray<Complex > &local_Fimgs_shifted, MultidimArray<Complex > &local_Fimgs_shifted_nomask,
		std::vector< long unsigned > &iover_transes, std::vector< long unsigned > &itranses, std::vector< long unsigned > &ihiddens,
		unsigned image_size)
{

	long unsigned translation_num(0), ihidden(0);

	for (long int itrans = itrans_min, iitrans = 0; itrans <= itrans_max; itrans++, ihidden++)
	{
		sampling.getTranslations(itrans, adaptive_oversampling,
				oversampled_translations_x, oversampled_translations_y, oversampled_translations_z);

		for (long int iover_trans = 0; iover_trans < nr_oversampled_trans; iover_trans++, iitrans++)
		{
			Complex* myAB;
			myAB = (adaptive_oversampling == 0 ) ? global_fftshifts_ab_current[iitrans].data : global_fftshifts_ab2_current[iitrans].data;


			FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(local_Fimgs_shifted)
			{
				FLOAT a = (*(myAB + n)).real;
				FLOAT b = (*(myAB + n)).imag;

				// Fimg_shift
				FLOAT real = a * (DIRECT_MULTIDIM_ELEM(local_Fimgs_shifted, n)).real
						- b *(DIRECT_MULTIDIM_ELEM(local_Fimgs_shifted, n)).imag;
				FLOAT imag = a * (DIRECT_MULTIDIM_ELEM(local_Fimgs_shifted, n)).imag
						+ b *(DIRECT_MULTIDIM_ELEM(local_Fimgs_shifted, n)).real;
				Fimgs_real[translation_num * image_size + n] = real;
				Fimgs_imag[translation_num * image_size + n] = imag;

				// Fimg_shift_nomask
				real = a * (DIRECT_MULTIDIM_ELEM(local_Fimgs_shifted_nomask, n)).real
						- b *(DIRECT_MULTIDIM_ELEM(local_Fimgs_shifted_nomask, n)).imag;
				imag = a * (DIRECT_MULTIDIM_ELEM(local_Fimgs_shifted_nomask, n)).imag
						+ b *(DIRECT_MULTIDIM_ELEM(local_Fimgs_shifted_nomask, n)).real;
				Fimgs_nomask_real[translation_num * image_size + n] = real;
				Fimgs_nomask_imag[translation_num * image_size + n] = imag;
			}

			translation_num ++;

			ihiddens.push_back(ihidden);
			itranses.push_back(itrans);
			iover_transes.push_back(iover_trans);
		}
	}

	Fimgs_real.size = translation_num * image_size;
	Fimgs_imag.size = translation_num * image_size;

	Fimgs_nomask_real.size = translation_num * image_size;
	Fimgs_nomask_imag.size = translation_num * image_size;

	return translation_num;
}


void generateEulerMatrices(
		FLOAT padding_factor,
		std::vector< double > &rots,
		std::vector< double > &tilts,
		std::vector< double > &psis,
		CudaGlobalPtr<FLOAT> &eulers,
		bool inverse)
{
	double alpha, beta, gamma;
    double ca, sa, cb, sb, cg, sg;
    double cc, cs, sc, ss;

	for (long int i = 0; i < rots.size(); i++)
	{
	    //TODO In a sense we're doing RAD2DEG just to do DEG2RAD here.
	    //The only place the degree value is actually used is in the metadata assignment.

	    alpha = DEG2RAD(rots[i]);
	    beta  = DEG2RAD(tilts[i]);
	    gamma = DEG2RAD(psis[i]);

	    sincos(alpha, &sa, &ca);
	    sincos(beta,  &sb, &cb);
	    sincos(gamma, &sg, &cg);

	    cc = cb * ca;
	    cs = cb * sa;
	    sc = sb * ca;
	    ss = sb * sa;

		if(inverse)
		{
		    eulers[9 * i + 0] = ( cg * cc - sg * sa) * padding_factor; //00
		    eulers[9 * i + 1] = (-sg * cc - cg * sa) * padding_factor; //10
		    eulers[9 * i + 2] = ( sc )               * padding_factor; //20
		    eulers[9 * i + 3] = ( cg * cs + sg * ca) * padding_factor; //01
		    eulers[9 * i + 4] = (-sg * cs + cg * ca) * padding_factor; //11
		    eulers[9 * i + 5] = ( ss )               * padding_factor; //21
		    eulers[9 * i + 6] = (-cg * sb )          * padding_factor; //02
		    eulers[9 * i + 7] = ( sg * sb )          * padding_factor; //12
		    eulers[9 * i + 8] = ( cb )               * padding_factor; //22
		}
		else
		{
		    eulers[9 * i + 0] = ( cg * cc - sg * sa) * padding_factor; //00
		    eulers[9 * i + 1] = ( cg * cs + sg * ca) * padding_factor; //01
		    eulers[9 * i + 2] = (-cg * sb )          * padding_factor; //02
		    eulers[9 * i + 3] = (-sg * cc - cg * sa) * padding_factor; //10
		    eulers[9 * i + 4] = (-sg * cs + cg * ca) * padding_factor; //11
		    eulers[9 * i + 5] = ( sg * sb )          * padding_factor; //12
		    eulers[9 * i + 6] = ( sc )               * padding_factor; //20
		    eulers[9 * i + 7] = ( ss )               * padding_factor; //21
		    eulers[9 * i + 8] = ( cb )               * padding_factor; //22
		}
	}
}


void generateEulerMatrices(
		std::vector< double > &psis,
		CudaGlobalPtr<FLOAT> &eulers,
		bool inverse)
{
    double gamma, c, s;

	for (long int i = 0; i < psis.size(); i++)
	{
	    //TODO In a sense we're doing RAD2DEG just to do DEG2RAD here.
	    //The only place the degree value is actually used is in the metadata assignment.

	    gamma = DEG2RAD(psis[i]);
	    sincos(gamma, &s, &c);

		if(inverse) //Noticed here that inverse actually yields the opposite (Hmmm)
		{
		    eulers[4 * i + 0] =  c; //00
		    eulers[4 * i + 1] = -s; //10
		    eulers[4 * i + 3] =  s; //01
		    eulers[4 * i + 4] =  c; //11
		}
		else
		{
		    eulers[4 * i + 0] =  c; //00
		    eulers[4 * i + 1] =  s; //01
		    eulers[4 * i + 3] = -s; //10
		    eulers[4 * i + 4] =  c; //11
		}
	}
}

long int generateProjectionSetup(
		OptimisationParamters &op,
		SamplingParameters &sp,
		MlOptimiser *baseMLO,
		bool coarse,
		unsigned iclass,
		std::vector< double > &rots,
		std::vector< double > &tilts,
		std::vector< double > &psis,
		std::vector< long unsigned > &iorientclasses,
		std::vector< long unsigned > &iover_rots)
{
	//Local variables
	std::vector< double > oversampled_rot, oversampled_tilt, oversampled_psi;
	long int orientation_num = 0;

	unsigned parts_size(sp.nr_psi * sp.nr_oversampled_rot);
	std::vector< double > rots_parts(parts_size);
	std::vector< double > tilts_parts(parts_size);
	std::vector< double > psis_parts(parts_size);
	std::vector< long unsigned > iorientclasses_parts(parts_size);
	std::vector< long unsigned > iover_rots_parts(parts_size);

	for (long int idir = sp.idir_min, iorient = 0; idir <= sp.idir_max; idir++)
	{
		for (long int ipsi = sp.ipsi_min, ipart = 0; ipsi <= sp.ipsi_max; ipsi++, iorient++)
		{
			long int iorientclass = iclass * sp.nr_dir * sp.nr_psi + iorient;

			// Get prior for this direction and skip calculation if prior==0
			double pdf_orientation;
			if (baseMLO->do_skip_align || baseMLO->do_skip_rotate)
			{
				pdf_orientation = baseMLO->mymodel.pdf_class[iclass];
			}
			else if (baseMLO->mymodel.orientational_prior_mode == NOPRIOR)
			{
				pdf_orientation = DIRECT_MULTIDIM_ELEM(baseMLO->mymodel.pdf_direction[iclass], idir);
			}
			else
			{
				pdf_orientation = op.directions_prior[idir] * op.psi_prior[ipsi];
			}
			// In the first pass, always proceed
			// In the second pass, check whether one of the translations for this orientation of any of the particles had a significant weight in the first pass
			// if so, proceed with projecting the reference in that direction

			bool do_proceed = coarse ? true :
					baseMLO->isSignificantAnyParticleAnyTranslation(iorientclass, sp.itrans_min, sp.itrans_max, op.Mcoarse_significant);

			if (do_proceed && pdf_orientation > 0.)
			{
				// Now get the oversampled (rot, tilt, psi) triplets
				// This will be only the original (rot,tilt,psi) triplet in the first pass (sp.current_oversampling==0)
				baseMLO->sampling.getOrientations(idir, ipsi, sp.current_oversampling, oversampled_rot, oversampled_tilt, oversampled_psi,
						op.pointer_dir_nonzeroprior, op.directions_prior, op.pointer_psi_nonzeroprior, op.psi_prior);

				// Loop over all oversampled orientations (only a single one in the first pass)
				for (long int iover_rot = 0; iover_rot < sp.nr_oversampled_rot; iover_rot++, ipart++)
				{
					iorientclasses_parts[ipart] = iorientclass;
					iover_rots_parts[ipart] = iover_rot;

					rots_parts[ipart] = oversampled_rot[iover_rot];
					tilts_parts[ipart] = oversampled_tilt[iover_rot];
					psis_parts[ipart] = oversampled_psi[iover_rot];

					orientation_num ++;
				}
			}
		}

		//TODO check that the following sort always works out

		if (sp.current_oversampling > 0)
		{
			int oversampling_per_psi = ROUND(std::pow(2., sp.current_oversampling));
			int oversampling_per_dir = ROUND(std::pow(4., sp.current_oversampling));

			//Sort the angles to have coalesced rot/tilt order
			for (unsigned i = 0; i < oversampling_per_dir; i++) //Loop over the perturbed dir pairs
			{
				for (unsigned j = 0; j < sp.nr_psi; j++)
				{
					for (unsigned k = 0; k < oversampling_per_psi; k++) //two psis per perturbed dir pair
					{
						unsigned ij = j*oversampling_per_psi*oversampling_per_dir + i*oversampling_per_psi + k;

						iorientclasses.push_back(iorientclasses_parts[ij]);
						iover_rots.push_back(iover_rots_parts[ij]);

						rots.push_back(rots_parts[ij]);
						tilts.push_back(tilts_parts[ij]);
						psis.push_back(psis_parts[ij]);
					}
				}
			}
		}
		else
		{
			for (unsigned i = 0; i < iorientclasses_parts.size(); i++)
			{
				iorientclasses.push_back(iorientclasses_parts[i]);
				iover_rots.push_back(iover_rots_parts[i]);

				rots.push_back(rots_parts[i]);
				tilts.push_back(tilts_parts[i]);
				psis.push_back(psis_parts[i]);
			}
		}
	}

	return orientation_num;
}


// uses global memory and explicit interpolation = can do double precision.
__global__ void cuda_kernel_projectAllViews_trilin_gloex( FLOAT *g_model_real,
													FLOAT *g_model_imag,
													FLOAT *g_eulers,
													FLOAT *g_Frefs_real,
													FLOAT *g_Frefs_imag,
													int my_r_max,
													int max_r2,
													int min_r2_nn,
													int image_size,
													int orientation_num,
													int XSIZE_img,
													int YSIZE_img,
													int XSIZE_mdl,
													int YSIZE_mdl,
													int STARTINGY_mdl,
													int STARTINGZ_mdl
												   	   )
{
	FLOAT fx, fy, fz, xp, yp, zp;
	int x0, x1, y0, y1, z0, z1; //y2;
	long int r2;
	int pixel;
	int YXSIZE_mdl = YSIZE_mdl * XSIZE_mdl;
	bool is_neg_x;
	CudaComplex d000, d001, d010, d011, d100, d101, d110, d111;
	CudaComplex dx00, dx01, dx10, dx11, dxy0, dxy1, val;
	int bid = blockIdx.y * gridDim.x + blockIdx.x;
	int tid = threadIdx.x;
	// inside the padded 2D orientation grid
	if( bid < orientation_num ) // we only need to make
	{
		unsigned pass_num(ceilf(   ((float)image_size) / (float)BLOCK_SIZE  ));
		long ref_pixel = bid*(image_size);
		for (unsigned pass = 0; pass < pass_num; pass++) // finish a reference proj in each block
		{
			pixel = (pass * BLOCK_SIZE) + tid;
			if(pixel<image_size)
			{
				int x = pixel % XSIZE_img;
				int y = (int)floorf( (float)pixel / (float)XSIZE_img);

				// Dont search beyond square with side max_r
				if (y > my_r_max)
				{
					if (y >= YSIZE_img - my_r_max)
						y = y - YSIZE_img ;
					else
						x=r2;
				}

				r2 = x*x + y*y;
				if (r2 <= max_r2)
				{
					xp = g_eulers[bid*9]   * x + g_eulers[bid*9+1] * y;  // FIXME: xp,yp,zp has has accuracy loss
					yp = g_eulers[bid*9+3] * x + g_eulers[bid*9+4] * y;  // compared to CPU-based projection. This
					zp = g_eulers[bid*9+6] * x + g_eulers[bid*9+7] * y;  // propagates to dx00, dx10, and so on.
					// Only asymmetric half is stored
					if (xp < 0)
					{
						// Get complex conjugated hermitian symmetry pair
						xp = -xp;
						yp = -yp;
						zp = -zp;
						is_neg_x = true;
					}
					else
					{
						is_neg_x = false;
					}
					// Trilinear interpolation (with physical coords)
					// Subtract STARTINGY and STARTINGZ to accelerate access to data (STARTINGX=0)
					// In that way use DIRECT_A3D_ELEM, rather than A3D_ELEM
					x0 = floorf(xp);
					fx = xp - x0;
					x1 = x0 + 1;
					xp = fx + x0;


					y0 = floorf(yp);
					fy = yp - y0;
					y0 -=  STARTINGY_mdl;
					y1 = y0 + 1;

					z0 = floorf(zp);
					fz = zp - z0;
					z0 -= STARTINGZ_mdl;
					z1 = z0 + 1;

					d000.real = g_model_real[z0*YXSIZE_mdl+y0*XSIZE_mdl+x0];
					d001.real = g_model_real[z0*YXSIZE_mdl+y0*XSIZE_mdl+x1];
					d010.real = g_model_real[z0*YXSIZE_mdl+y1*XSIZE_mdl+x0];
					d011.real = g_model_real[z0*YXSIZE_mdl+y1*XSIZE_mdl+x1];
					d100.real = g_model_real[z1*YXSIZE_mdl+y0*XSIZE_mdl+x0];
					d101.real = g_model_real[z1*YXSIZE_mdl+y0*XSIZE_mdl+x1];
					d110.real = g_model_real[z1*YXSIZE_mdl+y1*XSIZE_mdl+x0];
					d111.real = g_model_real[z1*YXSIZE_mdl+y1*XSIZE_mdl+x1];

					d000.imag = g_model_imag[z0*YXSIZE_mdl+y0*XSIZE_mdl+x0];
					d001.imag = g_model_imag[z0*YXSIZE_mdl+y0*XSIZE_mdl+x1];
					d010.imag = g_model_imag[z0*YXSIZE_mdl+y1*XSIZE_mdl+x0];
					d011.imag = g_model_imag[z0*YXSIZE_mdl+y1*XSIZE_mdl+x1];
					d100.imag = g_model_imag[z1*YXSIZE_mdl+y0*XSIZE_mdl+x0];
					d101.imag = g_model_imag[z1*YXSIZE_mdl+y0*XSIZE_mdl+x1];
					d110.imag = g_model_imag[z1*YXSIZE_mdl+y1*XSIZE_mdl+x0];
					d111.imag = g_model_imag[z1*YXSIZE_mdl+y1*XSIZE_mdl+x1];

					// Set the interpolated value in the 2D output array
					dx00 = d000 + (d001 - d000)*fx;
					dx01 = d100 + (d101 - d100)*fx;
					dx10 = d010 + (d011 - d010)*fx;
					dx11 = d110 + (d111 - d110)*fx;

					dxy0 = dx00 + (dx10 - dx00)*fy;
					dxy1 = dx01 + (dx11 - dx01)*fy;

					val = dxy0 + (dxy1 - dxy0)*fz;

					if (is_neg_x)
					{
						val.imag = -val.imag;
					}

				}
				else
				{
					val.real=0.0f;
					val.imag=0.0f;
				}
				g_Frefs_real[ref_pixel+ pixel] = val.real;
				g_Frefs_imag[ref_pixel+ pixel] = val.imag;
			}
		}
	}
}

#if !defined(CUDA_DOUBLE_PRECISION)
// uses texture memory and explicit interpolation = requires float precision.
__global__ void cuda_kernel_projectAllViews_trilin_texex( FLOAT *g_eulers,
														 FLOAT *g_Frefs_real,
														 FLOAT *g_Frefs_imag,
														 int my_r_max,
														 int max_r2,
														 int min_r2_nn,
														 int image_size,
														 int orientation_num,
														 int XSIZE_img,
														 int YSIZE_img,
														 int STARTINGY_mdl,
														 int STARTINGZ_mdl)
{
	FLOAT fx, fy, fz, xp, yp, zp;
	int x0, x1, y0, y1, z0, z1; //y2;
	long int r2;
	int pixel;
	bool is_neg_x;
	CudaComplex d000, d001, d010, d011, d100, d101, d110, d111;
	CudaComplex dx00, dx01, dx10, dx11, dxy0, dxy1, val;
	int bid = blockIdx.y * gridDim.x + blockIdx.x;
	int tid = threadIdx.x;
	// inside the padded 2D orientation grid
	if( bid < orientation_num ) // we only need to make
	{
		unsigned pass_num(ceilf(   ((float)image_size) / (float)BLOCK_SIZE  ));
		long ref_pixel = bid*(image_size);
		for (unsigned pass = 0; pass < pass_num; pass++) // finish a reference proj in each block
		{
			pixel = (pass * BLOCK_SIZE) + tid;
			if(pixel<image_size)
			{
				int x = pixel % XSIZE_img;
				int y = (int)floorf( (float)pixel / (float)XSIZE_img);

				// Dont search beyond square with side max_r
				if (y > my_r_max)
				{
					if (y >= YSIZE_img - my_r_max)
						y = y - YSIZE_img ;
					else
						x=r2;
				}

				r2 = x*x + y*y;
				if (r2 <= max_r2)
				{
					xp = g_eulers[bid*9]   * x + g_eulers[bid*9+1] * y;  // FIXME: xp,yp,zp has has accuracy loss
					yp = g_eulers[bid*9+3] * x + g_eulers[bid*9+4] * y;  // compared to CPU-based projection. This
					zp = g_eulers[bid*9+6] * x + g_eulers[bid*9+7] * y;  // propagates to dx00, dx10, and so on.
					// Only asymmetric half is stored
					if (xp < 0)
					{
						// Get complex conjugated hermitian symmetry pair
						xp = -xp;
						yp = -yp;
						zp = -zp;
						is_neg_x = true;
					}
					else
					{
						is_neg_x = false;
					}
					// Trilinear interpolation (with physical coords)
					// Subtract STARTINGY and STARTINGZ to accelerate access to data (STARTINGX=0)
					// In that way use DIRECT_A3D_ELEM, rather than A3D_ELEM
					x0 = floorf(xp);
					fx = xp - x0;
					x1 = x0 + 1;
					xp = fx + x0;


					y0 = floorf(yp);
					fy = yp - y0;
					y0 -=  STARTINGY_mdl;
					y1 = y0 + 1;
					yp -= STARTINGY_mdl;

					z0 = floorf(zp);
					fz = zp - z0;
					z0 -= STARTINGZ_mdl;
					z1 = z0 + 1;
					zp -= STARTINGZ_mdl;

					d000.real = tex3D(texModel_real,x0+0.5f,y0+0.5f,z0+0.5f);
					d001.real = tex3D(texModel_real,x1+0.5f,y0+0.5f,z0+0.5f);
					d010.real = tex3D(texModel_real,x0+0.5f,y1+0.5f,z0+0.5f);
					d011.real = tex3D(texModel_real,x1+0.5f,y1+0.5f,z0+0.5f);
					d100.real = tex3D(texModel_real,x0+0.5f,y0+0.5f,z1+0.5f);
					d101.real = tex3D(texModel_real,x1+0.5f,y0+0.5f,z1+0.5f);
					d110.real = tex3D(texModel_real,x0+0.5f,y1+0.5f,z1+0.5f);
					d111.real = tex3D(texModel_real,x1+0.5f,y1+0.5f,z1+0.5f);

					d000.imag = tex3D(texModel_imag,x0+0.5f,y0+0.5f,z0+0.5f);
					d001.imag = tex3D(texModel_imag,x1+0.5f,y0+0.5f,z0+0.5f);
					d010.imag = tex3D(texModel_imag,x0+0.5f,y1+0.5f,z0+0.5f);
					d011.imag = tex3D(texModel_imag,x1+0.5f,y1+0.5f,z0+0.5f);
					d100.imag = tex3D(texModel_imag,x0+0.5f,y0+0.5f,z1+0.5f);
					d101.imag = tex3D(texModel_imag,x1+0.5f,y0+0.5f,z1+0.5f);
					d110.imag = tex3D(texModel_imag,x0+0.5f,y1+0.5f,z1+0.5f);
					d111.imag = tex3D(texModel_imag,x1+0.5f,y1+0.5f,z1+0.5f);

					// Set the interpolated value in the 2D output array
					dx00 = d000 + (d001 - d000)*fx;
					dx01 = d100 + (d101 - d100)*fx;
					dx10 = d010 + (d011 - d010)*fx;
					dx11 = d110 + (d111 - d110)*fx;

					dxy0 = dx00 + (dx10 - dx00)*fy;
					dxy1 = dx01 + (dx11 - dx01)*fy;

					val = dxy0 + (dxy1 - dxy0)*fz;

					if (is_neg_x)
					{
						val.imag = -val.imag;
					}

				}
				else
				{
					val.real=0.0f;
					val.imag=0.0f;
				}
				g_Frefs_real[ref_pixel+ pixel] = val.real;
				g_Frefs_imag[ref_pixel+ pixel] = val.imag;
			}
		}
	}
}

// uses texture memory and implicit (texture) interpolation = requires float precision.
__global__ void cuda_kernel_projectAllViews_trilin_texim( FLOAT *g_eulers,
														 FLOAT *g_Frefs_real,
														 FLOAT *g_Frefs_imag,
														 int my_r_max,
														 int max_r2,
														 int min_r2_nn,
														 int image_size,
														 int orientation_num,
														 int XSIZE_img,
														 int YSIZE_img,
														 int STARTINGY_mdl,
														 int STARTINGZ_mdl)
{
	FLOAT xp, yp, zp;
	long int r2;
	int pixel;
	bool is_neg_x;
	CudaComplex val;
	int bid = blockIdx.y * gridDim.x + blockIdx.x;
	int tid = threadIdx.x;
	// inside the padded 2D orientation grid
	if( bid < orientation_num )
	{
		unsigned pass_num(ceilf(   ((float)image_size) / (float)BLOCK_SIZE  ));
		long ref_pixel = bid*(image_size);
		for (unsigned pass = 0; pass < pass_num; pass++) // finish a reference proj in each block
		{
			pixel = (pass * BLOCK_SIZE) + tid;
			if(pixel<image_size)
			{
				int x = pixel % XSIZE_img;
				int y = (int)floorf( (float)pixel / (float)XSIZE_img);

				// Dont search beyond square with side max_r
				if (y > my_r_max)
				{
					if (y >= YSIZE_img - my_r_max)
						y = y - YSIZE_img ;
					else
						x=r2;
				}

				r2 = x*x + y*y;
				if (r2 <= max_r2)
				{
					xp = __ldg(&g_eulers[bid*9])   * x + __ldg(&g_eulers[bid*9+1]) * y;  // FIXME: xp,yp,zp has has accuracy loss
					yp = __ldg(&g_eulers[bid*9+3]) * x + __ldg(&g_eulers[bid*9+4]) * y;  // compared to CPU-based projection. This
					zp = __ldg(&g_eulers[bid*9+6]) * x + __ldg(&g_eulers[bid*9+7]) * y;  // propagates to dx00, dx10, and so on.
					// Only asymmetric half is stored
					if (xp < 0)
					{
						// Get complex conjugated hermitian symmetry pair
						xp = -xp;
						yp = -yp;
						zp = -zp;
						is_neg_x = true;
					}
					else
					{
						is_neg_x = false;
					}
					yp -= STARTINGY_mdl;
					zp -= STARTINGZ_mdl;

					val.real=tex3D(texModel_real,xp+0.5f,yp+0.5f,zp+0.5f);
					val.imag=tex3D(texModel_imag,xp+0.5f,yp+0.5f,zp+0.5f);


					if (is_neg_x)
					{
						val.imag = -val.imag;
					}

				}
				else
				{
					val.real=0.0f;
					val.imag=0.0f;
				}
				g_Frefs_real[ref_pixel+ pixel] = val.real;
				g_Frefs_imag[ref_pixel+ pixel] = val.imag;
			}
		}
	}
}

// ===================================================
//     Combined Projection+Difference kernels are
// ===================================================
//	-PAV_TTI_D2      Texture Implicit   - single prec. only
//	-PAV_TTE_D2      Texture Explicit   - single prec. only (?)
//	-PAV_TGE_D2      Global  Explicit
//
//   PAV  =   Project All Views
//
//   FIXME: All should be available with suffix _CC  (cross-correlation algorithm)

__global__ void cuda_kernel_PAV_TTI_D2( FLOAT *g_eulers,
		                                FLOAT *g_imgs_real,
		                                FLOAT *g_imgs_imag,
										FLOAT *g_Minvsigma2,
										FLOAT *g_diff2s,
										int image_size,
										FLOAT sum_init,
										int orientation_num,
										int translation_num,
										int significant_num,
										unsigned long *d_rotidx,
										unsigned long *d_transidx,
										unsigned long *d_ihidden_overs,
										int my_r_max,
										int max_r2,
										int min_r2_nn,
										long int img_x,
										long int img_y,
										long int mdl_x,
										long int mdl_y,
										long int mdl_z,
										long int mdl_init_y,
										long int mdl_init_z
										)
{
	int bid = blockIdx.y * gridDim.x + blockIdx.x;
	int tid = threadIdx.x;

	FLOAT xp, yp, zp;
	long int r2;
	int pixel;
	bool is_neg_x;
	FLOAT ref_real;
	FLOAT ref_imag;

	// inside the padded 2D orientation grid
	if( bid < significant_num ) // we only need to make
	{
		__shared__ FLOAT s[BLOCK_SIZE];
		s[tid] = 0.0f;

		// index of comparison
		unsigned long int ix=d_rotidx[bid];
		unsigned long int iy=d_transidx[bid];

		unsigned pass_num(ceilf(   ((float)image_size) / (float)BLOCK_SIZE  ));
		unsigned long img_start(iy * image_size);
		unsigned long img_pixel_idx;

		for (unsigned pass = 0; pass < pass_num; pass++) // finish a reference proj in each block
		{
			pixel = (pass * BLOCK_SIZE) + tid;
			if(pixel<image_size)
			{
				int x = pixel % img_x;
				int y = (int)floorf( (float)pixel / (float)img_x);
				img_pixel_idx = img_start + pixel;

				// Dont search beyond square with side max_r
				if (y > my_r_max)
				{
					if (y >= img_y - my_r_max)
						y = y - img_y ;
					else
						x=r2;
				}

				r2 = x*x + y*y;
				if (r2 <= max_r2)
				{
					xp = __ldg(&g_eulers[ix*9])   * x + __ldg(&g_eulers[ix*9+1]) * y;  // FIXME: xp,yp,zp has has accuracy loss
					yp = __ldg(&g_eulers[ix*9+3]) * x + __ldg(&g_eulers[ix*9+4]) * y;  // compared to CPU-based projection. This
					zp = __ldg(&g_eulers[ix*9+6]) * x + __ldg(&g_eulers[ix*9+7]) * y;  // propagates to dx00, dx10, and so on.
					// Only asymmetric half is stored
					if (xp < 0)
					{
						// Get complex conjugated hermitian symmetry pair
						xp = -xp;
						yp = -yp;
						zp = -zp;
						is_neg_x = true;
					}
					else
					{
						is_neg_x = false;
					}
					yp -= mdl_init_y;
					zp -= mdl_init_z;

					ref_real=tex3D(texModel_real,xp+0.5f,yp+0.5f,zp+0.5f);
					ref_imag=tex3D(texModel_imag,xp+0.5f,yp+0.5f,zp+0.5f);

//					printf("%i, %i", x,y);
//					printf("%f, %f,%f", xp,yp,zp);
					if (is_neg_x)
					{
						ref_imag = -ref_imag;
					}
				}
				else
				{
					ref_real=0.0f;
					ref_imag=0.0f;
				}
				FLOAT diff_real =  ref_real - __ldg(&g_imgs_real[img_pixel_idx]); // TODO  Put g_img_* in texture (in such a way that fetching of next image might hit in cache)
				FLOAT diff_imag =  ref_imag - __ldg(&g_imgs_imag[img_pixel_idx]);

				s[tid] += (diff_real * diff_real + diff_imag * diff_imag) * 0.5f * __ldg(&g_Minvsigma2[pixel]);
//				printf(" diffs = %f, %f \n",ref_real,img_pixel_idx);
//				printf(" diffs = %i, %i ,%i \n",x,y);
			}
		}
		__syncthreads();

		for(int j=(BLOCK_SIZE/2); j>0; j>>=1)
		{
			if(tid<j)
			{
				s[tid] += s[tid+j];
			}
			__syncthreads();
		}
		if (tid == 0)
		{
			g_diff2s[ix * translation_num + iy] = s[0]+sum_init;
		}
	}
}
#endif

void generateModelProjections(
		CudaGlobalPtr<FLOAT > &model_real,
		CudaGlobalPtr<FLOAT > &model_imag,
		CudaGlobalPtr<FLOAT> &Frefs_real,
		CudaGlobalPtr<FLOAT> &Frefs_imag,
		CudaGlobalPtr<FLOAT> &eulers,
		long unsigned orientation_num,
		unsigned image_size,
		unsigned max_r,
		unsigned img_x,
		unsigned img_y,
		unsigned mdl_x,
		unsigned mdl_y,
		unsigned mdl_z,
		unsigned mdl_init_y,
		unsigned mdl_init_z)
{

	int max_r2 = max_r * max_r;
	int min_r2_nn = 0; // r_min_nn * r_min_nn;  //FIXME add nn-algorithm

	/*===========================
	 *      TEXTURE STUFF
	 * ==========================*/
	// TODO Use bindless textures to reduce some of this clutter

	hipArray*        modelArray_real;
	hipArray* 		  modelArray_imag;
	hipExtent        volumeSize = make_hipExtent(mdl_x, mdl_y, mdl_z);
	// create channel to describe data type (bits,bits,bits,bits,type)
	// TODO model should carry real & imag in separate channels of the same texture
	hipChannelFormatDesc channel = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

	//allocate device memory for cuda 3D array
	hipMalloc3DArray(&modelArray_real, &channel, volumeSize);
	hipMalloc3DArray(&modelArray_imag, &channel, volumeSize);

	//set cuda array copy parameters to be supplied to copy-command
	hipMemcpy3DParms copyParams = {0};
	copyParams.extent   = volumeSize;

	copyParams.kind     = hipMemcpyHostToDevice;
	copyParams.dstArray = modelArray_real;
	copyParams.srcPtr   = make_hipPitchedPtr(model_real.h_ptr,volumeSize.width*sizeof(FLOAT), volumeSize.height, volumeSize.depth);
	hipMemcpy3D(&copyParams);
	copyParams.kind     = hipMemcpyHostToDevice;
	copyParams.dstArray = modelArray_imag;
	copyParams.srcPtr   = make_hipPitchedPtr(model_imag.h_ptr,volumeSize.width*sizeof(FLOAT), volumeSize.height, volumeSize.depth);
	hipMemcpy3D(&copyParams);

	//set texture filter mode property, use hipFilterModePoint or hipFilterModeLinear
	texModel_real.normalized = false;
	texModel_real.filterMode = hipFilterModeLinear;
	texModel_imag.normalized = false;
	texModel_imag.filterMode = hipFilterModeLinear;

	//set texture address mode property, use hipAddressModeClamp , -Border, -Wrap, or -Mirror
	for(int n=0; n<3; n++)
	{
		texModel_real.addressMode[n]=hipAddressModeClamp;
		texModel_imag.addressMode[n]=hipAddressModeClamp;
	}

    //bind texture reference with cuda array
	hipBindTextureToArray(texModel_real, modelArray_real, channel);
	hipBindTextureToArray(texModel_imag, modelArray_imag, channel);

	Frefs_real.size = orientation_num * image_size;
	Frefs_real.device_alloc();
	Frefs_imag.size = orientation_num * image_size;
	Frefs_imag.device_alloc();

	unsigned int orient1, orient2;
	if(orientation_num>65535)
	{
		orient1 = ceil(sqrt(orientation_num));
		orient2 = orient1;
	}
	else
	{
		orient1 = orientation_num;
		orient2 = 1;
	}

	dim3 block_dim(orient1,orient2);
	std::cerr << "using block dimensions " << orient1 << "," << orient2 <<  std::endl;

#if !defined(CUDA_DOUBLE_PRECISION) && defined(USE_TEXINTERP)
	// we CAN use read-associated interpolation (fast, inaccurate)...
	cuda_kernel_projectAllViews_trilin_texim<<<block_dim,BLOCK_SIZE>>>(
															~eulers,
															~Frefs_real,
															~Frefs_imag,
															max_r,
															max_r2,
															min_r2_nn,
															image_size,
															orientation_num,
															img_x,
															img_y,
															mdl_init_y,
															mdl_init_z);
	hipFreeArray(modelArray_real);
	hipFreeArray(modelArray_imag);
#elif !defined(CUDA_DOUBLE_PRECISION)	// ...or explicit interpolation (slow, accurate)
	cuda_kernel_projectAllViews_trilin_texex<<<block_dim,BLOCK_SIZE>>>(
															~eulers,
															~Frefs_real,
															~Frefs_imag,
															max_r,
															max_r2,
															min_r2_nn,
															image_size,
															orientation_num,
															img_x,
															img_y,
															mdl_init_y,
															mdl_init_z);

	hipFreeArray(modelArray_real);
	hipFreeArray(modelArray_imag);
#else // under double precision, texture won't work.
	model_real.device_alloc();
    model_real.cp_to_device();
 	model_imag.device_alloc();
    model_imag.cp_to_device();
	cuda_kernel_projectAllViews_trilin_gloex<<<block_dim,BLOCK_SIZE>>>(
															~model_real,
															~model_imag,
															~eulers,
															~Frefs_real,
															~Frefs_imag,
															max_r,
															max_r2,
															min_r2_nn,
															image_size,
															orientation_num,
															img_x,
															img_y,
															mdl_x,
															mdl_y,
 															mdl_init_y,
 															mdl_init_z);

	model_real.free_device();
	model_imag.free_device();
#endif

	//unbind texture reference to free resource



}

__global__ void cuda_kernel_diff2(	FLOAT *g_refs_real,
									FLOAT *g_refs_imag,
									FLOAT *g_imgs_real,
									FLOAT *g_imgs_imag,
									FLOAT *g_Minvsigma2, FLOAT *g_diff2s,
									unsigned img_size, FLOAT sum_init,
									unsigned long significant_num,
									unsigned long translation_num,
									unsigned long *d_rotidx,
									unsigned long *d_transidx,
									unsigned long *d_ihidden_overs // TODO use it to map in here, get rid of collect_data_1
									)
{
	// blockid
	int ex = blockIdx.y * gridDim.x + blockIdx.x;
    int tid = threadIdx.x;

	// inside the padded 2D orientation grid
	if( ex < significant_num )
	{
		// index of comparison
		unsigned long int ix=d_rotidx[ex];
		unsigned long int iy=d_transidx[ex];

		__shared__ FLOAT s[BLOCK_SIZE];
		s[tid] = 0.0f;

		unsigned pass_num(ceilf((float)img_size/(float)BLOCK_SIZE)), pixel;

		unsigned long ref_start(ix * img_size);
		unsigned long img_start(iy * img_size);
		unsigned long ref_pixel_idx;
		unsigned long img_pixel_idx;

		for (unsigned pass = 0; pass < pass_num; pass ++)
		{
			pixel = pass * BLOCK_SIZE + tid;

			if (pixel < img_size) //Is inside image
			{
				ref_pixel_idx = ref_start + pixel;
				img_pixel_idx = img_start + pixel;

				FLOAT diff_real = __ldg(&g_refs_real[ref_pixel_idx]) - __ldg(&g_imgs_real[img_pixel_idx]); // TODO  Put g_img_* in texture (in such a way that fetching of next image might hit in cache)
				FLOAT diff_imag = __ldg(&g_refs_imag[ref_pixel_idx]) - __ldg(&g_imgs_imag[img_pixel_idx]);

				s[tid] += (diff_real * diff_real + diff_imag * diff_imag) * 0.5f * __ldg(&g_Minvsigma2[pixel]);
			}
		}
		__syncthreads();

		for(int j=(BLOCK_SIZE/2); j>0; j>>=1)
		{
			if(tid<j)
			{
				s[tid] += s[tid+j];
			}
			__syncthreads();
		}
//		if (threadIdx.x*ex == 0)
		{
			g_diff2s[ix * translation_num + iy] = s[0]+sum_init;
		}
		// -------------------------------------------------------------------------
	}
}

__global__ void cuda_kernel_cc_diff2(	FLOAT *g_refs_real,
										FLOAT *g_refs_imag,
										FLOAT *g_imgs_real,
										FLOAT *g_imgs_imag,
										FLOAT *g_Minvsigma2, FLOAT *g_diff2s,
										unsigned img_size, FLOAT exp_local_sqrtXi2,
										unsigned long significant_num,
										unsigned long translation_num,
										unsigned long *d_rotidx,
										unsigned long *d_transidx)
{
	// blockid
	int ex = blockIdx.y * gridDim.x + blockIdx.x;
	// inside the padded 2D orientation grid
	if( ex < significant_num )
	{
		// index of comparison
		unsigned long int ix=d_rotidx[ex];
		unsigned long int iy=d_transidx[ex];
		__shared__ double    s[BLOCK_SIZE];
		__shared__ double norm[BLOCK_SIZE];
		s[threadIdx.x] = 0;
		unsigned pass_num(ceilf((float)img_size/(float)BLOCK_SIZE));
		unsigned long pixel,
		ref_start(ix * img_size),
		img_start(iy * img_size);
		unsigned long ref_pixel_idx;
		unsigned long img_pixel_idx;
		for (unsigned pass = 0; pass < pass_num; pass ++)
		{
			pixel = pass * BLOCK_SIZE + threadIdx.x;

			if (pixel < img_size) //Is inside image
			{
				ref_pixel_idx = ref_start + pixel;
				img_pixel_idx = img_start + pixel;

				double diff_real = g_refs_real[ref_pixel_idx] * g_imgs_real[img_pixel_idx];
				double diff_imag = g_refs_imag[ref_pixel_idx] * g_imgs_imag[img_pixel_idx];

				double nR = g_refs_real[ref_pixel_idx]*g_refs_real[ref_pixel_idx];
				double nI = g_refs_imag[ref_pixel_idx]*g_refs_imag[ref_pixel_idx];

				s[threadIdx.x] -= (diff_real + diff_imag);
				norm[threadIdx.x] += nR+nI;
			}
		}
		// -------------------------------------------------------------------------
		__syncthreads();
		int trads = 32;
		int itr = BLOCK_SIZE/trads;
		if(threadIdx.x<trads)
		{
			for(int i=1; i<itr; i++)
			{
				s[threadIdx.x] += s[i*trads + threadIdx.x];
				norm[threadIdx.x] += norm[i*trads + threadIdx.x];
			}
		}
		for(int j=(trads/2); j>0; j/=2)
		{
			if(threadIdx.x<j)
			{
				s[threadIdx.x] += s[threadIdx.x+j];
				norm[threadIdx.x] += norm[threadIdx.x+j];
			}
		}
		__syncthreads();
		// -------------------------------------------------------------------------
		g_diff2s[ix * translation_num + iy] = s[0]/(sqrt(norm[0])*exp_local_sqrtXi2);
	}
}

void runDifferenceKernel(CudaGlobalPtr<FLOAT > &gpuMinvsigma2,
		CudaGlobalPtr<FLOAT > &Fimgs_real,
		CudaGlobalPtr<FLOAT > &Fimgs_imag,
		CudaGlobalPtr<FLOAT > &Frefs_real,
		CudaGlobalPtr<FLOAT > &Frefs_imag,
		CudaGlobalPtr<long unsigned > &rotidx,
		CudaGlobalPtr<long unsigned > &transidx,
		CudaGlobalPtr<long unsigned > &ihidden_overs,
		OptimisationParamters &op,
		MlOptimiser *baseMLO,
		long unsigned translation_num,
		long unsigned orientation_num,
		long unsigned significant_num,
		unsigned image_size,
		int ipart,
		int group_id,
		CudaGlobalPtr<FLOAT > &diff2s
		)
{
	/*====================================
	   Initiate Particle Related On GPU
	======================================*/
	CUDA_CPU_TIC("kernel_init_1");
	// Since we hijack Minvsigma to carry a bit more info into the GPU-kernel
	// we need to make a modified copy, since the global object shouldn't be
	// changed
	FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(op.local_Fimgs_shifted[ipart])
	{
		gpuMinvsigma2[n] = *(op.local_Minvsigma2s[ipart].data + n );
	}

	if (baseMLO->do_ctf_correction && baseMLO->refs_are_ctf_corrected)
	{
		FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(op.local_Fimgs_shifted[ipart])
		{
			gpuMinvsigma2[n] *= (DIRECT_MULTIDIM_ELEM(op.local_Fctfs[ipart], n)*DIRECT_MULTIDIM_ELEM(op.local_Fctfs[ipart], n));
		}
	}
	// TODO :    + Assure accuracy with the implemented GPU-based ctf-scaling
	//           + Make setting of myscale robust between here and above.
	//  (scale_correction turns off by default with only one group: ml_optimiser-line 1067,
	//   meaning small-scale test will probably not catch this malfunctioning when/if it breaks.)
	if (baseMLO->do_scale_correction)
	{
		FLOAT myscale = baseMLO->mymodel.scale_correction[group_id];
		FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(op.local_Fimgs_shifted[ipart])
		{
			gpuMinvsigma2[n] *= (myscale*myscale);
		}
	}

	gpuMinvsigma2.cp_to_device();

	Fimgs_real.size = translation_num * image_size;
	Fimgs_real.device_alloc();
	Fimgs_real.cp_to_device();
	Fimgs_imag.size = translation_num * image_size;
	Fimgs_imag.device_alloc();
	Fimgs_imag.cp_to_device();
	rotidx.size = significant_num;
	rotidx.device_alloc();
	rotidx.cp_to_device();
	transidx.size = significant_num;
	transidx.device_alloc();
	transidx.cp_to_device();
	ihidden_overs.size = significant_num;
	ihidden_overs.device_alloc();
	ihidden_overs.cp_to_device();

	/*====================================
				Kernel Calls
	======================================*/
	unsigned orient1, orient2;

	if(significant_num>65535)
	{
		orient1 = ceil(sqrt(significant_num));
		orient2 = orient1;
	}
	else
	{
		orient1 = significant_num;
		orient2 = 1;
	}
	dim3 block_dim(orient1,orient2);

	CUDA_CPU_TOC("kernel_init_1");
	CUDA_GPU_TIC("kernel_diff_noproj");
	// Could be used to automate __ldg() fallback runtime within cuda_kernel_diff2.
//				hipDeviceProp_t dP;
//				hipGetDeviceProperties(&dP, 0);
//				printf("-arch=sm_%d%d\n", dP.major, dP.minor);

	if ((baseMLO->iter == 1 && baseMLO->do_firstiter_cc) || baseMLO->do_always_cc) // do cross-correlation instead of diff
	{
		cuda_kernel_cc_diff2<<<block_dim,BLOCK_SIZE>>>(~Frefs_real, ~Frefs_imag, ~Fimgs_real, ~Fimgs_imag, ~gpuMinvsigma2,  ~diff2s,
														image_size, op.highres_Xi2_imgs[ipart],
														significant_num,
														translation_num,
														~rotidx,
														~transidx);
	}
	else
	{
		cuda_kernel_diff2<<<block_dim,BLOCK_SIZE>>>(~Frefs_real, ~Frefs_imag, ~Fimgs_real, ~Fimgs_imag, ~gpuMinvsigma2, ~diff2s,
													image_size, op.highres_Xi2_imgs[ipart] / 2.,
													significant_num,
													translation_num,
													~rotidx,
													~transidx,
													~ihidden_overs);
	}
	CUDA_GPU_TAC("kernel_diff_noproj");
	HANDLE_ERROR(hipDeviceSynchronize()); //TODO Apparently this is not required here
	CUDA_GPU_TOC("kernel_diff_noproj");
	size_t avail;
	size_t total;
	hipMemGetInfo( &avail, &total );
	float used = 100*((float)(total - avail)/(float)total);
	std::cerr << "Device memory used @ diff2: " << used << "%" << std::endl;
}

#if !defined(CUDA_DOUBLE_PRECISION)
void runProjAndDifferenceKernel(
		CudaGlobalPtr<FLOAT > &model_real,
		CudaGlobalPtr<FLOAT > &model_imag,
		CudaGlobalPtr<FLOAT > &gpuMinvsigma2,
		CudaGlobalPtr<FLOAT> &Fimgs_real,
		CudaGlobalPtr<FLOAT> &Fimgs_imag,
		CudaGlobalPtr<FLOAT> &eulers,
		CudaGlobalPtr<long unsigned> &rotidx,
		CudaGlobalPtr<long unsigned> &transidx,
		CudaGlobalPtr<long unsigned> &ihidden_overs,
		CudaGlobalPtr<FLOAT> &diff2s,
		OptimisationParamters op,
		MlOptimiser *baseMLO,
		long unsigned orientation_num,
		long unsigned translation_num,
		long unsigned significant_num,
		unsigned image_size,
		unsigned max_r,
		int ipart,
		int group_id,
		int exp_iclass)
{

	CUDA_CPU_TIC("kernel_init_1");
	int max_r2 = max_r * max_r;
	int min_r2_nn = 0; // r_min_nn * r_min_nn;  //FIXME add nn-algorithm

	/*===========================
	 *      TEXTURE STUFF
	 * ==========================*/
	// TODO Use bindless textures to reduce some of this clutter

	hipArray*        modelArray_real;
	hipArray* 		  modelArray_imag;
	hipExtent        volumeSize = make_hipExtent(baseMLO->mymodel.PPref[exp_iclass].data.xdim,
												   baseMLO->mymodel.PPref[exp_iclass].data.ydim,
												   baseMLO->mymodel.PPref[exp_iclass].data.zdim);
	// create channel to describe data type (bits,bits,bits,bits,type)
	// TODO model should carry real & imag in separate channels of the same texture
	hipChannelFormatDesc channel = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

	//allocate device memory for cuda 3D array
	hipMalloc3DArray(&modelArray_real, &channel, volumeSize);
	hipMalloc3DArray(&modelArray_imag, &channel, volumeSize);

	//set cuda array copy parameters to be supplied to copy-command
	hipMemcpy3DParms copyParams = {0};
	copyParams.extent   = volumeSize;

	copyParams.kind     = hipMemcpyHostToDevice;
	copyParams.dstArray = modelArray_real;
	copyParams.srcPtr   = make_hipPitchedPtr(model_real.h_ptr,volumeSize.width*sizeof(FLOAT), volumeSize.height, volumeSize.depth);
	hipMemcpy3D(&copyParams);
	copyParams.kind     = hipMemcpyHostToDevice;
	copyParams.dstArray = modelArray_imag;
	copyParams.srcPtr   = make_hipPitchedPtr(model_imag.h_ptr,volumeSize.width*sizeof(FLOAT), volumeSize.height, volumeSize.depth);
	hipMemcpy3D(&copyParams);

	//set texture filter mode property, use hipFilterModePoint or hipFilterModeLinear
	texModel_real.normalized = false;
	texModel_real.filterMode = hipFilterModeLinear;
	texModel_imag.normalized = false;
	texModel_imag.filterMode = hipFilterModeLinear;

	//set texture address mode property, use hipAddressModeClamp , -Border, -Wrap, or -Mirror
	for(int n=0; n<3; n++)
	{
		texModel_real.addressMode[n]=hipAddressModeClamp;
		texModel_imag.addressMode[n]=hipAddressModeClamp;
	}

    //bind texture reference with cuda array
	hipBindTextureToArray(texModel_real, modelArray_real, channel);
	hipBindTextureToArray(texModel_imag, modelArray_imag, channel);

	// Since we hijack Minvsigma to carry a bit more info into the GPU-kernel
	// we need to make a modified copy, since the global object shouldn't be
	// changed
	FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(op.local_Fimgs_shifted[ipart])
	{
		gpuMinvsigma2[n] = *(op.local_Minvsigma2s[ipart].data + n );
	}

	if (baseMLO->do_ctf_correction && baseMLO->refs_are_ctf_corrected)
	{
		FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(op.local_Fimgs_shifted[ipart])
		{
			gpuMinvsigma2[n] *= (DIRECT_MULTIDIM_ELEM(op.local_Fctfs[ipart], n)*DIRECT_MULTIDIM_ELEM(op.local_Fctfs[ipart], n));
		}
	}
	// TODO :    + Assure accuracy with the implemented GPU-based ctf-scaling
	//           + Make setting of myscale robust between here and above.
	//  (scale_correction turns off by default with only one group: ml_optimiser-line 1067,
	//   meaning small-scale test will probably not catch this malfunctioning when/if it breaks.)
	if (baseMLO->do_scale_correction)
	{
		FLOAT myscale = baseMLO->mymodel.scale_correction[group_id];
		FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(op.local_Fimgs_shifted[ipart])
		{
			gpuMinvsigma2[n] *= (myscale*myscale);
		}
	}

	gpuMinvsigma2.cp_to_device();

	Fimgs_real.size = translation_num * image_size;
	Fimgs_real.device_alloc();
	Fimgs_real.cp_to_device();
	Fimgs_imag.size = translation_num * image_size;
	Fimgs_imag.device_alloc();
	Fimgs_imag.cp_to_device();
	rotidx.size = significant_num;
	rotidx.device_alloc();
	rotidx.cp_to_device();
	transidx.size = significant_num;
	transidx.device_alloc();
	transidx.cp_to_device();
	ihidden_overs.size = significant_num;
	ihidden_overs.device_alloc();
	ihidden_overs.cp_to_device();

	/*====================================
				Kernel Calls
	======================================*/
	unsigned orient1, orient2;

	if(significant_num>65535)
	{
		orient1 = ceil(sqrt(significant_num));
		orient2 = orient1;
	}
	else
	{
		orient1 = significant_num;
		orient2 = 1;
	}
	dim3 block_dim(orient1,orient2);

	CUDA_CPU_TOC("kernel_init_1");
	CUDA_GPU_TIC("kernel_diff_proj");


// Could be used to automate __ldg() fallback runtime within cuda_kernel_diff2.
//				hipDeviceProp_t dP;
//				hipGetDeviceProperties(&dP, 0);
//				printf("-arch=sm_%d%d\n", dP.major, dP.minor);

	if ((baseMLO->iter == 1 && baseMLO->do_firstiter_cc) || baseMLO->do_always_cc) // do cross-correlation instead of diff
	{
		// FIXME  make _CC
		exit(0);
//		cuda_kernel_PAV_TTI_D2_CC<<<block_dim,BLOCK_SIZE>>>(~eulers,
//														 ~Fimgs_real,
//														 ~Fimgs_imag,
//														 ~gpuMinvsigma2,
//														 ~diff2s,
//														 image_size,
//														 op.highres_Xi2_imgs[ipart] / 2.,
//														 orientation_num,
//														 translation_num,
//														 significant_num,
//														 ~rotidx,
//														 ~transidx,
//														 ~ihidden_overs,
//														 max_r,
//													     max_r2,
//													     min_r2_nn,
//														 op.local_Minvsigma2s[0].xdim,
//														 op.local_Minvsigma2s[0].ydim,
//														 baseMLO->mymodel.PPref[exp_iclass].data.xdim,
//														 baseMLO->mymodel.PPref[exp_iclass].data.ydim,
//														 baseMLO->mymodel.PPref[exp_iclass].data.zdim,
//														 baseMLO->mymodel.PPref[exp_iclass].data.yinit,
//														 baseMLO->mymodel.PPref[exp_iclass].data.zinit);
//		hipFreeArray(modelArray_real);
//		hipFreeArray(modelArray_imag);
	}
	else
	{
		cuda_kernel_PAV_TTI_D2<<<block_dim,BLOCK_SIZE>>>(~eulers,
														 ~Fimgs_real,
														 ~Fimgs_imag,
														 ~gpuMinvsigma2,
														 ~diff2s,
														 image_size,
														 op.highres_Xi2_imgs[ipart] / 2.,
														 orientation_num,
														 translation_num,
														 significant_num,
														 ~rotidx,
														 ~transidx,
														 ~ihidden_overs,
														 max_r,
													     max_r2,
													     min_r2_nn,
													     op.local_Minvsigma2s[0].xdim,
														 op.local_Minvsigma2s[0].ydim,
														 baseMLO->mymodel.PPref[exp_iclass].data.xdim,
														 baseMLO->mymodel.PPref[exp_iclass].data.ydim,
														 baseMLO->mymodel.PPref[exp_iclass].data.zdim,
														 baseMLO->mymodel.PPref[exp_iclass].data.yinit,
														 baseMLO->mymodel.PPref[exp_iclass].data.zinit);
		size_t avail;
		size_t total;
		hipMemGetInfo( &avail, &total );
		float used = 100*((float)(total - avail)/(float)total);
		std::cerr << "Device memory used @ diff2: " << used << "%" << std::endl;

		hipFreeArray(modelArray_real);
		hipFreeArray(modelArray_imag);
	}
	CUDA_GPU_TAC("kernel_diff_proj");
	HANDLE_ERROR(hipDeviceSynchronize()); //TODO Apparently this is not required here

	CUDA_GPU_TOC("kernel_diff_proj");
	hipUnbindTexture(texModel_real);
    hipUnbindTexture(texModel_imag);
}
#endif


#define BACKPROJECTION4_BLOCK_SIZE 64
#define BACKPROJECTION4_GROUP_SIZE 16
#define BACKPROJECTION4_FETCH_COUNT 4

__global__ void cuda_kernel_backproject(
		int *g_xs,
		int *g_ys,
		int *g_zs,
		FLOAT *g_model_real,
		FLOAT *g_model_imag,
		FLOAT *g_weight,
		FLOAT *g_eulers,
		FLOAT *g_wavgs_real,
		FLOAT *g_wavgs_imag,
		FLOAT *g_Fweights,
		int max_r2, FLOAT scale2,
		unsigned img_xy, unsigned long img_count, unsigned img_x, unsigned img_y,
		unsigned mdl_x, unsigned mdl_y, int mdl_inity, int mdl_initz,
		int N)
{
	unsigned gid = threadIdx.x / 4;
	unsigned mid = threadIdx.x % 4;
	unsigned gm = gid * 4 + mid;
	unsigned pit = (gid * 4 + mid)*BACKPROJECTION4_FETCH_COUNT;
	unsigned global_idx = blockIdx.x * BACKPROJECTION4_GROUP_SIZE + gid;

	int X(0),Y(0),Z(0);

	if (global_idx < N)
	{
		X = g_xs[global_idx];
		Y = g_ys[global_idx];
		Z = g_zs[global_idx];
	}
	else
		X = mdl_x * 10; // Padding coordinate, place outside images

	int ax(0), ay(0);

	if (mid == 1)
		ax = 1;
	else if (mid == 2)
		ay = 1;
	else if (mid == 3)
	{
		ax = 1;
		ay = 1;
	}

	bool  is_neg_x;
	FLOAT d, w;
	FLOAT xp,yp,zp;
	int x,y,idx;

	__shared__ FLOAT s_e[BACKPROJECTION4_BLOCK_SIZE*BACKPROJECTION4_FETCH_COUNT];

	__shared__ FLOAT s_weight[BACKPROJECTION4_GROUP_SIZE*4];
	__shared__ FLOAT s_value_real[BACKPROJECTION4_GROUP_SIZE*4];
	__shared__ FLOAT s_value_imag[BACKPROJECTION4_GROUP_SIZE*4];

	s_weight[gm] = 0.0f;
	s_value_real[gm] = 0.0f;
	s_value_imag[gm] = 0.0f;

	for (int img = 0, b = BACKPROJECTION4_BLOCK_SIZE*BACKPROJECTION4_FETCH_COUNT; img < img_count; img ++, b += 9)
	{
		if (b+9 > BACKPROJECTION4_BLOCK_SIZE*BACKPROJECTION4_FETCH_COUNT)
		{
			__syncthreads();

			int img_9 = img*9+pit;
			if (img_9 < img_count*9)
			{
				s_e[pit+0] = g_eulers[img_9+0];
				s_e[pit+1] = g_eulers[img_9+1];
				s_e[pit+2] = g_eulers[img_9+2];
				s_e[pit+3] = g_eulers[img_9+3];
			}

			__syncthreads();
			b = 0;
		}

		zp = (s_e[b+6] * X + s_e[b+7] * Y + s_e[b+8] * Z);

		if (fabsf(zp) > 0.87f) continue; //Within the unit cube, sqrt(3)/2=0.866

		yp = (s_e[b+3] * X + s_e[b+4] * Y + s_e[b+5] * Z);
		xp = (s_e[b+0] * X + s_e[b+1] * Y + s_e[b+2] * Z);

		if (xp < 0.0f)
		{
			yp = -yp;
			xp = -xp;
			is_neg_x = true;
		}
		else
			is_neg_x = false;

		x = (int) floorf(xp) + ax;
		y = (int) floorf(yp) + ay;

		if (x * x + y * y > max_r2) continue;

		if (y < 0 && x == 0)
		{
			is_neg_x = !is_neg_x;
			y = -y;
		}

		xp = (s_e[b+0] * x + s_e[b+3] * y) * scale2;
		yp = (s_e[b+1] * x + s_e[b+4] * y) * scale2;
		zp = (s_e[b+2] * x + s_e[b+5] * y) * scale2;

		if (xp < 0.0f) //Flip sign
		{
			xp = fabsf(X+xp);
			yp = fabsf(Y+yp);
			zp = fabsf(Z+zp);
		}
		else
		{
			xp = fabsf(X-xp);
			yp = fabsf(Y-yp);
			zp = fabsf(Z-zp);
		}

		if (xp < 1.0f && yp < 1.0f && zp < 1.0f)
		{
			if (y < 0) y += img_y;
			idx = img*img_xy + y * img_x + x;
			w = g_Fweights[idx];

			if (w > 0.0f)
			{
				d = (1.0f - xp) * (1.0f - yp) * (1.0f - zp);

				s_weight[gm] += w * d;
				s_value_real[gm] += g_wavgs_real[idx] * d;
				if (is_neg_x) s_value_imag[gm] -= g_wavgs_imag[idx] * d;
				else          s_value_imag[gm] += g_wavgs_imag[idx] * d;
			}
		}
	}

	__syncthreads();

	if (mid == 0)
	{
		FLOAT sum = s_weight[gid*4 + 0] + s_weight[gid*4 + 1] + s_weight[gid*4 + 2] + s_weight[gid*4 + 3];
		if (sum != 0.0f)
			g_weight[(Z-mdl_initz)*mdl_x*mdl_y + (Y-mdl_inity)*mdl_x + X] = sum;
	}
	else if (mid == 1)
	{
		FLOAT sum = s_value_real[gid*4 + 0] + s_value_real[gid*4 + 1] + s_value_real[gid*4 + 2] + s_value_real[gid*4 + 3];
		if (sum != 0.0f)
			g_model_real[(Z-mdl_initz)*mdl_x*mdl_y + (Y-mdl_inity)*mdl_x + X] = sum;
	}
	else if (mid == 2)
	{
		FLOAT sum = s_value_imag[gid*4 + 0] + s_value_imag[gid*4 + 1] + s_value_imag[gid*4 + 2] + s_value_imag[gid*4 + 3];
		if (sum != 0.0f)
			g_model_imag[(Z-mdl_initz)*mdl_x*mdl_y + (Y-mdl_inity)*mdl_x + X] = sum;
	}
}

static void backproject(
		CudaGlobalPtr<FLOAT> &wavgs_real,
		CudaGlobalPtr<FLOAT> &wavgs_imag,
		CudaGlobalPtr<FLOAT> &Fweights,
		CudaGlobalPtr<FLOAT> &eulers,
		CudaGlobalPtr<FLOAT> &model_real,
		CudaGlobalPtr<FLOAT> &model_imag,
		CudaGlobalPtr<FLOAT> &weight,
		int max_r, FLOAT scale2, //grid scale 2D -> 3D squared
		int img_xy, long img_count, int img_x, int img_y,
		int mdl_x, int mdl_y, int mdl_z, int mdl_inity, int mdl_initz)
{
	int max_r2 = max_r * max_r;

	CudaGlobalPtr<int> xs(mdl_x*mdl_y*mdl_z); // >52% will actually be used, allocate some padding
	CudaGlobalPtr<int> ys(xs.size);
	CudaGlobalPtr<int> zs(xs.size);
	unsigned N(0);

	for (int x = 0; x < mdl_x; x ++)
	{
		for (int y = mdl_inity; y < mdl_y; y++)
		{
			for (int z = mdl_initz; z < mdl_z; z++)
			{
				if (x*x + y*y + z*z <= max_r2 * scale2 * 1.2f)
				{
					xs[N] = x;
					ys[N] = y;
					zs[N] = z;
					N ++;
				}
			}
		}
	}
	xs.size = N + N%BACKPROJECTION4_GROUP_SIZE;
	ys.size = xs.size;
	zs.size = xs.size;

	xs.device_alloc();
	ys.device_alloc();
	zs.device_alloc();

	xs.cp_to_device();
	ys.cp_to_device();
	zs.cp_to_device();

	int grid_dim = ceil((float)N / BACKPROJECTION4_GROUP_SIZE);
	dim3 block_dim( BACKPROJECTION4_GROUP_SIZE *4 );

	cuda_kernel_backproject<<<grid_dim,block_dim>>>(
			~xs,~ys,~zs,
			~model_real,
			~model_imag,
			~weight,
			~eulers,
			~wavgs_real,
			~wavgs_imag,
			~Fweights,
			max_r2,
			scale2,
			img_xy,
			img_count,
			img_x,
			img_y,
			mdl_x,
			mdl_y,
			mdl_inity,
			mdl_initz,
			N);
}









void MlOptimiserCuda::doThreadExpectationSomeParticles(unsigned thread_id)
{
	size_t first_ipart = 0, last_ipart = 0;
	while (baseMLO->exp_ipart_ThreadTaskDistributor->getTasks(first_ipart, last_ipart))
	{
		for (long unsigned ipart = first_ipart; ipart <= last_ipart; ipart++)
		{
			unsigned my_ori_particle = baseMLO->exp_my_first_ori_particle + ipart;
			SamplingParameters sp;
			sp.nr_particles = baseMLO->mydata.ori_particles[my_ori_particle].particles_id.size();

			OptimisationParamters op(sp.nr_particles, my_ori_particle);

			// In the first iteration, multiple seeds will be generated
			// A single random class is selected for each pool of images, and one does not marginalise over the orientations
			// The optimal orientation is based on signal-product (rather than the signal-intensity sensitive Gaussian)
			// If do_firstiter_cc, then first perform a single iteration with K=1 and cross-correlation criteria, afterwards

			// Decide which classes to integrate over (for random class assignment in 1st iteration)
			sp.iclass_max = baseMLO->mymodel.nr_classes - 1;
			// low-pass filter again and generate the seeds
			if (baseMLO->do_generate_seeds)
			{
				if (baseMLO->do_firstiter_cc && baseMLO->iter == 1)
				{
					// In first (CC) iter, use a single reference (and CC)
					sp.iclass_min = sp.iclass_max = 0;
				}
				else if ( (baseMLO->do_firstiter_cc && baseMLO->iter == 2) ||
						(!baseMLO->do_firstiter_cc && baseMLO->iter == 1))
				{
					// In second CC iter, or first iter without CC: generate the seeds
					// Now select a single random class
					// exp_part_id is already in randomized order (controlled by -seed)
					// WARNING: USING SAME iclass_min AND iclass_max FOR SomeParticles!!
					sp.iclass_min = sp.iclass_max = divide_equally_which_group(baseMLO->mydata.numberOfOriginalParticles(), baseMLO->mymodel.nr_classes, op.my_ori_particle);
				}
			}
			// Global exp_metadata array has metadata of all ori_particles. Where does my_ori_particle start?
			for (long int iori = baseMLO->exp_my_first_ori_particle; iori <= baseMLO->exp_my_last_ori_particle; iori++)
			{
				if (iori == my_ori_particle) break;
				op.metadata_offset += baseMLO->mydata.ori_particles[iori].particles_id.size();
			}

			baseMLO->getFourierTransformsAndCtfs(my_ori_particle, op.metadata_offset, op.Fimgs, op.Fimgs_nomask, op.Fctfs,
					op.old_offset, op.prior, op.power_imgs, op.highres_Xi2_imgs,
					op.pointer_dir_nonzeroprior, op.pointer_psi_nonzeroprior, op.directions_prior, op.psi_prior);

			if (baseMLO->do_realign_movies && baseMLO->movie_frame_running_avg_side > 0)
			{
				baseMLO->calculateRunningAveragesOfMovieFrames(my_ori_particle, op.Fimgs, op.power_imgs, op.highres_Xi2_imgs);
			}

			// To deal with skipped alignments/rotations
			if (baseMLO->do_skip_align)
			{
				sp.itrans_min = sp.itrans_max = sp.idir_min = sp.idir_max = sp.ipsi_min = sp.ipsi_max =
						my_ori_particle - baseMLO->exp_my_first_ori_particle;
			}
			else
			{
				sp.itrans_min = 0;
				sp.itrans_max = baseMLO->sampling.NrTranslationalSamplings() - 1;

				if (baseMLO->do_skip_rotate)
				{
					sp.idir_min = sp.idir_max = sp.ipsi_min = sp.ipsi_max =
							my_ori_particle - baseMLO->exp_my_first_ori_particle;
				}
				else
				{
					sp.idir_min = sp.ipsi_min = 0;
					sp.idir_max = baseMLO->sampling.NrDirections(0, &op.pointer_dir_nonzeroprior) - 1;
					sp.ipsi_max = baseMLO->sampling.NrPsiSamplings(0, &op.pointer_psi_nonzeroprior ) - 1;
				}
			}

			// Initialise significant weight to minus one, so that all coarse sampling points will be handled in the first pass
			op.significant_weight.resize(sp.nr_particles, -1.);

			// Only perform a second pass when using adaptive oversampling
			int nr_sampling_passes = (baseMLO->adaptive_oversampling > 0) ? 2 : 1;

			for (int ipass = 0; ipass < nr_sampling_passes; ipass++)
			{
				if (baseMLO->strict_highres_exp > 0.)
					// Use smaller images in both passes and keep a maximum on coarse_size, just like in FREALIGN
					sp.current_image_size = baseMLO->coarse_size;
				else if (baseMLO->adaptive_oversampling > 0)
					// Use smaller images in the first pass, larger ones in the second pass
					sp.current_image_size = (ipass == 0) ? baseMLO->coarse_size : baseMLO->mymodel.current_size;
				else
					sp.current_image_size = baseMLO->mymodel.current_size;

				// Use coarse sampling in the first pass, oversampled one the second pass
				sp.current_oversampling = (ipass == 0) ? 0 : baseMLO->adaptive_oversampling;

				sp.nr_dir = (baseMLO->do_skip_align || baseMLO->do_skip_rotate) ? 1 : baseMLO->sampling.NrDirections(0, &op.pointer_dir_nonzeroprior);
				sp.nr_psi = (baseMLO->do_skip_align || baseMLO->do_skip_rotate) ? 1 : baseMLO->sampling.NrPsiSamplings(0, &op.pointer_psi_nonzeroprior);
				sp.nr_trans = (baseMLO->do_skip_align) ? 1 : baseMLO->sampling.NrTranslationalSamplings();
				sp.nr_oversampled_rot = baseMLO->sampling.oversamplingFactorOrientations(sp.current_oversampling);
				sp.nr_oversampled_trans = baseMLO->sampling.oversamplingFactorTranslations(sp.current_oversampling);

				CUDA_CPU_TIC("getAllSquaredDifferences");
				getAllSquaredDifferences(ipass, op, sp);
				CUDA_CPU_TOC("getAllSquaredDifferences");
				CUDA_CPU_TIC("convertAllSquaredDifferencesToWeights");
				convertAllSquaredDifferencesToWeights(ipass, op, sp);
				CUDA_CPU_TOC("convertAllSquaredDifferencesToWeights");
			}

			// For the reconstruction step use mymodel.current_size!
			sp.current_image_size = baseMLO->mymodel.current_size;

			CUDA_CPU_TIC("storeWeightedSums");
			storeWeightedSums(op, sp);
			CUDA_CPU_TOC("storeWeightedSums");
		}
	}
}





void MlOptimiserCuda::getAllSquaredDifferences(unsigned exp_ipass, OptimisationParamters &op, SamplingParameters &sp)
{

	CUDA_CPU_TIC("diff_pre_gpu");

	//for scale_correction
	int group_id;

	//printf("sp.nr_oversampled_rot=%d\n", (unsigned)sp.nr_oversampled_rot);

	op.Mweight.resize(sp.nr_particles, baseMLO->mymodel.nr_classes * sp.nr_dir * sp.nr_psi * sp.nr_trans * sp.nr_oversampled_rot * sp.nr_oversampled_trans);
	op.Mweight.initConstant(-999.);
	if (exp_ipass==0)
	{
		op.Mcoarse_significant.clear();
	}

	op.min_diff2.clear();
	op.min_diff2.resize(sp.nr_particles, 99.e99);

	std::vector<MultidimArray<Complex > > dummy;
	baseMLO->precalculateShiftedImagesCtfsAndInvSigma2s(false, op.my_ori_particle, sp.current_image_size, sp.current_oversampling,
			sp.itrans_min, sp.itrans_max, op.Fimgs, dummy, op.Fctfs, op.local_Fimgs_shifted, dummy,
			op.local_Fctfs, op.local_sqrtXi2, op.local_Minvsigma2s);

	MultidimArray<Complex > Fref;
	Fref.resize(op.local_Minvsigma2s[0]);

	unsigned image_size = op.local_Minvsigma2s[0].nzyxdim;

	CUDA_CPU_TOC("diff_pre_gpu");

	// Loop only from sp.iclass_min to sp.iclass_max to deal with seed generation in first iteration
	for (int exp_iclass = sp.iclass_min; exp_iclass <= sp.iclass_max; exp_iclass++)
	{
		if (baseMLO->mymodel.pdf_class[exp_iclass] > 0.)
		{
			// Local variables
			std::vector< double > oversampled_rot, oversampled_tilt, oversampled_psi;
			std::vector< double > oversampled_translations_x, oversampled_translations_y, oversampled_translations_z;
			CudaGlobalPtr<FLOAT> gpuMinvsigma2(image_size);
			gpuMinvsigma2.device_alloc();

			// Mapping index look-up table
			std::vector< long unsigned > iorientclasses, iover_rots;
			std::vector< double > rots, tilts, psis;

			CUDA_CPU_TIC("projection_1");
			CUDA_CPU_TIC("generateProjectionSetup");
			long unsigned orientation_num = generateProjectionSetup(
					op,
					sp,
					baseMLO,
					exp_ipass == 0, //coarse
					exp_iclass,
					rots, tilts, psis,
					iorientclasses,
					iover_rots);

			CUDA_CPU_TOC("generateProjectionSetup");
			CUDA_CPU_TIC("generateEulerMatrices");
			CudaGlobalPtr<FLOAT> eulers(9 * orientation_num);

			generateEulerMatrices(
					baseMLO->mymodel.PPref[exp_iclass].padding_factor,
					rots,
					tilts,
					psis,
					eulers,
					!IS_NOT_INV);

		    eulers.device_alloc();
			eulers.cp_to_device();
			CUDA_CPU_TOC("generateEulerMatrices");
			CUDA_CPU_TIC("modelAssignment");
			CudaGlobalPtr<FLOAT > model_real((baseMLO->mymodel.PPref[exp_iclass]).data.nzyxdim);
			CudaGlobalPtr<FLOAT > model_imag((baseMLO->mymodel.PPref[exp_iclass]).data.nzyxdim);

			for(unsigned i = 0; i < model_real.size; i++)
			{
				model_real[i] = (FLOAT) baseMLO->mymodel.PPref[exp_iclass].data.data[i].real;
				model_imag[i] = (FLOAT) baseMLO->mymodel.PPref[exp_iclass].data.data[i].imag;
			}

			CudaGlobalPtr<FLOAT> Frefs_real;
		    CudaGlobalPtr<FLOAT> Frefs_imag;

			CUDA_CPU_TOC("modelAssignment");
			bool do_combineProjAndDiff = false; //TODO add control flag
			if(!do_combineProjAndDiff)
			{
				CUDA_CPU_TIC("generateModelProjections");
				generateModelProjections(
						model_real,
						model_imag,
						Frefs_real,
						Frefs_imag,
						eulers,
						orientation_num,
						image_size,
						XMIPP_MIN(baseMLO->mymodel.PPref[exp_iclass].r_max, op.local_Minvsigma2s[0].xdim - 1),
						op.local_Minvsigma2s[0].xdim,
						op.local_Minvsigma2s[0].ydim,
						baseMLO->mymodel.PPref[exp_iclass].data.xdim,
						baseMLO->mymodel.PPref[exp_iclass].data.ydim,
						baseMLO->mymodel.PPref[exp_iclass].data.zdim,
						baseMLO->mymodel.PPref[exp_iclass].data.yinit,
						baseMLO->mymodel.PPref[exp_iclass].data.zinit);
				CUDA_CPU_TOC("generateModelProjections");
			}
			CUDA_CPU_TOC("projection_1");

			/*=======================================================================================
			                                  	  Particle Iteration
			=========================================================================================*/

			for (long int ipart = 0; ipart < sp.nr_particles; ipart++)
			{
				/*====================================
				        Generate Translations
				======================================*/

				CUDA_CPU_TIC("translation_1");

				CudaGlobalPtr<FLOAT> Fimgs_real(image_size * sp.nr_trans * sp.nr_oversampled_trans);
				CudaGlobalPtr<FLOAT> Fimgs_imag(image_size * sp.nr_trans * sp.nr_oversampled_trans);

				long int part_id = baseMLO->mydata.ori_particles[op.my_ori_particle].particles_id[ipart];
				long unsigned translation_num(0), ihidden(0);
				std::vector< long unsigned > iover_transes, itranses, ihiddens;

				for (long int itrans = sp.itrans_min; itrans <= sp.itrans_max; itrans++, ihidden++)
				{
					baseMLO->sampling.getTranslations(itrans, sp.current_oversampling,
							oversampled_translations_x, oversampled_translations_y, oversampled_translations_z );

					for (long int iover_trans = 0; iover_trans < sp.nr_oversampled_trans; iover_trans++)
					{
						/// Now get the shifted image
						// Use a pointer to avoid copying the entire array again in this highly expensive loop
						Complex *myAB;
						if (sp.current_oversampling == 0)
						{
							myAB = (Fref.ydim == baseMLO->coarse_size) ? baseMLO->global_fftshifts_ab_coarse[itrans].data
									: baseMLO->global_fftshifts_ab_current[itrans].data;
						}
						else
						{
							int iitrans = itrans * sp.nr_oversampled_trans +  iover_trans;
							myAB = (baseMLO->strict_highres_exp > 0.) ? baseMLO->global_fftshifts_ab2_coarse[iitrans].data
									: baseMLO->global_fftshifts_ab2_current[iitrans].data;
						}


						FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(op.local_Fimgs_shifted[ipart])
						{
							FLOAT real = (*(myAB + n)).real * (DIRECT_MULTIDIM_ELEM(op.local_Fimgs_shifted[ipart], n)).real
									- (*(myAB + n)).imag *(DIRECT_MULTIDIM_ELEM(op.local_Fimgs_shifted[ipart], n)).imag;
							FLOAT imag = (*(myAB + n)).real * (DIRECT_MULTIDIM_ELEM(op.local_Fimgs_shifted[ipart], n)).imag
									+ (*(myAB + n)).imag *(DIRECT_MULTIDIM_ELEM(op.local_Fimgs_shifted[ipart], n)).real;

							//When on gpu, it makes more sense to ctf-correct translated images, rather than anti-ctf-correct ref-projections
							if (baseMLO->do_scale_correction)
							{
								//group_id = mydata.getGroupId(part_id);
								FLOAT myscale = baseMLO->mymodel.scale_correction[group_id];
								real /= myscale;
								imag /= myscale;
							}
							if (baseMLO->do_ctf_correction && baseMLO->refs_are_ctf_corrected)
							{
								real /= DIRECT_MULTIDIM_ELEM(op.local_Fctfs[ipart], n);
								imag /= DIRECT_MULTIDIM_ELEM(op.local_Fctfs[ipart], n);
							}
							Fimgs_real[translation_num * image_size + n] = real;
							Fimgs_imag[translation_num * image_size + n] = imag;
						}
						translation_num ++;

						ihiddens.push_back(ihidden);
						itranses.push_back(itrans);
						iover_transes.push_back(iover_trans);
					}
				}

				CUDA_CPU_TOC("translation_1");

				/*===========================================
				   Determine significant comparison indices
				=============================================*/
				//      This section is annoying to test because
				//		it can't complete on first pass, since
				//		the significance has never been set


				CUDA_CPU_TIC("pair_list_1");

				CudaGlobalPtr<long unsigned> transidx(orientation_num*translation_num), rotidx(orientation_num*translation_num);
				CudaGlobalPtr<long unsigned> ihidden_overs(orientation_num*translation_num);
				long unsigned coarse_num = sp.nr_dir*sp.nr_psi*sp.nr_trans;
				long unsigned significant_num(0);

				if (exp_ipass == 0)
				{
					op.Mcoarse_significant.resize(coarse_num, 1);
					for (long unsigned i = 0; i < orientation_num; i++)
					{
						for (long unsigned j = 0; j < translation_num; j++)
						{
							ihidden_overs[significant_num] = i * sp.nr_trans + j;
							rotidx[significant_num] = i;
							transidx[significant_num] = j;
							significant_num++;
						}
					}
				}
				else
				{
					for (long unsigned i = 0; i < orientation_num; i++)
					{
						long int iover_rot = iover_rots[i];
						long int coarse_rot = floor(i/sp.nr_oversampled_rot);
						for (long unsigned j = 0; j < translation_num; j++)
						{
							long int iover_trans = iover_transes[j];
							long int coarse_trans = floor(j/sp.nr_oversampled_trans);
							long int ihidden = iorientclasses[i] * sp.nr_trans + ihiddens[j];

							if(DIRECT_A2D_ELEM(op.Mcoarse_significant, ipart, ihidden)==1)
							{
								ihidden_overs[significant_num] = baseMLO->sampling.getPositionOversampledSamplingPoint(ihidden,
										                  sp.current_oversampling, iover_rot, iover_trans);

								rotidx[significant_num] = i;
								transidx[significant_num] = j;
								significant_num++;
							}
						}
					}
				}

				CUDA_CPU_TOC("pair_list_1");

//				std::cerr << "orientation_num "<< orientation_num << std::endl;
//				std::cerr << "translation_num "<< translation_num << std::endl;
//				std::cerr << "my_nr_significant_coarse_samples "<< DIRECT_A2D_ELEM(exp_metadata, metadata_offset + ipart, METADATA_NR_SIGN) << std::endl;
//				std::cerr << "significant_num "<< significant_num << std::endl;

				CudaGlobalPtr<FLOAT> diff2s(orientation_num*translation_num);
				diff2s.device_alloc();

#if !defined(CUDA_DOUBLE_PRECISION)
				if(do_combineProjAndDiff)
				{
					runProjAndDifferenceKernel(model_real,
											   model_imag,
											   gpuMinvsigma2,
										       Fimgs_real,
										       Fimgs_imag,
										       eulers,
										       rotidx,
										       transidx,
										       ihidden_overs,
										       diff2s,
										       op,
										       baseMLO,
										       orientation_num,
										       translation_num,
										       significant_num,
										       image_size,
											    XMIPP_MIN(baseMLO->mymodel.PPref[exp_iclass].r_max, op.local_Minvsigma2s[0].xdim - 1),
										       ipart,
										       group_id,
										       exp_iclass
											 );
				}
				else
#endif
				{
					runDifferenceKernel(gpuMinvsigma2,
										Fimgs_real,
										Fimgs_imag,
										Frefs_real,
										Frefs_imag,
										rotidx,
										transidx,
										ihidden_overs,
										op,
										baseMLO,
										translation_num,
										orientation_num,
										significant_num,
										image_size,
										ipart,
										group_id,
										diff2s
										);
				}
				/*====================================
				    	   Retrieve Results
				======================================*/

				diff2s.cp_to_host(); // FIXME may not be needed since we copy it back in ConvetToWeights()
//				for (long unsigned k = 0; k < 100; k++)
//				{
//					std::cerr << diff2s[k] << std::endl;
//				}
				if (exp_ipass == 0)
				{
					op.Mcoarse_significant.clear();
				}

				/*====================================
				    	Write To Destination
				======================================*/


				CUDA_CPU_TIC("collect_data_1");

				for (long unsigned k = 0; k < significant_num; k++)
				{
					long unsigned i = rotidx[k];
					long unsigned j = transidx[k];
					double diff2 = diff2s[i * translation_num + j];
					DIRECT_A2D_ELEM(op.Mweight, ipart, ihidden_overs[k]) = diff2; // TODO if we can write diff2 to the correct pos in the kernel we can just memcpy to a pointer and use thrust to find min
					// Keep track of minimum of all diff2, only for the last image in this series
					if (diff2 < op.min_diff2[ipart])
						op.min_diff2[ipart] = diff2;
				}

				CUDA_CPU_TOC("collect_data_1");

			} // end loop ipart

		} // end if class significant
	} // end loop iclass
}

__global__ void cuda_kernel_sumweight_oversampling(	FLOAT *g_pdf_orientation,
													FLOAT *g_pdf_offset,
													FLOAT *g_Mweight,
													FLOAT *g_thisparticle_sumweight,
													FLOAT min_diff2,
													int translation_num,
													int oversamples)
{
	__shared__ FLOAT s_sumweight[SUM_BLOCK_SIZE];
	// blockid
	int ex  = blockIdx.x * gridDim.y + blockIdx.y;
	//threadid
	int tid = threadIdx.x;
	s_sumweight[tid]=0;

	// passes to take care of all fine samples in a coarse sample
	int pass_num = ceil((float)oversamples / (float)SUM_BLOCK_SIZE);
	//Where to start in g_Mweight to find all data for this *coarse* orientation
	long int ref_Mweight_idx = ex * ( translation_num*oversamples );

	// Go over all *coarse* translations, reducing in place
	for (int itrans=0; itrans<translation_num; itrans++)
	{
		//Where to start in g_Mweights to find all fine samples for this *coarse* translation
		int pos = ref_Mweight_idx + itrans*oversamples + tid;
		for (int pass = 0; pass < pass_num; pass++, pos+=SUM_BLOCK_SIZE)
		{
			if( g_Mweight[pos] < 0.0f ) //TODO Might be slow (divergent threads)
			{
				g_Mweight[pos] = 0.0f;
			}
			else
			{
				FLOAT weight = g_pdf_orientation[ex] * g_pdf_offset[itrans];          	// Same      for all threads - TODO: should be done once for all trans through warp-parallel execution
				FLOAT diff2 = g_Mweight[pos] - min_diff2;								// Different for all threads
				// next line because of numerical precision of exp-function
				if (diff2 > 700.0f)
					weight = 0.0f;
				else weight *= exp(-diff2);  // TODO: use tabulated exp function? / Sjors  TODO: exp, expf, or __exp in CUDA? /Bjorn

				// Store the weight for each fine sample in this coarse pair
				g_Mweight[pos] = weight; // TODO put in shared mem

				// Reduce weights for each fine sample in this coarse pair
				s_sumweight[tid] += weight;
			}
		}
	}
	// Reduction of all fine samples in this coarse orientation
	for(int j=(SUM_BLOCK_SIZE/2); j>0; j/=2)
	{
		if(tid<j)
		{
			s_sumweight[tid] += s_sumweight[tid+j];
		}
	}
	__syncthreads();
	g_thisparticle_sumweight[ex]=s_sumweight[0];
}

void MlOptimiserCuda::convertAllSquaredDifferencesToWeights(unsigned exp_ipass, OptimisationParamters &op, SamplingParameters &sp)
{
	op.sum_weight.clear();
	op.sum_weight.resize(sp.nr_particles, 0.);

	// loop over all particles inside this ori_particle
	for (long int ipart = 0; ipart < sp.nr_particles; ipart++)
	{
		long int part_id = baseMLO->mydata.ori_particles[op.my_ori_particle].particles_id[ipart];
		double exp_thisparticle_sumweight = 0.;

		double old_offset_z;
		double old_offset_x = XX(op.old_offset[ipart]);
		double old_offset_y = YY(op.old_offset[ipart]);
		if (baseMLO->mymodel.data_dim == 3)
			old_offset_z = ZZ(op.old_offset[ipart]);

		if ((baseMLO->iter == 1 && baseMLO->do_firstiter_cc) || baseMLO->do_always_cc)
		{
			// Binarize the squared differences array to skip marginalisation
			double mymindiff2 = 99.e10;
			long int myminidx = -1;
			// Find the smallest element in this row of op.Mweight
			for (long int i = 0; i < XSIZE(op.Mweight); i++)
			{

				double cc = DIRECT_A2D_ELEM(op.Mweight, ipart, i);
				// ignore non-determined cc
				if (cc == -999.)
					continue;

				// just search for the maximum
				if (cc < mymindiff2)
				{
					mymindiff2 = cc;
					myminidx = i;
				}
			}
			// Set all except for the best hidden variable to zero and the smallest element to 1
			for (long int i = 0; i < XSIZE(op.Mweight); i++)
				DIRECT_A2D_ELEM(op.Mweight, ipart, i)= 0.;

			DIRECT_A2D_ELEM(op.Mweight, ipart, myminidx)= 1.;
			exp_thisparticle_sumweight += 1.;

		}
		else
		{
			// Loop from iclass_min to iclass_max to deal with seed generation in first iteration
			for (int exp_iclass = sp.iclass_min; exp_iclass <= sp.iclass_max; exp_iclass++)
			{

				// Make PdfOffset calculation much faster...
				double myprior_x, myprior_y, myprior_z;
				if (baseMLO->mymodel.ref_dim == 2)
				{
					myprior_x = XX(baseMLO->mymodel.prior_offset_class[exp_iclass]);
					myprior_y = YY(baseMLO->mymodel.prior_offset_class[exp_iclass]);
				}
				else
				{
					myprior_x = XX(op.prior[ipart]);
					myprior_y = YY(op.prior[ipart]);
					if (baseMLO->mymodel.data_dim == 3)
						myprior_z = ZZ(op.prior[ipart]);
				}

				/*=========================================
						Fetch+generate Orientation data
				===========================================*/
				CudaGlobalPtr<FLOAT >  pdf_orientation(sp.nr_dir * sp.nr_psi);
				pdf_orientation.size = sp.nr_dir * sp.nr_psi;
				for (long int idir = sp.idir_min, iorient = 0; idir <= sp.idir_max; idir++)
				{
					for (long int ipsi = sp.ipsi_min; ipsi <= sp.ipsi_max; ipsi++, iorient++)
					{
						//std::cerr << "orient "  << idir << "," << iorient <<  std::endl;
						// Get prior for this direction
						if (baseMLO->do_skip_align || baseMLO->do_skip_rotate)
						{
							pdf_orientation[iorient] = baseMLO->mymodel.pdf_class[exp_iclass];
						}
						else if (baseMLO->mymodel.orientational_prior_mode == NOPRIOR)
						{
							pdf_orientation[iorient] = DIRECT_MULTIDIM_ELEM(baseMLO->mymodel.pdf_direction[exp_iclass], idir);
						}
						else
						{
							// P(orientation) = P(idir|dir_prior) * P(ipsi|psi_prior)
							// This is the probability of the orientation, given the gathered
							// statistics of all assigned orientations of the dataset, since we
							// are assigning a gaussian prior to all parameters.
							pdf_orientation[iorient] = op.directions_prior[idir] * op.psi_prior[ipsi];
						}
					}
				}
//				long int ihidden = iorientclass * sp.nr_trans;

				/*=========================================
						Fetch+generate Translation data
				===========================================*/
				CudaGlobalPtr<FLOAT >  pdf_offset(sp.nr_trans);

				int jtrans=0;
				for (long int itrans = sp.itrans_min; itrans <= sp.itrans_max; itrans++,jtrans++)
				{
					//std::cerr << "trans " << itrans << "," << jtrans <<  std::endl;
			        // To speed things up, only calculate pdf_offset at the coarse sampling.
					// That should not matter much, and that way one does not need to calculate all the OversampledTranslations
					double offset_x = old_offset_x + baseMLO->sampling.translations_x[itrans];
					double offset_y = old_offset_y + baseMLO->sampling.translations_y[itrans];
					double tdiff2 = (offset_x - myprior_x) * (offset_x - myprior_x) + (offset_y - myprior_y) * (offset_y - myprior_y);
					if (baseMLO->mymodel.data_dim == 3)
					{
						double offset_z = old_offset_z + baseMLO->sampling.translations_z[itrans];
						tdiff2 += (offset_z - myprior_z) * (offset_z - myprior_z);
					}
					// P(offset|sigma2_offset)
					// This is the probability of the offset, given the model offset and variance.
					if (baseMLO->mymodel.sigma2_offset < 0.0001)
						pdf_offset[jtrans] = ( tdiff2 > 0.) ? 0. : 1.;
					else
						pdf_offset[jtrans] = exp ( tdiff2 / (-2. * baseMLO->mymodel.sigma2_offset) ) / ( 2. * PI * baseMLO->mymodel.sigma2_offset );
				}

// TODO : Put back when  convertAllSquaredDifferencesToWeights is GPU-parallel.
//							// TMP DEBUGGING
//							if (baseMLO->mymodel.orientational_prior_mode != NOPRIOR && (pdf_offset==0. || pdf_orientation==0.))
//							{
//								pthread_mutex_lock(&global_mutex);
//								std::cerr << " pdf_offset= " << pdf_offset << " pdf_orientation= " << pdf_orientation << std::endl;
//								std::cerr << " ipart= " << ipart << " part_id= " << part_id << std::endl;
//								std::cerr << " iorient= " << iorient << " idir= " << idir << " ipsi= " << ipsi << std::endl;
//								//std::cerr << " sp.nr_psi= " << sp.nr_psi << " exp_nr_dir= " << exp_nr_dir << " sp.nr_trans= " << sp.nr_trans << std::endl;
//								for (long int i = 0; i < op.directions_prior.size(); i++)
//									std::cerr << " op.directions_prior["<<i<<"]= " << op.directions_prior[i] << std::endl;
//								for (long int i = 0; i < op.psi_prior.size(); i++)
//									std::cerr << " op.psi_prior["<<i<<"]= " << op.psi_prior[i] << std::endl;
//								REPORT_ERROR("ERROR! pdf_offset==0.|| pdf_orientation==0.");
//								//pthread_mutex_unlock(&global_mutex);
//							}
//							if (sp.nr_oversampled_rot == 0)
//								REPORT_ERROR("sp.nr_oversampled_rot == 0");
//							if (sp.nr_oversampled_trans == 0)
//								REPORT_ERROR("sp.nr_oversampled_trans == 0");

				// Now first loop over iover_rot, because that is the order in op.Mweight as well
//				long int ihidden_over = ihidden * sp.nr_oversampled_rot * sp.nr_oversampled_trans;

				/*=========================================
					  Kernel call over all combinations
				===========================================*/

				// One block will be started for each (coarse) orientation, and will process all (coarse) transes,
				// and since oversmapling is by factors of 2 on 5 dofs, we get 2^5=32 fine comparisons per coarse.
				// In case of higher oversampling this is simply factors of 32, making it warp-perfect. Having 21
				// coarse transes allows a block to finish in 21, 100% utilized, warp passes.

				int oversamples = sp.nr_oversampled_trans * sp.nr_oversampled_rot;

				bool do_gpu_sumweight = true;  //TODO add control flag
				if(oversamples>=SUM_BLOCK_SIZE && do_gpu_sumweight) // Send task to GPU where warps can access automatically coalesced oversamples
				{
					//std::cerr << "summing weights on GPU... baseMLO->mymodel.pdf_class[exp_iclass] = " << baseMLO->mymodel.pdf_class[sp.iclass_min] <<  std::endl;
					pdf_orientation.device_alloc();
					pdf_orientation.cp_to_device();
					pdf_offset.device_alloc();
					pdf_offset.cp_to_device();

					CudaGlobalPtr<FLOAT >  thisparticle_sumweight(sp.nr_dir * sp.nr_psi);  // This will be reduced in a second step.
					thisparticle_sumweight.device_alloc();

					CudaGlobalPtr<FLOAT >  Mweight( &(op.Mweight.data[(ipart)*(op.Mweight).xdim]),
													sp.nr_dir * sp.nr_psi * sp.nr_trans * oversamples);
					Mweight.device_alloc();
					Mweight.cp_to_device();

					dim3 block_dim(sp.nr_dir,sp.nr_psi);
					//std::cerr << "using block dimensions " << sp.nr_dir << "," << sp.nr_psi <<  std::endl;
					cuda_kernel_sumweight_oversampling<<<block_dim,SUM_BLOCK_SIZE>>>(	~pdf_orientation,
																						~pdf_offset,
																						~Mweight,
																						~thisparticle_sumweight,
																						op.min_diff2[ipart],
																						sp.nr_trans,
																						oversamples
																					 );

					Mweight.cp_to_host(); //FIXME make wider in scope; pass to storeWsums() to be used in collect-step. Needs som coordination with else() below.
					Mweight.free_device();  //FIXME see line above
					thisparticle_sumweight.cp_to_host();
					thisparticle_sumweight.free_device();

					// The reduced entity *MUST* be double to avoid loss of information// TODO better reduction
					for (long int n = 0; n < sp.nr_dir * sp.nr_psi; n++)
					{
						exp_thisparticle_sumweight += (double)thisparticle_sumweight[n];
					}
				}
				else // Not enough oversamples to utilize GPU resources effciently with current CUDA-kernel.
				{
					//std::cerr << "summing weights on CPU... " <<  std::endl;
					for (long int idir = sp.idir_min, iorient = 0; idir <= sp.idir_max; idir++)
					{
						for (long int ipsi = sp.ipsi_min; ipsi <= sp.ipsi_max; ipsi++, iorient++)
						{
							long int iorientclass = exp_iclass * sp.nr_dir * sp.nr_psi + iorient;
							long int ihidden = iorientclass * sp.nr_trans;
							for (long int itrans = sp.itrans_min; itrans <= sp.itrans_max; itrans++, ihidden++)
							{
								long int ihidden_over = ihidden * sp.nr_oversampled_rot * sp.nr_oversampled_trans;
								for (long int iover_rot = 0; iover_rot < sp.nr_oversampled_rot; iover_rot++)
								{
									// Then loop over iover_trans
									for (long int iover_trans = 0; iover_trans < sp.nr_oversampled_trans; iover_trans++, ihidden_over++)
									{
										// Only exponentiate for determined values of op.Mweight
										// (this is always true in the first pass, but not so in the second pass)
										// Only deal with this sampling point if its weight was significant
										if (DIRECT_A2D_ELEM(op.Mweight, ipart, ihidden_over) < 0.)
										{
											DIRECT_A2D_ELEM(op.Mweight, ipart, ihidden_over) = 0.;
										}
										else
										{
											// Set the weight base to the probability of the parameters given the prior
											double weight = pdf_orientation[iorient] * pdf_offset[itrans];
											double diff2 = DIRECT_A2D_ELEM(op.Mweight, ipart, ihidden_over) - op.min_diff2[ipart];
											// next line because of numerical precision of exp-function
											if (diff2 > 700.) weight = 0.;
											// TODO: use tabulated exp function?
											else weight *= exp(-diff2);
											// Store the weight
											DIRECT_A2D_ELEM(op.Mweight, ipart, ihidden_over) = weight;

											// Keep track of sum and maximum of all weights for this particle
											// Later add all to exp_thisparticle_sumweight, but inside this loop sum to local thisthread_sumweight first
											exp_thisparticle_sumweight += weight;
										} // end if/else op.Mweight < 0.
									} // end loop iover_trans
								}// end loop iover_rot
							} // end loop itrans
						} // end loop ipsi
					} // end loop idir
				}                            //endif do_gpu_sumweight
			} // end loop exp_iclass
		} // end if iter==1

		//Store parameters for this particle
		op.sum_weight[ipart] = exp_thisparticle_sumweight;

#if defined(DEBUG_CUDA) && defined(__linux__)
		if (exp_thisparticle_sumweight == 0. || std::isnan(exp_thisparticle_sumweight))
		{
			printf("DEBUG_ERROR: zero sum of weights.\n");
			exit( EXIT_FAILURE );
		}
#endif

	} // end loop ipart

	if (exp_ipass==0)
	{
		op.Mcoarse_significant.resize(sp.nr_particles, XSIZE(op.Mweight));
	}

	CUDA_CPU_TIC("convert_post_kernel");
	// Now, for each particle,  find the exp_significant_weight that encompasses adaptive_fraction of op.sum_weight
	op.significant_weight.clear();
	op.significant_weight.resize(sp.nr_particles, 0.);
	for (long int ipart = 0; ipart < sp.nr_particles; ipart++)
	{
		long int part_id = baseMLO->mydata.ori_particles[op.my_ori_particle].particles_id[ipart];
		MultidimArray<FLOAT> sorted_weight;
		// Get the relevant row for this particle
		op.Mweight.getRow(ipart, sorted_weight);

		// Only select non-zero probabilities to speed up sorting
		long int np = 0;
		FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(sorted_weight)
		{
			if (DIRECT_MULTIDIM_ELEM(sorted_weight, n) > 0.)
			{
				DIRECT_MULTIDIM_ELEM(sorted_weight, np) = DIRECT_MULTIDIM_ELEM(sorted_weight, n);
				np++;
			}
		}
		sorted_weight.resize(np);

		// Sort from low to high values
		CUDA_CPU_TIC("sort");
#if defined(USE_THRUST) // Thrust seems incredibly slow in debug build this is clearly a FIXME
		thrust::sort(sorted_weight.data, sorted_weight.data + np);
#else
		sorted_weight.sort();
#endif
		CUDA_CPU_TOC("sort");

		double frac_weight = 0.;
		double my_significant_weight;
		long int my_nr_significant_coarse_samples = 0;
		for (long int i = XSIZE(sorted_weight) - 1; i >= 0; i--)
		{
			if (exp_ipass==0) my_nr_significant_coarse_samples++;
			my_significant_weight = DIRECT_A1D_ELEM(sorted_weight, i);
			frac_weight += my_significant_weight;
			if (frac_weight > baseMLO->adaptive_fraction * op.sum_weight[ipart])
				break;
		}

		if (exp_ipass==0 && my_nr_significant_coarse_samples == 0)
		{
			std::cerr << " ipart= " << ipart << " adaptive_fraction= " << baseMLO->adaptive_fraction << std::endl;
			std::cerr << " frac-weight= " << frac_weight << std::endl;
			std::cerr << " op.sum_weight[ipart]= " << op.sum_weight[ipart] << std::endl;
			Image<FLOAT> It;
			std::cerr << " XSIZE(op.Mweight)= " << XSIZE(op.Mweight) << std::endl;
			It()=op.Mweight;
			It() *= 10000;
			It.write("Mweight2.spi");
			std::cerr << "written Mweight2.spi" << std::endl;
			std::cerr << " np= " << np << std::endl;
			It()=sorted_weight;
			It() *= 10000;
			std::cerr << " XSIZE(sorted_weight)= " << XSIZE(sorted_weight) << std::endl;
			if (XSIZE(sorted_weight) > 0)
			{
				It.write("sorted_weight.spi");
				std::cerr << "written sorted_weight.spi" << std::endl;
			}
			REPORT_ERROR("my_nr_significant_coarse_samples == 0");
		}

		if (exp_ipass==0)
		{
			// Store nr_significant_coarse_samples for this particle
			DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_NR_SIGN) = (double)my_nr_significant_coarse_samples;

			// Keep track of which coarse samplings were significant were significant for this particle
			for (int ihidden = 0; ihidden < XSIZE(op.Mcoarse_significant); ihidden++)
			{
				if (DIRECT_A2D_ELEM(op.Mweight, ipart, ihidden) >= my_significant_weight)
					DIRECT_A2D_ELEM(op.Mcoarse_significant, ipart, ihidden) = true;
				else
					DIRECT_A2D_ELEM(op.Mcoarse_significant, ipart, ihidden) = false;
			}

		}
		op.significant_weight[ipart] = my_significant_weight;
	} // end loop ipart
	CUDA_CPU_TOC("convert_post_kernel");

}

__global__ void cuda_kernel_wavg(
		FLOAT *g_refs_real,
		FLOAT *g_refs_imag,
		FLOAT *g_imgs_real,
		FLOAT *g_imgs_imag,
		FLOAT *g_imgs_nomask_real,
		FLOAT *g_imgs_nomask_imag,
		FLOAT* g_weights,
		FLOAT* g_ctfs,
		FLOAT* g_Minvsigma2s,
		FLOAT *g_wdiff2s_parts,
		FLOAT *g_wavgs_real,
		FLOAT *g_wavgs_imag,
		FLOAT* g_Fweights,
		unsigned long translation_num,
		FLOAT weight_norm,
		FLOAT significant_weight,
		unsigned image_size,
		bool refs_are_ctf_corrected)
{
	unsigned long iorient = blockIdx.y*gridDim.x + blockIdx.x;
	unsigned tid = threadIdx.x;

	unsigned pass_num(ceilf((float)image_size/(float)BLOCK_SIZE)),pixel;
	FLOAT Fweight;
	__shared__ FLOAT s_wavgs_real[BLOCK_SIZE];
	__shared__ FLOAT s_wavgs_imag[BLOCK_SIZE];
	__shared__ FLOAT s_wdiff2s_parts[BLOCK_SIZE];
	__shared__ FLOAT s_Minvsigma2s[BLOCK_SIZE];
	for (unsigned pass = 0; pass < pass_num; pass ++)
	{
		s_wavgs_real[tid]  = 0.0f;
		s_wavgs_imag[tid]  = 0.0f;
		s_wdiff2s_parts[tid] = 0.0f;
		Fweight = 0;

		pixel = pass * BLOCK_SIZE + tid;
		s_Minvsigma2s[tid]=g_Minvsigma2s[pixel];

		if (pixel < image_size)
		{
			unsigned long orientation_pixel = iorient * image_size + pixel;
			FLOAT ref_real = g_refs_real[orientation_pixel];
			FLOAT ref_imag = g_refs_imag[orientation_pixel];
			if (refs_are_ctf_corrected) //FIXME Create two kernels for the different cases
			{
				ref_real *= __ldg(&g_ctfs[pixel]);
				ref_imag *= __ldg(&g_ctfs[pixel]);
			}

			for (unsigned long itrans = 0; itrans < translation_num; itrans++)
			{
				FLOAT weight = __ldg(&g_weights[iorient * translation_num + itrans]);

				if (weight >= significant_weight)
				{
					weight /= weight_norm;

					unsigned long img_pixel_idx = itrans * image_size + pixel;

					FLOAT diff_real = ref_real - g_imgs_real[img_pixel_idx];    // TODO  Put in texture (in such a way that fetching of next image might hit in cache)
					FLOAT diff_imag = ref_imag - g_imgs_imag[img_pixel_idx];

					s_wdiff2s_parts[tid] += weight * (diff_real*diff_real + diff_imag*diff_imag);

					FLOAT weightxinvsigma2 = weight * __ldg(&g_ctfs[pixel]) * s_Minvsigma2s[tid];

					s_wavgs_real[tid] += g_imgs_nomask_real[img_pixel_idx] * weightxinvsigma2;    // TODO  Put in texture (in such a way that fetching of next image might hit in cache)
					s_wavgs_imag[tid] += g_imgs_nomask_imag[img_pixel_idx] * weightxinvsigma2;

					Fweight += weightxinvsigma2 * __ldg(&g_ctfs[pixel]);
				}
			}

			g_wavgs_real[orientation_pixel] += s_wavgs_real[tid];
			g_wavgs_imag[orientation_pixel] += s_wavgs_imag[tid];
			g_wdiff2s_parts[orientation_pixel] = s_wdiff2s_parts[tid]; //TODO this could be further reduced in here
			g_Fweights[orientation_pixel] += Fweight; //TODO should be buffered into shared
		}
	}
}


__global__ void cuda_kernel_wavg_fast(
		FLOAT *g_refs_real,
		FLOAT *g_refs_imag,
		FLOAT *g_imgs_real,
		FLOAT *g_imgs_imag,
		FLOAT *g_imgs_nm_real,
		FLOAT *g_imgs_nm_imag,
		FLOAT* g_weights,
		FLOAT* g_ctfs,
		FLOAT* g_Minvsigma2s,
		FLOAT *g_wdiff2s_parts,
		FLOAT *g_wavgs_real,
		FLOAT *g_wavgs_imag,
		FLOAT* g_Fweights,
		unsigned long translation_num, FLOAT weight_norm,
		FLOAT significant_weight, unsigned image_size,
		bool refs_are_ctf_corrected)
{
	// Internal
	__shared__ FLOAT s_wavgs_real[REF_GROUP_SIZE*BLOCK_SIZE];
	__shared__ FLOAT s_wavgs_imag[REF_GROUP_SIZE*BLOCK_SIZE];
	__shared__ FLOAT s_Fweight[REF_GROUP_SIZE*BLOCK_SIZE];
	__shared__ FLOAT s_wdiff2s_parts[REF_GROUP_SIZE*BLOCK_SIZE];

	// External to load in
	__shared__ FLOAT s_refs_real[REF_GROUP_SIZE*BLOCK_SIZE];
	__shared__ FLOAT s_refs_imag[REF_GROUP_SIZE*BLOCK_SIZE];
	__shared__ FLOAT s_imgs_real[BLOCK_SIZE];
	__shared__ FLOAT s_imgs_imag[BLOCK_SIZE];
	__shared__ FLOAT s_imgs_nm_real[BLOCK_SIZE];
	__shared__ FLOAT s_imgs_nm_imag[BLOCK_SIZE];

	// TODO Consider Mresol_fine to speed up this kernel

    FLOAT val;
	unsigned long iorient = blockIdx.y*gridDim.x + blockIdx.x;
	unsigned tid = threadIdx.x;
	unsigned pass_num(ceilf((float)image_size/(float)BLOCK_SIZE)),pixel;


	for (unsigned pass = 0; pass < pass_num; pass ++)
	{
		//Zero everythin that is reduced (+=) in any one pass
		for (int iref=0; iref<REF_GROUP_SIZE; iref++)
		{
			s_wdiff2s_parts[iref*BLOCK_SIZE+tid]= 0;
			s_Fweight[iref*BLOCK_SIZE+tid]      = 0;
			s_wavgs_real[iref*BLOCK_SIZE+tid]   = 0;
			s_wavgs_imag[iref*BLOCK_SIZE+tid]   = 0;
		}
		pixel = pass * BLOCK_SIZE + tid;
		FLOAT ctf = g_ctfs[pixel]; // TODO Put in read-only cache
		if (pixel < image_size)
		{
			// Make REF_GROUP_SIZE*image_size the new basic unit and load in BLOCK_SIZE
			// elements from REF_GROUP_SIZE references into shared memory. iorient no
			// longer indexes orientations, but blocks of REF_GROUP_SIZE orientations.
			unsigned long orientation_pixel = iorient * (REF_GROUP_SIZE*image_size) + pixel;
			for (int iref=0; iref<REF_GROUP_SIZE; iref++)
			{
				s_refs_real[iref*BLOCK_SIZE+tid] = g_refs_real[orientation_pixel+iref*image_size];   //---------------------------
				s_refs_imag[iref*BLOCK_SIZE+tid] = g_refs_imag[orientation_pixel+iref*image_size];   //---------------------------
				if (refs_are_ctf_corrected) // Correct if needed FIXME Create two kernels for the different cases
				{
					s_refs_real[iref*BLOCK_SIZE+tid] *= ctf;
					s_refs_imag[iref*BLOCK_SIZE+tid] *= ctf;
				}
			}
			// Now go through all translations, reducing to REF_GROUP_SIZE number of outputs at each one
			for (unsigned long itrans = 0; itrans < translation_num; itrans++)
			{
				unsigned long img_pixel_idx = itrans * image_size + pixel;
				s_imgs_real[tid]    = g_imgs_real[img_pixel_idx];  //---------------------------
				s_imgs_imag[tid]    = g_imgs_imag[img_pixel_idx];  //---------------------------
				s_imgs_nm_real[tid] = g_imgs_nm_real[img_pixel_idx];  //---------------------------
				s_imgs_nm_imag[tid] = g_imgs_nm_imag[img_pixel_idx];  //---------------------------

				for (int iref=0; iref<REF_GROUP_SIZE; iref++)
				{
					FLOAT weight = g_weights[(iorient*REF_GROUP_SIZE+iref) * translation_num + itrans]; //TODO load all neccessary weights at once to avoid global access inside a deep loop OR put in read-only cache
					if (weight >= significant_weight)
					{
						weight /= weight_norm;

						val = s_refs_real[iref*BLOCK_SIZE+tid] - s_imgs_real[tid];
						s_wdiff2s_parts[iref*BLOCK_SIZE+tid] += weight * val*val;
						val = s_refs_imag[iref*BLOCK_SIZE+tid] - s_imgs_imag[tid];
						s_wdiff2s_parts[iref*BLOCK_SIZE+tid] += weight * val*val;

						val = weight * ctf * g_Minvsigma2s[pixel];

						s_wavgs_real[iref*BLOCK_SIZE+tid] += s_imgs_nm_real[tid] * val;
						s_wavgs_imag[iref*BLOCK_SIZE+tid] += s_imgs_nm_imag[tid] * val;

						s_Fweight[iref*BLOCK_SIZE+tid] += val * ctf;
					}
				}
			}
			// Now write shared memory data to global memory for all REF_GROUP_SIZE images in this block
			for (int iref=0; iref<REF_GROUP_SIZE; iref++)
			{
				g_wavgs_real[orientation_pixel+iref*image_size]    += s_wavgs_real[iref*BLOCK_SIZE+tid]; //TODO should be buffered into shared    // **
				g_wavgs_imag[orientation_pixel+iref*image_size]    += s_wavgs_imag[iref*BLOCK_SIZE+tid]; //TODO should be buffered into shared    // **
				g_wdiff2s_parts[orientation_pixel+iref*image_size] = s_wdiff2s_parts[iref*BLOCK_SIZE+tid]; //TODO this could be further reduced in here
				g_Fweights[orientation_pixel+iref*image_size]      += s_Fweight[iref*BLOCK_SIZE+tid]; //TODO should be buffered into shared
			}
		} // endif(pixel < image__size)
	} // endfor(pass)
}

// Stacks images in place, reducing at most 2*gridDim.x images down to gridDim.x images.
// Ex; 19 -> 16 or 32 -> 16,
__global__ void cuda_kernel_reduce_wdiff2s(FLOAT *g_wdiff2s_parts,
										   long int orientation_num,
										   int image_size,
										   int current_block_num)
{
	unsigned long bid = blockIdx.y*gridDim.x + blockIdx.x;
	unsigned tid = threadIdx.x;
	unsigned pass_num(ceilf((float)image_size/(float)BLOCK_SIZE)),pixel;
	if((current_block_num+bid)<orientation_num)
	{
		for (unsigned pass = 0; pass < pass_num; pass++)
		{
			pixel = pass * BLOCK_SIZE + tid;
			if(pixel<image_size)
				g_wdiff2s_parts[bid*image_size+pixel] += g_wdiff2s_parts[(current_block_num+bid)*image_size+pixel];
		}
	}
}

__global__ void cuda_kernel_collect2(	FLOAT *g_oo_otrans_x,          // otrans-size -> make const
										FLOAT *g_oo_otrans_y,          // otrans-size -> make const
										FLOAT *g_myp_oo_otrans_x2y2z2, // otrans-size -> make const
										FLOAT *g_Mweight,
										FLOAT op_significant_weight,    // TODO Put in const
										FLOAT op_sum_weight,            // TODO Put in const
										int   coarse_trans,
										int   oversamples_trans,
										int   oversamples_orient,
										int   oversamples,
										bool  do_ignore_pdf_direction,
										FLOAT *g_weights,
										FLOAT *g_thr_wsum_prior_offsetx_class,
										FLOAT *g_thr_wsum_prior_offsety_class,
										FLOAT *g_thr_wsum_sigma2_offset
										)
{
	// objects reduced in this kernel, which need to be further reduced for all blocks
	// after the kernel has finished. Notice that all reductions span fine sampling =>
	// a block can treat all fine samples in a coarse orientation and output a single
	// floating point value for each reduction. We do however list the dimension of
	// post-kernel reduction for all reductions here:
	__shared__ FLOAT                      s_weights[SUM_BLOCK_SIZE];
	__shared__ FLOAT s_thr_wsum_prior_offsetx_class[SUM_BLOCK_SIZE];
	__shared__ FLOAT s_thr_wsum_prior_offsety_class[SUM_BLOCK_SIZE];
	__shared__ FLOAT       s_thr_wsum_sigma2_offset[SUM_BLOCK_SIZE];

	int ex  = blockIdx.x * gridDim.y + blockIdx.y;            // coarse orientation
	int tid = threadIdx.x;
	int pass_num = ceil((float)oversamples / (float)SUM_BLOCK_SIZE);
	//Where to start in g_Mweight to find all data for this *coarse* orientation
	long int ref_Mweight_idx = ex * ( coarse_trans*oversamples );

	int iover_trans = tid % oversamples_trans;
	int iover_rot = floor((float) tid / (float)oversamples_trans);
	s_weights[tid]                      = 0.0f;
	s_thr_wsum_prior_offsetx_class[tid] = 0.0f;
	s_thr_wsum_prior_offsety_class[tid] = 0.0f;
	s_thr_wsum_sigma2_offset[tid]       = 0.0f;

	// Go over all (21 typically) *coarse* translations, reducing in place
	for (int itrans=0; itrans<coarse_trans; itrans++)
	{
		//Where to start in g_Mweights to find all fine samples for this *coarse* translation
		int pos = ref_Mweight_idx + itrans*oversamples + iover_rot*oversamples_trans + iover_trans;
		for (int pass = 0; pass < pass_num; pass++, pos+=SUM_BLOCK_SIZE)
		{
			FLOAT weight = g_Mweight[pos];
			if( weight >= op_significant_weight ) //TODO Might be slow (divergent threads)
				weight /= op_sum_weight;
			else
				weight = 0.0f;

			s_weights[tid] += weight;
			s_thr_wsum_prior_offsetx_class[tid] +=    weight * g_oo_otrans_x[iover_trans + itrans*oversamples_trans];    // precalc otrans_y, only overtrans-size => const
			s_thr_wsum_prior_offsety_class[tid] +=    weight * g_oo_otrans_y[iover_trans + itrans*oversamples_trans];    // precalc otrans_y, only overtrans-size => const
			s_thr_wsum_sigma2_offset[tid] += weight * g_myp_oo_otrans_x2y2z2[iover_trans + itrans*oversamples_trans];    // precalc x2y2z2,   only overtrans-size => const
		}
	}
	// Reduction of all fine samples in this coarse orientation
	for(int j=(SUM_BLOCK_SIZE/2); j>0; j/=2)
	{
		if(tid<j)
		{
			s_weights[tid]                      += s_weights[tid+j];
			s_thr_wsum_prior_offsetx_class[tid] += s_thr_wsum_prior_offsetx_class[tid+j];
			s_thr_wsum_prior_offsety_class[tid] += s_thr_wsum_prior_offsety_class[tid+j];
			s_thr_wsum_sigma2_offset[tid]       += s_thr_wsum_sigma2_offset[tid+j];
		}
		__syncthreads();
	}
	// write pre-reduced (for all fine samples and itrans) to global mem.
	g_weights[ex]			           = s_weights[0];
	g_thr_wsum_prior_offsetx_class[ex] = s_thr_wsum_prior_offsetx_class[0];
	g_thr_wsum_prior_offsety_class[ex] = s_thr_wsum_prior_offsety_class[0];
	g_thr_wsum_sigma2_offset[ex]       = s_thr_wsum_sigma2_offset[0];
}

dim3 runWavgKernel(CudaGlobalPtr<FLOAT> &Frefs_real,
				   CudaGlobalPtr<FLOAT> &Frefs_imag,
				   CudaGlobalPtr<FLOAT> &Fimgs_real,
				   CudaGlobalPtr<FLOAT> &Fimgs_imag,
				   CudaGlobalPtr<FLOAT> &Fimgs_nomask_real,
				   CudaGlobalPtr<FLOAT> &Fimgs_nomask_imag,
				   CudaGlobalPtr<FLOAT> &sorted_weights,
				   CudaGlobalPtr<FLOAT> &ctfs,
				   CudaGlobalPtr<FLOAT> &Minvsigma2s,
				   CudaGlobalPtr<FLOAT> &wdiff2s_parts,
				   CudaGlobalPtr<FLOAT> &wavgs_real,
				   CudaGlobalPtr<FLOAT> &wavgs_imag,
				   CudaGlobalPtr<FLOAT> &Fweights,
				   OptimisationParamters op,
				   MlOptimiser *baseMLO,
				   long unsigned orientation_num,
				   long unsigned translation_num,
				   unsigned image_size,
				   long int ipart,
				   int group_id,
				   int exp_iclass)
{
	/*======================================================
			            KERNEL CALL
	======================================================*/

	if (baseMLO->do_map)
		for (unsigned i = 0; i < image_size; i++)
			Minvsigma2s[i] = op.local_Minvsigma2s[ipart].data[i];
	else //TODO should be handled by memset
		for (unsigned i = 0; i < image_size; i++)
			Minvsigma2s[i] = 1;

	Minvsigma2s.cp_to_device();

	unsigned orient1, orient2;
	//We only want as many blocks as there are chunks of orientations to be treated
	//within the same block (this is done to reduce memory loads in the kernel).
	unsigned orientation_chunks = orientation_num;//ceil((float)orientation_num/(float)REF_GROUP_SIZE);
	if(orientation_chunks>65535)
	{
		orient1 = ceil(sqrt(orientation_chunks));
		orient2 = orient1;
	}
	else
	{
		orient1 = orientation_chunks;
		orient2 = 1;
	}
	dim3 block_dim(orient1,orient2);

	CUDA_GPU_TIC("cuda_kernel_wavg");

	//hipFuncSetCacheConfig(reinterpret_cast<const void*>(cuda_kernel_wavg_fast), hipFuncCachePreferShared);
	cuda_kernel_wavg<<<block_dim,BLOCK_SIZE>>>(
										~Frefs_real, ~Frefs_imag, ~Fimgs_real, ~Fimgs_imag,
										~Fimgs_nomask_real, ~Fimgs_nomask_imag,
										~sorted_weights, ~ctfs, ~Minvsigma2s,
										~wdiff2s_parts,
										~wavgs_real,
										~wavgs_imag,
										~Fweights,
										translation_num,
										(FLOAT) op.sum_weight[ipart],
										(FLOAT) op.significant_weight[ipart],
										image_size,
										baseMLO->refs_are_ctf_corrected
										);
	size_t avail;
	size_t total;
	hipMemGetInfo( &avail, &total );
	float used = 100*((float)(total - avail)/(float)total);
	std::cerr << "Device memory used @ wavg: " << used << "%" << std::endl;
	CUDA_GPU_TAC("cuda_kernel_wavg");

	HANDLE_ERROR(hipDeviceSynchronize()); //TODO Apparently this is not required here

	CUDA_GPU_TOC("cuda_kernel_wavg");

	Fimgs_real.free_device();
	Fimgs_imag.free_device();
	Fimgs_nomask_real.free_device();
	Fimgs_nomask_imag.free_device();

	sorted_weights.free_device();
	ctfs.free_device();
	Minvsigma2s.free_device();
	return(block_dim);
}

void MlOptimiserCuda::storeWeightedSums(OptimisationParamters &op, SamplingParameters &sp)
{
	CUDA_CPU_TIC("store_pre_gpu");

	// Re-do below because now also want unmasked images AND if (stricht_highres_exp >0.) then may need to resize
	baseMLO->precalculateShiftedImagesCtfsAndInvSigma2s(true, op.my_ori_particle, sp.current_image_size, sp.current_oversampling,
			sp.itrans_min, sp.itrans_max, op.Fimgs, op.Fimgs_nomask, op.Fctfs, op.local_Fimgs_shifted, op.local_Fimgs_shifted_nomask,
			op.local_Fctfs, op.local_sqrtXi2, op.local_Minvsigma2s);

	// In doThreadPrecalculateShiftedImagesCtfsAndInvSigma2s() the origin of the op.local_Minvsigma2s was omitted.
	// Set those back here
	for (long int ipart = 0; ipart < sp.nr_particles; ipart++)
	{
		long int part_id = baseMLO->mydata.ori_particles[op.my_ori_particle].particles_id[ipart];
		int group_id = baseMLO->mydata.getGroupId(part_id);
		DIRECT_MULTIDIM_ELEM(op.local_Minvsigma2s[ipart], 0) = 1. / (baseMLO->sigma2_fudge * DIRECT_A1D_ELEM(baseMLO->mymodel.sigma2_noise[group_id], 0));
	}

	// Initialise the maximum of all weights to a negative value
	op.max_weight.clear();
	op.max_weight.resize(sp.nr_particles, -1.);

	// For norm_correction and scale_correction of all particles of this ori_particle
	std::vector<double> exp_wsum_norm_correction;
	std::vector<MultidimArray<double> > exp_wsum_scale_correction_XA, exp_wsum_scale_correction_AA;
	std::vector<MultidimArray<double> > thr_wsum_signal_product_spectra, thr_wsum_reference_power_spectra;
	exp_wsum_norm_correction.resize(sp.nr_particles, 0.);

	// For scale_correction
	if (baseMLO->do_scale_correction)
	{
		MultidimArray<double> aux;
		aux.initZeros(baseMLO->mymodel.ori_size/2 + 1);
		exp_wsum_scale_correction_XA.resize(sp.nr_particles, aux);
		exp_wsum_scale_correction_AA.resize(sp.nr_particles, aux);
		thr_wsum_signal_product_spectra.resize(baseMLO->mymodel.nr_groups, aux);
		thr_wsum_reference_power_spectra.resize(baseMLO->mymodel.nr_groups, aux);
	}


	std::vector<double> oversampled_translations_x, oversampled_translations_y, oversampled_translations_z;
	Matrix2D<double> A;
	MultidimArray<FLOAT > Fimg_real, Fimg_imag;
	MultidimArray<Complex > Fimg, Fimg_otfshift_nomask;  //TODO remove, currently needed for Fourier stuff, which is based on the complex class
	MultidimArray<double> Fweight, Minvsigma2, Mctf;
	bool have_warned_small_scale = false;

	Fimg_real.resize(op.Fimgs[0]);
	Fimg_imag.resize(op.Fimgs[0]);
	Fimg.resize(op.Fimgs[0]);
	Fweight.resize(op.Fimgs[0]);

	// Initialise Mctf to all-1 for if !do_ctf_corection
	Mctf.resize(op.Fimgs[0]);
	Mctf.initConstant(1.);
	// Initialise Minvsigma2 to all-1 for if !do_map
	Minvsigma2.resize(op.Fimgs[0]);
	Minvsigma2.initConstant(1.);

	// Make local copies of weighted sums (except BPrefs, which are too big)
	// so that there are not too many mutex locks below
	std::vector<MultidimArray<double> > thr_wsum_sigma2_noise, thr_wsum_pdf_direction;
	std::vector<double> thr_wsum_norm_correction, thr_sumw_group, thr_wsum_pdf_class, thr_wsum_prior_offsetx_class, thr_wsum_prior_offsety_class;
	double thr_wsum_sigma2_offset;
	MultidimArray<double> thr_metadata, zeroArray;
	// Wsum_sigma_noise2 is a 1D-spectrum for each group
	zeroArray.initZeros(baseMLO->mymodel.ori_size/2 + 1);
	thr_wsum_sigma2_noise.resize(baseMLO->mymodel.nr_groups, zeroArray);
	// wsum_pdf_direction is a 1D-array (of length sampling.NrDirections()) for each class
	zeroArray.initZeros(baseMLO->sampling.NrDirections());
	thr_wsum_pdf_direction.resize(baseMLO->mymodel.nr_classes, zeroArray);
	// sumw_group is a double for each group
	thr_sumw_group.resize(baseMLO->mymodel.nr_groups, 0.);
	// wsum_pdf_class is a double for each class
	thr_wsum_pdf_class.resize(baseMLO->mymodel.nr_classes, 0.);
	if (baseMLO->mymodel.ref_dim == 2)
	{
		thr_wsum_prior_offsetx_class.resize(baseMLO->mymodel.nr_classes, 0.);
		thr_wsum_prior_offsety_class.resize(baseMLO->mymodel.nr_classes, 0.);
	}
	// wsum_sigma2_offset is just a double
	thr_wsum_sigma2_offset = 0.;

	unsigned image_size = op.Fimgs[0].nzyxdim;

	CUDA_CPU_TOC("store_pre_gpu");

	// Loop from iclass_min to iclass_max to deal with seed generation in first iteration
	for (int exp_iclass = sp.iclass_min; exp_iclass <= sp.iclass_max; exp_iclass++)
	{

		/*=======================================================================================
		                            REFERENCE PROJECTION GENERATION
		=======================================================================================*/

		// Since we will need the euler matrices for all projections in the data_collect stage,
		// we might as well make it wider in scope and retain it on the GPU until then. When we
		// switch from pair to bool, there won't be any need to remake it every class, but for
		// now we create only those matrices corresponding to significant orientations, which IS  * class-specific *

		std::vector< long unsigned > iorientclasses, iover_rots;
		std::vector< double > rots, tilts, psis;

		CUDA_CPU_TIC("projection_2");

		long unsigned orientation_num = generateProjectionSetup(
					op,
					sp,
					baseMLO,
					false,  //coarse
					exp_iclass,
					rots, tilts, psis,
					iorientclasses,
					iover_rots);


		CudaGlobalPtr<FLOAT> eulers(9 * orientation_num);

		generateEulerMatrices(
				baseMLO->mymodel.PPref[exp_iclass].padding_factor,
				rots,
				tilts,
				psis,
				eulers,
				!IS_NOT_INV);

	    eulers.device_alloc();
		eulers.cp_to_device();

		CudaGlobalPtr<FLOAT > model_real((baseMLO->mymodel.PPref[exp_iclass]).data.nzyxdim);
		CudaGlobalPtr<FLOAT > model_imag((baseMLO->mymodel.PPref[exp_iclass]).data.nzyxdim);

		for(unsigned i = 0; i < model_real.size; i++)
		{
			model_real[i] = (FLOAT) baseMLO->mymodel.PPref[exp_iclass].data.data[i].real;
			model_imag[i] = (FLOAT) baseMLO->mymodel.PPref[exp_iclass].data.data[i].imag;
		}

		CudaGlobalPtr<FLOAT> Frefs_real;
		CudaGlobalPtr<FLOAT> Frefs_imag;


		generateModelProjections(
				model_real,
				model_imag,
				Frefs_real,
				Frefs_imag,
				eulers,
				orientation_num,
				image_size,
				XMIPP_MIN(baseMLO->mymodel.PPref[exp_iclass].r_max, op.local_Minvsigma2s[0].xdim - 1),
				op.local_Minvsigma2s[0].xdim,
				op.local_Minvsigma2s[0].ydim,
				baseMLO->mymodel.PPref[exp_iclass].data.xdim,
				baseMLO->mymodel.PPref[exp_iclass].data.ydim,
				baseMLO->mymodel.PPref[exp_iclass].data.zdim,
				baseMLO->mymodel.PPref[exp_iclass].data.yinit,
				baseMLO->mymodel.PPref[exp_iclass].data.zinit);

		model_real.free_device();
		model_imag.free_device();
		eulers.free();


		CUDA_CPU_TOC("projection_2");

		CudaGlobalPtr<FLOAT> wavgs_real(orientation_num * image_size);
		wavgs_real.device_alloc();
		wavgs_real.device_init(0);
		CudaGlobalPtr<FLOAT> wavgs_imag(orientation_num * image_size);
		wavgs_imag.device_alloc();
		wavgs_imag.device_init(0);

		CudaGlobalPtr<FLOAT> Fweights(orientation_num * image_size);
		Fweights.device_alloc();
		Fweights.device_init(0);

		/*=======================================================================================
										  PARTICLE ITERATION
		=======================================================================================*/

		/// Now that reference projection has been made loop over all particles inside this ori_particle
		for (long int ipart = 0; ipart < sp.nr_particles; ipart++)
		{
			long int part_id = baseMLO->mydata.ori_particles[op.my_ori_particle].particles_id[ipart];
			int group_id = baseMLO->mydata.getGroupId(part_id);

			double myprior_x, myprior_y, myprior_z;
			double old_offset_x = XX(op.old_offset[ipart]);
			double old_offset_y = YY(op.old_offset[ipart]);
			double old_offset_z;

			if (baseMLO->mymodel.ref_dim == 2)
			{
				myprior_x = XX(baseMLO->mymodel.prior_offset_class[exp_iclass]);
				myprior_y = YY(baseMLO->mymodel.prior_offset_class[exp_iclass]);
			}
			else
			{
				myprior_x = XX(op.prior[ipart]);
				myprior_y = YY(op.prior[ipart]);
				if (baseMLO->mymodel.data_dim == 3)
				{
					myprior_z = ZZ(op.prior[ipart]);
					old_offset_z = ZZ(op.old_offset[ipart]);
				}
			}


			/*======================================================
								 TRANSLATIONS
			======================================================*/

			CUDA_CPU_TIC("translation_2");

			CudaGlobalPtr<FLOAT> Fimgs_real(image_size * sp.nr_trans * sp.nr_oversampled_trans);
			CudaGlobalPtr<FLOAT> Fimgs_imag(Fimgs_real.size);
			CudaGlobalPtr<FLOAT> Fimgs_nomask_real(Fimgs_real.size);
			CudaGlobalPtr<FLOAT> Fimgs_nomask_imag(Fimgs_real.size);

			std::vector< long unsigned > iover_transes, itranses, ihiddens;

			long unsigned translation_num = imageTranslation(
					Fimgs_real,
					Fimgs_imag,
					Fimgs_nomask_real,
					Fimgs_nomask_imag,
					sp.itrans_min,
					sp.itrans_max,
					baseMLO->adaptive_oversampling ,
					baseMLO->sampling,
					oversampled_translations_x,
					oversampled_translations_y,
					oversampled_translations_z,
					sp.nr_oversampled_trans,
					baseMLO->global_fftshifts_ab_current,
					baseMLO->global_fftshifts_ab2_current,
					op.local_Fimgs_shifted[ipart],
					op.local_Fimgs_shifted_nomask[ipart],
					iover_transes,
					itranses,
					ihiddens,
					image_size);

			Fimgs_real.device_alloc();
			Fimgs_real.cp_to_device();
			Fimgs_imag.device_alloc();
			Fimgs_imag.cp_to_device();

			Fimgs_nomask_real.device_alloc();
			Fimgs_nomask_real.cp_to_device();
			Fimgs_nomask_imag.device_alloc();
			Fimgs_nomask_imag.cp_to_device();

			CUDA_CPU_TOC("translation_2");


			/*======================================================
					            	SCALE
			======================================================*/

			CUDA_CPU_TIC("scale_ctf");
			FLOAT part_scale(1.);

			if (baseMLO->do_scale_correction)
			{
				part_scale = baseMLO->mymodel.scale_correction[group_id];
				if (part_scale > 10000.)
				{
					std::cerr << " rlnMicrographScaleCorrection= " << part_scale << " group= " << group_id + 1 << std::endl;
					REPORT_ERROR("ERROR: rlnMicrographScaleCorrection is very high. Did you normalize your data?");
				}
				else if (part_scale < 0.001)
				{
					if (!have_warned_small_scale)
					{
						std::cout << " WARNING: ignoring group " << group_id + 1 << " with very small or negative scale (" << part_scale <<
								"); Use larger groups for more stable scale estimates." << std::endl;
						have_warned_small_scale = true;
					}
					part_scale = 0.001;
				}
			}

			CudaGlobalPtr<FLOAT> ctfs(image_size); //TODO Same size for all iparts, should be allocated once
			ctfs.device_alloc();

			if (baseMLO->do_ctf_correction)
			{
				for (unsigned i = 0; i < image_size; i++)
					ctfs[i] = (FLOAT) op.local_Fctfs[ipart].data[i] * part_scale;
			}
			else //TODO should be handled by memset
				for (unsigned i = 0; i < image_size; i++)
					ctfs[i] = part_scale;

			ctfs.cp_to_device();
			CUDA_CPU_TOC("scale_ctf");

			/*======================================================
					            MAP WEIGHTS
			======================================================*/

			CUDA_CPU_TIC("map");
			CudaGlobalPtr<FLOAT> sorted_weights(orientation_num * translation_num);

			mapWeights(
					sorted_weights,
					orientation_num,
					translation_num,
					baseMLO->sampling,
					ipart,
					iover_transes,
					ihiddens,
					iorientclasses,
					iover_rots,
					op.Mweight,
					sp.current_oversampling,
					sp.nr_trans);

			sorted_weights.device_alloc();
			sorted_weights.cp_to_device();
			sorted_weights.free_host();

			CUDA_CPU_TOC("map");

			/*======================================================
								KERNEL CALL
			======================================================*/

			// The below allocations are kept outside runWavgKernel(...) in case we decide to make them global.
			CudaGlobalPtr<FLOAT> Minvsigma2s(image_size); //TODO Same size for all iparts, should be allocated once
			Minvsigma2s.device_alloc();
			CudaGlobalPtr<FLOAT> wdiff2s_parts(orientation_num * image_size); //TODO Almost same size for all iparts, should be allocated once
			wdiff2s_parts.device_alloc();

			dim3 block_dim = runWavgKernel(Frefs_real,
						                   Frefs_imag,
						                   Fimgs_real,
						                   Fimgs_imag,
						                   Fimgs_nomask_real,
						                   Fimgs_nomask_imag,
						                   sorted_weights,
						                   ctfs,
						                   Minvsigma2s,
						                   wdiff2s_parts,
						                   wavgs_real,
						                   wavgs_imag,
						                   Fweights,
						                   op,
						                   baseMLO,
						                   orientation_num,
						                   translation_num,
						                   image_size,
						                   ipart,
						                   group_id,
						                   exp_iclass);

			/*======================================================
								COLLECT DATA
			======================================================*/

			CUDA_CPU_TIC("reduce_wdiff2s");
			// reduction_block_num = the highest possible power of two that covers more than or exactly half of all images to be reduced
			int num_reductions = (int)floor(log2((float)orientation_num));
			int reduction_block_num = pow(2,num_reductions);
			if(reduction_block_num==orientation_num) // (possibly) very special case where orientation_num is a power of 2
				reduction_block_num /= 2;

			CUDA_GPU_TIC("cuda_kernels_reduce_wdiff2s");
			unsigned orient1, orient2;
			for(int k=reduction_block_num; k>=1; k/=2) //invoke kernel repeatedly until all images have been stacked into the first image position
			{
				if(k>65535)
				{
					orient1 = ceil(sqrt(k));
					orient2 = orient1 + (orient1 % 2);  // For some reason the "optimal" values in the METADATA ooutput is sensitive to the choice of block-grid dims,
					orient1 +=          (orient1 % 2);  // and seems to work properly only when even numbers are used. // TODO examine why
				}
				else
				{
					orient1 = k;
					orient2 = 1;
				}
				dim3 block_dim_wd(orient1,orient2);
				 // TODO **OF VERY LITTLE IMPORTANCE**  One block treating just 2 images is a very innefficient amount of loads per store
				cuda_kernel_reduce_wdiff2s<<<block_dim_wd,BLOCK_SIZE>>>(~wdiff2s_parts,orientation_num,image_size,k);
			}
			CUDA_GPU_TOC("cuda_kernels_reduce_wdiff2s");

			wdiff2s_parts.size = image_size; //temporarily set the size to the single image we have now reduced, to not copy more than necessary
			wdiff2s_parts.cp_to_host();
			wdiff2s_parts.size = orientation_num * image_size;
			wdiff2s_parts.free_device();

			for (long int j = 0; j < image_size; j++)
			{
				int ires = DIRECT_MULTIDIM_ELEM(baseMLO->Mresol_fine, j);
				if (ires > -1)
				{
					thr_wsum_sigma2_noise[group_id].data[ires] += (double) wdiff2s_parts[j];
					exp_wsum_norm_correction[ipart] += (double) wdiff2s_parts[j];
				}
			}

			wdiff2s_parts.free_host();

			CUDA_CPU_TOC("reduce_wdiff2s");

			CUDA_CPU_TIC("collect_data_2");
			CUDA_CPU_TIC("collect_data_2_pre_kernel");
			//TODO should be replaced with loop over pairs of projections and translations (like in the getAllSquaredDifferences-function)

			std::vector< double> oversampled_rot, oversampled_tilt, oversampled_psi;

			int oversamples = sp.nr_oversampled_trans * sp.nr_oversampled_rot;
			CudaGlobalPtr<FLOAT >  Mweight( &(op.Mweight.data[(ipart)*(op.Mweight).xdim]),
											sp.nr_dir * sp.nr_psi * sp.nr_trans * oversamples);
			int nr_transes = sp.nr_trans*sp.nr_oversampled_trans;
			CudaGlobalPtr<FLOAT>     oo_otrans_x(nr_transes); // old_offset_oversampled_trans_x
			CudaGlobalPtr<FLOAT>     oo_otrans_y(nr_transes);
			CudaGlobalPtr<FLOAT> myp_oo_otrans_x2y2z2(nr_transes); // my_prior_old_offs....x^2*y^2*z^2

			//Pregenerate oversampled translation objects for kernel-call
			for (long int itrans = 0, iitrans = 0; itrans < sp.nr_trans; itrans++)
			{
				baseMLO->sampling.getTranslations(itrans, baseMLO->adaptive_oversampling,
						oversampled_translations_x, oversampled_translations_y, oversampled_translations_z);
				for (long int iover_trans = 0; iover_trans < sp.nr_oversampled_trans; iover_trans++, iitrans++)
				{
					oo_otrans_x[iitrans] = old_offset_x + oversampled_translations_x[iover_trans];
					oo_otrans_y[iitrans] = old_offset_y + oversampled_translations_y[iover_trans];
					double diffx = myprior_x - oo_otrans_x[iitrans];
					double diffy = myprior_y - oo_otrans_y[iitrans];
					if (baseMLO->mymodel.data_dim == 3)
					{
						double diffz = myprior_z - (old_offset_z + oversampled_translations_z[iover_trans]);
						myp_oo_otrans_x2y2z2[iitrans] = diffx*diffx + diffy*diffy + diffz*diffz ;
					}
					else
					{
						myp_oo_otrans_x2y2z2[iitrans] = diffx*diffx + diffy*diffy ;
					}
				}
			}

			Mweight.device_alloc();
			Mweight.cp_to_device();
			oo_otrans_x.device_alloc();
			oo_otrans_x.cp_to_device();
			oo_otrans_y.device_alloc();
			oo_otrans_y.cp_to_device();
			myp_oo_otrans_x2y2z2.device_alloc();
			myp_oo_otrans_x2y2z2.cp_to_device();

			CudaGlobalPtr<FLOAT>                      p_weights(block_dim.x * block_dim.y);
			CudaGlobalPtr<FLOAT> p_thr_wsum_prior_offsetx_class(block_dim.x * block_dim.y);
			CudaGlobalPtr<FLOAT> p_thr_wsum_prior_offsety_class(block_dim.x * block_dim.y);
			CudaGlobalPtr<FLOAT>       p_thr_wsum_sigma2_offset(block_dim.x * block_dim.y);

			p_weights.device_alloc();
			p_thr_wsum_prior_offsetx_class.device_alloc();
			p_thr_wsum_prior_offsety_class.device_alloc();
			p_thr_wsum_sigma2_offset.device_alloc();

			block_dim.x=sp.nr_dir;
			block_dim.y=sp.nr_psi;
			CUDA_CPU_TOC("collect_data_2_pre_kernel");
			for (int exp_iclass = sp.iclass_min; exp_iclass <= sp.iclass_max; exp_iclass++)
			{
				cuda_kernel_collect2<<<block_dim,SUM_BLOCK_SIZE>>>( ~oo_otrans_x,          // otrans-size -> make const
																	~oo_otrans_y,          // otrans-size -> make const
																	~myp_oo_otrans_x2y2z2, // otrans-size -> make const
																	~Mweight,
																	(FLOAT)op.significant_weight[ipart],
																	(FLOAT)op.sum_weight[ipart],
																	sp.nr_trans,
																	sp.nr_oversampled_trans,
																	sp.nr_oversampled_rot,
																	oversamples,
																	(baseMLO->do_skip_align || baseMLO->do_skip_rotate ),
																	~p_weights,
																	~p_thr_wsum_prior_offsetx_class,
																	~p_thr_wsum_prior_offsety_class,
																	~p_thr_wsum_sigma2_offset
																   );
				HANDLE_ERROR(hipDeviceSynchronize());

				// TODO further reduce the below 4 arrays while data is still on gpu
				p_weights.cp_to_host();
				p_thr_wsum_prior_offsetx_class.cp_to_host();
				p_thr_wsum_prior_offsety_class.cp_to_host();
				p_thr_wsum_sigma2_offset.cp_to_host();

				thr_wsum_sigma2_offset = 0.0;
				int iorient = 0;
				for (long int idir = 0; idir < sp.nr_dir; idir++)
				{
					for (long int ipsi = 0; ipsi < sp.nr_psi; ipsi++, iorient++)
					{
						long int iorientclass = exp_iclass * sp.nr_dir * sp.nr_psi + iorient;
						// Only proceed if any of the particles had any significant coarsely sampled translation

						if (baseMLO->isSignificantAnyParticleAnyTranslation(iorientclass, sp.itrans_min, sp.itrans_max, op.Mcoarse_significant))
						{
							long int mydir;
							if (baseMLO->mymodel.orientational_prior_mode == NOPRIOR)
								mydir = idir;
							else
								mydir = op.pointer_dir_nonzeroprior[idir];

							// store partials according to indices of the relevant dimension
							DIRECT_MULTIDIM_ELEM(thr_wsum_pdf_direction[exp_iclass], mydir) += p_weights[iorient];
							thr_sumw_group[group_id]                 						+= p_weights[iorient];
							thr_wsum_pdf_class[exp_iclass]           						+= p_weights[iorient];
							thr_wsum_sigma2_offset                   						+= p_thr_wsum_sigma2_offset[iorient];

							if (baseMLO->mymodel.ref_dim == 2)
							{
								thr_wsum_prior_offsetx_class[exp_iclass] 	+= p_thr_wsum_prior_offsetx_class[iorient];
								thr_wsum_prior_offsety_class[exp_iclass] 	+= p_thr_wsum_prior_offsety_class[iorient];
							}
						}
					}
				}
			}
			CUDA_CPU_TIC("collect_data_2_post_kernel");
			Mweight.free_device();
			p_weights.free();
			p_thr_wsum_sigma2_offset.free();
			p_thr_wsum_prior_offsetx_class.free();
			p_thr_wsum_prior_offsety_class.free();

			oo_otrans_y.free();
			oo_otrans_x.free();
			myp_oo_otrans_x2y2z2.free();

			//Get index of max element using GPU-tool thrust
			Indices max_index;
			max_index.fineIdx = thrust::max_element(&DIRECT_A2D_ELEM(op.Mweight, ipart, 0),&DIRECT_A2D_ELEM(op.Mweight, ipart+1, 0)) - &DIRECT_A2D_ELEM(op.Mweight, ipart, 0);
			op.max_weight[ipart] = DIRECT_A2D_ELEM(op.Mweight, ipart, max_index.fineIdx);
			max_index.fineIndexToFineIndices(sp); // set partial indices corresponding to the found max_index, to be used below

			baseMLO->sampling.getTranslations(max_index.itrans, baseMLO->adaptive_oversampling,
					oversampled_translations_x, oversampled_translations_y, oversampled_translations_z);
			baseMLO->sampling.getOrientations(max_index.idir, max_index.ipsi, baseMLO->adaptive_oversampling, oversampled_rot, oversampled_tilt, oversampled_psi,
					op.pointer_dir_nonzeroprior, op.directions_prior, op.pointer_psi_nonzeroprior, op.psi_prior);
			double rot = oversampled_rot[max_index.ioverrot];
			double tilt = oversampled_tilt[max_index.ioverrot];
			double psi = oversampled_psi[max_index.ioverrot];
			DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_ROT) = rot;
			DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_TILT) = tilt;
			DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_PSI) = psi;
			DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_XOFF) = XX(op.old_offset[ipart]) + oversampled_translations_x[max_index.iovertrans];
			DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_YOFF) = YY(op.old_offset[ipart]) + oversampled_translations_y[max_index.iovertrans];
			if (baseMLO->mymodel.data_dim == 3)
				DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_ZOFF) = ZZ(op.old_offset[ipart]) + oversampled_translations_z[max_index.iovertrans];
			DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_CLASS) = (double)max_index.iclass + 1;
			DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_PMAX) = op.max_weight[ipart]/op.sum_weight[ipart];
			CUDA_CPU_TOC("collect_data_2_post_kernel");
			CUDA_CPU_TOC("collect_data_2");

		} // end loop ipart

		Frefs_real.free_device();
		Frefs_imag.free_device();

		/*=======================================================================================
										   BACKPROJECTION
		=======================================================================================*/

		CUDA_CPU_TIC("backprojection");

		CudaGlobalPtr<FLOAT> bp_model_real(baseMLO->wsum_model.BPref[exp_iclass].data.nzyxdim);
		bp_model_real.device_alloc();
		bp_model_real.device_init(0);
		CudaGlobalPtr<FLOAT> bp_model_imag(bp_model_real.size);
		bp_model_imag.device_alloc();
		bp_model_imag.device_init(0);
		CudaGlobalPtr<FLOAT> bp_weight(bp_model_real.size);
		bp_weight.device_alloc();
		bp_weight.device_init(0);


		CudaGlobalPtr<FLOAT> bp_eulers(9 * orientation_num);

		FLOAT padding_factor = baseMLO->wsum_model.BPref[exp_iclass].padding_factor;

		generateEulerMatrices(
				1/padding_factor, //Why squared scale factor is given in backprojection
				rots,
				tilts,
				psis,
				bp_eulers,
				IS_NOT_INV);

		bp_eulers.device_alloc();
	    bp_eulers.cp_to_device();
	    bp_eulers.free_host();


		backproject(
				wavgs_real,
				wavgs_imag,
				Fweights,
				bp_eulers,
				bp_model_real,
				bp_model_imag,
				bp_weight,
				baseMLO->wsum_model.BPref[exp_iclass].r_max,
				padding_factor * padding_factor,
				image_size,
				orientation_num,
				op.local_Minvsigma2s[0].xdim,
				op.local_Minvsigma2s[0].ydim,
				baseMLO->wsum_model.BPref[exp_iclass].data.xdim,
				baseMLO->wsum_model.BPref[exp_iclass].data.ydim,
				baseMLO->wsum_model.BPref[exp_iclass].data.zdim,
				baseMLO->wsum_model.BPref[exp_iclass].data.yinit,
				baseMLO->wsum_model.BPref[exp_iclass].data.zinit);

		bp_model_real.cp_to_host();
		bp_model_imag.cp_to_host();
		bp_weight.cp_to_host();

		HANDLE_ERROR(hipDeviceSynchronize()); //TODO Optimize concurrency

		bp_model_real.free_device();
		bp_model_imag.free_device();
		bp_weight.free_device();

//#define PRINT_BACKPROJECTION_RESULTS
#ifdef PRINT_BACKPROJECTION_RESULTS

		FILE *fPtr1 = fopen("gpu_backproj_values.dat","w");
		for (unsigned i = 0; i < bp_model_real.size; i ++)
			fprintf(fPtr1, "%.1e %.1e\n", bp_model_real[i], bp_model_imag[i]);
		fclose(fPtr1);

		FILE *fPtr2 = fopen("gpu_backproj_weights.dat","w");
		for (unsigned i = 0; i < bp_weight.size; i ++)
			fprintf(fPtr2, "%.1e\n", bp_weight[i]);
		fclose(fPtr2);

		wavgs_real.cp_to_host();
		wavgs_imag.cp_to_host();
		Fweights.cp_to_host();

		for (long int i = 0; i < orientation_num; i++)
		{
			Euler_angles2matrix(rots[i], tilts[i], psis[i], A);

			for (unsigned j = 0; j < image_size; j++)
			{
				Fimg.data[j].real = (double) wavgs_real[i * image_size + j];
				Fimg.data[j].imag = (double) wavgs_imag[i * image_size + j];
				Fweight.data[j] = (double) Fweights[i * image_size + j];
			}

			int my_mutex = exp_iclass % NR_CLASS_MUTEXES;
			pthread_mutex_lock(&global_mutex2[my_mutex]);
			(baseMLO->wsum_model.BPref[exp_iclass]).set2DFourierTransform(Fimg, A, IS_NOT_INV, &Fweight);
			pthread_mutex_unlock(&global_mutex2[my_mutex]);

		}

		FILE *fPtr3 = fopen("cpu_backproj_values.dat","w");
		for (unsigned i = 0; i < (baseMLO->wsum_model.BPref[exp_iclass]).data.nzyxdim; i ++)
			fprintf(fPtr3, "%.1e %.1e\n", (baseMLO->wsum_model.BPref[exp_iclass]).data.data[i].real, (baseMLO->wsum_model.BPref[exp_iclass]).data.data[i].imag);
		fclose(fPtr3);

		FILE *fPtr4 = fopen("cpu_backproj_weights.dat","w");
		for (unsigned i = 0; i < (baseMLO->wsum_model.BPref[exp_iclass]).data.nzyxdim; i ++)
			fprintf(fPtr4, "%.1e\n", (baseMLO->wsum_model.BPref[exp_iclass]).bp_weight.data[i]);
		fclose(fPtr4);

		exit(0);
#endif

		Fweights.free();
		wavgs_real.free();
		wavgs_imag.free();

		int my_mutex = exp_iclass % NR_CLASS_MUTEXES;
		pthread_mutex_lock(&global_mutex2[my_mutex]);

		for (long unsigned i = 0; i < bp_model_real.size; i++)
		{
			baseMLO->wsum_model.BPref[exp_iclass].data.data[i].real += bp_model_real[i];
			baseMLO->wsum_model.BPref[exp_iclass].data.data[i].imag += bp_model_imag[i];
			baseMLO->wsum_model.BPref[exp_iclass].weight.data[i] += bp_weight[i];
		}

		pthread_mutex_unlock(&global_mutex2[my_mutex]);

		CUDA_CPU_TOC("backprojection");

	} // end loop iclass

	CUDA_CPU_TIC("store_post_gpu");

	// Extend norm_correction and sigma2_noise estimation to higher resolutions for all particles
	// Also calculate dLL for each particle and store in metadata
	// loop over all particles inside this ori_particle
	double thr_avg_norm_correction = 0.;
	double thr_sum_dLL = 0., thr_sum_Pmax = 0.;
	for (long int ipart = 0; ipart < sp.nr_particles; ipart++)
	{
		long int part_id = baseMLO->mydata.ori_particles[op.my_ori_particle].particles_id[ipart];
		int group_id = baseMLO->mydata.getGroupId(part_id);

		// If the current images were smaller than the original size, fill the rest of wsum_model.sigma2_noise with the power_class spectrum of the images
		for (int ires = baseMLO->mymodel.current_size/2 + 1; ires < baseMLO->mymodel.ori_size/2 + 1; ires++)
		{
			DIRECT_A1D_ELEM(thr_wsum_sigma2_noise[group_id], ires) += DIRECT_A1D_ELEM(op.power_imgs[ipart], ires);
			// Also extend the weighted sum of the norm_correction
			exp_wsum_norm_correction[ipart] += DIRECT_A1D_ELEM(op.power_imgs[ipart], ires);
		}

		// Store norm_correction
		// Multiply by old value because the old norm_correction term was already applied to the image
		if (baseMLO->do_norm_correction)
		{
			double old_norm_correction = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_NORM);
			old_norm_correction /= baseMLO->mymodel.avg_norm_correction;
			// The factor two below is because exp_wsum_norm_correctiom is similar to sigma2_noise, which is the variance for the real/imag components
			// The variance of the total image (on which one normalizes) is twice this value!
			double normcorr = old_norm_correction * sqrt(exp_wsum_norm_correction[ipart] * 2.);
			thr_avg_norm_correction += normcorr;
			// Now set the new norm_correction in the relevant position of exp_metadata
			DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_NORM) = normcorr;


			// Print warning for strange norm-correction values
			if (!(baseMLO->iter == 1 && baseMLO->do_firstiter_cc) && DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_NORM) > 10.)
			{
				std::cout << " WARNING: norm_correction= "<< DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_NORM) << " for particle " << part_id << " in group " << group_id + 1 << "; Are your groups large enough?" << std::endl;
			}

		}

		// Store weighted sums for scale_correction
		if (baseMLO->do_scale_correction)
		{
			// Divide XA by the old scale_correction and AA by the square of that, because was incorporated into Fctf
			exp_wsum_scale_correction_XA[ipart] /= baseMLO->mymodel.scale_correction[group_id];
			exp_wsum_scale_correction_AA[ipart] /= baseMLO->mymodel.scale_correction[group_id] * baseMLO->mymodel.scale_correction[group_id];

			thr_wsum_signal_product_spectra[group_id] += exp_wsum_scale_correction_XA[ipart];
			thr_wsum_reference_power_spectra[group_id] += exp_wsum_scale_correction_AA[ipart];
		}

		// Calculate DLL for each particle
		double logsigma2 = 0.;
		FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(baseMLO->Mresol_fine)
		{
			int ires = DIRECT_MULTIDIM_ELEM(baseMLO->Mresol_fine, n);
			// Note there is no sqrt in the normalisation term because of the 2-dimensionality of the complex-plane
			// Also exclude origin from logsigma2, as this will not be considered in the P-calculations
			if (ires > 0)
				logsigma2 += log( 2. * PI * DIRECT_A1D_ELEM(baseMLO->mymodel.sigma2_noise[group_id], ires));
		}
		if (op.sum_weight[ipart]==0)
		{
			std::cerr << " part_id= " << part_id << std::endl;
			std::cerr << " ipart= " << ipart << std::endl;
			std::cerr << " op.min_diff2[ipart]= " << op.min_diff2[ipart] << std::endl;
			std::cerr << " logsigma2= " << logsigma2 << std::endl;
			int group_id = baseMLO->mydata.getGroupId(part_id);
			std::cerr << " group_id= " << group_id << std::endl;
			std::cerr << " ml_model.scale_correction[group_id]= " << baseMLO->mymodel.scale_correction[group_id] << std::endl;
			std::cerr << " exp_significant_weight[ipart]= " << op.significant_weight[ipart] << std::endl;
			std::cerr << " exp_max_weight[ipart]= " << op.max_weight[ipart] << std::endl;
			std::cerr << " ml_model.sigma2_noise[group_id]= " << baseMLO->mymodel.sigma2_noise[group_id] << std::endl;
			REPORT_ERROR("ERROR: op.sum_weight[ipart]==0");
		}
		double dLL;
		if ((baseMLO->iter==1 && baseMLO->do_firstiter_cc) || baseMLO->do_always_cc)
			dLL = -op.min_diff2[ipart];
		else
			dLL = log(op.sum_weight[ipart]) - op.min_diff2[ipart] - logsigma2;

		// Store dLL of each image in the output array, and keep track of total sum
		DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_DLL) = dLL;
		thr_sum_dLL += dLL;

		// Also store sum of Pmax
		thr_sum_Pmax += DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_PMAX);

	}

	// Now, inside a global_mutex, update the other weighted sums among all threads
	if (!baseMLO->do_skip_maximization)
	{
		pthread_mutex_lock(&global_mutex);
		for (int n = 0; n < baseMLO->mymodel.nr_groups; n++)
		{
			baseMLO->wsum_model.sigma2_noise[n] += thr_wsum_sigma2_noise[n];
			baseMLO->wsum_model.sumw_group[n] += thr_sumw_group[n];
			if (baseMLO->do_scale_correction)
			{
				baseMLO->wsum_model.wsum_signal_product_spectra[n] += thr_wsum_signal_product_spectra[n];
				baseMLO->wsum_model.wsum_reference_power_spectra[n] += thr_wsum_reference_power_spectra[n];
			}
		}
		for (int n = 0; n < baseMLO->mymodel.nr_classes; n++)
		{
			baseMLO->wsum_model.pdf_class[n] += thr_wsum_pdf_class[n];
			if (baseMLO->mymodel.ref_dim == 2)
			{
				XX(baseMLO->wsum_model.prior_offset_class[n]) += thr_wsum_prior_offsetx_class[n];
				YY(baseMLO->wsum_model.prior_offset_class[n]) += thr_wsum_prior_offsety_class[n];
			}

			if (!(baseMLO->do_skip_align || baseMLO->do_skip_rotate) )
				baseMLO->wsum_model.pdf_direction[n] += thr_wsum_pdf_direction[n];
		}
		baseMLO->wsum_model.sigma2_offset += thr_wsum_sigma2_offset;
		if (baseMLO->do_norm_correction)
			baseMLO->wsum_model.avg_norm_correction += thr_avg_norm_correction;
		baseMLO->wsum_model.LL += thr_sum_dLL;
		baseMLO->wsum_model.ave_Pmax += thr_sum_Pmax;
		pthread_mutex_unlock(&global_mutex);
	} // end if !do_skip_maximization

	CUDA_CPU_TOC("store_post_gpu");
}
