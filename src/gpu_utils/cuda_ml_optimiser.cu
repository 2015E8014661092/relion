#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <ctime>
#include <iostream>
#include "src/gpu_utils/cuda_ml_optimiser.h"
#include "src/gpu_utils/cuda_img_operations.h"
#include "src/complex.h"
#include <fstream>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 128         // This is optimally set as big as possible without its ceil:ed multiple exceeding imagesize by too much.
#define NR_CLASS_MUTEXES 5

static pthread_mutex_t global_mutex2[NR_CLASS_MUTEXES] = { PTHREAD_MUTEX_INITIALIZER };
static pthread_mutex_t global_mutex = PTHREAD_MUTEX_INITIALIZER;

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "CUDA ERROR: %s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

class CudaComplex
{
public:
	double real, imag;

	inline
	__device__ __host__ CudaComplex(): real(), imag() {};
	inline
	__device__ __host__ CudaComplex(double real, double imag): real(real), imag(imag) {};
};

class CudaImages
{
public:
	unsigned x,y,xy,num,max_num;
	CudaComplex* start;

	inline
	CudaImages(long unsigned x, long unsigned y, long unsigned max_num):
			x(x), y(y), num(0), max_num(max_num), xy(x*y), start(new CudaComplex[xy*max_num])
	{};

	inline
	CudaComplex* current() { return start + (num*xy); };

	inline
	void increment() { num++; };

	inline
	CudaComplex* operator [](long unsigned i) { return start + (i*((long unsigned) xy)); };

	inline
	long unsigned alloc_size() { return ((long unsigned) num)*((long unsigned) xy); };

	inline
	CudaComplex* data_to_device()
	{
		CudaComplex* d_ptr(0);
		HANDLE_ERROR(hipMalloc( (void**) &d_ptr, alloc_size() * sizeof(CudaComplex)));
		HANDLE_ERROR(hipMemcpy( d_ptr, start, alloc_size() * sizeof(CudaComplex), hipMemcpyHostToDevice));
		return d_ptr;
	}

	inline
	void clear() { delete[] start; }

	inline
	~CudaImages() { delete[] start; }
};

__global__ void cuda_kernel_massive_diff2(	CudaComplex *g_refs, CudaComplex *g_imgs,
									double *g_Minvsigma2, double *g_diff2s,
									const unsigned img_size, const double sum_init,
									bool *g_exp_Mcoarse_significant,
									long int orientation_num,
									long int translation_num,
									long int exp_nr_oversampled_rot,
									long int exp_nr_oversampled_trans)
{
//	int ex = blockIdx.x % orientation_num;
//	int ey = (blockIdx.x - ex) / orientation_num;
	int ex = blockIdx.y * gridDim.x + blockIdx.x;
	//int ey = blockIdx.y;
	int ez = blockIdx.z;

	unsigned long int coarse_rot_idx   = floorf(ex/exp_nr_oversampled_rot);
	unsigned long int coarse_trans_idx = floorf(ez/exp_nr_oversampled_trans);

	// 		Check if it is significant
	//          		AND
	// inside the padded 2D orientation grid
	if(g_exp_Mcoarse_significant + ex + ez*coarse_rot_idx && ex < orientation_num )
	{
		__shared__ double s[BLOCK_SIZE];
		s[threadIdx.x] = 0;

		unsigned pass_num(ceilf((float)img_size/(float)BLOCK_SIZE));
		unsigned long pixel,
		ref_start(ex * img_size),
		img_start(ez * img_size);

		unsigned long ref_pixel_idx;
		unsigned long img_pixel_idx;

		for (unsigned pass = 0; pass < pass_num; pass ++)
		{
			pixel = pass * BLOCK_SIZE + threadIdx.x;

			if (pixel < img_size) //Is inside image
			{
				ref_pixel_idx = ref_start + pixel;
				img_pixel_idx = img_start + pixel;

				double diff_real = g_refs[ref_pixel_idx].real - g_imgs[img_pixel_idx].real;
				double diff_imag = g_refs[ref_pixel_idx].imag - g_imgs[img_pixel_idx].imag;

				s[threadIdx.x] += (diff_real * diff_real + diff_imag * diff_imag) * 0.5 * g_Minvsigma2[pixel];
			}
		}

		// This version should run in             BLOCK_SIZE                  cycles
		// -------------------------------------------------------------------------
	//		if (threadIdx.x == 0)
	//		{
	//			double sum(sum_init);
	//			for (unsigned i = 0; i < BLOCK_SIZE; i++)
	//				sum += s[i];
	//
	//			g_diff2s[ex * translation_num + ey] = sum;
	//		}
		// -------------------------------------------------------------------------

		// This version should run in     BLOCK_SIZE/trads + log2(trads)      cycles
		// ( Runs ~2x as fast as the above one for BLOCK_SIZE=32 )
		// -------------------------------------------------------------------------
		__syncthreads();
		int trads = 32;
		int itr = BLOCK_SIZE/trads;
		if(threadIdx.x<trads)
		{
			for(int i=1; i<itr; i++)
			{
				s[threadIdx.x] += s[i*trads + threadIdx.x];
				//__syncthreads();
			}
		}

		for(int j=(trads/2); j>0; j/=2)
		{
			if(threadIdx.x<j)
			{
				s[threadIdx.x] += s[threadIdx.x+j];
			}
		}
		__syncthreads();
//		if (threadIdx.x*ex == 0)
		{
			g_diff2s[ex * translation_num + ez] = s[0]+sum_init;
		}
		// -------------------------------------------------------------------------
	}
//	else
//	{
//		g_diff2s[ex * translation_num + ey] = 0; //(float)g_exp_Mcoarse_significant[blockIdx.x+blockIdx.y*coarse_rot_idx];
//	}


}

//  Takes a boolean N-by-M matrix and returns pointer pairs to coordinates in two corresponding objects
//__global__ void cuda_kernel_boolToPointers(	bool *matrix,
//												int yLength,
//												int** yPoints)
//{
//	//save the current index of the partial array to a shared location
//	__shared__  long int  length[blockDim.x*BLOCK_SIZE];
//	length[threadIdx.x]=0;
//
//	unsigned yiter(ceilf((float)yLength/(float)BLOCK_SIZE));
//
//	for(i=0; i<yiter; i++)
//	{
//		int pos = ylength*blockIdx.x + i*BLOCK_SIZE + threadIdx.x
//		if(matrix[pos]==1)
//		{
//			yPoints[blockIdx.x][length[blockIdx.x*BLOCK_SIZE+threadidx.x]]=blockIdx.x;
//			length[blockIdx.x*BLOCK_SIZE+threadidx.x]+=1;
//		}
//	}
//
//}

void MlOptimiserCUDA::getAllSquaredDifferences(
		long int my_ori_particle, int exp_current_image_size,
		int exp_ipass, int exp_current_oversampling, int metadata_offset,
		int exp_idir_min, int exp_idir_max, int exp_ipsi_min, int exp_ipsi_max,
		int exp_itrans_min, int exp_itrans_max, int exp_iclass_min, int exp_iclass_max,
		std::vector<double> &exp_min_diff2,
		std::vector<double> &exp_highres_Xi2_imgs,
		std::vector<MultidimArray<Complex > > &exp_Fimgs,
		std::vector<MultidimArray<double> > &exp_Fctfs,
		MultidimArray<double> &exp_Mweight,
		MultidimArray<bool> &exp_Mcoarse_significant,
		std::vector<int> &exp_pointer_dir_nonzeroprior, std::vector<int> &exp_pointer_psi_nonzeroprior,
		std::vector<double> &exp_directions_prior, std::vector<double> &exp_psi_prior,
		std::vector<MultidimArray<Complex> > &exp_local_Fimgs_shifted,
		std::vector<MultidimArray<double> > &exp_local_Minvsigma2s,
		std::vector<MultidimArray<double> > &exp_local_Fctfs,
		std::vector<double> &exp_local_sqrtXi2)
{
	// Initialise min_diff and exp_Mweight for this pass
	int exp_nr_particles = mydata.ori_particles[my_ori_particle].particles_id.size();
	long int exp_nr_dir = (do_skip_align || do_skip_rotate) ? 1 : sampling.NrDirections(0, &exp_pointer_dir_nonzeroprior);
	long int exp_nr_psi = (do_skip_align || do_skip_rotate) ? 1 : sampling.NrPsiSamplings(0, &exp_pointer_psi_nonzeroprior);
	long int exp_nr_trans = (do_skip_align) ? 1 : sampling.NrTranslationalSamplings();
	long int exp_nr_oversampled_rot = sampling.oversamplingFactorOrientations(exp_current_oversampling);
	long int exp_nr_oversampled_trans = sampling.oversamplingFactorTranslations(exp_current_oversampling);

	//for scale_correction
	int group_id;
	double myscale;

	//printf("exp_nr_oversampled_rot=%d\n", (unsigned)exp_nr_oversampled_rot);

	exp_Mweight.resize(exp_nr_particles, mymodel.nr_classes * exp_nr_dir * exp_nr_psi * exp_nr_trans * exp_nr_oversampled_rot * exp_nr_oversampled_trans);
	exp_Mweight.initConstant(-999.);
	if (exp_ipass==0)
		exp_Mcoarse_significant.clear();

	exp_min_diff2.clear();
	exp_min_diff2.resize(exp_nr_particles, 99.e99);

	std::vector<MultidimArray<Complex > > dummy;
	precalculateShiftedImagesCtfsAndInvSigma2s(false, my_ori_particle, exp_current_image_size, exp_current_oversampling,
			exp_itrans_min, exp_itrans_max, exp_Fimgs, dummy, exp_Fctfs, exp_local_Fimgs_shifted, dummy,
			exp_local_Fctfs, exp_local_sqrtXi2, exp_local_Minvsigma2s);

	// Loop only from exp_iclass_min to exp_iclass_max to deal with seed generation in first iteration
	for (int exp_iclass = exp_iclass_min; exp_iclass <= exp_iclass_max; exp_iclass++)
	{
		if (mymodel.pdf_class[exp_iclass] > 0.)
		{
			// Local variables
			std::vector< double > oversampled_rot, oversampled_tilt, oversampled_psi;
			std::vector< double > oversampled_translations_x, oversampled_translations_y, oversampled_translations_z;
			MultidimArray<Complex > Fref;
			double *Minvsigma2;
			Matrix2D<double> A;

			CudaImages Frefs(exp_local_Minvsigma2s[0].xdim, exp_local_Minvsigma2s[0].ydim,
					(exp_idir_max - exp_idir_min + 1) * (exp_ipsi_max - exp_ipsi_min + 1) * exp_nr_oversampled_rot);

			// Mapping index look-up table
			std::vector< long unsigned > iorientclasses, iover_rots;
			long unsigned orientation_num(0);

			/*=======================================================================================
			                           Generate Reference Projections
			=========================================================================================*/

			//printf("Generate Reference Projections\n");

			Fref.resize(exp_local_Minvsigma2s[0]); //TODO remove this
			Complex* FrefBag = Fref.data; //TODO remove this

			for (long int idir = exp_idir_min, iorient = 0; idir <= exp_idir_max; idir++)
			{
				for (long int ipsi = exp_ipsi_min; ipsi <= exp_ipsi_max; ipsi++, iorient++)
				{
					long int iorientclass = exp_iclass * exp_nr_dir * exp_nr_psi + iorient;

					// Get prior for this direction and skip calculation if prior==0
					double pdf_orientation;
					if (do_skip_align || do_skip_rotate)
					{
						pdf_orientation = mymodel.pdf_class[exp_iclass];
					}
					else if (mymodel.orientational_prior_mode == NOPRIOR)
					{
						pdf_orientation = DIRECT_MULTIDIM_ELEM(mymodel.pdf_direction[exp_iclass], idir);
					}
					else
					{
						pdf_orientation = exp_directions_prior[idir] * exp_psi_prior[ipsi];
					}
					// In the first pass, always proceed
					// In the second pass, check whether one of the translations for this orientation of any of the particles had a significant weight in the first pass
					// if so, proceed with projecting the reference in that direction
					bool do_proceed = (exp_ipass==0) ? true :
						isSignificantAnyParticleAnyTranslation(iorientclass, exp_itrans_min, exp_itrans_max, exp_Mcoarse_significant);
					if (do_proceed && pdf_orientation > 0.)
					{
						// Now get the oversampled (rot, tilt, psi) triplets
						// This will be only the original (rot,tilt,psi) triplet in the first pass (exp_current_oversampling==0)
						sampling.getOrientations(idir, ipsi, exp_current_oversampling, oversampled_rot, oversampled_tilt, oversampled_psi,
								exp_pointer_dir_nonzeroprior, exp_directions_prior, exp_pointer_psi_nonzeroprior, exp_psi_prior);
						// Loop over all oversampled orientations (only a single one in the first pass)
						for (long int iover_rot = 0; iover_rot < exp_nr_oversampled_rot; iover_rot++)
						{
							// Get the Euler matrix
							Euler_angles2matrix(oversampled_rot[iover_rot],
												oversampled_tilt[iover_rot],
												oversampled_psi[iover_rot], A);

							Fref.data = (Complex*) Frefs.current();

							// Project the reference map (into Fref)
							(mymodel.PPref[exp_iclass]).get2DFourierTransform(Fref, A, IS_NOT_INV);

							//TODO REMOVE ONCE YOU KNOW THIS IS ALLWAYS TRUE
							if (Frefs.x != Fref.xdim || Frefs.y != Fref.ydim)
								std::cerr << "!!!!!!! BAD Fref size x:" << Fref.xdim << ":" << Frefs.x << " y:" << Fref.ydim << ":" << Frefs.y << std::endl;

							Frefs.increment();

							orientation_num ++;
							iorientclasses.push_back(iorientclass);
							iover_rots.push_back(iover_rot);
						}
					}
				}
			}
			//printf("Finished generating reference projections\n");

			Fref.data = FrefBag; //TODO remove this

			CudaComplex *d_Frefs = Frefs.data_to_device();

			/*=======================================================================================
			                                  	  Particle Iteration
			=========================================================================================*/

			for (long int ipart = 0; ipart < mydata.ori_particles[my_ori_particle].particles_id.size(); ipart++)
			{
				/*====================================
				        Generate Translations
				======================================*/

				CudaImages Fimgs(Frefs.x, Frefs.y,
						( exp_itrans_max - exp_itrans_min + 1) * exp_nr_oversampled_trans);

				long int part_id = mydata.ori_particles[my_ori_particle].particles_id[ipart];
				long unsigned translation_num(0), ihidden(0);
				std::vector< long unsigned > iover_transes, itranses, ihiddens;

				//printf("Generating translations \n");

				for (long int itrans = exp_itrans_min; itrans <= exp_itrans_max; itrans++, ihidden++)
				{
					sampling.getTranslations(itrans, exp_current_oversampling,
							oversampled_translations_x, oversampled_translations_y, oversampled_translations_z );

					for (long int iover_trans = 0; iover_trans < exp_nr_oversampled_trans; iover_trans++)
					{
						/// Now get the shifted image
						// Use a pointer to avoid copying the entire array again in this highly expensive loop
						Complex *myAB;
						if (exp_current_oversampling == 0)
						{
							myAB = (Frefs.y == coarse_size) ? global_fftshifts_ab_coarse[itrans].data
									: global_fftshifts_ab_current[itrans].data;
						}
						else
						{
							int iitrans = itrans * exp_nr_oversampled_trans +  iover_trans;
							myAB = (strict_highres_exp > 0.) ? global_fftshifts_ab2_coarse[iitrans].data
									: global_fftshifts_ab2_current[iitrans].data;
						}
						FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(exp_local_Fimgs_shifted[ipart])
						{
							double real = (*(myAB + n)).real * (DIRECT_MULTIDIM_ELEM(exp_local_Fimgs_shifted[ipart], n)).real
									- (*(myAB + n)).imag *(DIRECT_MULTIDIM_ELEM(exp_local_Fimgs_shifted[ipart], n)).imag;
							double imag = (*(myAB + n)).real * (DIRECT_MULTIDIM_ELEM(exp_local_Fimgs_shifted[ipart], n)).imag
									+ (*(myAB + n)).imag *(DIRECT_MULTIDIM_ELEM(exp_local_Fimgs_shifted[ipart], n)).real;

							//When on gpu, it makes more sense to ctf-correct translated images, rather than anti-ctf-correct ref-projections
							if (do_scale_correction)
							{
								//group_id = mydata.getGroupId(part_id);
								float myscale = mymodel.scale_correction[group_id];
								real /= myscale;
								imag /= myscale;
							}
							if (do_ctf_correction && refs_are_ctf_corrected)
							{
								real /= DIRECT_MULTIDIM_ELEM(exp_local_Fctfs[ipart], n);
								imag /= DIRECT_MULTIDIM_ELEM(exp_local_Fctfs[ipart], n);
							}
							*(Fimgs.current() + n) = CudaComplex(real, imag);
						}
						Fimgs.increment();
						translation_num ++;

						ihiddens.push_back(ihidden);
						itranses.push_back(itrans);
						iover_transes.push_back(iover_trans);
					}
				}
				//printf("Generating translations finished \n");

				/*===========================================
				   Determine significant comparison indices
				=============================================*/
				//      This section is annoying to test because
				//		it can't complete on first pass, since
				//		the significance has never been set

				long int coarse_num = exp_nr_dir*exp_nr_psi*exp_nr_trans;
				long int significant_num=0;
				std::cerr << "exp_ipass "<< exp_ipass << std::endl;
				if (exp_ipass == 0)
				{
					exp_Mcoarse_significant.resize(coarse_num, 1);
					for (long int i = 0; i < coarse_num; i++)
					{
						DIRECT_A2D_ELEM(exp_Mcoarse_significant, ipart, i)=1;
//						std::cerr << "exp_Mcoarse_significant("<< i <<") = " <<    DIRECT_A2D_ELEM(exp_Mcoarse_significant, ipart, i) << std::endl;
//						std::cerr << "exp_Mcoarse_significant("<< i <<") = " << *(&DIRECT_A2D_ELEM(exp_Mcoarse_significant, ipart, 0)+i*sizeof(bool)) << std::endl;
					}
					significant_num = coarse_num;
				}
				else
				{
					std::vector< long unsigned > transidx, rotidx;
					for (long int i = 0; i < orientation_num; i++)
					{
						long int iover_rot = iover_rots[i];
//						long int iover_rot = i % exp_nr_oversampled_rot
						long int coarse_rot = floor(i/exp_nr_oversampled_rot);
						for (long int j = 0; j < translation_num; j++)
						{
							long int iover_trans = iover_transes[j];
//							long int iover_trans = j % exp_nr_oversampled_trans
							long int coarse_trans = floor(j/exp_nr_oversampled_trans);
							long int ihidden = iorientclasses[i] * exp_nr_trans + ihiddens[j];
							if(DIRECT_A2D_ELEM(exp_Mcoarse_significant, ipart, ihidden)==1)
							{
								 long int ihidden_over = sampling.getPositionOversampledSamplingPoint(ihidden,
										                  exp_current_oversampling, iover_rot, iover_trans);
								 transidx.push_back(i);
								 rotidx.push_back(j);
								 significant_num++;
							}
						}
					}
				}
				std::cerr << "orientation_num "<< orientation_num << std::endl;
				std::cerr << "translation_num "<< translation_num << std::endl;
				std::cerr << "my_nr_significant_coarse_samples "<< DIRECT_A2D_ELEM(exp_metadata, metadata_offset + ipart, METADATA_NR_SIGN) << std::endl;
				std::cerr << "significant_num "<< significant_num << std::endl;

				/*====================================
				   Initiate Particle Related On GPU
				======================================*/

				//When on gpu, it makes more sense to ctf-correct translated images, rather than anti-ctf-correct ref-projections
				if (do_ctf_correction && refs_are_ctf_corrected)
				{
					FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(exp_local_Fimgs_shifted[ipart])
					{
						DIRECT_MULTIDIM_ELEM(exp_local_Minvsigma2s[ipart], n) *= (DIRECT_MULTIDIM_ELEM(exp_local_Fctfs[ipart], n)*DIRECT_MULTIDIM_ELEM(exp_local_Fctfs[ipart], n));
					}
				}
				// TODO :    + Assure accuracy with the implemented GPU-based ctf-scaling
				//           + Make setting of myscale robust between here and above.
				//  (scale_correction turns off by default with only one group: ml_optimiser-line 1067,
				//   meaning small-scale test will probably not catch this malfunctioning when/if it breaks.)
				if (do_scale_correction)
				{
					float myscale = mymodel.scale_correction[group_id];
					FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(exp_local_Fimgs_shifted[ipart])
					{
						DIRECT_MULTIDIM_ELEM(exp_local_Minvsigma2s[ipart], n) *= (myscale*myscale);
					}
				}

				Minvsigma2 = exp_local_Minvsigma2s[ipart].data;
				double *d_Minvsigma2(0);

				CudaComplex *d_Fimgs = Fimgs.data_to_device();

				HANDLE_ERROR(hipMalloc( (void**) &d_Minvsigma2, Fimgs.xy * sizeof(double)));
				HANDLE_ERROR(hipMemcpy( d_Minvsigma2, exp_local_Minvsigma2s[ipart].data, Fimgs.xy * sizeof(double), hipMemcpyHostToDevice));

				double *d_diff2s(0);
				HANDLE_ERROR(hipMalloc( (void**) &d_diff2s, orientation_num*translation_num * sizeof(double)));
				//HANDLE_ERROR(hipMemset(d_diff2s, exp_highres_Xi2_imgs[ipart] / 2., orientation_num*translation_num * sizeof(double))); //Initiate diff2 values with zeros


				bool *d_exp_Mcoarse_significant(0);

				HANDLE_ERROR(hipMalloc( (void**) &d_exp_Mcoarse_significant, coarse_num * sizeof(bool)));
				HANDLE_ERROR(hipMemcpy( d_exp_Mcoarse_significant, &(exp_Mcoarse_significant.data),  coarse_num * sizeof(bool), hipMemcpyHostToDevice));
//
//				int *d_rotidx(0);
//				HANDLE_ERROR(hipMalloc( (void**) &d_rotidx, significant_num * sizeof(int)));
//				HANDLE_ERROR(hipMemcpy( d_rotidx, rotidx,  significant_num * sizeof(int), hipMemcpyHostToDevice));

				/*====================================
				    		Kernel Calls
				======================================*/
				short int orient1, orient2;

				if(orientation_num>65535)
				{
					orient1 = ceil(sqrt(orientation_num));
					orient2 = orient1;
				}
				else
				{
					orient1 = orientation_num;
					orient2 = 1;
				}
				dim3 block_dim(orient1,orient2,translation_num);

				hipEvent_t start, stop;
				float time;
				hipEventCreate(&start);
				hipEventCreate(&stop);
				hipEventRecord(start, 0);
				//printf("Calling kernel with <<(%d,%d), %d>> \n", block_dim.x, block_dim.y, BLOCK_SIZE);
				cuda_kernel_massive_diff2<<<block_dim,BLOCK_SIZE>>>(d_Frefs, d_Fimgs, d_Minvsigma2, d_diff2s,
																	Frefs.xy, exp_highres_Xi2_imgs[ipart] / 2.,
																	d_exp_Mcoarse_significant,
																	orientation_num,
																	translation_num,
																	exp_nr_oversampled_rot,
																	exp_nr_oversampled_trans);

				hipEventRecord(stop, 0);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&time, start, stop);
				hipEventDestroy(start);
				hipEventDestroy(stop);
//				for (long unsigned i = 0; i < orientation_num; i ++)
//				{
//					for (long unsigned j = 0; j < translation_num; j ++)
//					{
//						cuda_diff2_deviceImage( Frefs.xy, (double*) ( d_Frefs + (i * Frefs.xy) ), (double*) ( d_Fimgs + (j * Fimgs.xy) ), d_Minvsigma2, d_diff2s + (i * translation_num + j));
//					}
//				}

				/*====================================
				    	   Retrieve Results
				======================================*/

				HANDLE_ERROR(hipDeviceSynchronize());
				//printf("Kernel call finished \n");

				std::cerr << "It took "<< time <<" msecs."<< std::endl;
				double* diff2s = new double[orientation_num*translation_num];
				if (exp_ipass == 0)
				{
					exp_Mcoarse_significant.clear();
				}
				HANDLE_ERROR(hipMemcpy( diff2s, d_diff2s, orientation_num*translation_num*sizeof(double), hipMemcpyDeviceToHost ));

				/*====================================
				    	Write To Destination
				======================================*/


//				if (exp_current_oversampling > 1)
//				{
//				std::ofstream myfile;
//				std::stringstream sstm;
//				sstm << "diff2s/gpu_part.dat";
//				myfile.open(sstm.str().c_str(), std::ios_base::app);
//				}

				//printf("Writing to destination \n");
				for (long int i = 0; i < orientation_num; i++)
				{
					long int iover_rot = iover_rots[i];

					for (long int j = 0; j < translation_num; j++)
					{
						long int ihidden = iorientclasses[i] * exp_nr_trans + ihiddens[j];
						long int iover_trans = iover_transes[j];

						long int ihidden_over = sampling.getPositionOversampledSamplingPoint(ihidden, exp_current_oversampling,
																							iover_rot, iover_trans);

						double diff2 = diff2s[i * translation_num + j];
						//diff2 += exp_highres_Xi2_imgs[ipart] / 2.;
//
//						if (exp_current_oversampling > 1)
//							myfile << ihidden_over << " " << diff2 << std::endl;

						DIRECT_A2D_ELEM(exp_Mweight, ipart, ihidden_over) = diff2;

						// Keep track of minimum of all diff2, only for the last image in this series
						if (diff2 < exp_min_diff2[ipart])
							exp_min_diff2[ipart] = diff2;
					}
				}
				//printf("Writing to destination finished \n");


				hipFree(d_Fimgs);
				hipFree(d_diff2s);
				delete [] diff2s;

			} // end loop ipart

			hipFree(d_Frefs);

		} // end if class significant
	} // end loop iclass
}

void MlOptimiserCUDA::storeWeightedSums(long int my_ori_particle, int exp_current_image_size,
		int exp_current_oversampling, int metadata_offset,
		int exp_idir_min, int exp_idir_max, int exp_ipsi_min, int exp_ipsi_max,
		int exp_itrans_min, int exp_itrans_max, int exp_iclass_min, int exp_iclass_max,
		std::vector<double> &exp_min_diff2,
		std::vector<double> &exp_highres_Xi2_imgs,
		std::vector<MultidimArray<Complex > > &exp_Fimgs,
		std::vector<MultidimArray<Complex > > &exp_Fimgs_nomask,
		std::vector<MultidimArray<double> > &exp_Fctfs,
		std::vector<MultidimArray<double> > &exp_power_imgs,
		std::vector<Matrix1D<double> > &exp_old_offset,
		std::vector<Matrix1D<double> > &exp_prior,
		MultidimArray<double> &exp_Mweight,
		MultidimArray<bool> &exp_Mcoarse_significant,
		std::vector<double> &exp_significant_weight,
		std::vector<double> &exp_sum_weight,
		std::vector<double> &exp_max_weight,
		std::vector<int> &exp_pointer_dir_nonzeroprior, std::vector<int> &exp_pointer_psi_nonzeroprior,
		std::vector<double> &exp_directions_prior, std::vector<double> &exp_psi_prior,
		std::vector<MultidimArray<Complex > > &exp_local_Fimgs_shifted,
		std::vector<MultidimArray<Complex > > &exp_local_Fimgs_shifted_nomask,
		std::vector<MultidimArray<double> > &exp_local_Minvsigma2s,
		std::vector<MultidimArray<double> > &exp_local_Fctfs,
		std::vector<double> &exp_local_sqrtXi2)
{

	int exp_nr_particles = mydata.ori_particles[my_ori_particle].particles_id.size();
	long int exp_nr_dir = (do_skip_align || do_skip_rotate) ? 1 : sampling.NrDirections(0, &exp_pointer_dir_nonzeroprior);
	long int exp_nr_psi = (do_skip_align || do_skip_rotate) ? 1 : sampling.NrPsiSamplings(0, &exp_pointer_psi_nonzeroprior);
	long int exp_nr_trans = (do_skip_align) ? 1 : sampling.NrTranslationalSamplings();
	long int exp_nr_oversampled_rot = sampling.oversamplingFactorOrientations(exp_current_oversampling);
	long int exp_nr_oversampled_trans = sampling.oversamplingFactorTranslations(exp_current_oversampling);

	// Re-do below because now also want unmasked images AND if (stricht_highres_exp >0.) then may need to resize
	precalculateShiftedImagesCtfsAndInvSigma2s(true, my_ori_particle, exp_current_image_size, exp_current_oversampling,
			exp_itrans_min, exp_itrans_max, exp_Fimgs, exp_Fimgs_nomask, exp_Fctfs, exp_local_Fimgs_shifted, exp_local_Fimgs_shifted_nomask,
			exp_local_Fctfs, exp_local_sqrtXi2, exp_local_Minvsigma2s);

	// In doThreadPrecalculateShiftedImagesCtfsAndInvSigma2s() the origin of the exp_local_Minvsigma2s was omitted.
	// Set those back here
	for (long int ipart = 0; ipart < mydata.ori_particles[my_ori_particle].particles_id.size(); ipart++)
	{
		long int part_id = mydata.ori_particles[my_ori_particle].particles_id[ipart];
		int group_id = mydata.getGroupId(part_id);
		DIRECT_MULTIDIM_ELEM(exp_local_Minvsigma2s[ipart], 0) = 1. / (sigma2_fudge * DIRECT_A1D_ELEM(mymodel.sigma2_noise[group_id], 0));
	}

	// Initialise the maximum of all weights to a negative value
	exp_max_weight.clear();
	exp_max_weight.resize(exp_nr_particles, -1.);

	// For norm_correction and scale_correction of all particles of this ori_particle
	std::vector<double> exp_wsum_norm_correction;
	std::vector<MultidimArray<double> > exp_wsum_scale_correction_XA, exp_wsum_scale_correction_AA;
	std::vector<MultidimArray<double> > thr_wsum_signal_product_spectra, thr_wsum_reference_power_spectra;
	exp_wsum_norm_correction.resize(exp_nr_particles, 0.);

	// For scale_correction
	if (do_scale_correction)
	{
		MultidimArray<double> aux;
		aux.initZeros(mymodel.ori_size/2 + 1);
		exp_wsum_scale_correction_XA.resize(exp_nr_particles, aux);
		exp_wsum_scale_correction_AA.resize(exp_nr_particles, aux);
		thr_wsum_signal_product_spectra.resize(mymodel.nr_groups, aux);
		thr_wsum_reference_power_spectra.resize(mymodel.nr_groups, aux);
	}

	std::vector< double> oversampled_rot, oversampled_tilt, oversampled_psi;
	std::vector<double> oversampled_translations_x, oversampled_translations_y, oversampled_translations_z;
	Matrix2D<double> A;
	MultidimArray<Complex > Fimg, Fref, Frefctf, Fimg_otfshift, Fimg_otfshift_nomask;
	MultidimArray<double> Minvsigma2, Mctf, Fweight;
	double rot, tilt, psi;
	bool have_warned_small_scale = false;
	// Initialising... exp_Fimgs[0] has mymodel.current_size (not coarse_size!)
	Fref.resize(exp_Fimgs[0]);
	Frefctf.resize(exp_Fimgs[0]);
	Fweight.resize(exp_Fimgs[0]);
	Fimg.resize(exp_Fimgs[0]);
	// Initialise Mctf to all-1 for if !do_ctf_corection
	Mctf.resize(exp_Fimgs[0]);
	Mctf.initConstant(1.);
	// Initialise Minvsigma2 to all-1 for if !do_map
	Minvsigma2.resize(exp_Fimgs[0]);
	Minvsigma2.initConstant(1.);
	if (do_shifts_onthefly)
	{
		Fimg_otfshift.resize(Frefctf);
		Fimg_otfshift_nomask.resize(Frefctf);
	}

	// Make local copies of weighted sums (except BPrefs, which are too big)
	// so that there are not too many mutex locks below
	std::vector<MultidimArray<double> > thr_wsum_sigma2_noise, thr_wsum_pdf_direction;
	std::vector<double> thr_wsum_norm_correction, thr_sumw_group, thr_wsum_pdf_class, thr_wsum_prior_offsetx_class, thr_wsum_prior_offsety_class;
	double thr_wsum_sigma2_offset;
	MultidimArray<double> thr_metadata, zeroArray;
	// Wsum_sigma_noise2 is a 1D-spectrum for each group
	zeroArray.initZeros(mymodel.ori_size/2 + 1);
	thr_wsum_sigma2_noise.resize(mymodel.nr_groups, zeroArray);
	// wsum_pdf_direction is a 1D-array (of length sampling.NrDirections()) for each class
	zeroArray.initZeros(sampling.NrDirections());
	thr_wsum_pdf_direction.resize(mymodel.nr_classes, zeroArray);
	// sumw_group is a double for each group
	thr_sumw_group.resize(mymodel.nr_groups, 0.);
	// wsum_pdf_class is a double for each class
	thr_wsum_pdf_class.resize(mymodel.nr_classes, 0.);
	if (mymodel.ref_dim == 2)
	{
		thr_wsum_prior_offsetx_class.resize(mymodel.nr_classes, 0.);
		thr_wsum_prior_offsety_class.resize(mymodel.nr_classes, 0.);
	}
	// wsum_sigma2_offset is just a double
	thr_wsum_sigma2_offset = 0.;


	printf("Entering crazy-ass loop\n");

	// Loop from iclass_min to iclass_max to deal with seed generation in first iteration
	for (int exp_iclass = exp_iclass_min; exp_iclass <= exp_iclass_max; exp_iclass++)
	{
		for (long int idir = exp_idir_min, iorient = 0; idir <= exp_idir_max; idir++)
		{
			for (long int ipsi = exp_ipsi_min; ipsi <= exp_ipsi_max; ipsi++, iorient++)
			{
				long int iorientclass = exp_iclass * exp_nr_dir * exp_nr_psi + iorient;

				// Only proceed if any of the particles had any significant coarsely sampled translation
				if (isSignificantAnyParticleAnyTranslation(iorientclass, exp_itrans_min, exp_itrans_max, exp_Mcoarse_significant))
				{
					// Now get the oversampled (rot, tilt, psi) triplets
					// This will be only the original (rot,tilt,psi) triplet if (adaptive_oversampling==0)
					sampling.getOrientations(idir, ipsi, adaptive_oversampling, oversampled_rot, oversampled_tilt, oversampled_psi,
							exp_pointer_dir_nonzeroprior, exp_directions_prior, exp_pointer_psi_nonzeroprior, exp_psi_prior);
					// Loop over all oversampled orientations (only a single one in the first pass)
					for (long int iover_rot = 0; iover_rot < exp_nr_oversampled_rot; iover_rot++)
					{
						rot = oversampled_rot[iover_rot];
						tilt = oversampled_tilt[iover_rot];
						psi = oversampled_psi[iover_rot];
						// Get the Euler matrix
						Euler_angles2matrix(rot, tilt, psi, A);
						// Project the reference map (into Fref)
						if (!do_skip_maximization)
							(mymodel.PPref[exp_iclass]).get2DFourierTransform(Fref, A, IS_NOT_INV);
						// Inside the loop over all translations and all part_id sum all shift Fimg's and their weights
						// Then outside this loop do the actual backprojection
						Fimg.initZeros();
						Fweight.initZeros();
						/// Now that reference projection has been made loop over all particles inside this ori_particle
						for (long int ipart = 0; ipart < mydata.ori_particles[my_ori_particle].particles_id.size(); ipart++)
						{
							// This is an attempt to speed up illogically slow updates of wsum_sigma2_offset....
							// It seems to make a big difference!
							double myprior_x, myprior_y, myprior_z, old_offset_z;
							double old_offset_x = XX(exp_old_offset[ipart]);
							double old_offset_y = YY(exp_old_offset[ipart]);
							if (mymodel.ref_dim == 2)
							{
								myprior_x = XX(mymodel.prior_offset_class[exp_iclass]);
								myprior_y = YY(mymodel.prior_offset_class[exp_iclass]);
							}
							else
							{
								myprior_x = XX(exp_prior[ipart]);
								myprior_y = YY(exp_prior[ipart]);
								if (mymodel.data_dim == 3)
								{
									myprior_z = ZZ(exp_prior[ipart]);
									old_offset_z = ZZ(exp_old_offset[ipart]);
								}
							}

							long int part_id = mydata.ori_particles[my_ori_particle].particles_id[ipart];
							int group_id = mydata.getGroupId(part_id);
							if (!do_skip_maximization)
							{
								if (do_map)
									Minvsigma2 = exp_local_Minvsigma2s[ipart];
								// else Minvsigma2 was initialised to ones
								// Apply CTF to reference projection
								if (do_ctf_correction)
								{
									Mctf = exp_local_Fctfs[ipart];
									if (refs_are_ctf_corrected)
									{
										FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(Fref)
										{
											DIRECT_MULTIDIM_ELEM(Frefctf, n) = DIRECT_MULTIDIM_ELEM(Fref, n) * DIRECT_MULTIDIM_ELEM(Mctf, n);
										}
									}
									else
									{
										Frefctf = Fref;
									}
								}
								else
								{
									// initialise because there are multiple particles and Mctf gets selfMultiplied for scale_correction
									Mctf.initConstant(1.);
									Frefctf = Fref;
								}
								if (do_scale_correction)
								{
									double myscale = mymodel.scale_correction[group_id];
									if (myscale > 10000.)
									{
										std::cerr << " rlnMicrographScaleCorrection= " << myscale << " group= " << group_id + 1 << std::endl;
										REPORT_ERROR("ERROR: rlnMicrographScaleCorrection is very high. Did you normalize your data?");
									}
									else if (myscale < 0.001)
									{
										if (!have_warned_small_scale)
										{
											std::cout << " WARNING: ignoring group " << group_id + 1 << " with very small or negative scale (" << myscale <<
													"); Use larger groups for more stable scale estimates." << std::endl;
											have_warned_small_scale = true;
										}
										myscale = 0.001;
									}
									FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(Frefctf)
									{
										DIRECT_MULTIDIM_ELEM(Frefctf, n) *= myscale;
									}
									// For CTF-terms in BP
									Mctf *= myscale;
								}
							} // end if !do_skip_maximization

							long int ihidden = iorientclass * exp_nr_trans;
							for (long int itrans = exp_itrans_min, iitrans = 0; itrans <= exp_itrans_max; itrans++, ihidden++)
							{
								sampling.getTranslations(itrans, adaptive_oversampling,
										oversampled_translations_x, oversampled_translations_y, oversampled_translations_z);
								for (long int iover_trans = 0; iover_trans < exp_nr_oversampled_trans; iover_trans++, iitrans++)
								{
									// Only deal with this sampling point if its weight was significant
									long int ihidden_over = ihidden * exp_nr_oversampled_trans * exp_nr_oversampled_rot +
											iover_rot * exp_nr_oversampled_trans + iover_trans;
									double weight = DIRECT_A2D_ELEM(exp_Mweight, ipart, ihidden_over);
									// Only sum weights for non-zero weights
									if (weight >= exp_significant_weight[ipart])
									{
										// Normalise the weight (do this after the comparison with exp_significant_weight!)
										weight /= exp_sum_weight[ipart];
										if (!do_skip_maximization)
										{

											/// Now get the shifted image
											// Use a pointer to avoid copying the entire array again in this highly expensive loop
											Complex *Fimg_shift, *Fimg_shift_nomask;
											if (!do_shifts_onthefly)
											{
												long int ishift = ipart * exp_nr_oversampled_trans * exp_nr_trans + iitrans;
												Fimg_shift = exp_local_Fimgs_shifted[ishift].data;
												Fimg_shift_nomask = exp_local_Fimgs_shifted_nomask[ishift].data;
											}
											else
											{
												Complex* myAB;
												myAB = (adaptive_oversampling == 0 ) ? global_fftshifts_ab_current[iitrans].data : global_fftshifts_ab2_current[iitrans].data;
												FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(exp_local_Fimgs_shifted[ipart])
												{
													double a = (*(myAB + n)).real;
													double b = (*(myAB + n)).imag;
													// Fimg_shift
													double real = a * (DIRECT_MULTIDIM_ELEM(exp_local_Fimgs_shifted[ipart], n)).real
															- b *(DIRECT_MULTIDIM_ELEM(exp_local_Fimgs_shifted[ipart], n)).imag;
													double imag = a * (DIRECT_MULTIDIM_ELEM(exp_local_Fimgs_shifted[ipart], n)).imag
															+ b *(DIRECT_MULTIDIM_ELEM(exp_local_Fimgs_shifted[ipart], n)).real;
													DIRECT_MULTIDIM_ELEM(Fimg_otfshift, n) = Complex(real, imag);
													// Fimg_shift_nomask
													real = a * (DIRECT_MULTIDIM_ELEM(exp_local_Fimgs_shifted_nomask[ipart], n)).real
															- b *(DIRECT_MULTIDIM_ELEM(exp_local_Fimgs_shifted_nomask[ipart], n)).imag;
													imag = a * (DIRECT_MULTIDIM_ELEM(exp_local_Fimgs_shifted_nomask[ipart], n)).imag
															+ b *(DIRECT_MULTIDIM_ELEM(exp_local_Fimgs_shifted_nomask[ipart], n)).real;
													DIRECT_MULTIDIM_ELEM(Fimg_otfshift_nomask, n) = Complex(real, imag);
												}
												Fimg_shift = Fimg_otfshift.data;
												Fimg_shift_nomask = Fimg_otfshift_nomask.data;
											}

											// Store weighted sum of squared differences for sigma2_noise estimation
											// Suggestion Robert Sinkovitz: merge difference and scale steps to make better use of cache
											FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(Mresol_fine)
											{
												int ires = DIRECT_MULTIDIM_ELEM(Mresol_fine, n);
												if (ires > -1)
												{
													// Use FT of masked image for noise estimation!
													double diff_real = (DIRECT_MULTIDIM_ELEM(Frefctf, n)).real - (*(Fimg_shift + n)).real;
													double diff_imag = (DIRECT_MULTIDIM_ELEM(Frefctf, n)).imag - (*(Fimg_shift + n)).imag;
													double wdiff2 = weight * (diff_real*diff_real + diff_imag*diff_imag);
													// group-wise sigma2_noise
													DIRECT_MULTIDIM_ELEM(thr_wsum_sigma2_noise[group_id], ires) += wdiff2;
													// For norm_correction
													exp_wsum_norm_correction[ipart] += wdiff2;
												}
											    if (do_scale_correction && DIRECT_A1D_ELEM(mymodel.data_vs_prior_class[exp_iclass], ires) > 3.)
												{
											    	double sumXA, sumA2;
											    	sumXA = (DIRECT_MULTIDIM_ELEM(Frefctf, n)).real * (*(Fimg_shift + n)).real;
											    	sumXA += (DIRECT_MULTIDIM_ELEM(Frefctf, n)).imag * (*(Fimg_shift + n)).imag;
											    	DIRECT_A1D_ELEM(exp_wsum_scale_correction_XA[ipart], ires) += weight * sumXA;
											    	sumA2 = (DIRECT_MULTIDIM_ELEM(Frefctf, n)).real * (DIRECT_MULTIDIM_ELEM(Frefctf, n)).real;
											    	sumA2 += (DIRECT_MULTIDIM_ELEM(Frefctf, n)).imag * (DIRECT_MULTIDIM_ELEM(Frefctf, n)).imag;
											    	DIRECT_A1D_ELEM(exp_wsum_scale_correction_AA[ipart], ires) += weight * sumA2;
												}
											}

											// Store sum of weights for this group
											thr_sumw_group[group_id] += weight;
											// Store weights for this class and orientation
											thr_wsum_pdf_class[exp_iclass] += weight;

											// The following goes MUCH faster than the original lines below....
											if (mymodel.ref_dim == 2)
											{
												thr_wsum_prior_offsetx_class[exp_iclass] += weight * (old_offset_x + oversampled_translations_x[iover_trans]);
												thr_wsum_prior_offsety_class[exp_iclass] += weight * (old_offset_y + oversampled_translations_y[iover_trans]);
											}
											double diffx = myprior_x - old_offset_x - oversampled_translations_x[iover_trans];
											double diffy = myprior_y - old_offset_y - oversampled_translations_y[iover_trans];
											if (mymodel.data_dim == 3)
											{
												double diffz  = myprior_z - old_offset_z - oversampled_translations_z[iover_trans];
												thr_wsum_sigma2_offset += weight * (diffx*diffx + diffy*diffy + diffz*diffz);
											}
											else
											{
												thr_wsum_sigma2_offset += weight * (diffx*diffx + diffy*diffy);
											}

											// Store weight for this direction of this class
											if (do_skip_align || do_skip_rotate )
											{
												//ignore pdf_direction
											}
											else if (mymodel.orientational_prior_mode == NOPRIOR)
											{
												DIRECT_MULTIDIM_ELEM(thr_wsum_pdf_direction[exp_iclass], idir) += weight;
											}
											else
											{
												// In the case of orientational priors, get the original number of the direction back
												long int mydir = exp_pointer_dir_nonzeroprior[idir];
												DIRECT_MULTIDIM_ELEM(thr_wsum_pdf_direction[exp_iclass], mydir) += weight;
											}
											// Store sum of weight*SSNR*Fimg in data and sum of weight*SSNR in weight
											// Use the FT of the unmasked image to back-project in order to prevent reconstruction artefacts! SS 25oct11
											FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(Fimg)
											{
												double myctf = DIRECT_MULTIDIM_ELEM(Mctf, n);
												// Note that weightxinvsigma2 already contains the CTF!
												double weightxinvsigma2 = weight * myctf * DIRECT_MULTIDIM_ELEM(Minvsigma2, n);
												// now Fimg stores sum of all shifted w*Fimg
												(DIRECT_MULTIDIM_ELEM(Fimg, n)).real += (*(Fimg_shift_nomask + n)).real * weightxinvsigma2;
												(DIRECT_MULTIDIM_ELEM(Fimg, n)).imag += (*(Fimg_shift_nomask + n)).imag * weightxinvsigma2;
												// now Fweight stores sum of all w
												// Note that CTF needs to be squared in Fweight, weightxinvsigma2 already contained one copy
												DIRECT_MULTIDIM_ELEM(Fweight, n) += weightxinvsigma2 * myctf;
											}
										} // end if !do_skip_maximization

										// Keep track of max_weight and the corresponding optimal hidden variables
										if (weight > exp_max_weight[ipart])
										{
											// Store optimal image parameters
											exp_max_weight[ipart] = weight;

											// TODO: remove, for now to maintain exact numerical version of old threads....
											A = A.inv();
											A = A.inv();
											Euler_matrix2angles(A, rot, tilt, psi);

											DIRECT_A2D_ELEM(exp_metadata, metadata_offset + ipart, METADATA_ROT) = rot;
											DIRECT_A2D_ELEM(exp_metadata, metadata_offset + ipart, METADATA_TILT) = tilt;
											DIRECT_A2D_ELEM(exp_metadata, metadata_offset + ipart, METADATA_PSI) = psi;
											DIRECT_A2D_ELEM(exp_metadata, metadata_offset + ipart, METADATA_XOFF) = XX(exp_old_offset[ipart]) + oversampled_translations_x[iover_trans];
											DIRECT_A2D_ELEM(exp_metadata, metadata_offset + ipart, METADATA_YOFF) = YY(exp_old_offset[ipart]) + oversampled_translations_y[iover_trans];
											if (mymodel.data_dim == 3)
												DIRECT_A2D_ELEM(exp_metadata, metadata_offset + ipart, METADATA_ZOFF) = ZZ(exp_old_offset[ipart]) + oversampled_translations_z[iover_trans];
											DIRECT_A2D_ELEM(exp_metadata, metadata_offset + ipart, METADATA_CLASS) = (double)exp_iclass + 1;
											DIRECT_A2D_ELEM(exp_metadata, metadata_offset + ipart, METADATA_PMAX) = exp_max_weight[ipart];
										}
									} // end if weight >= exp_significant_weight
								} // end loop iover_trans
							} // end loop itrans
						} // end loop ipart

						if (!do_skip_maximization)
						{
							// Perform the actual back-projection.
							// This is done with the sum of all (in-plane) shifted Fimg's
							// Perform this inside a mutex
							int my_mutex = exp_iclass % NR_CLASS_MUTEXES;
							pthread_mutex_lock(&global_mutex2[my_mutex]);
							(wsum_model.BPref[exp_iclass]).set2DFourierTransform(Fimg, A, IS_NOT_INV, &Fweight);
							pthread_mutex_unlock(&global_mutex2[my_mutex]);
						} // end if !do_skip_maximization
					} // end loop iover_rot
				}// end loop do_proceed
			} // end loop ipsi
		} // end loop idir
	} // end loop iclass

	printf("Exiting crazy-ass loop\n");

	// Extend norm_correction and sigma2_noise estimation to higher resolutions for all particles
	// Also calculate dLL for each particle and store in metadata
	// loop over all particles inside this ori_particle
	double thr_avg_norm_correction = 0.;
	double thr_sum_dLL = 0., thr_sum_Pmax = 0.;
	for (long int ipart = 0; ipart < mydata.ori_particles[my_ori_particle].particles_id.size(); ipart++)
	{
		long int part_id = mydata.ori_particles[my_ori_particle].particles_id[ipart];
		int group_id = mydata.getGroupId(part_id);

		// If the current images were smaller than the original size, fill the rest of wsum_model.sigma2_noise with the power_class spectrum of the images
		for (int ires = mymodel.current_size/2 + 1; ires < mymodel.ori_size/2 + 1; ires++)
		{
			DIRECT_A1D_ELEM(thr_wsum_sigma2_noise[group_id], ires) += DIRECT_A1D_ELEM(exp_power_imgs[ipart], ires);
			// Also extend the weighted sum of the norm_correction
			exp_wsum_norm_correction[ipart] += DIRECT_A1D_ELEM(exp_power_imgs[ipart], ires);
		}

		// Store norm_correction
		// Multiply by old value because the old norm_correction term was already applied to the image
		if (do_norm_correction)
		{
			double old_norm_correction = DIRECT_A2D_ELEM(exp_metadata, metadata_offset + ipart, METADATA_NORM);
			old_norm_correction /= mymodel.avg_norm_correction;
			// The factor two below is because exp_wsum_norm_correctiom is similar to sigma2_noise, which is the variance for the real/imag components
			// The variance of the total image (on which one normalizes) is twice this value!
			double normcorr = old_norm_correction * sqrt(exp_wsum_norm_correction[ipart] * 2.);
			thr_avg_norm_correction += normcorr;
			// Now set the new norm_correction in the relevant position of exp_metadata
			DIRECT_A2D_ELEM(exp_metadata, metadata_offset + ipart, METADATA_NORM) = normcorr;

			// Print warning for strange norm-correction values
			if (!(iter == 1 && do_firstiter_cc) && DIRECT_A2D_ELEM(exp_metadata, metadata_offset + ipart, METADATA_NORM) > 10.)
			{
				std::cout << " WARNING: norm_correction= "<< DIRECT_A2D_ELEM(exp_metadata, metadata_offset + ipart, METADATA_NORM) << " for particle " << part_id << " in group " << group_id + 1 << "; Are your groups large enough?" << std::endl;
			}

		}

		// Store weighted sums for scale_correction
		if (do_scale_correction)
		{
			// Divide XA by the old scale_correction and AA by the square of that, because was incorporated into Fctf
			exp_wsum_scale_correction_XA[ipart] /= mymodel.scale_correction[group_id];
			exp_wsum_scale_correction_AA[ipart] /= mymodel.scale_correction[group_id] * mymodel.scale_correction[group_id];

			thr_wsum_signal_product_spectra[group_id] += exp_wsum_scale_correction_XA[ipart];
			thr_wsum_reference_power_spectra[group_id] += exp_wsum_scale_correction_AA[ipart];
		}

		// Calculate DLL for each particle
		double logsigma2 = 0.;
		FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(Mresol_fine)
		{
			int ires = DIRECT_MULTIDIM_ELEM(Mresol_fine, n);
			// Note there is no sqrt in the normalisation term because of the 2-dimensionality of the complex-plane
			// Also exclude origin from logsigma2, as this will not be considered in the P-calculations
			if (ires > 0)
				logsigma2 += log( 2. * PI * DIRECT_A1D_ELEM(mymodel.sigma2_noise[group_id], ires));
		}
		if (exp_sum_weight[ipart]==0)
		{
			std::cerr << " part_id= " << part_id << std::endl;
			std::cerr << " ipart= " << ipart << std::endl;
			std::cerr << " exp_min_diff2[ipart]= " << exp_min_diff2[ipart] << std::endl;
			std::cerr << " logsigma2= " << logsigma2 << std::endl;
			int group_id = mydata.getGroupId(part_id);
			std::cerr << " group_id= " << group_id << std::endl;
			std::cerr << " ml_model.scale_correction[group_id]= " << mymodel.scale_correction[group_id] << std::endl;
			std::cerr << " exp_significant_weight[ipart]= " << exp_significant_weight[ipart] << std::endl;
			std::cerr << " exp_max_weight[ipart]= " << exp_max_weight[ipart] << std::endl;
			std::cerr << " ml_model.sigma2_noise[group_id]= " << mymodel.sigma2_noise[group_id] << std::endl;
			REPORT_ERROR("ERROR: exp_sum_weight[ipart]==0");
		}
		double dLL;
		if ((iter==1 && do_firstiter_cc) || do_always_cc)
			dLL = -exp_min_diff2[ipart];
		else
			dLL = log(exp_sum_weight[ipart]) - exp_min_diff2[ipart] - logsigma2;

		// Store dLL of each image in the output array, and keep track of total sum
		DIRECT_A2D_ELEM(exp_metadata, metadata_offset + ipart, METADATA_DLL) = dLL;
		thr_sum_dLL += dLL;

		// Also store sum of Pmax
		thr_sum_Pmax += DIRECT_A2D_ELEM(exp_metadata, metadata_offset + ipart, METADATA_PMAX);

	}

	// Now, inside a global_mutex, update the other weighted sums among all threads
	if (!do_skip_maximization)
	{
		pthread_mutex_lock(&global_mutex);
		for (int n = 0; n < mymodel.nr_groups; n++)
		{
			wsum_model.sigma2_noise[n] += thr_wsum_sigma2_noise[n];
			wsum_model.sumw_group[n] += thr_sumw_group[n];
			if (do_scale_correction)
			{
				wsum_model.wsum_signal_product_spectra[n] += thr_wsum_signal_product_spectra[n];
				wsum_model.wsum_reference_power_spectra[n] += thr_wsum_reference_power_spectra[n];
			}
		}
		for (int n = 0; n < mymodel.nr_classes; n++)
		{
			wsum_model.pdf_class[n] += thr_wsum_pdf_class[n];
			if (mymodel.ref_dim == 2)
			{
				XX(wsum_model.prior_offset_class[n]) += thr_wsum_prior_offsetx_class[n];
				YY(wsum_model.prior_offset_class[n]) += thr_wsum_prior_offsety_class[n];
			}
			if (!(do_skip_align || do_skip_rotate) )
				wsum_model.pdf_direction[n] += thr_wsum_pdf_direction[n];
		}
		wsum_model.sigma2_offset += thr_wsum_sigma2_offset;
		if (do_norm_correction)
			wsum_model.avg_norm_correction += thr_avg_norm_correction;
		wsum_model.LL += thr_sum_dLL;
		wsum_model.ave_Pmax += thr_sum_Pmax;
		pthread_mutex_unlock(&global_mutex);
	} // end if !do_skip_maximization

	printf("Done doing other stuff\n");
}

//void MlOptimiserCUDA::precalculateModelProjectionsCtfsAndInvSigma2s(bool do_also_unmasked,
//	    int exp_current_image_size, int exp_current_oversampling,
//		std::vector<MultidimArray<Complex > > &model,
//		std::vector<MultidimArray<Complex > > &exp_Fimgs_nomask,
//		std::vector<MultidimArray<double> > &exp_Fctfs,
//		std::vector<MultidimArray<Complex > > &exp_local_Fimgs_shifted,
//		std::vector<MultidimArray<Complex > > &exp_local_Fimgs_shifted_nomask,
//		std::vector<MultidimArray<double> > &exp_local_Fctfs,
//		std::vector<double> &exp_local_sqrtXi2,
//		std::vector<MultidimArray<double> > &exp_local_Minvsigma2s)
//{
//
//}

void MlOptimiserCUDA::precalculateShiftedImagesCtfsAndInvSigma2s(bool do_also_unmasked,
		long int my_ori_particle, int exp_current_image_size, int exp_current_oversampling,
		int exp_itrans_min, int exp_itrans_max,
		std::vector<MultidimArray<Complex > > &exp_Fimgs,
		std::vector<MultidimArray<Complex > > &exp_Fimgs_nomask,
		std::vector<MultidimArray<double> > &exp_Fctfs,
		std::vector<MultidimArray<Complex > > &exp_local_Fimgs_shifted,
		std::vector<MultidimArray<Complex > > &exp_local_Fimgs_shifted_nomask,
		std::vector<MultidimArray<double> > &exp_local_Fctfs,
		std::vector<double> &exp_local_sqrtXi2,
		std::vector<MultidimArray<double> > &exp_local_Minvsigma2s)
{

	int exp_nr_particles = mydata.ori_particles[my_ori_particle].particles_id.size();
	int nr_shifts = (do_shifts_onthefly || do_skip_align) ? exp_nr_particles : exp_nr_particles * sampling.NrTranslationalSamplings(exp_current_oversampling);
	// Don't re-do if nothing has changed....
	bool do_ctf_invsig = (exp_local_Fctfs.size() > 0) ? YSIZE(exp_local_Fctfs[0])  != exp_current_image_size : true; // size has changed
	bool do_masked_shifts = (do_ctf_invsig || nr_shifts != exp_local_Fimgs_shifted.size()); // size or nr_shifts has changed

	// Use pre-sized vectors instead of push_backs!!
	exp_local_Fimgs_shifted.resize(nr_shifts);
	if (do_also_unmasked)
		exp_local_Fimgs_shifted_nomask.resize(nr_shifts);
	exp_local_Minvsigma2s.resize(exp_nr_particles);
	exp_local_Fctfs.resize(exp_nr_particles);
	exp_local_sqrtXi2.resize(exp_nr_particles);

	MultidimArray<Complex > Fimg, Fimg_nomask;
	for (int ipart = 0, my_trans_image = 0; ipart < mydata.ori_particles[my_ori_particle].particles_id.size(); ipart++)
	{
		long int part_id = mydata.ori_particles[my_ori_particle].particles_id[ipart];
		int group_id = mydata.getGroupId(part_id);

		if (do_masked_shifts)
			windowFourierTransform(exp_Fimgs[ipart], Fimg, exp_current_image_size);
		if (do_also_unmasked)
			windowFourierTransform(exp_Fimgs_nomask[ipart], Fimg_nomask, exp_current_image_size);

		if (do_ctf_invsig)
		{
			// Also precalculate the sqrt of the sum of all Xi2
			// Could exp_current_image_size ever be different from mymodel.current_size?
			// Probably therefore do it here rather than in getFourierTransforms
			if ((iter == 1 && do_firstiter_cc) || do_always_cc)
			{
				double sumxi2 = 0.;
				FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(Fimg)
				{
					sumxi2 += norm(DIRECT_MULTIDIM_ELEM(Fimg, n));
				}
				// Normalised cross-correlation coefficient: divide by power of reference (power of image is a constant)
				exp_local_sqrtXi2[ipart] = sqrt(sumxi2);
			}

			// Also store downsized Fctfs
			// In the second pass of the adaptive approach this will have no effect,
			// since then exp_current_image_size will be the same as the size of exp_Fctfs
			windowFourierTransform(exp_Fctfs[ipart], exp_local_Fctfs[ipart], exp_current_image_size);

			// Also prepare Minvsigma2
			if (mymodel.data_dim == 3)
				exp_local_Minvsigma2s[ipart].initZeros(ZSIZE(Fimg), YSIZE(Fimg), XSIZE(Fimg));
			else
				exp_local_Minvsigma2s[ipart].initZeros(YSIZE(Fimg), XSIZE(Fimg));

			int *myMresol = (YSIZE(Fimg) == coarse_size) ? Mresol_coarse.data : Mresol_fine.data;
			// With group_id and relevant size of Fimg, calculate inverse of sigma^2 for relevant parts of Mresol
			FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(exp_local_Minvsigma2s[ipart])
			{
				int ires = *(myMresol + n);
				// Exclude origin (ires==0) from the Probability-calculation
				// This way we are invariant to additive factors
				if (ires > 0)
					DIRECT_MULTIDIM_ELEM(exp_local_Minvsigma2s[ipart], n) = 1. / (sigma2_fudge * DIRECT_A1D_ELEM(mymodel.sigma2_noise[group_id], ires));
			}

		}

		if (do_shifts_onthefly)
		{
			// Store a single, down-sized version of exp_Fimgs[ipart] in exp_local_Fimgs_shifted
			if (do_masked_shifts)
				exp_local_Fimgs_shifted[ipart] = Fimg;
			if (do_also_unmasked)
				exp_local_Fimgs_shifted_nomask[ipart] = Fimg_nomask;
		}
		else
		{
			// Store all translated variants of Fimg
			for (long int itrans = exp_itrans_min; itrans <= exp_itrans_max; itrans++)
			{
				// First get the non-oversampled translations as defined by the sampling object
				std::vector<double > oversampled_translations_x, oversampled_translations_y, oversampled_translations_z;
				sampling.getTranslations(itrans, exp_current_oversampling, oversampled_translations_x,
						oversampled_translations_y, oversampled_translations_z);
				// Then loop over all its oversampled relatives
				for (long int iover_trans = 0; iover_trans < oversampled_translations_x.size(); iover_trans++, my_trans_image++)
				{
					// Shift through phase-shifts in the Fourier transform
					// Note that the shift search range is centered around (exp_old_xoff, exp_old_yoff)
					if (do_masked_shifts)
					{
						exp_local_Fimgs_shifted[my_trans_image].resize(Fimg);
						if (mymodel.data_dim ==2)
							shiftImageInFourierTransform(Fimg, exp_local_Fimgs_shifted[my_trans_image],
									tab_sin, tab_cos, (double)mymodel.ori_size,
									oversampled_translations_x[iover_trans],
									oversampled_translations_y[iover_trans]);
						else
							shiftImageInFourierTransform(Fimg, exp_local_Fimgs_shifted[my_trans_image],
									tab_sin, tab_cos, (double)mymodel.ori_size,
									oversampled_translations_x[iover_trans],
									oversampled_translations_y[iover_trans],
									oversampled_translations_z[iover_trans]);
					}
					if (do_also_unmasked)
					{
						exp_local_Fimgs_shifted_nomask[my_trans_image].resize(Fimg_nomask);
						if (mymodel.data_dim ==2)
							shiftImageInFourierTransform(Fimg_nomask, exp_local_Fimgs_shifted_nomask[my_trans_image],
								tab_sin, tab_cos, (double)mymodel.ori_size,
								oversampled_translations_x[iover_trans],
								oversampled_translations_y[iover_trans]);
						else
							shiftImageInFourierTransform(Fimg_nomask, exp_local_Fimgs_shifted_nomask[my_trans_image],
								tab_sin, tab_cos, (double)mymodel.ori_size,
								oversampled_translations_x[iover_trans],
								oversampled_translations_y[iover_trans],
								oversampled_translations_z[iover_trans]);
					}
				}
			}
		}
	}
}

bool MlOptimiserCUDA::isSignificantAnyParticleAnyTranslation(long int iorient, int exp_itrans_min, int exp_itrans_max, MultidimArray<bool> &exp_Mcoarse_significant)
{

	long int exp_nr_trans = exp_itrans_max - exp_itrans_min + 1;
	for (long int ipart = 0; ipart < YSIZE(exp_Mcoarse_significant); ipart++)
	{
		long int ihidden = iorient * exp_nr_trans;
		for (long int itrans = exp_itrans_min; itrans <= exp_itrans_max; itrans++, ihidden++)
		{
			if (DIRECT_A2D_ELEM(exp_Mcoarse_significant, ipart, ihidden))
				return true;
		}
	}
	return false;

}
