#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <ctime>
#include <iostream>
#include "src/gpu_utils/cuda_ml_optimiser.h"
#include "src/gpu_utils/cuda_helper_kernels.cuh"
#include "src/gpu_utils/cuda_projection_kernels.cuh"
#include "src/gpu_utils/cuda_difference_kernels.cuh"
#include "src/gpu_utils/cuda_ProjDiff_kernels.cuh"
#include "src/gpu_utils/cuda_utils.cuh"
#include "src/complex.h"
#include <fstream>
#include <hip/hip_runtime.h>
#include "src/parallel.h"
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/extrema.h>


static pthread_mutex_t global_mutex2[NR_CLASS_MUTEXES] = { PTHREAD_MUTEX_INITIALIZER };
static pthread_mutex_t global_mutex = PTHREAD_MUTEX_INITIALIZER;

/*
 * Maps weights to a decoupled indexing of translations and orientations
 */
inline
void mapWeights(CudaGlobalPtr<FLOAT> &mapped_weights, unsigned orientation_num, unsigned translation_num,
		HealpixSampling &sampling, long int ipart,
		std::vector< long unsigned > &iover_transes, std::vector< long unsigned > &ihiddens,
		std::vector< long unsigned > &iorientclasses, std::vector< long unsigned > &iover_rots,
		MultidimArray<FLOAT> &Mweight, unsigned long current_oversampling, unsigned long nr_trans)
{

	for (long unsigned i = 0; i < orientation_num; i++)
	{
		long unsigned iover_rot = iover_rots[i];
		for (long unsigned j = 0; j < translation_num; j++)
		{
			long unsigned iover_trans = iover_transes[j];
			long unsigned ihidden = iorientclasses[i] * nr_trans + ihiddens[j];
			long unsigned ihidden_over = sampling.getPositionOversampledSamplingPoint(ihidden,
									  current_oversampling, iover_rot, iover_trans);
			mapped_weights[(long unsigned) i * translation_num + j] =
					DIRECT_A2D_ELEM(Mweight, ipart, ihidden_over);
			//Mweight[(i)*(v).xdim+(j)]
		}
	}
}


inline
long unsigned imageTranslation(
		CudaGlobalPtr<FLOAT> &Fimgs_real, CudaGlobalPtr<FLOAT> &Fimgs_imag,
		CudaGlobalPtr<FLOAT> &Fimgs_nomask_real, CudaGlobalPtr<FLOAT> &Fimgs_nomask_imag,
		long int itrans_min, long int itrans_max, int adaptive_oversampling , HealpixSampling &sampling,
		std::vector<double> &oversampled_translations_x, std::vector<double> &oversampled_translations_y, std::vector<double> &oversampled_translations_z,
		unsigned long nr_oversampled_trans, std::vector<MultidimArray<Complex> > &global_fftshifts_ab_current, std::vector<MultidimArray<Complex> > &global_fftshifts_ab2_current,
		MultidimArray<Complex > &local_Fimgs_shifted, MultidimArray<Complex > &local_Fimgs_shifted_nomask,
		std::vector< long unsigned > &iover_transes, std::vector< long unsigned > &itranses, std::vector< long unsigned > &ihiddens,
		unsigned image_size)
{

	long unsigned translation_num(0), ihidden(0);

	for (long int itrans = itrans_min, iitrans = 0; itrans <= itrans_max; itrans++, ihidden++)
	{
		sampling.getTranslations(itrans, adaptive_oversampling,
				oversampled_translations_x, oversampled_translations_y, oversampled_translations_z);

		for (long int iover_trans = 0; iover_trans < nr_oversampled_trans; iover_trans++, iitrans++)
		{
			Complex* myAB;
			myAB = (adaptive_oversampling == 0 ) ? global_fftshifts_ab_current[iitrans].data : global_fftshifts_ab2_current[iitrans].data;


			FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(local_Fimgs_shifted)
			{
				FLOAT a = (*(myAB + n)).real;
				FLOAT b = (*(myAB + n)).imag;

				// Fimg_shift
				FLOAT real = a * (DIRECT_MULTIDIM_ELEM(local_Fimgs_shifted, n)).real
						- b *(DIRECT_MULTIDIM_ELEM(local_Fimgs_shifted, n)).imag;
				FLOAT imag = a * (DIRECT_MULTIDIM_ELEM(local_Fimgs_shifted, n)).imag
						+ b *(DIRECT_MULTIDIM_ELEM(local_Fimgs_shifted, n)).real;
				Fimgs_real[translation_num * image_size + n] = real;
				Fimgs_imag[translation_num * image_size + n] = imag;

				// Fimg_shift_nomask
				real = a * (DIRECT_MULTIDIM_ELEM(local_Fimgs_shifted_nomask, n)).real
						- b *(DIRECT_MULTIDIM_ELEM(local_Fimgs_shifted_nomask, n)).imag;
				imag = a * (DIRECT_MULTIDIM_ELEM(local_Fimgs_shifted_nomask, n)).imag
						+ b *(DIRECT_MULTIDIM_ELEM(local_Fimgs_shifted_nomask, n)).real;
				Fimgs_nomask_real[translation_num * image_size + n] = real;
				Fimgs_nomask_imag[translation_num * image_size + n] = imag;
			}

			translation_num ++;

			ihiddens.push_back(ihidden);
			itranses.push_back(itrans);
			iover_transes.push_back(iover_trans);
		}
	}

	Fimgs_real.size = translation_num * image_size;
	Fimgs_imag.size = translation_num * image_size;

	Fimgs_nomask_real.size = translation_num * image_size;
	Fimgs_nomask_imag.size = translation_num * image_size;

	return translation_num;
}


void generateEulerMatrices(
		FLOAT padding_factor,
		std::vector< double > &rots,
		std::vector< double > &tilts,
		std::vector< double > &psis,
		CudaGlobalPtr<FLOAT> &eulers,
		bool inverse)
{
	double alpha, beta, gamma;
    double ca, sa, cb, sb, cg, sg;
    double cc, cs, sc, ss;

	for (long int i = 0; i < rots.size(); i++)
	{
	    //TODO In a sense we're doing RAD2DEG just to do DEG2RAD here.
	    //The only place the degree value is actually used is in the metadata assignment.

	    alpha = DEG2RAD(rots[i]);
	    beta  = DEG2RAD(tilts[i]);
	    gamma = DEG2RAD(psis[i]);

	    sincos(alpha, &sa, &ca);
	    sincos(beta,  &sb, &cb);
	    sincos(gamma, &sg, &cg);

	    cc = cb * ca;
	    cs = cb * sa;
	    sc = sb * ca;
	    ss = sb * sa;

		if(inverse)
		{
		    eulers[9 * i + 0] = ( cg * cc - sg * sa) * padding_factor; //00
		    eulers[9 * i + 1] = (-sg * cc - cg * sa) * padding_factor; //10
		    eulers[9 * i + 2] = ( sc )               * padding_factor; //20
		    eulers[9 * i + 3] = ( cg * cs + sg * ca) * padding_factor; //01
		    eulers[9 * i + 4] = (-sg * cs + cg * ca) * padding_factor; //11
		    eulers[9 * i + 5] = ( ss )               * padding_factor; //21
		    eulers[9 * i + 6] = (-cg * sb )          * padding_factor; //02
		    eulers[9 * i + 7] = ( sg * sb )          * padding_factor; //12
		    eulers[9 * i + 8] = ( cb )               * padding_factor; //22
		}
		else
		{
		    eulers[9 * i + 0] = ( cg * cc - sg * sa) * padding_factor; //00
		    eulers[9 * i + 1] = ( cg * cs + sg * ca) * padding_factor; //01
		    eulers[9 * i + 2] = (-cg * sb )          * padding_factor; //02
		    eulers[9 * i + 3] = (-sg * cc - cg * sa) * padding_factor; //10
		    eulers[9 * i + 4] = (-sg * cs + cg * ca) * padding_factor; //11
		    eulers[9 * i + 5] = ( sg * sb )          * padding_factor; //12
		    eulers[9 * i + 6] = ( sc )               * padding_factor; //20
		    eulers[9 * i + 7] = ( ss )               * padding_factor; //21
		    eulers[9 * i + 8] = ( cb )               * padding_factor; //22
		}
	}
}


void generateEulerMatrices(
		std::vector< double > &psis,
		CudaGlobalPtr<FLOAT> &eulers,
		bool inverse)
{
    double gamma, c, s;

	for (long int i = 0; i < psis.size(); i++)
	{
	    //TODO In a sense we're doing RAD2DEG just to do DEG2RAD here.
	    //The only place the degree value is actually used is in the metadata assignment.

	    gamma = DEG2RAD(psis[i]);
	    sincos(gamma, &s, &c);

		if(inverse) //Noticed here that inverse actually yields the opposite (Hmmm)
		{
		    eulers[4 * i + 0] =  c; //00
		    eulers[4 * i + 1] = -s; //10
		    eulers[4 * i + 3] =  s; //01
		    eulers[4 * i + 4] =  c; //11
		}
		else
		{
		    eulers[4 * i + 0] =  c; //00
		    eulers[4 * i + 1] =  s; //01
		    eulers[4 * i + 3] = -s; //10
		    eulers[4 * i + 4] =  c; //11
		}
	}
}

long int generateProjectionSetup(
		OptimisationParamters &op,
		SamplingParameters &sp,
		MlOptimiser *baseMLO,
		bool coarse,
		unsigned iclass,
		std::vector< double > &rots,
		std::vector< double > &tilts,
		std::vector< double > &psis,
		std::vector< long unsigned > &iorientclasses,
		std::vector< long unsigned > &iover_rots)
{
	//Local variables
	std::vector< double > oversampled_rot, oversampled_tilt, oversampled_psi;
	long int orientation_num = 0;

	unsigned parts_size(sp.nr_psi * sp.nr_oversampled_rot);
	std::vector< double > rots_parts(parts_size);
	std::vector< double > tilts_parts(parts_size);
	std::vector< double > psis_parts(parts_size);
	std::vector< long unsigned > iorientclasses_parts(parts_size);
	std::vector< long unsigned > iover_rots_parts(parts_size);

	for (long int idir = sp.idir_min, iorient = 0; idir <= sp.idir_max; idir++)
	{
		for (long int ipsi = sp.ipsi_min, ipart = 0; ipsi <= sp.ipsi_max; ipsi++, iorient++)
		{
			long int iorientclass = iclass * sp.nr_dir * sp.nr_psi + iorient;

			// Get prior for this direction and skip calculation if prior==0
			double pdf_orientation;
			if (baseMLO->do_skip_align || baseMLO->do_skip_rotate)
			{
				pdf_orientation = baseMLO->mymodel.pdf_class[iclass];
			}
			else if (baseMLO->mymodel.orientational_prior_mode == NOPRIOR)
			{
				pdf_orientation = DIRECT_MULTIDIM_ELEM(baseMLO->mymodel.pdf_direction[iclass], idir);
			}
			else
			{
				pdf_orientation = op.directions_prior[idir] * op.psi_prior[ipsi];
			}
			// In the first pass, always proceed
			// In the second pass, check whether one of the translations for this orientation of any of the particles had a significant weight in the first pass
			// if so, proceed with projecting the reference in that direction

			bool do_proceed = coarse ? true :
					baseMLO->isSignificantAnyParticleAnyTranslation(iorientclass, sp.itrans_min, sp.itrans_max, op.Mcoarse_significant);

			if (do_proceed && pdf_orientation > 0.)
			{
				// Now get the oversampled (rot, tilt, psi) triplets
				// This will be only the original (rot,tilt,psi) triplet in the first pass (sp.current_oversampling==0)
				baseMLO->sampling.getOrientations(idir, ipsi, sp.current_oversampling, oversampled_rot, oversampled_tilt, oversampled_psi,
						op.pointer_dir_nonzeroprior, op.directions_prior, op.pointer_psi_nonzeroprior, op.psi_prior);

				// Loop over all oversampled orientations (only a single one in the first pass)
				for (long int iover_rot = 0; iover_rot < sp.nr_oversampled_rot; iover_rot++, ipart++)
				{
					iorientclasses_parts[ipart] = iorientclass;
					iover_rots_parts[ipart] = iover_rot;

					rots_parts[ipart] = oversampled_rot[iover_rot];
					tilts_parts[ipart] = oversampled_tilt[iover_rot];
					psis_parts[ipart] = oversampled_psi[iover_rot];

					orientation_num ++;
				}
			}
		}

		//TODO check that the following sort always works out

		if (sp.current_oversampling > 0)
		{
			int oversampling_per_psi = ROUND(std::pow(2., sp.current_oversampling));
			int oversampling_per_dir = ROUND(std::pow(4., sp.current_oversampling));

			//Sort the angles to have coalesced rot/tilt order
			for (unsigned i = 0; i < oversampling_per_dir; i++) //Loop over the perturbed dir pairs
			{
				for (unsigned j = 0; j < sp.nr_psi; j++)
				{
					for (unsigned k = 0; k < oversampling_per_psi; k++) //two psis per perturbed dir pair
					{
						unsigned ij = j*oversampling_per_psi*oversampling_per_dir + i*oversampling_per_psi + k;

						iorientclasses.push_back(iorientclasses_parts[ij]);
						iover_rots.push_back(iover_rots_parts[ij]);

						rots.push_back(rots_parts[ij]);
						tilts.push_back(tilts_parts[ij]);
						psis.push_back(psis_parts[ij]);
					}
				}
			}
		}
		else
		{
			for (unsigned i = 0; i < iorientclasses_parts.size(); i++)
			{
				iorientclasses.push_back(iorientclasses_parts[i]);
				iover_rots.push_back(iover_rots_parts[i]);

				rots.push_back(rots_parts[i]);
				tilts.push_back(tilts_parts[i]);
				psis.push_back(psis_parts[i]);
			}
		}
	}

	return orientation_num;
}

void generateModelProjections(
		CudaGlobalPtr<FLOAT > &model_real,
		CudaGlobalPtr<FLOAT > &model_imag,
		CudaGlobalPtr<FLOAT> &Frefs_real,
		CudaGlobalPtr<FLOAT> &Frefs_imag,
		CudaGlobalPtr<FLOAT> &eulers,
		long unsigned orientation_num,
		unsigned image_size,
		unsigned max_r,
		unsigned img_x,
		unsigned img_y,
		unsigned mdl_x,
		unsigned mdl_y,
		unsigned mdl_z,
		unsigned mdl_init_y,
		unsigned mdl_init_z)
{

	int max_r2 = max_r * max_r;
	int min_r2_nn = 0; // r_min_nn * r_min_nn;  //FIXME add nn-algorithm

	/*===========================
	 *      TEXTURE STUFF
	 * ==========================*/

	// create channel to describe data type (bits,bits,bits,bits,type)
	// TODO model should carry real & imag in separate channels of the same texture
	hipChannelFormatDesc channel = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipArray*        modelArray_real;
	hipArray* 		  modelArray_imag;
	hipExtent        volumeSize = make_hipExtent(mdl_x, mdl_y, mdl_z);

	//allocate device memory for cuda 3D array
	hipMalloc3DArray(&modelArray_real, &channel, volumeSize);
	hipMalloc3DArray(&modelArray_imag, &channel, volumeSize);

	//set cuda array copy parameters to be supplied to copy-command
	hipMemcpy3DParms copyParams = {0};
	copyParams.extent   = volumeSize;

	copyParams.kind     = hipMemcpyHostToDevice;
	copyParams.dstArray = modelArray_real;
	copyParams.srcPtr   = make_hipPitchedPtr(model_real.h_ptr,volumeSize.width*sizeof(FLOAT), volumeSize.height, volumeSize.depth);
	hipMemcpy3D(&copyParams);
	copyParams.kind     = hipMemcpyHostToDevice;
	copyParams.dstArray = modelArray_imag;
	copyParams.srcPtr   = make_hipPitchedPtr(model_imag.h_ptr,volumeSize.width*sizeof(FLOAT), volumeSize.height, volumeSize.depth);
	hipMemcpy3D(&copyParams);

	// Create texture object// Specify texture
    struct hipResourceDesc resDesc_real,resDesc_imag;
    memset(&resDesc_real, 0, sizeof(resDesc_real));
    memset(&resDesc_imag, 0, sizeof(resDesc_imag));
    resDesc_real.resType = hipResourceTypeArray;
    resDesc_imag.resType = hipResourceTypeArray;
    resDesc_real.res.array.array = modelArray_real;
    resDesc_imag.res.array.array = modelArray_imag;

    struct hipTextureDesc texDesc_real, texDesc_imag;
    memset(&texDesc_real, 0, sizeof(texDesc_real));
    memset(&texDesc_imag, 0, sizeof(texDesc_imag));
    for(int n=0; n<3; n++)
	{
    	texDesc_real.addressMode[n]=hipAddressModeClamp;
    	texDesc_imag.addressMode[n]=hipAddressModeClamp;
	}
    texDesc_real.filterMode       = hipFilterModeLinear;
    texDesc_real.readMode         = hipReadModeElementType;
    texDesc_real.normalizedCoords = false;
    texDesc_imag.filterMode       = hipFilterModeLinear;
    texDesc_imag.readMode         = hipReadModeElementType;
    texDesc_real.normalizedCoords = false;

	hipTextureObject_t texModel_real = 0;
	hipCreateTextureObject(&texModel_real, &resDesc_real, &texDesc_real, NULL);
	hipTextureObject_t texModel_imag = 0;
	hipCreateTextureObject(&texModel_imag, &resDesc_imag, &texDesc_imag, NULL);

	Frefs_real.size = orientation_num * image_size;
	Frefs_real.device_alloc();
	Frefs_imag.size = orientation_num * image_size;
	Frefs_imag.device_alloc();

	unsigned int orient1, orient2;
	if(orientation_num>65535)
	{
		orient1 = ceil(sqrt(orientation_num));
		orient2 = orient1;
	}
	else
	{
		orient1 = orientation_num;
		orient2 = 1;
	}

	dim3 block_dim(orient1,orient2);
	std::cerr << "using block dimensions " << orient1 << "," << orient2 <<  std::endl;

#if !defined(CUDA_DOUBLE_PRECISION) && defined(USE_TEXINTERP)
	// we CAN use read-associated interpolation (fast, inaccurate)...
	cuda_kernel_PAV_TTI<<<block_dim,BLOCK_SIZE>>>(
													~eulers,
													~Frefs_real,
													~Frefs_imag,
													texModel_real,
													texModel_imag,
													max_r,
													max_r2,
													min_r2_nn,
													image_size,
													orientation_num,
													img_x,
													img_y,
													mdl_init_y,
													mdl_init_z);
	hipDestroyTextureObject(texModel_real);
	hipDestroyTextureObject(texModel_imag);
	hipFreeArray(modelArray_real);
	hipFreeArray(modelArray_imag);
#elif !defined(CUDA_DOUBLE_PRECISION)	// ...or explicit interpolation (slow, accurate)
	cuda_kernel_PAV_TTE<<<block_dim,BLOCK_SIZE>>>(
													~eulers,
													~Frefs_real,
													~Frefs_imag,
													texModel_real,
													texModel_imag,
													max_r,
													max_r2,
													min_r2_nn,
													image_size,
													orientation_num,
													img_x,
													img_y,
													mdl_init_y,
													mdl_init_z);

	hipDestroyTextureObject(texModel_real);
	hipDestroyTextureObject(texModel_imag);
	hipFreeArray(modelArray_real);
	hipFreeArray(modelArray_imag);
#else // under double precision, texture won't work.
	model_real.device_alloc();
    model_real.cp_to_device();
 	model_imag.device_alloc();
    model_imag.cp_to_device();
	cuda_kernel_PAV_TGE<<<block_dim,BLOCK_SIZE>>>(
													~model_real,
													~model_imag,
													~eulers,
													~Frefs_real,
													~Frefs_imag,
													max_r,
													max_r2,
													min_r2_nn,
													image_size,
													orientation_num,
													img_x,
													img_y,
													mdl_x,
													mdl_y,
													mdl_init_y,
													mdl_init_z);
	model_real.free_device();
	model_imag.free_device();
#endif

	//unbind texture reference to free resource



}


void runDifferenceKernel(CudaGlobalPtr<FLOAT > &gpuMinvsigma2,
		CudaGlobalPtr<FLOAT > &Fimgs_real,
		CudaGlobalPtr<FLOAT > &Fimgs_imag,
		CudaGlobalPtr<FLOAT > &Frefs_real,
		CudaGlobalPtr<FLOAT > &Frefs_imag,
		CudaGlobalPtr<long unsigned > &rotidx,
		CudaGlobalPtr<long unsigned > &transidx,
		CudaGlobalPtr<long unsigned > &ihidden_overs,
		OptimisationParamters &op,
		MlOptimiser *baseMLO,
		long unsigned translation_num,
		long unsigned orientation_num,
		long unsigned significant_num,
		unsigned image_size,
		int ipart,
		int group_id,
		CudaGlobalPtr<FLOAT > &diff2s
		)
{
	/*====================================
	   Initiate Particle Related On GPU
	======================================*/
	CUDA_CPU_TIC("kernel_init_1");
	// Since we hijack Minvsigma to carry a bit more info into the GPU-kernel
	// we need to make a modified copy, since the global object shouldn't be
	// changed
	FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(op.local_Fimgs_shifted[ipart])
	{
		gpuMinvsigma2[n] = *(op.local_Minvsigma2s[ipart].data + n );
	}

	if (baseMLO->do_ctf_correction && baseMLO->refs_are_ctf_corrected)
	{
		FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(op.local_Fimgs_shifted[ipart])
		{
			gpuMinvsigma2[n] *= (DIRECT_MULTIDIM_ELEM(op.local_Fctfs[ipart], n)*DIRECT_MULTIDIM_ELEM(op.local_Fctfs[ipart], n));
		}
	}
	// TODO :    + Assure accuracy with the implemented GPU-based ctf-scaling
	//           + Make setting of myscale robust between here and above.
	//  (scale_correction turns off by default with only one group: ml_optimiser-line 1067,
	//   meaning small-scale test will probably not catch this malfunctioning when/if it breaks.)
	if (baseMLO->do_scale_correction)
	{
		FLOAT myscale = baseMLO->mymodel.scale_correction[group_id];
		FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(op.local_Fimgs_shifted[ipart])
		{
			gpuMinvsigma2[n] *= (myscale*myscale);
		}
	}

	gpuMinvsigma2.cp_to_device();

	Fimgs_real.size = translation_num * image_size;
	Fimgs_real.device_alloc();
	Fimgs_real.cp_to_device();
	Fimgs_imag.size = translation_num * image_size;
	Fimgs_imag.device_alloc();
	Fimgs_imag.cp_to_device();
	rotidx.size = significant_num;
	rotidx.device_alloc();
	rotidx.cp_to_device();
	transidx.size = significant_num;
	transidx.device_alloc();
	transidx.cp_to_device();
	ihidden_overs.size = significant_num;
	ihidden_overs.device_alloc();
	ihidden_overs.cp_to_device();

	/*====================================
				Kernel Calls
	======================================*/
	unsigned orient1, orient2;

	if(significant_num>65535)
	{
		orient1 = ceil(sqrt(significant_num));
		orient2 = orient1;
	}
	else
	{
		orient1 = significant_num;
		orient2 = 1;
	}
	dim3 block_dim(orient1,orient2);

	CUDA_CPU_TOC("kernel_init_1");
	CUDA_GPU_TIC("kernel_diff_noproj");
	// Could be used to automate __ldg() fallback runtime within cuda_kernel_diff2.
//				hipDeviceProp_t dP;
//				hipGetDeviceProperties(&dP, 0);
//				printf("-arch=sm_%d%d\n", dP.major, dP.minor);

	if ((baseMLO->iter == 1 && baseMLO->do_firstiter_cc) || baseMLO->do_always_cc) // do cross-correlation instead of diff
	{
		cuda_kernel_D2_CC<<<block_dim,BLOCK_SIZE>>>(~Frefs_real, ~Frefs_imag, ~Fimgs_real, ~Fimgs_imag, ~gpuMinvsigma2,  ~diff2s,
														image_size, op.highres_Xi2_imgs[ipart],
														significant_num,
														translation_num,
														~rotidx,
														~transidx);
	}
	else
	{
		cuda_kernel_D2<<<block_dim,BLOCK_SIZE>>>(~Frefs_real, ~Frefs_imag, ~Fimgs_real, ~Fimgs_imag, ~gpuMinvsigma2, ~diff2s,
													image_size, op.highres_Xi2_imgs[ipart] / 2.,
													significant_num,
													translation_num,
													~rotidx,
													~transidx,
													~ihidden_overs);
	}
	CUDA_GPU_TAC("kernel_diff_noproj");
	HANDLE_ERROR(hipDeviceSynchronize()); //TODO Apparently this is not required here
	CUDA_GPU_TOC("kernel_diff_noproj");
	size_t avail;
	size_t total;
	hipMemGetInfo( &avail, &total );
	float used = 100*((float)(total - avail)/(float)total);
	std::cerr << "Device memory used @ diff2: " << used << "%" << std::endl;
}

#if !defined(CUDA_DOUBLE_PRECISION)
void runProjAndDifferenceKernel(
		CudaGlobalPtr<FLOAT > &model_real,
		CudaGlobalPtr<FLOAT > &model_imag,
		CudaGlobalPtr<FLOAT > &gpuMinvsigma2,
		CudaGlobalPtr<FLOAT> &Fimgs_real,
		CudaGlobalPtr<FLOAT> &Fimgs_imag,
		CudaGlobalPtr<FLOAT> &eulers,
		CudaGlobalPtr<long unsigned> &rotidx,
		CudaGlobalPtr<long unsigned> &transidx,
		CudaGlobalPtr<long unsigned> &trans_num,
		CudaGlobalPtr<long unsigned> &ihidden_overs,
		CudaGlobalPtr<FLOAT> &diff2s,
		OptimisationParamters op,
		MlOptimiser *baseMLO,
		long unsigned orientation_num,
		long unsigned translation_num,
		long unsigned significant_num,
		unsigned image_size,
		unsigned max_r,
		int ipart,
		int group_id,
		int exp_iclass)
{

	CUDA_CPU_TIC("kernel_init_1");
	int max_r2 = max_r * max_r;
	int min_r2_nn = 0; // r_min_nn * r_min_nn;  //FIXME add nn-algorithm

	/*===========================
	 *      TEXTURE STUFF
	 * ==========================*/

	// create channel to describe data type (bits,bits,bits,bits,type)
	// TODO model should carry real & imag in separate channels of the same texture
	hipChannelFormatDesc channel = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipArray*        modelArray_real;
	hipArray* 		  modelArray_imag;
	hipExtent        volumeSize = make_hipExtent(baseMLO->mymodel.PPref[exp_iclass].data.xdim,
			   	   	   	   	   	   	   	   	       baseMLO->mymodel.PPref[exp_iclass].data.ydim,
			   	   	   	   	   	   	   	   	       baseMLO->mymodel.PPref[exp_iclass].data.zdim);

	//allocate device memory for cuda 3D array
	hipMalloc3DArray(&modelArray_real, &channel, volumeSize);
	hipMalloc3DArray(&modelArray_imag, &channel, volumeSize);

	//set cuda array copy parameters to be supplied to copy-command
	hipMemcpy3DParms copyParams = {0};
	copyParams.extent   = volumeSize;

	copyParams.kind     = hipMemcpyHostToDevice;
	copyParams.dstArray = modelArray_real;
	copyParams.srcPtr   = make_hipPitchedPtr(model_real.h_ptr,volumeSize.width*sizeof(FLOAT), volumeSize.height, volumeSize.depth);
	hipMemcpy3D(&copyParams);
	copyParams.kind     = hipMemcpyHostToDevice;
	copyParams.dstArray = modelArray_imag;
	copyParams.srcPtr   = make_hipPitchedPtr(model_imag.h_ptr,volumeSize.width*sizeof(FLOAT), volumeSize.height, volumeSize.depth);
	hipMemcpy3D(&copyParams);

	// Create texture object// Specify texture
	struct hipResourceDesc resDesc_real,resDesc_imag;
	memset(&resDesc_real, 0, sizeof(resDesc_real));
	memset(&resDesc_imag, 0, sizeof(resDesc_imag));
	resDesc_real.resType = hipResourceTypeArray;
	resDesc_imag.resType = hipResourceTypeArray;
	resDesc_real.res.array.array = modelArray_real;
	resDesc_imag.res.array.array = modelArray_imag;

	struct hipTextureDesc texDesc_real, texDesc_imag;
	memset(&texDesc_real, 0, sizeof(texDesc_real));
	memset(&texDesc_imag, 0, sizeof(texDesc_imag));
	for(int n=0; n<3; n++)
	{
		texDesc_real.addressMode[n]=hipAddressModeClamp;
		texDesc_imag.addressMode[n]=hipAddressModeClamp;
	}
	texDesc_real.filterMode       = hipFilterModeLinear;
	texDesc_real.readMode         = hipReadModeElementType;
	texDesc_real.normalizedCoords = false;
	texDesc_imag.filterMode       = hipFilterModeLinear;
	texDesc_imag.readMode         = hipReadModeElementType;
	texDesc_real.normalizedCoords = false;

	hipTextureObject_t texModel_real = 0;
	hipCreateTextureObject(&texModel_real, &resDesc_real, &texDesc_real, NULL);
	hipTextureObject_t texModel_imag = 0;
	hipCreateTextureObject(&texModel_imag, &resDesc_imag, &texDesc_imag, NULL);


	// Since we hijack Minvsigma to carry a bit more info into the GPU-kernel
	// we need to make a modified copy, since the global object shouldn't be
	// changed
	FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(op.local_Fimgs_shifted[ipart])
	{
		gpuMinvsigma2[n] = *(op.local_Minvsigma2s[ipart].data + n );
	}

	if (baseMLO->do_ctf_correction && baseMLO->refs_are_ctf_corrected)
	{
		FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(op.local_Fimgs_shifted[ipart])
		{
			gpuMinvsigma2[n] *= (DIRECT_MULTIDIM_ELEM(op.local_Fctfs[ipart], n)*DIRECT_MULTIDIM_ELEM(op.local_Fctfs[ipart], n));
		}
	}
	// TODO :    + Assure accuracy with the implemented GPU-based ctf-scaling
	//           + Make setting of myscale robust between here and above.
	//  (scale_correction turns off by default with only one group: ml_optimiser-line 1067,
	//   meaning small-scale test will probably not catch this malfunctioning when/if it breaks.)
	if (baseMLO->do_scale_correction)
	{
		FLOAT myscale = baseMLO->mymodel.scale_correction[group_id];
		FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(op.local_Fimgs_shifted[ipart])
		{
			gpuMinvsigma2[n] *= (myscale*myscale);
		}
	}

	gpuMinvsigma2.cp_to_device();

	unsigned orient1, orient2;
    unsigned long block_num = trans_num.size;//significant_num;

	if(block_num>65535)
	{
		orient1 = ceil(sqrt(block_num));
		orient2 = orient1;
	}
	else
	{
		orient1 = block_num;
		orient2 = 1;
	}
	dim3 block_dim(orient1,orient2);

	Fimgs_real.size = translation_num * image_size;
	Fimgs_real.device_alloc();
	Fimgs_real.cp_to_device();
	Fimgs_imag.size = translation_num * image_size;
	Fimgs_imag.device_alloc();
	Fimgs_imag.cp_to_device();
	rotidx.size = block_num;
	rotidx.device_alloc();
	rotidx.cp_to_device();
	transidx.size = block_num;
	transidx.device_alloc();
	transidx.cp_to_device();
//  trans_num.size = block_num; // already set
    trans_num.device_alloc();
    trans_num.cp_to_device();
	ihidden_overs.size = block_num;
	ihidden_overs.device_alloc();
	ihidden_overs.cp_to_device();


	CUDA_CPU_TOC("kernel_init_1");
	CUDA_GPU_TIC("kernel_diff_proj");


// Could be used to automate __ldg() fallback runtime within cuda_kernel_diff2.
//				hipDeviceProp_t dP;
//				hipGetDeviceProperties(&dP, 0);
//				printf("-arch=sm_%d%d\n", dP.major, dP.minor);

	if ((baseMLO->iter == 1 && baseMLO->do_firstiter_cc) || baseMLO->do_always_cc) // do cross-correlation instead of diff
	{
		// FIXME  make _CC
		exit(0);
//		cuda_kernel_PAV_TTI_D2_CC<<<block_dim,BLOCK_SIZE>>>(~eulers,
//														 ~Fimgs_real,
//														 ~Fimgs_imag,
//														 texModel_real,
//													 	 texModel_imag,
//														 ~gpuMinvsigma2,
//														 ~diff2s,
//														 image_size,
//														 op.highres_Xi2_imgs[ipart] / 2.,
//														 orientation_num,
//														 translation_num,
//														 significant_num,
//														 ~rotidx,
//														 ~transidx,
//		 	 	 	 	 	 	 	 	 	 	 	 	 ~trans_num,
//														 ~ihidden_overs,
//														 max_r,
//													     max_r2,
//													     min_r2_nn,
//														 op.local_Minvsigma2s[0].xdim,
//														 op.local_Minvsigma2s[0].ydim,
//														 baseMLO->mymodel.PPref[exp_iclass].data.yinit,
//														 baseMLO->mymodel.PPref[exp_iclass].data.zinit);
//		hipDestroyTextureObject(texModel_real);
//		hipDestroyTextureObject(texModel_imag);
//		hipFreeArray(modelArray_real);
//		hipFreeArray(modelArray_imag);
	}
	else
	{
		cuda_kernel_PAV_TTI_D2<<<block_dim,BLOCK_SIZE>>>(~eulers,
														 ~Fimgs_real,
														 ~Fimgs_imag,
														 texModel_real,
														 texModel_imag,
														 ~gpuMinvsigma2,
														 ~diff2s,
														 image_size,
														 op.highres_Xi2_imgs[ipart] / 2.,
														 orientation_num,
														 translation_num,
														 block_num, //significant_num,
														 ~rotidx,
														 ~transidx,
														 ~trans_num,
														 ~ihidden_overs,
														 max_r,
														 max_r2,
														 min_r2_nn,
														 op.local_Minvsigma2s[0].xdim,
														 op.local_Minvsigma2s[0].ydim,
														 baseMLO->mymodel.PPref[exp_iclass].data.yinit,
														 baseMLO->mymodel.PPref[exp_iclass].data.zinit);
		size_t avail;
		size_t total;
		hipMemGetInfo( &avail, &total );
		float used = 100*((float)(total - avail)/(float)total);
		std::cerr << "Device memory used @ diff2: " << used << "%" << std::endl;
		hipDestroyTextureObject(texModel_real);
		hipDestroyTextureObject(texModel_imag);
		hipFreeArray(modelArray_real);
		hipFreeArray(modelArray_imag);
	}
	CUDA_GPU_TAC("kernel_diff_proj");
	HANDLE_ERROR(hipDeviceSynchronize()); //TODO Apparently this is not required here

	CUDA_GPU_TOC("kernel_diff_proj");

}
#endif


#define BACKPROJECTION4_BLOCK_SIZE 64
#define BACKPROJECTION4_GROUP_SIZE 16
#define BACKPROJECTION4_FETCH_COUNT 4

__global__ void cuda_kernel_backproject(
		int *g_xs,
		int *g_ys,
		int *g_zs,
		FLOAT *g_model_real,
		FLOAT *g_model_imag,
		FLOAT *g_weight,
		FLOAT *g_eulers,
		FLOAT *g_wavgs_real,
		FLOAT *g_wavgs_imag,
		FLOAT *g_Fweights,
		int max_r2, FLOAT scale2,
		unsigned img_xy, unsigned long img_count, unsigned img_x, unsigned img_y,
		unsigned mdl_x, unsigned mdl_y, int mdl_inity, int mdl_initz,
		int N)
{
	unsigned gid = threadIdx.x / 4;
	unsigned mid = threadIdx.x % 4;
	unsigned gm = gid * 4 + mid;
	unsigned pit = (gid * 4 + mid)*BACKPROJECTION4_FETCH_COUNT;
	unsigned global_idx = blockIdx.x * BACKPROJECTION4_GROUP_SIZE + gid;

	int X(0),Y(0),Z(0);

	if (global_idx < N)
	{
		X = g_xs[global_idx];
		Y = g_ys[global_idx];
		Z = g_zs[global_idx];
	}
	else
		X = mdl_x * 10; // Padding coordinate, place outside images

	int ax(0), ay(0);

	if (mid == 1)
		ax = 1;
	else if (mid == 2)
		ay = 1;
	else if (mid == 3)
	{
		ax = 1;
		ay = 1;
	}

	bool  is_neg_x;
	FLOAT d, w;
	FLOAT xp,yp,zp;
	int x,y,idx;

	__shared__ FLOAT s_e[BACKPROJECTION4_BLOCK_SIZE*BACKPROJECTION4_FETCH_COUNT];

	__shared__ FLOAT s_weight[BACKPROJECTION4_GROUP_SIZE*4];
	__shared__ FLOAT s_value_real[BACKPROJECTION4_GROUP_SIZE*4];
	__shared__ FLOAT s_value_imag[BACKPROJECTION4_GROUP_SIZE*4];

	s_weight[gm] = 0.0f;
	s_value_real[gm] = 0.0f;
	s_value_imag[gm] = 0.0f;

	for (int img = 0, b = BACKPROJECTION4_BLOCK_SIZE*BACKPROJECTION4_FETCH_COUNT; img < img_count; img ++, b += 9)
	{
		if (b+9 > BACKPROJECTION4_BLOCK_SIZE*BACKPROJECTION4_FETCH_COUNT)
		{
			__syncthreads();

			int img_9 = img*9+pit;
			if (img_9 < img_count*9)
			{
				s_e[pit+0] = g_eulers[img_9+0];
				s_e[pit+1] = g_eulers[img_9+1];
				s_e[pit+2] = g_eulers[img_9+2];
				s_e[pit+3] = g_eulers[img_9+3];
			}

			__syncthreads();
			b = 0;
		}

		zp = (s_e[b+6] * X + s_e[b+7] * Y + s_e[b+8] * Z);

		if (fabsf(zp) > 0.87f) continue; //Within the unit cube, sqrt(3)/2=0.866

		yp = (s_e[b+3] * X + s_e[b+4] * Y + s_e[b+5] * Z);
		xp = (s_e[b+0] * X + s_e[b+1] * Y + s_e[b+2] * Z);

		if (xp < 0.0f)
		{
			yp = -yp;
			xp = -xp;
			is_neg_x = true;
		}
		else
			is_neg_x = false;

		x = (int) floorf(xp) + ax;
		y = (int) floorf(yp) + ay;

		if (x * x + y * y > max_r2) continue;

		if (y < 0 && x == 0)
		{
			is_neg_x = !is_neg_x;
			y = -y;
		}

		xp = (s_e[b+0] * x + s_e[b+3] * y) * scale2;
		yp = (s_e[b+1] * x + s_e[b+4] * y) * scale2;
		zp = (s_e[b+2] * x + s_e[b+5] * y) * scale2;

		if (xp < 0.0f) //Flip sign
		{
			xp = fabsf(X+xp);
			yp = fabsf(Y+yp);
			zp = fabsf(Z+zp);
		}
		else
		{
			xp = fabsf(X-xp);
			yp = fabsf(Y-yp);
			zp = fabsf(Z-zp);
		}

		if (xp < 1.0f && yp < 1.0f && zp < 1.0f)
		{
			if (y < 0) y += img_y;
			idx = img*img_xy + y * img_x + x;
			w = g_Fweights[idx];

			if (w > 0.0f)
			{
				d = (1.0f - xp) * (1.0f - yp) * (1.0f - zp);

				s_weight[gm] += w * d;
				s_value_real[gm] += g_wavgs_real[idx] * d;
				if (is_neg_x) s_value_imag[gm] -= g_wavgs_imag[idx] * d;
				else          s_value_imag[gm] += g_wavgs_imag[idx] * d;
			}
		}
	}

	__syncthreads();

	if (mid == 0)
	{
		FLOAT sum = s_weight[gid*4 + 0] + s_weight[gid*4 + 1] + s_weight[gid*4 + 2] + s_weight[gid*4 + 3];
		if (sum != 0.0f)
			g_weight[(Z-mdl_initz)*mdl_x*mdl_y + (Y-mdl_inity)*mdl_x + X] = sum;
	}
	else if (mid == 1)
	{
		FLOAT sum = s_value_real[gid*4 + 0] + s_value_real[gid*4 + 1] + s_value_real[gid*4 + 2] + s_value_real[gid*4 + 3];
		if (sum != 0.0f)
			g_model_real[(Z-mdl_initz)*mdl_x*mdl_y + (Y-mdl_inity)*mdl_x + X] = sum;
	}
	else if (mid == 2)
	{
		FLOAT sum = s_value_imag[gid*4 + 0] + s_value_imag[gid*4 + 1] + s_value_imag[gid*4 + 2] + s_value_imag[gid*4 + 3];
		if (sum != 0.0f)
			g_model_imag[(Z-mdl_initz)*mdl_x*mdl_y + (Y-mdl_inity)*mdl_x + X] = sum;
	}
}

static void runBackprojectKernel(
		CudaGlobalPtr<FLOAT> &wavgs_real,
		CudaGlobalPtr<FLOAT> &wavgs_imag,
		CudaGlobalPtr<FLOAT> &Fweights,
		CudaGlobalPtr<FLOAT> &eulers,
		CudaGlobalPtr<FLOAT> &model_real,
		CudaGlobalPtr<FLOAT> &model_imag,
		CudaGlobalPtr<FLOAT> &weight,
		int max_r, FLOAT scale2, //grid scale 2D -> 3D squared
		int img_xy, long img_count, int img_x, int img_y,
		int mdl_x, int mdl_y, int mdl_z, int mdl_inity, int mdl_initz)
{
	int max_r2 = max_r * max_r;

	CudaGlobalPtr<int> xs(mdl_x*mdl_y*mdl_z); // >52% will actually be used, allocate some padding
	CudaGlobalPtr<int> ys(xs.size);
	CudaGlobalPtr<int> zs(xs.size);
	unsigned N(0);

	for (int x = 0; x < mdl_x; x ++)
	{
		for (int y = mdl_inity; y < mdl_y; y++)
		{
			for (int z = mdl_initz; z < mdl_z; z++)
			{
				if (x*x + y*y + z*z <= max_r2 * scale2 * 1.2f)
				{
					xs[N] = x;
					ys[N] = y;
					zs[N] = z;
					N ++;
				}
			}
		}
	}
	xs.size = N + N%BACKPROJECTION4_GROUP_SIZE;
	ys.size = xs.size;
	zs.size = xs.size;

	xs.device_alloc();
	ys.device_alloc();
	zs.device_alloc();

	xs.cp_to_device();
	ys.cp_to_device();
	zs.cp_to_device();

	int grid_dim = ceil((float)N / BACKPROJECTION4_GROUP_SIZE);
	dim3 block_dim( BACKPROJECTION4_GROUP_SIZE *4 );

	cuda_kernel_backproject<<<grid_dim,block_dim>>>(
			~xs,~ys,~zs,
			~model_real,
			~model_imag,
			~weight,
			~eulers,
			~wavgs_real,
			~wavgs_imag,
			~Fweights,
			max_r2,
			scale2,
			img_xy,
			img_count,
			img_x,
			img_y,
			mdl_x,
			mdl_y,
			mdl_inity,
			mdl_initz,
			N);
}









void MlOptimiserCuda::doThreadExpectationSomeParticles(unsigned thread_id)
{
	size_t first_ipart = 0, last_ipart = 0;
	while (baseMLO->exp_ipart_ThreadTaskDistributor->getTasks(first_ipart, last_ipart))
	{
		for (long unsigned ipart = first_ipart; ipart <= last_ipart; ipart++)
		{
			unsigned my_ori_particle = baseMLO->exp_my_first_ori_particle + ipart;
			SamplingParameters sp;
			sp.nr_particles = baseMLO->mydata.ori_particles[my_ori_particle].particles_id.size();

			OptimisationParamters op(sp.nr_particles, my_ori_particle);

			// In the first iteration, multiple seeds will be generated
			// A single random class is selected for each pool of images, and one does not marginalise over the orientations
			// The optimal orientation is based on signal-product (rather than the signal-intensity sensitive Gaussian)
			// If do_firstiter_cc, then first perform a single iteration with K=1 and cross-correlation criteria, afterwards

			// Decide which classes to integrate over (for random class assignment in 1st iteration)
			sp.iclass_max = baseMLO->mymodel.nr_classes - 1;
			// low-pass filter again and generate the seeds
			if (baseMLO->do_generate_seeds)
			{
				if (baseMLO->do_firstiter_cc && baseMLO->iter == 1)
				{
					// In first (CC) iter, use a single reference (and CC)
					sp.iclass_min = sp.iclass_max = 0;
				}
				else if ( (baseMLO->do_firstiter_cc && baseMLO->iter == 2) ||
						(!baseMLO->do_firstiter_cc && baseMLO->iter == 1))
				{
					// In second CC iter, or first iter without CC: generate the seeds
					// Now select a single random class
					// exp_part_id is already in randomized order (controlled by -seed)
					// WARNING: USING SAME iclass_min AND iclass_max FOR SomeParticles!!
					sp.iclass_min = sp.iclass_max = divide_equally_which_group(baseMLO->mydata.numberOfOriginalParticles(), baseMLO->mymodel.nr_classes, op.my_ori_particle);
				}
			}
			// Global exp_metadata array has metadata of all ori_particles. Where does my_ori_particle start?
			for (long int iori = baseMLO->exp_my_first_ori_particle; iori <= baseMLO->exp_my_last_ori_particle; iori++)
			{
				if (iori == my_ori_particle) break;
				op.metadata_offset += baseMLO->mydata.ori_particles[iori].particles_id.size();
			}

			baseMLO->getFourierTransformsAndCtfs(my_ori_particle, op.metadata_offset, op.Fimgs, op.Fimgs_nomask, op.Fctfs,
					op.old_offset, op.prior, op.power_imgs, op.highres_Xi2_imgs,
					op.pointer_dir_nonzeroprior, op.pointer_psi_nonzeroprior, op.directions_prior, op.psi_prior);

			if (baseMLO->do_realign_movies && baseMLO->movie_frame_running_avg_side > 0)
			{
				baseMLO->calculateRunningAveragesOfMovieFrames(my_ori_particle, op.Fimgs, op.power_imgs, op.highres_Xi2_imgs);
			}

			// To deal with skipped alignments/rotations
			if (baseMLO->do_skip_align)
			{
				sp.itrans_min = sp.itrans_max = sp.idir_min = sp.idir_max = sp.ipsi_min = sp.ipsi_max =
						my_ori_particle - baseMLO->exp_my_first_ori_particle;
			}
			else
			{
				sp.itrans_min = 0;
				sp.itrans_max = baseMLO->sampling.NrTranslationalSamplings() - 1;

				if (baseMLO->do_skip_rotate)
				{
					sp.idir_min = sp.idir_max = sp.ipsi_min = sp.ipsi_max =
							my_ori_particle - baseMLO->exp_my_first_ori_particle;
				}
				else
				{
					sp.idir_min = sp.ipsi_min = 0;
					sp.idir_max = baseMLO->sampling.NrDirections(0, &op.pointer_dir_nonzeroprior) - 1;
					sp.ipsi_max = baseMLO->sampling.NrPsiSamplings(0, &op.pointer_psi_nonzeroprior ) - 1;
				}
			}

			// Initialise significant weight to minus one, so that all coarse sampling points will be handled in the first pass
			op.significant_weight.resize(sp.nr_particles, -1.);

			// Only perform a second pass when using adaptive oversampling
			int nr_sampling_passes = (baseMLO->adaptive_oversampling > 0) ? 2 : 1;

			for (int ipass = 0; ipass < nr_sampling_passes; ipass++)
			{
				if (baseMLO->strict_highres_exp > 0.)
					// Use smaller images in both passes and keep a maximum on coarse_size, just like in FREALIGN
					sp.current_image_size = baseMLO->coarse_size;
				else if (baseMLO->adaptive_oversampling > 0)
					// Use smaller images in the first pass, larger ones in the second pass
					sp.current_image_size = (ipass == 0) ? baseMLO->coarse_size : baseMLO->mymodel.current_size;
				else
					sp.current_image_size = baseMLO->mymodel.current_size;

				// Use coarse sampling in the first pass, oversampled one the second pass
				sp.current_oversampling = (ipass == 0) ? 0 : baseMLO->adaptive_oversampling;

				sp.nr_dir = (baseMLO->do_skip_align || baseMLO->do_skip_rotate) ? 1 : baseMLO->sampling.NrDirections(0, &op.pointer_dir_nonzeroprior);
				sp.nr_psi = (baseMLO->do_skip_align || baseMLO->do_skip_rotate) ? 1 : baseMLO->sampling.NrPsiSamplings(0, &op.pointer_psi_nonzeroprior);
				sp.nr_trans = (baseMLO->do_skip_align) ? 1 : baseMLO->sampling.NrTranslationalSamplings();
				sp.nr_oversampled_rot = baseMLO->sampling.oversamplingFactorOrientations(sp.current_oversampling);
				sp.nr_oversampled_trans = baseMLO->sampling.oversamplingFactorTranslations(sp.current_oversampling);

				CUDA_CPU_TIC("getAllSquaredDifferences");
				getAllSquaredDifferences(ipass, op, sp);
				CUDA_CPU_TOC("getAllSquaredDifferences");
				CUDA_CPU_TIC("convertAllSquaredDifferencesToWeights");
				convertAllSquaredDifferencesToWeights(ipass, op, sp);
				CUDA_CPU_TOC("convertAllSquaredDifferencesToWeights");
			}

			// For the reconstruction step use mymodel.current_size!
			sp.current_image_size = baseMLO->mymodel.current_size;

			CUDA_CPU_TIC("storeWeightedSums");
			storeWeightedSums(op, sp);
			CUDA_CPU_TOC("storeWeightedSums");
		}
	}
}





void MlOptimiserCuda::getAllSquaredDifferences(unsigned exp_ipass, OptimisationParamters &op, SamplingParameters &sp)
{

	CUDA_CPU_TIC("diff_pre_gpu");

	//for scale_correction
	int group_id;

	//printf("sp.nr_oversampled_rot=%d\n", (unsigned)sp.nr_oversampled_rot);

	op.Mweight.resize(sp.nr_particles, baseMLO->mymodel.nr_classes * sp.nr_dir * sp.nr_psi * sp.nr_trans * sp.nr_oversampled_rot * sp.nr_oversampled_trans);
	op.Mweight.initConstant(-999.);
	if (exp_ipass==0)
	{
		op.Mcoarse_significant.clear();
	}

	op.min_diff2.clear();
	op.min_diff2.resize(sp.nr_particles, 99.e99);

	std::vector<MultidimArray<Complex > > dummy;
	baseMLO->precalculateShiftedImagesCtfsAndInvSigma2s(false, op.my_ori_particle, sp.current_image_size, sp.current_oversampling,
			sp.itrans_min, sp.itrans_max, op.Fimgs, dummy, op.Fctfs, op.local_Fimgs_shifted, dummy,
			op.local_Fctfs, op.local_sqrtXi2, op.local_Minvsigma2s);

	MultidimArray<Complex > Fref;
	Fref.resize(op.local_Minvsigma2s[0]);

	unsigned image_size = op.local_Minvsigma2s[0].nzyxdim;

	CUDA_CPU_TOC("diff_pre_gpu");

	// Loop only from sp.iclass_min to sp.iclass_max to deal with seed generation in first iteration
	for (int exp_iclass = sp.iclass_min; exp_iclass <= sp.iclass_max; exp_iclass++)
	{
		if (baseMLO->mymodel.pdf_class[exp_iclass] > 0.)
		{
			// Local variables
			std::vector< double > oversampled_rot, oversampled_tilt, oversampled_psi;
			std::vector< double > oversampled_translations_x, oversampled_translations_y, oversampled_translations_z;
			CudaGlobalPtr<FLOAT> gpuMinvsigma2(image_size);
			gpuMinvsigma2.device_alloc();

			// Mapping index look-up table
			std::vector< long unsigned > iorientclasses, iover_rots;
			std::vector< double > rots, tilts, psis;

			CUDA_CPU_TIC("projection_1");
			CUDA_CPU_TIC("generateProjectionSetup");
			long unsigned orientation_num = generateProjectionSetup(
					op,
					sp,
					baseMLO,
					exp_ipass == 0, //coarse
					exp_iclass,
					rots, tilts, psis,
					iorientclasses,
					iover_rots);

			CUDA_CPU_TOC("generateProjectionSetup");
			CUDA_CPU_TIC("generateEulerMatrices");
			CudaGlobalPtr<FLOAT> eulers(9 * orientation_num);

			generateEulerMatrices(
					baseMLO->mymodel.PPref[exp_iclass].padding_factor,
					rots,
					tilts,
					psis,
					eulers,
					!IS_NOT_INV);

		    eulers.device_alloc();
			eulers.cp_to_device();
			CUDA_CPU_TOC("generateEulerMatrices");
			CUDA_CPU_TIC("modelAssignment");

//			CudaGlobalPtr<FLOAT > model_real;
//			float* holder = new float[(baseMLO->mymodel.PPref[exp_iclass]).data.nzyxdim];
//			model_real.h_ptr = holder;
//			model_real.size=(baseMLO->mymodel.PPref[exp_iclass]).data.nzyxdim;
//			model_real.h_do_free = true;

			CudaGlobalPtr<FLOAT > model_real((baseMLO->mymodel.PPref[exp_iclass]).data.nzyxdim);
			CudaGlobalPtr<FLOAT > model_imag((baseMLO->mymodel.PPref[exp_iclass]).data.nzyxdim);

			for(unsigned i = 0; i < model_real.size; i++)
			{
				model_real[i] = (FLOAT) baseMLO->mymodel.PPref[exp_iclass].data.data[i].real;
				model_imag[i] = (FLOAT) baseMLO->mymodel.PPref[exp_iclass].data.data[i].imag;
			}

			CudaGlobalPtr<FLOAT> Frefs_real;
		    CudaGlobalPtr<FLOAT> Frefs_imag;

			CUDA_CPU_TOC("modelAssignment");
			bool do_combineProjAndDiff = true; //TODO add control flag
			if(!do_combineProjAndDiff)
			{
				CUDA_CPU_TIC("generateModelProjections_diff");
				generateModelProjections(
						model_real,
						model_imag,
						Frefs_real,
						Frefs_imag,
						eulers,
						orientation_num,
						image_size,
						XMIPP_MIN(baseMLO->mymodel.PPref[exp_iclass].r_max, op.local_Minvsigma2s[0].xdim - 1),
						op.local_Minvsigma2s[0].xdim,
						op.local_Minvsigma2s[0].ydim,
						baseMLO->mymodel.PPref[exp_iclass].data.xdim,
						baseMLO->mymodel.PPref[exp_iclass].data.ydim,
						baseMLO->mymodel.PPref[exp_iclass].data.zdim,
						baseMLO->mymodel.PPref[exp_iclass].data.yinit,
						baseMLO->mymodel.PPref[exp_iclass].data.zinit);
				model_real.free_device();
				model_imag.free_device();
				eulers.free_device();
				CUDA_CPU_TOC("generateModelProjections_diff");
			}
			CUDA_CPU_TOC("projection_1");

			/*=======================================================================================
			                                  	  Particle Iteration
			=========================================================================================*/

			for (long int ipart = 0; ipart < sp.nr_particles; ipart++)
			{
				/*====================================
				        Generate Translations
				======================================*/

				CUDA_CPU_TIC("translation_1");

				CudaGlobalPtr<FLOAT> Fimgs_real(image_size * sp.nr_trans * sp.nr_oversampled_trans);
				CudaGlobalPtr<FLOAT> Fimgs_imag(image_size * sp.nr_trans * sp.nr_oversampled_trans);

				long int part_id = baseMLO->mydata.ori_particles[op.my_ori_particle].particles_id[ipart];
				long unsigned translation_num(0), ihidden(0);
				std::vector< long unsigned > iover_transes, itranses, ihiddens;

				for (long int itrans = sp.itrans_min; itrans <= sp.itrans_max; itrans++, ihidden++)
				{
					baseMLO->sampling.getTranslations(itrans, sp.current_oversampling,
							oversampled_translations_x, oversampled_translations_y, oversampled_translations_z );

					for (long int iover_trans = 0; iover_trans < sp.nr_oversampled_trans; iover_trans++)
					{
						/// Now get the shifted image
						// Use a pointer to avoid copying the entire array again in this highly expensive loop
						Complex *myAB;
						if (sp.current_oversampling == 0)
						{
							myAB = (Fref.ydim == baseMLO->coarse_size) ? baseMLO->global_fftshifts_ab_coarse[itrans].data
									: baseMLO->global_fftshifts_ab_current[itrans].data;
						}
						else
						{
							int iitrans = itrans * sp.nr_oversampled_trans +  iover_trans;
							myAB = (baseMLO->strict_highres_exp > 0.) ? baseMLO->global_fftshifts_ab2_coarse[iitrans].data
									: baseMLO->global_fftshifts_ab2_current[iitrans].data;
						}


						FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(op.local_Fimgs_shifted[ipart])
						{
							FLOAT real = (*(myAB + n)).real * (DIRECT_MULTIDIM_ELEM(op.local_Fimgs_shifted[ipart], n)).real
									- (*(myAB + n)).imag *(DIRECT_MULTIDIM_ELEM(op.local_Fimgs_shifted[ipart], n)).imag;
							FLOAT imag = (*(myAB + n)).real * (DIRECT_MULTIDIM_ELEM(op.local_Fimgs_shifted[ipart], n)).imag
									+ (*(myAB + n)).imag *(DIRECT_MULTIDIM_ELEM(op.local_Fimgs_shifted[ipart], n)).real;

							//When on gpu, it makes more sense to ctf-correct translated images, rather than anti-ctf-correct ref-projections
							if (baseMLO->do_scale_correction)
							{
								//group_id = mydata.getGroupId(part_id);
								FLOAT myscale = baseMLO->mymodel.scale_correction[group_id];
								real /= myscale;
								imag /= myscale;
							}
							if (baseMLO->do_ctf_correction && baseMLO->refs_are_ctf_corrected)
							{
								real /= DIRECT_MULTIDIM_ELEM(op.local_Fctfs[ipart], n);
								imag /= DIRECT_MULTIDIM_ELEM(op.local_Fctfs[ipart], n);
							}
							Fimgs_real[translation_num * image_size + n] = real;
							Fimgs_imag[translation_num * image_size + n] = imag;
						}
						translation_num ++;

						ihiddens.push_back(ihidden);
						itranses.push_back(itrans);
						iover_transes.push_back(iover_trans);
					}
				}

				CUDA_CPU_TOC("translation_1");

				/*===========================================
				   Determine significant comparison indices
				=============================================*/
				//      This section is annoying to test because
				//		it can't complete on first pass, since
				//		the significance has never been set


				CUDA_CPU_TIC("pair_list_1");

				CudaGlobalPtr<long unsigned> transidx(orientation_num*translation_num), rotidx(orientation_num*translation_num);
				CudaGlobalPtr<long unsigned> ihidden_overs(orientation_num*translation_num);
				CudaGlobalPtr<long unsigned> trans_num(orientation_num*translation_num);
				long unsigned coarse_num = sp.nr_dir*sp.nr_psi*sp.nr_trans;
				long unsigned significant_num(0);
//				long int check_num=0;
				long unsigned k=0;
				if (exp_ipass == 0)
				{
					op.Mcoarse_significant.resize(coarse_num, 1);
					for (long unsigned i = 0; i < orientation_num; i++)
					{
						trans_num[k]=0;
						transidx[k]=translation_num+1;//set higher than max(j) so that XMIPP_MIN() sets
						int tk=0;
						for (long unsigned j = 0; j < translation_num; j++)
						{
							ihidden_overs[significant_num] = i * sp.nr_trans + j;
							if(tk>=PROJDIFF_CHUNK_SIZE)
							{
								tk=0;             // reset counter
//								check_num+=trans_num[k];
								k++;              // use new element
								trans_num[k]=0;   // prepare next element for ++ incrementing
								transidx[k]=translation_num+1; //set higher than max(j) so that XMIPP_MIN() sets
							}
							tk++;                 // increment limit
							trans_num[k]++;       // increment number of transes this ProjDiff-block
							rotidx[k] = i;
							transidx[k] = XMIPP_MIN(j,transidx[k]);
							significant_num++;
						}
//						check_num+=trans_num[k];
						k++;   // use new element
					}
					trans_num.size=k;
				}
				else
				{
					for (long unsigned i = 0; i < orientation_num; i++)
					{
						trans_num[k]=0;
						transidx[k]=translation_num+1;//set higher than max(j) so that XMIPP_MIN() sets
						int tk=0;

						long int iover_rot = iover_rots[i];
						long int coarse_rot = floor(i/sp.nr_oversampled_rot);
						for (long unsigned j = 0; j < translation_num; j++)
						{
							long int iover_trans = iover_transes[j];
							long int coarse_trans = floor(j/sp.nr_oversampled_trans);
							long int ihidden = iorientclasses[i] * sp.nr_trans + ihiddens[j];

							if(DIRECT_A2D_ELEM(op.Mcoarse_significant, ipart, ihidden)==1)
							{
								ihidden_overs[significant_num] = baseMLO->sampling.getPositionOversampledSamplingPoint(ihidden,
										                  sp.current_oversampling, iover_rot, iover_trans);
								if(tk>=PROJDIFF_CHUNK_SIZE)
								{
									tk=0;             // reset counter
//									check_num+=trans_num[k];
									k++;              // use new element
									trans_num[k]=0;   // prepare next element for ++ incrementing
									transidx[k]=translation_num+1; //set higher than max(j) so that XMIPP_MIN() sets
								}
								tk++;                 // increment limit
								trans_num[k]++;       // increment number of transes this ProjDiff-block
								rotidx[k] = i;
								transidx[k] = XMIPP_MIN(j,transidx[k]);
								significant_num++;
							}
							else if(tk!=0) // start a new one - we expect transes to be sequential.
							{
								tk=0;             // reset counter
//								check_num+=trans_num[k];
								k++;              // use new element
								trans_num[k]=0;   // prepare next element for ++ incrementing
								transidx[k]=translation_num+1; //set higher than max(j) so that XMIPP_MIN() sets
							}

						}
//						check_num+=trans_num[k];
						k++;   // use new element
					}
					trans_num.size=k;
				}
				//  check_num should equal significant_num here, and be less or equal to  PROJDIFF_CHUNK_SIZE*trans_num.size

				CUDA_CPU_TOC("pair_list_1");
//				std::cerr << "orientation_num "<< orientation_num << std::endl;
//				std::cerr << "translation_num "<< translation_num << std::endl;
//				std::cerr << "my_nr_significant_coarse_samples "<< DIRECT_A2D_ELEM(exp_metadata, metadata_offset + ipart, METADATA_NR_SIGN) << std::endl;
//				std::cerr << "significant_num "<< significant_num << std::endl;

				CudaGlobalPtr<FLOAT> diff2s(orientation_num*translation_num);
				diff2s.device_alloc();

#if !defined(CUDA_DOUBLE_PRECISION)
				if(do_combineProjAndDiff)
				{
					runProjAndDifferenceKernel(model_real,
											   model_imag,
											   gpuMinvsigma2,
										       Fimgs_real,
										       Fimgs_imag,
										       eulers,
										       rotidx,
										       transidx,
										       trans_num,
										       ihidden_overs,
										       diff2s,
										       op,
										       baseMLO,
										       orientation_num,
										       translation_num,
										       significant_num,
										       image_size,
											    XMIPP_MIN(baseMLO->mymodel.PPref[exp_iclass].r_max, op.local_Minvsigma2s[0].xdim - 1),
										       ipart,
										       group_id,
										       exp_iclass
											 );
					eulers.free_device();
				}
				else
#endif
				{
					runDifferenceKernel(gpuMinvsigma2,
										Fimgs_real,
										Fimgs_imag,
										Frefs_real,
										Frefs_imag,
										rotidx,
										transidx,
										ihidden_overs,
										op,
										baseMLO,
										translation_num,
										orientation_num,
										significant_num,
										image_size,
										ipart,
										group_id,
										diff2s
										);
				}
				/*====================================
				    	   Retrieve Results
				======================================*/

				diff2s.cp_to_host(); // FIXME may not be needed since we copy it back in ConvetToWeights()
//				for (long unsigned k = 0; k < 100; k++)
//				{
//					std::cerr << diff2s[k] << std::endl;
//				}
				if (exp_ipass == 0)
				{
					op.Mcoarse_significant.clear();
				}

				/*====================================
				    	Write To Destination
				======================================*/


				CUDA_CPU_TIC("collect_data_1");

//				freopen(text,"w",stdout);
				long unsigned m=0;
				for (long unsigned k = 0; k < trans_num.size; k++)
				{
					for (int itrans=0;  itrans < trans_num[k]; itrans++, m++)
					{
						long unsigned i = rotidx[k];
						long unsigned j = transidx[k]+itrans;
						double diff2 = diff2s[i * translation_num + j];
//						printf("%4.8f \n",DIRECT_A2D_ELEM(op.Mweight, ipart, n));
//						printf("%4.8f, %i, %i \n",diff2,i,j);// << std::endl;

						DIRECT_A2D_ELEM(op.Mweight, ipart, ihidden_overs[m]) = diff2; // TODO if we can write diff2 to the correct pos in the kernel we can just memcpy to a pointer and use thrust to find min
						// Keep track of minimum of all diff2, only for the last image in this series
						if (diff2 < op.min_diff2[ipart])
							op.min_diff2[ipart] = diff2;
					}
				}
//				fclose(stdout);

				CUDA_CPU_TOC("collect_data_1");
			} // end loop ipart
		} // end if class significant
	} // end loop iclass
}

void MlOptimiserCuda::convertAllSquaredDifferencesToWeights(unsigned exp_ipass, OptimisationParamters &op, SamplingParameters &sp)
{
	op.sum_weight.clear();
	op.sum_weight.resize(sp.nr_particles, 0.);

	// loop over all particles inside this ori_particle
	for (long int ipart = 0; ipart < sp.nr_particles; ipart++)
	{
		long int part_id = baseMLO->mydata.ori_particles[op.my_ori_particle].particles_id[ipart];
		double exp_thisparticle_sumweight = 0.;

		double old_offset_z;
		double old_offset_x = XX(op.old_offset[ipart]);
		double old_offset_y = YY(op.old_offset[ipart]);
		if (baseMLO->mymodel.data_dim == 3)
			old_offset_z = ZZ(op.old_offset[ipart]);

		if ((baseMLO->iter == 1 && baseMLO->do_firstiter_cc) || baseMLO->do_always_cc)
		{
			// Binarize the squared differences array to skip marginalisation
			double mymindiff2 = 99.e10;
			long int myminidx = -1;
			// Find the smallest element in this row of op.Mweight
			for (long int i = 0; i < XSIZE(op.Mweight); i++)
			{

				double cc = DIRECT_A2D_ELEM(op.Mweight, ipart, i);
				// ignore non-determined cc
				if (cc == -999.)
					continue;

				// just search for the maximum
				if (cc < mymindiff2)
				{
					mymindiff2 = cc;
					myminidx = i;
				}
			}
			// Set all except for the best hidden variable to zero and the smallest element to 1
			for (long int i = 0; i < XSIZE(op.Mweight); i++)
				DIRECT_A2D_ELEM(op.Mweight, ipart, i)= 0.;

			DIRECT_A2D_ELEM(op.Mweight, ipart, myminidx)= 1.;
			exp_thisparticle_sumweight += 1.;

		}
		else
		{
			// Loop from iclass_min to iclass_max to deal with seed generation in first iteration
			for (int exp_iclass = sp.iclass_min; exp_iclass <= sp.iclass_max; exp_iclass++)
			{

				// Make PdfOffset calculation much faster...
				double myprior_x, myprior_y, myprior_z;
				if (baseMLO->mymodel.ref_dim == 2)
				{
					myprior_x = XX(baseMLO->mymodel.prior_offset_class[exp_iclass]);
					myprior_y = YY(baseMLO->mymodel.prior_offset_class[exp_iclass]);
				}
				else
				{
					myprior_x = XX(op.prior[ipart]);
					myprior_y = YY(op.prior[ipart]);
					if (baseMLO->mymodel.data_dim == 3)
						myprior_z = ZZ(op.prior[ipart]);
				}

				/*=========================================
						Fetch+generate Orientation data
				===========================================*/
				CudaGlobalPtr<FLOAT >  pdf_orientation(sp.nr_dir * sp.nr_psi);
				pdf_orientation.size = sp.nr_dir * sp.nr_psi;
				for (long int idir = sp.idir_min, iorient = 0; idir <= sp.idir_max; idir++)
				{
					for (long int ipsi = sp.ipsi_min; ipsi <= sp.ipsi_max; ipsi++, iorient++)
					{
						//std::cerr << "orient "  << idir << "," << iorient <<  std::endl;
						// Get prior for this direction
						if (baseMLO->do_skip_align || baseMLO->do_skip_rotate)
						{
							pdf_orientation[iorient] = baseMLO->mymodel.pdf_class[exp_iclass];
						}
						else if (baseMLO->mymodel.orientational_prior_mode == NOPRIOR)
						{
							pdf_orientation[iorient] = DIRECT_MULTIDIM_ELEM(baseMLO->mymodel.pdf_direction[exp_iclass], idir);
						}
						else
						{
							// P(orientation) = P(idir|dir_prior) * P(ipsi|psi_prior)
							// This is the probability of the orientation, given the gathered
							// statistics of all assigned orientations of the dataset, since we
							// are assigning a gaussian prior to all parameters.
							pdf_orientation[iorient] = op.directions_prior[idir] * op.psi_prior[ipsi];
						}
					}
				}
//				long int ihidden = iorientclass * sp.nr_trans;

				/*=========================================
						Fetch+generate Translation data
				===========================================*/
				CudaGlobalPtr<FLOAT >  pdf_offset(sp.nr_trans);

				int jtrans=0;
				for (long int itrans = sp.itrans_min; itrans <= sp.itrans_max; itrans++,jtrans++)
				{
					//std::cerr << "trans " << itrans << "," << jtrans <<  std::endl;
			        // To speed things up, only calculate pdf_offset at the coarse sampling.
					// That should not matter much, and that way one does not need to calculate all the OversampledTranslations
					double offset_x = old_offset_x + baseMLO->sampling.translations_x[itrans];
					double offset_y = old_offset_y + baseMLO->sampling.translations_y[itrans];
					double tdiff2 = (offset_x - myprior_x) * (offset_x - myprior_x) + (offset_y - myprior_y) * (offset_y - myprior_y);
					if (baseMLO->mymodel.data_dim == 3)
					{
						double offset_z = old_offset_z + baseMLO->sampling.translations_z[itrans];
						tdiff2 += (offset_z - myprior_z) * (offset_z - myprior_z);
					}
					// P(offset|sigma2_offset)
					// This is the probability of the offset, given the model offset and variance.
					if (baseMLO->mymodel.sigma2_offset < 0.0001)
						pdf_offset[jtrans] = ( tdiff2 > 0.) ? 0. : 1.;
					else
						pdf_offset[jtrans] = exp ( tdiff2 / (-2. * baseMLO->mymodel.sigma2_offset) ) / ( 2. * PI * baseMLO->mymodel.sigma2_offset );
				}

// TODO : Put back when  convertAllSquaredDifferencesToWeights is GPU-parallel.
//							// TMP DEBUGGING
//							if (baseMLO->mymodel.orientational_prior_mode != NOPRIOR && (pdf_offset==0. || pdf_orientation==0.))
//							{
//								pthread_mutex_lock(&global_mutex);
//								std::cerr << " pdf_offset= " << pdf_offset << " pdf_orientation= " << pdf_orientation << std::endl;
//								std::cerr << " ipart= " << ipart << " part_id= " << part_id << std::endl;
//								std::cerr << " iorient= " << iorient << " idir= " << idir << " ipsi= " << ipsi << std::endl;
//								//std::cerr << " sp.nr_psi= " << sp.nr_psi << " exp_nr_dir= " << exp_nr_dir << " sp.nr_trans= " << sp.nr_trans << std::endl;
//								for (long int i = 0; i < op.directions_prior.size(); i++)
//									std::cerr << " op.directions_prior["<<i<<"]= " << op.directions_prior[i] << std::endl;
//								for (long int i = 0; i < op.psi_prior.size(); i++)
//									std::cerr << " op.psi_prior["<<i<<"]= " << op.psi_prior[i] << std::endl;
//								REPORT_ERROR("ERROR! pdf_offset==0.|| pdf_orientation==0.");
//								//pthread_mutex_unlock(&global_mutex);
//							}
//							if (sp.nr_oversampled_rot == 0)
//								REPORT_ERROR("sp.nr_oversampled_rot == 0");
//							if (sp.nr_oversampled_trans == 0)
//								REPORT_ERROR("sp.nr_oversampled_trans == 0");

				// Now first loop over iover_rot, because that is the order in op.Mweight as well
//				long int ihidden_over = ihidden * sp.nr_oversampled_rot * sp.nr_oversampled_trans;

				/*=========================================
					  Kernel call over all combinations
				===========================================*/

				// One block will be started for each (coarse) orientation, and will process all (coarse) transes,
				// and since oversmapling is by factors of 2 on 5 dofs, we get 2^5=32 fine comparisons per coarse.
				// In case of higher oversampling this is simply factors of 32, making it warp-perfect. Having 21
				// coarse transes allows a block to finish in 21, 100% utilized, warp passes.

				int oversamples = sp.nr_oversampled_trans * sp.nr_oversampled_rot;

				bool do_gpu_sumweight = true;  //TODO add control flag
				if(oversamples>=SUM_BLOCK_SIZE && do_gpu_sumweight) // Send task to GPU where warps can access automatically coalesced oversamples
				{
					//std::cerr << "summing weights on GPU... baseMLO->mymodel.pdf_class[exp_iclass] = " << baseMLO->mymodel.pdf_class[sp.iclass_min] <<  std::endl;
					pdf_orientation.device_alloc();
					pdf_orientation.cp_to_device();
					pdf_offset.device_alloc();
					pdf_offset.cp_to_device();

					CudaGlobalPtr<FLOAT >  thisparticle_sumweight(sp.nr_dir * sp.nr_psi);  // This will be reduced in a second step.
					thisparticle_sumweight.device_alloc();

					CudaGlobalPtr<FLOAT >  Mweight( &(op.Mweight.data[(ipart)*(op.Mweight).xdim]),
													sp.nr_dir * sp.nr_psi * sp.nr_trans * oversamples);
					Mweight.device_alloc();
					Mweight.cp_to_device();

					dim3 block_dim(sp.nr_dir,sp.nr_psi);
					//std::cerr << "using block dimensions " << sp.nr_dir << "," << sp.nr_psi <<  std::endl;
					cuda_kernel_sumweight_oversampling<<<block_dim,SUM_BLOCK_SIZE>>>(	~pdf_orientation,
																						~pdf_offset,
																						~Mweight,
																						~thisparticle_sumweight,
																						op.min_diff2[ipart],
																						sp.nr_trans,
																						oversamples
																					 );

					Mweight.cp_to_host(); //FIXME make wider in scope; pass to storeWsums() to be used in collect-step. Needs som coordination with else() below.
					Mweight.free_device();  //FIXME see line above
					thisparticle_sumweight.cp_to_host();
					thisparticle_sumweight.free_device();

					// The reduced entity *MUST* be double to avoid loss of information// TODO better reduction
					for (long int n = 0; n < sp.nr_dir * sp.nr_psi; n++)
					{
						exp_thisparticle_sumweight += (double)thisparticle_sumweight[n];
					}
				}
				else // Not enough oversamples to utilize GPU resources effciently with current CUDA-kernel.
				{
					//std::cerr << "summing weights on CPU... " <<  std::endl;
					for (long int idir = sp.idir_min, iorient = 0; idir <= sp.idir_max; idir++)
					{
						for (long int ipsi = sp.ipsi_min; ipsi <= sp.ipsi_max; ipsi++, iorient++)
						{
							long int iorientclass = exp_iclass * sp.nr_dir * sp.nr_psi + iorient;
							long int ihidden = iorientclass * sp.nr_trans;
							for (long int itrans = sp.itrans_min; itrans <= sp.itrans_max; itrans++, ihidden++)
							{
								long int ihidden_over = ihidden * sp.nr_oversampled_rot * sp.nr_oversampled_trans;
								for (long int iover_rot = 0; iover_rot < sp.nr_oversampled_rot; iover_rot++)
								{
									// Then loop over iover_trans
									for (long int iover_trans = 0; iover_trans < sp.nr_oversampled_trans; iover_trans++, ihidden_over++)
									{
										// Only exponentiate for determined values of op.Mweight
										// (this is always true in the first pass, but not so in the second pass)
										// Only deal with this sampling point if its weight was significant
										if (DIRECT_A2D_ELEM(op.Mweight, ipart, ihidden_over) < 0.)
										{
											DIRECT_A2D_ELEM(op.Mweight, ipart, ihidden_over) = 0.;
										}
										else
										{
											// Set the weight base to the probability of the parameters given the prior
											double weight = pdf_orientation[iorient] * pdf_offset[itrans];
											double diff2 = DIRECT_A2D_ELEM(op.Mweight, ipart, ihidden_over) - op.min_diff2[ipart];
											// next line because of numerical precision of exp-function
											if (diff2 > 700.) weight = 0.;
											// TODO: use tabulated exp function?
											else weight *= exp(-diff2);
											// Store the weight
											DIRECT_A2D_ELEM(op.Mweight, ipart, ihidden_over) = weight;

											// Keep track of sum and maximum of all weights for this particle
											// Later add all to exp_thisparticle_sumweight, but inside this loop sum to local thisthread_sumweight first
											exp_thisparticle_sumweight += weight;
										} // end if/else op.Mweight < 0.
									} // end loop iover_trans
								}// end loop iover_rot
							} // end loop itrans
						} // end loop ipsi
					} // end loop idir
				}                            //endif do_gpu_sumweight
			} // end loop exp_iclass
		} // end if iter==1

		//Store parameters for this particle
		op.sum_weight[ipart] = exp_thisparticle_sumweight;

#if defined(DEBUG_CUDA) && defined(__linux__)
		if (exp_thisparticle_sumweight == 0. || std::isnan(exp_thisparticle_sumweight))
		{
			printf("DEBUG_ERROR: zero sum of weights.\n");
			exit( EXIT_FAILURE );
		}
#endif

	} // end loop ipart

	if (exp_ipass==0)
	{
		op.Mcoarse_significant.resize(sp.nr_particles, XSIZE(op.Mweight));
	}

	CUDA_CPU_TIC("convert_post_kernel");
	// Now, for each particle,  find the exp_significant_weight that encompasses adaptive_fraction of op.sum_weight
	op.significant_weight.clear();
	op.significant_weight.resize(sp.nr_particles, 0.);
	for (long int ipart = 0; ipart < sp.nr_particles; ipart++)
	{
		long int part_id = baseMLO->mydata.ori_particles[op.my_ori_particle].particles_id[ipart];
		MultidimArray<FLOAT> sorted_weight;
		// Get the relevant row for this particle
		op.Mweight.getRow(ipart, sorted_weight);

		// Only select non-zero probabilities to speed up sorting
		long int np = 0;
		FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(sorted_weight)
		{
			if (DIRECT_MULTIDIM_ELEM(sorted_weight, n) > 0.)
			{
				DIRECT_MULTIDIM_ELEM(sorted_weight, np) = DIRECT_MULTIDIM_ELEM(sorted_weight, n);
				np++;
			}
		}
		sorted_weight.resize(np);

		// Sort from low to high values
		CUDA_CPU_TIC("sort");
#if defined(USE_THRUST) // Thrust seems incredibly slow in debug build this is clearly a FIXME
		thrust::sort(sorted_weight.data, sorted_weight.data + np);
#else
		sorted_weight.sort();
#endif
		CUDA_CPU_TOC("sort");

		double frac_weight = 0.;
		double my_significant_weight;
		long int my_nr_significant_coarse_samples = 0;
		for (long int i = XSIZE(sorted_weight) - 1; i >= 0; i--)
		{
			if (exp_ipass==0) my_nr_significant_coarse_samples++;
			my_significant_weight = DIRECT_A1D_ELEM(sorted_weight, i);
			frac_weight += my_significant_weight;
			if (frac_weight > baseMLO->adaptive_fraction * op.sum_weight[ipart])
				break;
		}

		if (exp_ipass==0 && my_nr_significant_coarse_samples == 0)
		{
			std::cerr << " ipart= " << ipart << " adaptive_fraction= " << baseMLO->adaptive_fraction << std::endl;
			std::cerr << " frac-weight= " << frac_weight << std::endl;
			std::cerr << " op.sum_weight[ipart]= " << op.sum_weight[ipart] << std::endl;
			Image<FLOAT> It;
			std::cerr << " XSIZE(op.Mweight)= " << XSIZE(op.Mweight) << std::endl;
			It()=op.Mweight;
			It() *= 10000;
			It.write("Mweight2.spi");
			std::cerr << "written Mweight2.spi" << std::endl;
			std::cerr << " np= " << np << std::endl;
			It()=sorted_weight;
			It() *= 10000;
			std::cerr << " XSIZE(sorted_weight)= " << XSIZE(sorted_weight) << std::endl;
			if (XSIZE(sorted_weight) > 0)
			{
				It.write("sorted_weight.spi");
				std::cerr << "written sorted_weight.spi" << std::endl;
			}
			REPORT_ERROR("my_nr_significant_coarse_samples == 0");
		}

		if (exp_ipass==0)
		{
			// Store nr_significant_coarse_samples for this particle
			DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_NR_SIGN) = (double)my_nr_significant_coarse_samples;

			// Keep track of which coarse samplings were significant were significant for this particle
			for (int ihidden = 0; ihidden < XSIZE(op.Mcoarse_significant); ihidden++)
			{
				if (DIRECT_A2D_ELEM(op.Mweight, ipart, ihidden) >= my_significant_weight)
					DIRECT_A2D_ELEM(op.Mcoarse_significant, ipart, ihidden) = true;
				else
					DIRECT_A2D_ELEM(op.Mcoarse_significant, ipart, ihidden) = false;
			}

		}
		op.significant_weight[ipart] = my_significant_weight;
	} // end loop ipart
	CUDA_CPU_TOC("convert_post_kernel");

}

// __global__ void cuda_kernel_wavg_fast   // REMOVED in commit
#if !defined(CUDA_DOUBLE_PRECISION)
__global__ void cuda_kernel_ProjAndWavg(
		FLOAT *g_eulers,
		hipTextureObject_t texModel_real,
		hipTextureObject_t texModel_imag,
		unsigned my_r_max,
		int max_r2,
		int min_r2_nn,
		unsigned image_size,
		unsigned long orientation_num,
	 	long int XSIZE_img,
	 	long int YSIZE_img,
	 	long int STARTINGY_mdl,
	 	long int STARTINGZ_mdl,
		FLOAT *g_imgs_real,
		FLOAT *g_imgs_imag,
		FLOAT *g_imgs_nomask_real,
		FLOAT *g_imgs_nomask_imag,
		FLOAT* g_weights,
		FLOAT* g_ctfs,
		FLOAT* g_Minvsigma2s,
		FLOAT *g_wdiff2s_parts,
		FLOAT *g_wavgs_real,
		FLOAT *g_wavgs_imag,
		FLOAT* g_Fweights,
		unsigned long translation_num,
		FLOAT weight_norm,
		FLOAT significant_weight,
		bool refs_are_ctf_corrected)
{
	FLOAT xp, yp, zp;
	long int r2;
	bool is_neg_x;
	FLOAT ref_real, ref_imag;
	int bid = blockIdx.y * gridDim.x + blockIdx.x;
	int tid = threadIdx.x;
	// inside the padded 2D orientation grid
//	if( bid < orientation_num )
//	{
		unsigned pass_num(ceilf(   ((float)image_size) / (float)BLOCK_SIZE  )),pixel;
		FLOAT Fweight;
		__shared__ FLOAT s_wavgs_real[BLOCK_SIZE];
		__shared__ FLOAT s_wavgs_imag[BLOCK_SIZE];
		__shared__ FLOAT s_wdiff2s_parts[BLOCK_SIZE];
		__shared__ FLOAT s_Minvsigma2s[BLOCK_SIZE];
		for (unsigned pass = 0; pass < pass_num; pass++) // finish a reference proj in each block
		{
			s_wavgs_real[tid]  = 0.0f;
			s_wavgs_imag[tid]  = 0.0f;
			s_wdiff2s_parts[tid] = 0.0f;
			Fweight = 0;

			pixel = pass * BLOCK_SIZE + tid;
			s_Minvsigma2s[tid]=g_Minvsigma2s[pixel];

			if(pixel<image_size)
			{
				unsigned long ref_pixel = bid * image_size + pixel;
				// Now istead of loading pre-calculated ref, we project it out from the texture-model
				//----------------------------------------------------------------------------------- =>
				int x = pixel % XSIZE_img;
				int y = (int)floorf( (float)pixel / (float)XSIZE_img);

				// Dont search beyond square with side max_r
				if (y > my_r_max)
				{
					if (y >= YSIZE_img - my_r_max)
						y = y - YSIZE_img ;
					else
						x=r2;
				}

				r2 = x*x + y*y;
				if (r2 <= max_r2)
				{
					xp = __ldg(&g_eulers[bid*9])   * x + __ldg(&g_eulers[bid*9+1]) * y;  // FIXME: xp,yp,zp has has accuracy loss
					yp = __ldg(&g_eulers[bid*9+3]) * x + __ldg(&g_eulers[bid*9+4]) * y;  // compared to CPU-based projection. This
					zp = __ldg(&g_eulers[bid*9+6]) * x + __ldg(&g_eulers[bid*9+7]) * y;  // propagates to dx00, dx10, and so on.
					// Only asymmetric half is stored
					if (xp < 0)
					{
						// Get complex conjugated hermitian symmetry pair
						xp = -xp;
						yp = -yp;
						zp = -zp;
						is_neg_x = true;
					}
					else
					{
						is_neg_x = false;
					}
					yp -= STARTINGY_mdl;
					zp -= STARTINGZ_mdl;

					ref_real=tex3D<FLOAT>(texModel_real,xp+0.5f,yp+0.5f,zp+0.5f);
					ref_imag=tex3D<FLOAT>(texModel_imag,xp+0.5f,yp+0.5f,zp+0.5f);


					if (is_neg_x)
					{
						ref_imag = -ref_imag;
					}

				}
				else
				{
					ref_real=0.0f;
					ref_imag=0.0f;
				}
				//-----------------------------------------------------------------------------------  <=
				if (refs_are_ctf_corrected) //FIXME Create two kernels for the different cases
				{
					ref_real *= __ldg(&g_ctfs[pixel]);
					ref_imag *= __ldg(&g_ctfs[pixel]);
				}

				for (unsigned long itrans = 0; itrans < translation_num; itrans++)
				{
					FLOAT weight = __ldg(&g_weights[bid * translation_num + itrans]);

					if (weight >= significant_weight)
					{
						weight /= weight_norm;

						unsigned long img_pixel_idx = itrans * image_size + pixel;

						FLOAT diff_real = ref_real - g_imgs_real[img_pixel_idx];    // TODO  Put in texture (in such a way that fetching of next image might hit in cache)
						FLOAT diff_imag = ref_imag - g_imgs_imag[img_pixel_idx];

						s_wdiff2s_parts[tid] += weight * (diff_real*diff_real + diff_imag*diff_imag);

						FLOAT weightxinvsigma2 = weight * __ldg(&g_ctfs[pixel]) * s_Minvsigma2s[tid];

						s_wavgs_real[tid] += g_imgs_nomask_real[img_pixel_idx] * weightxinvsigma2;    // TODO  Put in texture (in such a way that fetching of next image might hit in cache)
						s_wavgs_imag[tid] += g_imgs_nomask_imag[img_pixel_idx] * weightxinvsigma2;

						Fweight += weightxinvsigma2 * __ldg(&g_ctfs[pixel]);
					}
				}
				g_wavgs_real[ref_pixel] += s_wavgs_real[tid];
				g_wavgs_imag[ref_pixel] += s_wavgs_imag[tid];
				g_wdiff2s_parts[ref_pixel] = s_wdiff2s_parts[tid]; //TODO this could be further reduced in here
				g_Fweights[ref_pixel] += Fweight; //TODO should be buffered into shared
			}
		}
//	}
}
#endif

void runWavgKernel(CudaGlobalPtr<FLOAT> &Frefs_real,
				   CudaGlobalPtr<FLOAT> &Frefs_imag,
				   CudaGlobalPtr<FLOAT> &Fimgs_real,
				   CudaGlobalPtr<FLOAT> &Fimgs_imag,
				   CudaGlobalPtr<FLOAT> &Fimgs_nomask_real,
				   CudaGlobalPtr<FLOAT> &Fimgs_nomask_imag,
				   CudaGlobalPtr<FLOAT> &sorted_weights,
				   CudaGlobalPtr<FLOAT> &ctfs,
				   CudaGlobalPtr<FLOAT> &Minvsigma2s,
				   CudaGlobalPtr<FLOAT> &wdiff2s_parts,
				   CudaGlobalPtr<FLOAT> &wavgs_real,
				   CudaGlobalPtr<FLOAT> &wavgs_imag,
				   CudaGlobalPtr<FLOAT> &Fweights,
				   OptimisationParamters op,
				   MlOptimiser *baseMLO,
				   long unsigned orientation_num,
				   long unsigned translation_num,
				   unsigned image_size,
				   long int ipart,
				   int group_id,
				   int exp_iclass)
{

	unsigned orient1, orient2;
	//We only want as many blocks as there are chunks of orientations to be treated
	//within the same block (this is done to reduce memory loads in the kernel).
	unsigned orientation_chunks = orientation_num;//ceil((float)orientation_num/(float)REF_GROUP_SIZE);
	if(orientation_chunks>65535)
	{
		orient1 = ceil(sqrt(orientation_chunks));
		orient2 = orient1;
	}
	else
	{
		orient1 = orientation_chunks;
		orient2 = 1;
	}
	dim3 block_dim(orient1,orient2);

	CUDA_GPU_TIC("cuda_kernel_wavg");

	//hipFuncSetCacheConfig(reinterpret_cast<const void*>(cuda_kernel_wavg_fast), hipFuncCachePreferShared);
	cuda_kernel_wavg<<<block_dim,BLOCK_SIZE>>>(
										~Frefs_real, ~Frefs_imag, ~Fimgs_real, ~Fimgs_imag,
										~Fimgs_nomask_real, ~Fimgs_nomask_imag,
										~sorted_weights, ~ctfs, ~Minvsigma2s,
										~wdiff2s_parts,
										~wavgs_real,
										~wavgs_imag,
										~Fweights,
										orientation_num,
										translation_num,
										(FLOAT) op.sum_weight[ipart],
										(FLOAT) op.significant_weight[ipart],
										image_size,
										baseMLO->refs_are_ctf_corrected
										);
	size_t avail;
	size_t total;
	hipMemGetInfo( &avail, &total );
	float used = 100*((float)(total - avail)/(float)total);
	std::cerr << "Device memory used @ wavg: " << used << "%" << std::endl;
	CUDA_GPU_TAC("cuda_kernel_wavg");

	HANDLE_ERROR(hipDeviceSynchronize()); //TODO Apparently this is not required here

	CUDA_GPU_TOC("cuda_kernel_wavg");

//	Fimgs_real.free_device();
//	Fimgs_imag.free_device();
	Fimgs_nomask_real.free_device();
	Fimgs_nomask_imag.free_device();

	sorted_weights.free_device();
	ctfs.free_device();
	Minvsigma2s.free_device();
}


#if !defined(CUDA_DOUBLE_PRECISION)
void runProjAndWavgKernel(
		CudaGlobalPtr<FLOAT> &model_real,
		CudaGlobalPtr<FLOAT> &model_imag,
		CudaGlobalPtr<FLOAT> &eulers,
		CudaGlobalPtr<FLOAT> &Fimgs_real,
	    CudaGlobalPtr<FLOAT> &Fimgs_imag,
	    CudaGlobalPtr<FLOAT> &Fimgs_nomask_real,
 	    CudaGlobalPtr<FLOAT> &Fimgs_nomask_imag,
 	    CudaGlobalPtr<FLOAT> &sorted_weights,
 	    CudaGlobalPtr<FLOAT> &ctfs,
 	    CudaGlobalPtr<FLOAT> &Minvsigma2s,
 	    CudaGlobalPtr<FLOAT> &wdiff2s_parts,
 	    CudaGlobalPtr<FLOAT> &wavgs_real,
	    CudaGlobalPtr<FLOAT> &wavgs_imag,
	    CudaGlobalPtr<FLOAT> &Fweights,
	    OptimisationParamters op,
	    MlOptimiser *baseMLO,
	    long unsigned orientation_num,
	    long unsigned translation_num,
	    unsigned image_size,
	    unsigned max_r,
	    long int ipart,
	    int group_id,
	    int exp_iclass
	   )
{
	int max_r2 = max_r * max_r;
	int min_r2_nn = 0; // r_min_nn * r_min_nn;  //FIXME add nn-algorithm

	/*===========================
	 *      TEXTURE STUFF
	 * ==========================*/

	// create channel to describe data type (bits,bits,bits,bits,type)
	// TODO model should carry real & imag in separate channels of the same texture
	hipChannelFormatDesc channel = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipArray*        modelArray_real;
	hipArray* 		  modelArray_imag;
	hipExtent        volumeSize = make_hipExtent(baseMLO->mymodel.PPref[exp_iclass].data.xdim,
			   	   	   	   	   	   	   	   	       baseMLO->mymodel.PPref[exp_iclass].data.ydim,
			   	   	   	   	   	   	   	   	       baseMLO->mymodel.PPref[exp_iclass].data.zdim);

	//allocate device memory for cuda 3D array
	hipMalloc3DArray(&modelArray_real, &channel, volumeSize);
	hipMalloc3DArray(&modelArray_imag, &channel, volumeSize);

	//set cuda array copy parameters to be supplied to copy-command
	hipMemcpy3DParms copyParams = {0};
	copyParams.extent   = volumeSize;

	copyParams.kind     = hipMemcpyHostToDevice;
	copyParams.dstArray = modelArray_real;
	copyParams.srcPtr   = make_hipPitchedPtr(model_real.h_ptr,volumeSize.width*sizeof(FLOAT), volumeSize.height, volumeSize.depth);
	hipMemcpy3D(&copyParams);
	copyParams.kind     = hipMemcpyHostToDevice;
	copyParams.dstArray = modelArray_imag;
	copyParams.srcPtr   = make_hipPitchedPtr(model_imag.h_ptr,volumeSize.width*sizeof(FLOAT), volumeSize.height, volumeSize.depth);
	hipMemcpy3D(&copyParams);

	// Create texture object// Specify texture
    struct hipResourceDesc resDesc_real,resDesc_imag;
    memset(&resDesc_real, 0, sizeof(resDesc_real));
    memset(&resDesc_imag, 0, sizeof(resDesc_imag));
    resDesc_real.resType = hipResourceTypeArray;
    resDesc_imag.resType = hipResourceTypeArray;
    resDesc_real.res.array.array = modelArray_real;
    resDesc_imag.res.array.array = modelArray_imag;

    struct hipTextureDesc texDesc_real, texDesc_imag;
    memset(&texDesc_real, 0, sizeof(texDesc_real));
    memset(&texDesc_imag, 0, sizeof(texDesc_imag));
    for(int n=0; n<3; n++)
	{
    	texDesc_real.addressMode[n]=hipAddressModeClamp;
    	texDesc_imag.addressMode[n]=hipAddressModeClamp;
	}
    texDesc_real.filterMode       = hipFilterModeLinear;
    texDesc_real.readMode         = hipReadModeElementType;
    texDesc_real.normalizedCoords = false;
    texDesc_imag.filterMode       = hipFilterModeLinear;
    texDesc_imag.readMode         = hipReadModeElementType;
    texDesc_real.normalizedCoords = false;

	hipTextureObject_t texModel_real = 0;
	hipCreateTextureObject(&texModel_real, &resDesc_real, &texDesc_real, NULL);
	hipTextureObject_t texModel_imag = 0;
	hipCreateTextureObject(&texModel_imag, &resDesc_imag, &texDesc_imag, NULL);
	unsigned orient1, orient2;
	//We only want as many blocks as there are chunks of orientations to be treated
	//within the same block (this is done to reduce memory loads in the kernel).
	unsigned orientation_chunks = orientation_num;//ceil((float)orientation_num/(float)REF_GROUP_SIZE);
	if(orientation_chunks>65535)
	{
		orient1 = ceil(sqrt(orientation_chunks));
		orient2 = orient1;
	}
	else
	{
		orient1 = orientation_chunks;
		orient2 = 1;
	}
	dim3 block_dim(orient1,orient2);

	CUDA_GPU_TIC("cuda_kernel_wavg");

	//hipFuncSetCacheConfig(reinterpret_cast<const void*>(cuda_kernel_wavg_fast), hipFuncCachePreferShared);
	cuda_kernel_ProjAndWavg<<<block_dim,BLOCK_SIZE>>>(~eulers,
													  texModel_real,
													  texModel_imag,
													  max_r,
													  max_r2,
													  min_r2_nn,
													  image_size,
													  orientation_num,
													  op.local_Minvsigma2s[0].xdim,
													  op.local_Minvsigma2s[0].ydim,
													  baseMLO->mymodel.PPref[exp_iclass].data.yinit,
													  baseMLO->mymodel.PPref[exp_iclass].data.zinit,
													  ~Fimgs_real, ~Fimgs_imag,
													  ~Fimgs_nomask_real, ~Fimgs_nomask_imag,
													  ~sorted_weights, ~ctfs, ~Minvsigma2s,
													  ~wdiff2s_parts,
													  ~wavgs_real,
													  ~wavgs_imag,
													  ~Fweights,
													  translation_num,
													  (FLOAT) op.sum_weight[ipart],
													  (FLOAT) op.significant_weight[ipart],
													  baseMLO->refs_are_ctf_corrected
													);
	hipDestroyTextureObject(texModel_real);
	hipDestroyTextureObject(texModel_imag);
	hipFreeArray(modelArray_real);
	hipFreeArray(modelArray_imag);

	size_t avail;
	size_t total;
	hipMemGetInfo( &avail, &total );
	float used = 100*((float)(total - avail)/(float)total);
	std::cerr << "Device memory used @ wavg: " << used << "%" << std::endl;
	CUDA_GPU_TAC("cuda_kernel_wavg");

	HANDLE_ERROR(hipDeviceSynchronize()); //TODO Apparently this is not required here

	CUDA_GPU_TOC("cuda_kernel_wavg");

//	Fimgs_real.free_device();
//	Fimgs_imag.free_device();
	Fimgs_nomask_real.free_device();
	Fimgs_nomask_imag.free_device();

	sorted_weights.free_device();
	ctfs.free_device();
	Minvsigma2s.free_device();
}
#endif

void MlOptimiserCuda::storeWeightedSums(OptimisationParamters &op, SamplingParameters &sp)
{
	CUDA_CPU_TIC("store_pre_gpu");

	// Re-do below because now also want unmasked images AND if (stricht_highres_exp >0.) then may need to resize
	baseMLO->precalculateShiftedImagesCtfsAndInvSigma2s(true, op.my_ori_particle, sp.current_image_size, sp.current_oversampling,
			sp.itrans_min, sp.itrans_max, op.Fimgs, op.Fimgs_nomask, op.Fctfs, op.local_Fimgs_shifted, op.local_Fimgs_shifted_nomask,
			op.local_Fctfs, op.local_sqrtXi2, op.local_Minvsigma2s);

	// In doThreadPrecalculateShiftedImagesCtfsAndInvSigma2s() the origin of the op.local_Minvsigma2s was omitted.
	// Set those back here
	for (long int ipart = 0; ipart < sp.nr_particles; ipart++)
	{
		long int part_id = baseMLO->mydata.ori_particles[op.my_ori_particle].particles_id[ipart];
		int group_id = baseMLO->mydata.getGroupId(part_id);
		DIRECT_MULTIDIM_ELEM(op.local_Minvsigma2s[ipart], 0) = 1. / (baseMLO->sigma2_fudge * DIRECT_A1D_ELEM(baseMLO->mymodel.sigma2_noise[group_id], 0));
	}

	// Initialise the maximum of all weights to a negative value
	op.max_weight.clear();
	op.max_weight.resize(sp.nr_particles, -1.);

	// For norm_correction and scale_correction of all particles of this ori_particle
	std::vector<double> exp_wsum_norm_correction;
	std::vector<MultidimArray<double> > exp_wsum_scale_correction_XA, exp_wsum_scale_correction_AA;
	std::vector<MultidimArray<double> > thr_wsum_signal_product_spectra, thr_wsum_reference_power_spectra;
	exp_wsum_norm_correction.resize(sp.nr_particles, 0.);

	// For scale_correction
	if (baseMLO->do_scale_correction)
	{
		MultidimArray<double> aux;
		aux.initZeros(baseMLO->mymodel.ori_size/2 + 1);
		exp_wsum_scale_correction_XA.resize(sp.nr_particles, aux);
		exp_wsum_scale_correction_AA.resize(sp.nr_particles, aux);
		thr_wsum_signal_product_spectra.resize(baseMLO->mymodel.nr_groups, aux);
		thr_wsum_reference_power_spectra.resize(baseMLO->mymodel.nr_groups, aux);
	}


	std::vector<double> oversampled_translations_x, oversampled_translations_y, oversampled_translations_z;
	Matrix2D<double> A;
	MultidimArray<FLOAT > Fimg_real, Fimg_imag;
	MultidimArray<Complex > Fimg, Fimg_otfshift_nomask;  //TODO remove, currently needed for Fourier stuff, which is based on the complex class
	MultidimArray<double> Fweight, Minvsigma2, Mctf;
	bool have_warned_small_scale = false;

	Fimg_real.resize(op.Fimgs[0]);
	Fimg_imag.resize(op.Fimgs[0]);
	Fimg.resize(op.Fimgs[0]);
	Fweight.resize(op.Fimgs[0]);

	// Initialise Mctf to all-1 for if !do_ctf_corection
	Mctf.resize(op.Fimgs[0]);
	Mctf.initConstant(1.);
	// Initialise Minvsigma2 to all-1 for if !do_map
	Minvsigma2.resize(op.Fimgs[0]);
	Minvsigma2.initConstant(1.);

	// Make local copies of weighted sums (except BPrefs, which are too big)
	// so that there are not too many mutex locks below
	std::vector<MultidimArray<double> > thr_wsum_sigma2_noise, thr_wsum_pdf_direction;
	std::vector<double> thr_wsum_norm_correction, thr_sumw_group, thr_wsum_pdf_class, thr_wsum_prior_offsetx_class, thr_wsum_prior_offsety_class;
	double thr_wsum_sigma2_offset;
	MultidimArray<double> thr_metadata, zeroArray;
	// Wsum_sigma_noise2 is a 1D-spectrum for each group
	zeroArray.initZeros(baseMLO->mymodel.ori_size/2 + 1);
	thr_wsum_sigma2_noise.resize(baseMLO->mymodel.nr_groups, zeroArray);
	// wsum_pdf_direction is a 1D-array (of length sampling.NrDirections()) for each class
	zeroArray.initZeros(baseMLO->sampling.NrDirections());
	thr_wsum_pdf_direction.resize(baseMLO->mymodel.nr_classes, zeroArray);
	// sumw_group is a double for each group
	thr_sumw_group.resize(baseMLO->mymodel.nr_groups, 0.);
	// wsum_pdf_class is a double for each class
	thr_wsum_pdf_class.resize(baseMLO->mymodel.nr_classes, 0.);
	if (baseMLO->mymodel.ref_dim == 2)
	{
		thr_wsum_prior_offsetx_class.resize(baseMLO->mymodel.nr_classes, 0.);
		thr_wsum_prior_offsety_class.resize(baseMLO->mymodel.nr_classes, 0.);
	}
	// wsum_sigma2_offset is just a double
	thr_wsum_sigma2_offset = 0.;

	unsigned image_size = op.Fimgs[0].nzyxdim;

	CUDA_CPU_TOC("store_pre_gpu");

	// Loop from iclass_min to iclass_max to deal with seed generation in first iteration
	for (int exp_iclass = sp.iclass_min; exp_iclass <= sp.iclass_max; exp_iclass++)
	{

		/*=======================================================================================
		                            REFERENCE PROJECTION GENERATION
		=======================================================================================*/

		// Since we will need the euler matrices for all projections in the data_collect stage,
		// we might as well make it wider in scope and retain it on the GPU until then. When we
		// switch from pair to bool, there won't be any need to remake it every class, but for
		// now we create only those matrices corresponding to significant orientations, which IS  * class-specific *

		std::vector< long unsigned > iorientclasses, iover_rots;
		std::vector< double > rots, tilts, psis;

		CUDA_CPU_TIC("projection_2");

		long unsigned orientation_num = generateProjectionSetup(
					op,
					sp,
					baseMLO,
					false,  //coarse
					exp_iclass,
					rots, tilts, psis,
					iorientclasses,
					iover_rots);


		CudaGlobalPtr<FLOAT> eulers(9 * orientation_num);

		generateEulerMatrices(
				baseMLO->mymodel.PPref[exp_iclass].padding_factor,
				rots,
				tilts,
				psis,
				eulers,
				!IS_NOT_INV);

	    eulers.device_alloc();
		eulers.cp_to_device();

		CudaGlobalPtr<FLOAT > model_real((baseMLO->mymodel.PPref[exp_iclass]).data.nzyxdim);
		CudaGlobalPtr<FLOAT > model_imag((baseMLO->mymodel.PPref[exp_iclass]).data.nzyxdim);

		for(unsigned i = 0; i < model_real.size; i++)
		{
			model_real[i] = (FLOAT) baseMLO->mymodel.PPref[exp_iclass].data.data[i].real;
			model_imag[i] = (FLOAT) baseMLO->mymodel.PPref[exp_iclass].data.data[i].imag;
		}

		CudaGlobalPtr<FLOAT> Frefs_real;
		CudaGlobalPtr<FLOAT> Frefs_imag;

		bool do_combineProjAndWavg = true; //TODO add control flag
#if !defined(CUDA_DOUBLE_PRECISION)
		if(!do_combineProjAndWavg)
#endif
		{
			CUDA_CPU_TIC("generateModelProjections_wavg");
			generateModelProjections(
					model_real,
					model_imag,
					Frefs_real,
					Frefs_imag,
					eulers,
					orientation_num,
					image_size,
					XMIPP_MIN(baseMLO->mymodel.PPref[exp_iclass].r_max, op.local_Minvsigma2s[0].xdim - 1),
					op.local_Minvsigma2s[0].xdim,
					op.local_Minvsigma2s[0].ydim,
					baseMLO->mymodel.PPref[exp_iclass].data.xdim,
					baseMLO->mymodel.PPref[exp_iclass].data.ydim,
					baseMLO->mymodel.PPref[exp_iclass].data.zdim,
					baseMLO->mymodel.PPref[exp_iclass].data.yinit,
					baseMLO->mymodel.PPref[exp_iclass].data.zinit);
			model_real.free_device();
			model_imag.free_device();
			eulers.free();
			CUDA_CPU_TOC("generateModelProjections_wavg");
		}
		CUDA_CPU_TOC("projection_2");

		CudaGlobalPtr<FLOAT> wavgs_real(orientation_num * image_size);
		wavgs_real.device_alloc();
		wavgs_real.device_init(0);
		CudaGlobalPtr<FLOAT> wavgs_imag(orientation_num * image_size);
		wavgs_imag.device_alloc();
		wavgs_imag.device_init(0);
		CudaGlobalPtr<FLOAT> Fweights(orientation_num * image_size);
		Fweights.device_alloc();
		Fweights.device_init(0);

		/*=======================================================================================
										  PARTICLE ITERATION
		=======================================================================================*/

		/// Now that reference projection has been made loop over all particles inside this ori_particle
		for (long int ipart = 0; ipart < sp.nr_particles; ipart++)
		{
			long int part_id = baseMLO->mydata.ori_particles[op.my_ori_particle].particles_id[ipart];
			int group_id = baseMLO->mydata.getGroupId(part_id);

			double myprior_x, myprior_y, myprior_z;
			double old_offset_x = XX(op.old_offset[ipart]);
			double old_offset_y = YY(op.old_offset[ipart]);
			double old_offset_z;

			if (baseMLO->mymodel.ref_dim == 2)
			{
				myprior_x = XX(baseMLO->mymodel.prior_offset_class[exp_iclass]);
				myprior_y = YY(baseMLO->mymodel.prior_offset_class[exp_iclass]);
			}
			else
			{
				myprior_x = XX(op.prior[ipart]);
				myprior_y = YY(op.prior[ipart]);
				if (baseMLO->mymodel.data_dim == 3)
				{
					myprior_z = ZZ(op.prior[ipart]);
					old_offset_z = ZZ(op.old_offset[ipart]);
				}
			}


			/*======================================================
								 TRANSLATIONS
			======================================================*/

			CUDA_CPU_TIC("translation_2");

			CudaGlobalPtr<FLOAT> Fimgs_real(image_size * sp.nr_trans * sp.nr_oversampled_trans);
			CudaGlobalPtr<FLOAT> Fimgs_imag(Fimgs_real.size);
			CudaGlobalPtr<FLOAT> Fimgs_nomask_real(Fimgs_real.size);
			CudaGlobalPtr<FLOAT> Fimgs_nomask_imag(Fimgs_real.size);

			std::vector< long unsigned > iover_transes, itranses, ihiddens;

			long unsigned translation_num = imageTranslation(
					Fimgs_real,
					Fimgs_imag,
					Fimgs_nomask_real,
					Fimgs_nomask_imag,
					sp.itrans_min,
					sp.itrans_max,
					baseMLO->adaptive_oversampling ,
					baseMLO->sampling,
					oversampled_translations_x,
					oversampled_translations_y,
					oversampled_translations_z,
					sp.nr_oversampled_trans,
					baseMLO->global_fftshifts_ab_current,
					baseMLO->global_fftshifts_ab2_current,
					op.local_Fimgs_shifted[ipart],
					op.local_Fimgs_shifted_nomask[ipart],
					iover_transes,
					itranses,
					ihiddens,
					image_size);

			Fimgs_real.device_alloc();
			Fimgs_real.cp_to_device();
			Fimgs_imag.device_alloc();
			Fimgs_imag.cp_to_device();

			Fimgs_nomask_real.device_alloc();
			Fimgs_nomask_real.cp_to_device();
			Fimgs_nomask_imag.device_alloc();
			Fimgs_nomask_imag.cp_to_device();

			CUDA_CPU_TOC("translation_2");


			/*======================================================
					            	SCALE
			======================================================*/

			CUDA_CPU_TIC("scale_ctf");
			FLOAT part_scale(1.);

			if (baseMLO->do_scale_correction)
			{
				part_scale = baseMLO->mymodel.scale_correction[group_id];
				if (part_scale > 10000.)
				{
					std::cerr << " rlnMicrographScaleCorrection= " << part_scale << " group= " << group_id + 1 << std::endl;
					REPORT_ERROR("ERROR: rlnMicrographScaleCorrection is very high. Did you normalize your data?");
				}
				else if (part_scale < 0.001)
				{
					if (!have_warned_small_scale)
					{
						std::cout << " WARNING: ignoring group " << group_id + 1 << " with very small or negative scale (" << part_scale <<
								"); Use larger groups for more stable scale estimates." << std::endl;
						have_warned_small_scale = true;
					}
					part_scale = 0.001;
				}
			}

			CudaGlobalPtr<FLOAT> ctfs(image_size); //TODO Same size for all iparts, should be allocated once
			ctfs.device_alloc();

			if (baseMLO->do_ctf_correction)
			{
				for (unsigned i = 0; i < image_size; i++)
					ctfs[i] = (FLOAT) op.local_Fctfs[ipart].data[i] * part_scale;
			}
			else //TODO should be handled by memset
				for (unsigned i = 0; i < image_size; i++)
					ctfs[i] = part_scale;

			ctfs.cp_to_device();
			CUDA_CPU_TOC("scale_ctf");

			/*======================================================
					            MAP WEIGHTS
			======================================================*/

			CUDA_CPU_TIC("map");
			CudaGlobalPtr<FLOAT> sorted_weights(orientation_num * translation_num);

			mapWeights(
					sorted_weights,
					orientation_num,
					translation_num,
					baseMLO->sampling,
					ipart,
					iover_transes,
					ihiddens,
					iorientclasses,
					iover_rots,
					op.Mweight,
					sp.current_oversampling,
					sp.nr_trans);

			sorted_weights.device_alloc();
			sorted_weights.cp_to_device();
			sorted_weights.free_host();

			CUDA_CPU_TOC("map");

			/*======================================================
								KERNEL CALL
			======================================================*/

			// The below allocations are kept outside runWavgKernel(...) in case we decide to make them global.
			CudaGlobalPtr<FLOAT> Minvsigma2s(image_size); //TODO Same size for all iparts, should be allocated once
			Minvsigma2s.device_alloc();
			CudaGlobalPtr<FLOAT> wdiff2s_parts(orientation_num * image_size); //TODO Almost same size for all iparts, should be allocated once
			wdiff2s_parts.device_alloc();

			if (baseMLO->do_map)
				for (unsigned i = 0; i < image_size; i++)
					Minvsigma2s[i] = op.local_Minvsigma2s[ipart].data[i];
			else //TODO should be handled by memset
				for (unsigned i = 0; i < image_size; i++)
					Minvsigma2s[i] = 1;

			Minvsigma2s.cp_to_device();
			
#if !defined(CUDA_DOUBLE_PRECISION)
			if(do_combineProjAndWavg)
			{
				runProjAndWavgKernel(
						model_real,
						model_imag,
						eulers,
						Fimgs_real,
						Fimgs_imag,
						Fimgs_nomask_real,
						Fimgs_nomask_imag,
						sorted_weights,
						ctfs,
						Minvsigma2s,
						wdiff2s_parts,
						wavgs_real,
						wavgs_imag,
						Fweights,
						op,
						baseMLO,
						orientation_num,
						translation_num,
						image_size,
						XMIPP_MIN(baseMLO->mymodel.PPref[exp_iclass].r_max, op.local_Minvsigma2s[0].xdim - 1),
						ipart,
						group_id,
						exp_iclass
						);
			}
			else
#endif
			{
				runWavgKernel(
						Frefs_real,
						Frefs_imag,
						Fimgs_real,
						Fimgs_imag,
						Fimgs_nomask_real,
						Fimgs_nomask_imag,
						sorted_weights,
						ctfs,
						Minvsigma2s,
						wdiff2s_parts,
						wavgs_real,
						wavgs_imag,
						Fweights,
						op,
						baseMLO,
						orientation_num,
						translation_num,
						image_size,
						ipart,
						group_id,
						exp_iclass);

				Frefs_real.free_device();
				Frefs_imag.free_device();
			}
//			wdiff2s_parts.cp_to_host();
//			for (long unsigned k = 0; k < 100; k++)
//			{
//				std::cerr << wdiff2s_parts[k] << std::endl;
//			}
			/*======================================================
								COLLECT DATA
			======================================================*/

			CUDA_CPU_TIC("reduce_wdiff2s");
			// reduction_block_num = the highest possible power of two that covers more than or exactly half of all images to be reduced
			int num_reductions = (int)floor(log2((float)orientation_num));
			int reduction_block_num = pow(2,num_reductions);
			if(reduction_block_num==orientation_num) // (possibly) very special case where orientation_num is a power of 2
				reduction_block_num /= 2;

			CUDA_GPU_TIC("cuda_kernels_reduce_wdiff2s");
			unsigned orient1, orient2;
			for(int k=reduction_block_num; k>=1; k/=2) //invoke kernel repeatedly until all images have been stacked into the first image position
			{
				if(k>65535)
				{
					orient1 = ceil(sqrt(k));
					orient2 = orient1 + (orient1 % 2);  // For some reason the "optimal" values in the METADATA ooutput is sensitive to the choice of block-grid dims,
					orient1 +=          (orient1 % 2);  // and seems to work properly only when even numbers are used. // TODO examine why
				}
				else
				{
					orient1 = k;
					orient2 = 1;
				}
				dim3 grid_dim_wd(orient1,orient2);
				 // TODO **OF VERY LITTLE IMPORTANCE**  One block treating just 2 images is a very innefficient amount of loads per store
				cuda_kernel_reduce_wdiff2s<<<grid_dim_wd,BLOCK_SIZE>>>(~wdiff2s_parts,orientation_num,image_size,k);
			}
			CUDA_GPU_TOC("cuda_kernels_reduce_wdiff2s");

			wdiff2s_parts.size = image_size; //temporarily set the size to the single image we have now reduced, to not copy more than necessary
			wdiff2s_parts.cp_to_host();
			wdiff2s_parts.size = orientation_num * image_size;
			wdiff2s_parts.free_device();

			for (long int j = 0; j < image_size; j++)
			{
				int ires = DIRECT_MULTIDIM_ELEM(baseMLO->Mresol_fine, j);
				if (ires > -1)
				{
					thr_wsum_sigma2_noise[group_id].data[ires] += (double) wdiff2s_parts[j];
					exp_wsum_norm_correction[ipart] += (double) wdiff2s_parts[j];
				}
			}

			wdiff2s_parts.free_host();

			CUDA_CPU_TOC("reduce_wdiff2s");

			CUDA_CPU_TIC("collect_data_2");
			CUDA_CPU_TIC("collect_data_2_pre_kernel");
			//TODO should be replaced with loop over pairs of projections and translations (like in the getAllSquaredDifferences-function)

			std::vector< double> oversampled_rot, oversampled_tilt, oversampled_psi;

			int oversamples = sp.nr_oversampled_trans * sp.nr_oversampled_rot;
			CudaGlobalPtr<FLOAT >  Mweight( &(op.Mweight.data[(ipart)*(op.Mweight).xdim]),
											sp.nr_dir * sp.nr_psi * sp.nr_trans * oversamples);
			int nr_transes = sp.nr_trans*sp.nr_oversampled_trans;
			CudaGlobalPtr<FLOAT>     oo_otrans_x(nr_transes); // old_offset_oversampled_trans_x
			CudaGlobalPtr<FLOAT>     oo_otrans_y(nr_transes);
			CudaGlobalPtr<FLOAT> myp_oo_otrans_x2y2z2(nr_transes); // my_prior_old_offs....x^2*y^2*z^2

			//Pregenerate oversampled translation objects for kernel-call
			for (long int itrans = 0, iitrans = 0; itrans < sp.nr_trans; itrans++)
			{
				baseMLO->sampling.getTranslations(itrans, baseMLO->adaptive_oversampling,
						oversampled_translations_x, oversampled_translations_y, oversampled_translations_z);
				for (long int iover_trans = 0; iover_trans < sp.nr_oversampled_trans; iover_trans++, iitrans++)
				{
					oo_otrans_x[iitrans] = old_offset_x + oversampled_translations_x[iover_trans];
					oo_otrans_y[iitrans] = old_offset_y + oversampled_translations_y[iover_trans];
					double diffx = myprior_x - oo_otrans_x[iitrans];
					double diffy = myprior_y - oo_otrans_y[iitrans];
					if (baseMLO->mymodel.data_dim == 3)
					{
						double diffz = myprior_z - (old_offset_z + oversampled_translations_z[iover_trans]);
						myp_oo_otrans_x2y2z2[iitrans] = diffx*diffx + diffy*diffy + diffz*diffz ;
					}
					else
					{
						myp_oo_otrans_x2y2z2[iitrans] = diffx*diffx + diffy*diffy ;
					}
				}
			}

			Mweight.device_alloc();
			Mweight.cp_to_device();
			oo_otrans_x.device_alloc();
			oo_otrans_x.cp_to_device();
			oo_otrans_y.device_alloc();
			oo_otrans_y.cp_to_device();
			myp_oo_otrans_x2y2z2.device_alloc();
			myp_oo_otrans_x2y2z2.cp_to_device();

			CudaGlobalPtr<FLOAT>                      p_weights(orientation_num);
			CudaGlobalPtr<FLOAT> p_thr_wsum_prior_offsetx_class(orientation_num);
			CudaGlobalPtr<FLOAT> p_thr_wsum_prior_offsety_class(orientation_num);
			CudaGlobalPtr<FLOAT>       p_thr_wsum_sigma2_offset(orientation_num);

			p_weights.device_alloc();
			p_thr_wsum_prior_offsetx_class.device_alloc();
			p_thr_wsum_prior_offsety_class.device_alloc();
			p_thr_wsum_sigma2_offset.device_alloc();

			dim3 grid_dim_collect2(sp.nr_dir, sp.nr_psi);
			CUDA_CPU_TOC("collect_data_2_pre_kernel");

			cuda_kernel_collect2<<<grid_dim_collect2,SUM_BLOCK_SIZE>>>(
					~oo_otrans_x,          // otrans-size -> make const
					~oo_otrans_y,          // otrans-size -> make const
					~myp_oo_otrans_x2y2z2, // otrans-size -> make const
					~Mweight,
					(FLOAT)op.significant_weight[ipart],
					(FLOAT)op.sum_weight[ipart],
					sp.nr_trans,
					sp.nr_oversampled_trans,
					sp.nr_oversampled_rot,
					oversamples,
					(baseMLO->do_skip_align || baseMLO->do_skip_rotate ),
					~p_weights,
					~p_thr_wsum_prior_offsetx_class,
					~p_thr_wsum_prior_offsety_class,
					~p_thr_wsum_sigma2_offset
				   );
			HANDLE_ERROR(hipDeviceSynchronize());

			// TODO further reduce the below 4 arrays while data is still on gpu
			p_weights.cp_to_host();
			p_thr_wsum_prior_offsetx_class.cp_to_host();
			p_thr_wsum_prior_offsety_class.cp_to_host();
			p_thr_wsum_sigma2_offset.cp_to_host();

			thr_wsum_sigma2_offset = 0.0;
			int iorient = 0;
			for (long int idir = 0; idir < sp.nr_dir; idir++)
			{
				for (long int ipsi = 0; ipsi < sp.nr_psi; ipsi++, iorient++)
				{
					long int iorientclass = exp_iclass * sp.nr_dir * sp.nr_psi + iorient;
					// Only proceed if any of the particles had any significant coarsely sampled translation

					if (baseMLO->isSignificantAnyParticleAnyTranslation(iorientclass, sp.itrans_min, sp.itrans_max, op.Mcoarse_significant))
					{
						long int mydir;
						if (baseMLO->mymodel.orientational_prior_mode == NOPRIOR)
							mydir = idir;
						else
							mydir = op.pointer_dir_nonzeroprior[idir];

						// store partials according to indices of the relevant dimension
						DIRECT_MULTIDIM_ELEM(thr_wsum_pdf_direction[exp_iclass], mydir) += p_weights[iorient];
						thr_sumw_group[group_id]                 						+= p_weights[iorient];
						thr_wsum_pdf_class[exp_iclass]           						+= p_weights[iorient];
						thr_wsum_sigma2_offset                   						+= p_thr_wsum_sigma2_offset[iorient];

						if (baseMLO->mymodel.ref_dim == 2)
						{
							thr_wsum_prior_offsetx_class[exp_iclass] 	+= p_thr_wsum_prior_offsetx_class[iorient];
							thr_wsum_prior_offsety_class[exp_iclass] 	+= p_thr_wsum_prior_offsety_class[iorient];
						}
					}
				}
			}

			CUDA_CPU_TIC("collect_data_2_post_kernel");
			Mweight.free_device();
			p_weights.free();
			p_thr_wsum_sigma2_offset.free();
			p_thr_wsum_prior_offsetx_class.free();
			p_thr_wsum_prior_offsety_class.free();

			oo_otrans_y.free();
			oo_otrans_x.free();
			myp_oo_otrans_x2y2z2.free();

			//Get index of max element using GPU-tool thrust
			Indices max_index;
			max_index.fineIdx = thrust::max_element(&DIRECT_A2D_ELEM(op.Mweight, ipart, 0),&DIRECT_A2D_ELEM(op.Mweight, ipart+1, 0)) - &DIRECT_A2D_ELEM(op.Mweight, ipart, 0);
			op.max_weight[ipart] = DIRECT_A2D_ELEM(op.Mweight, ipart, max_index.fineIdx);
			max_index.fineIndexToFineIndices(sp); // set partial indices corresponding to the found max_index, to be used below

			baseMLO->sampling.getTranslations(max_index.itrans, baseMLO->adaptive_oversampling,
					oversampled_translations_x, oversampled_translations_y, oversampled_translations_z);
			baseMLO->sampling.getOrientations(max_index.idir, max_index.ipsi, baseMLO->adaptive_oversampling, oversampled_rot, oversampled_tilt, oversampled_psi,
					op.pointer_dir_nonzeroprior, op.directions_prior, op.pointer_psi_nonzeroprior, op.psi_prior);
			double rot = oversampled_rot[max_index.ioverrot];
			double tilt = oversampled_tilt[max_index.ioverrot];
			double psi = oversampled_psi[max_index.ioverrot];
			DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_ROT) = rot;
			DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_TILT) = tilt;
			DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_PSI) = psi;
			DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_XOFF) = XX(op.old_offset[ipart]) + oversampled_translations_x[max_index.iovertrans];
			DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_YOFF) = YY(op.old_offset[ipart]) + oversampled_translations_y[max_index.iovertrans];
			if (baseMLO->mymodel.data_dim == 3)
				DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_ZOFF) = ZZ(op.old_offset[ipart]) + oversampled_translations_z[max_index.iovertrans];
			DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_CLASS) = (double)max_index.iclass + 1;
			DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_PMAX) = op.max_weight[ipart]/op.sum_weight[ipart];
			CUDA_CPU_TOC("collect_data_2_post_kernel");
			CUDA_CPU_TOC("collect_data_2");

		} // end loop ipart

		/*=======================================================================================
										   BACKPROJECTION
		=======================================================================================*/

		CUDA_CPU_TIC("backprojection");

		CudaGlobalPtr<FLOAT> bp_model_real(baseMLO->wsum_model.BPref[exp_iclass].data.nzyxdim);
		bp_model_real.device_alloc();
		bp_model_real.device_init(0);
		CudaGlobalPtr<FLOAT> bp_model_imag(bp_model_real.size);
		bp_model_imag.device_alloc();
		bp_model_imag.device_init(0);
		CudaGlobalPtr<FLOAT> bp_weight(bp_model_real.size);
		bp_weight.device_alloc();
		bp_weight.device_init(0);


		CudaGlobalPtr<FLOAT> bp_eulers(9 * orientation_num);

		FLOAT padding_factor = baseMLO->wsum_model.BPref[exp_iclass].padding_factor;

		generateEulerMatrices(
				1/padding_factor, //Why squared scale factor is given in backprojection
				rots,
				tilts,
				psis,
				bp_eulers,
				IS_NOT_INV);

		bp_eulers.device_alloc();
	    bp_eulers.cp_to_device();
	    bp_eulers.free_host();


	    runBackprojectKernel(
				wavgs_real,
				wavgs_imag,
				Fweights,
				bp_eulers,
				bp_model_real,
				bp_model_imag,
				bp_weight,
				baseMLO->wsum_model.BPref[exp_iclass].r_max,
				padding_factor * padding_factor,
				image_size,
				orientation_num,
				op.local_Minvsigma2s[0].xdim,
				op.local_Minvsigma2s[0].ydim,
				baseMLO->wsum_model.BPref[exp_iclass].data.xdim,
				baseMLO->wsum_model.BPref[exp_iclass].data.ydim,
				baseMLO->wsum_model.BPref[exp_iclass].data.zdim,
				baseMLO->wsum_model.BPref[exp_iclass].data.yinit,
				baseMLO->wsum_model.BPref[exp_iclass].data.zinit);

		bp_model_real.cp_to_host();
		bp_model_imag.cp_to_host();
		bp_weight.cp_to_host();

		HANDLE_ERROR(hipDeviceSynchronize()); //TODO Optimize concurrency

		bp_model_real.free_device();
		bp_model_imag.free_device();
		bp_weight.free_device();

//#define PRINT_BACKPROJECTION_RESULTS
#ifdef PRINT_BACKPROJECTION_RESULTS

		FILE *fPtr1 = fopen("gpu_backproj_values.dat","w");
		for (unsigned i = 0; i < bp_model_real.size; i ++)
			fprintf(fPtr1, "%.1e %.1e\n", bp_model_real[i], bp_model_imag[i]);
		fclose(fPtr1);

		FILE *fPtr2 = fopen("gpu_backproj_weights.dat","w");
		for (unsigned i = 0; i < bp_weight.size; i ++)
			fprintf(fPtr2, "%.1e\n", bp_weight[i]);
		fclose(fPtr2);

		wavgs_real.cp_to_host();
		wavgs_imag.cp_to_host();
		Fweights.cp_to_host();

		for (long int i = 0; i < orientation_num; i++)
		{
			Euler_angles2matrix(rots[i], tilts[i], psis[i], A);

			for (unsigned j = 0; j < image_size; j++)
			{
				Fimg.data[j].real = (double) wavgs_real[i * image_size + j];
				Fimg.data[j].imag = (double) wavgs_imag[i * image_size + j];
				Fweight.data[j] = (double) Fweights[i * image_size + j];
			}

			int my_mutex = exp_iclass % NR_CLASS_MUTEXES;
			pthread_mutex_lock(&global_mutex2[my_mutex]);
			(baseMLO->wsum_model.BPref[exp_iclass]).set2DFourierTransform(Fimg, A, IS_NOT_INV, &Fweight);
			pthread_mutex_unlock(&global_mutex2[my_mutex]);

		}

		FILE *fPtr3 = fopen("cpu_backproj_values.dat","w");
		for (unsigned i = 0; i < (baseMLO->wsum_model.BPref[exp_iclass]).data.nzyxdim; i ++)
			fprintf(fPtr3, "%.1e %.1e\n", (baseMLO->wsum_model.BPref[exp_iclass]).data.data[i].real, (baseMLO->wsum_model.BPref[exp_iclass]).data.data[i].imag);
		fclose(fPtr3);

		FILE *fPtr4 = fopen("cpu_backproj_weights.dat","w");
		for (unsigned i = 0; i < (baseMLO->wsum_model.BPref[exp_iclass]).data.nzyxdim; i ++)
			fprintf(fPtr4, "%.1e\n", (baseMLO->wsum_model.BPref[exp_iclass]).bp_weight.data[i]);
		fclose(fPtr4);

		exit(0);
#endif

		Fweights.free();
		wavgs_real.free();
		wavgs_imag.free();

		int my_mutex = exp_iclass % NR_CLASS_MUTEXES;
		pthread_mutex_lock(&global_mutex2[my_mutex]);

		for (long unsigned i = 0; i < bp_model_real.size; i++)
		{
			baseMLO->wsum_model.BPref[exp_iclass].data.data[i].real += bp_model_real[i];
			baseMLO->wsum_model.BPref[exp_iclass].data.data[i].imag += bp_model_imag[i];
			baseMLO->wsum_model.BPref[exp_iclass].weight.data[i] += bp_weight[i];
		}

		pthread_mutex_unlock(&global_mutex2[my_mutex]);

		CUDA_CPU_TOC("backprojection");

	} // end loop iclass

	CUDA_CPU_TIC("store_post_gpu");

	// Extend norm_correction and sigma2_noise estimation to higher resolutions for all particles
	// Also calculate dLL for each particle and store in metadata
	// loop over all particles inside this ori_particle
	double thr_avg_norm_correction = 0.;
	double thr_sum_dLL = 0., thr_sum_Pmax = 0.;
	for (long int ipart = 0; ipart < sp.nr_particles; ipart++)
	{
		long int part_id = baseMLO->mydata.ori_particles[op.my_ori_particle].particles_id[ipart];
		int group_id = baseMLO->mydata.getGroupId(part_id);

		// If the current images were smaller than the original size, fill the rest of wsum_model.sigma2_noise with the power_class spectrum of the images
		for (int ires = baseMLO->mymodel.current_size/2 + 1; ires < baseMLO->mymodel.ori_size/2 + 1; ires++)
		{
			DIRECT_A1D_ELEM(thr_wsum_sigma2_noise[group_id], ires) += DIRECT_A1D_ELEM(op.power_imgs[ipart], ires);
			// Also extend the weighted sum of the norm_correction
			exp_wsum_norm_correction[ipart] += DIRECT_A1D_ELEM(op.power_imgs[ipart], ires);
		}

		// Store norm_correction
		// Multiply by old value because the old norm_correction term was already applied to the image
		if (baseMLO->do_norm_correction)
		{
			double old_norm_correction = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_NORM);
			old_norm_correction /= baseMLO->mymodel.avg_norm_correction;
			// The factor two below is because exp_wsum_norm_correctiom is similar to sigma2_noise, which is the variance for the real/imag components
			// The variance of the total image (on which one normalizes) is twice this value!
			double normcorr = old_norm_correction * sqrt(exp_wsum_norm_correction[ipart] * 2.);
			thr_avg_norm_correction += normcorr;
			// Now set the new norm_correction in the relevant position of exp_metadata
			DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_NORM) = normcorr;


			// Print warning for strange norm-correction values
			if (!(baseMLO->iter == 1 && baseMLO->do_firstiter_cc) && DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_NORM) > 10.)
			{
				std::cout << " WARNING: norm_correction= "<< DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_NORM) << " for particle " << part_id << " in group " << group_id + 1 << "; Are your groups large enough?" << std::endl;
			}

		}

		// Store weighted sums for scale_correction
		if (baseMLO->do_scale_correction)
		{
			// Divide XA by the old scale_correction and AA by the square of that, because was incorporated into Fctf
			exp_wsum_scale_correction_XA[ipart] /= baseMLO->mymodel.scale_correction[group_id];
			exp_wsum_scale_correction_AA[ipart] /= baseMLO->mymodel.scale_correction[group_id] * baseMLO->mymodel.scale_correction[group_id];

			thr_wsum_signal_product_spectra[group_id] += exp_wsum_scale_correction_XA[ipart];
			thr_wsum_reference_power_spectra[group_id] += exp_wsum_scale_correction_AA[ipart];
		}

		// Calculate DLL for each particle
		double logsigma2 = 0.;
		FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(baseMLO->Mresol_fine)
		{
			int ires = DIRECT_MULTIDIM_ELEM(baseMLO->Mresol_fine, n);
			// Note there is no sqrt in the normalisation term because of the 2-dimensionality of the complex-plane
			// Also exclude origin from logsigma2, as this will not be considered in the P-calculations
			if (ires > 0)
				logsigma2 += log( 2. * PI * DIRECT_A1D_ELEM(baseMLO->mymodel.sigma2_noise[group_id], ires));
		}
		if (op.sum_weight[ipart]==0)
		{
			std::cerr << " part_id= " << part_id << std::endl;
			std::cerr << " ipart= " << ipart << std::endl;
			std::cerr << " op.min_diff2[ipart]= " << op.min_diff2[ipart] << std::endl;
			std::cerr << " logsigma2= " << logsigma2 << std::endl;
			int group_id = baseMLO->mydata.getGroupId(part_id);
			std::cerr << " group_id= " << group_id << std::endl;
			std::cerr << " ml_model.scale_correction[group_id]= " << baseMLO->mymodel.scale_correction[group_id] << std::endl;
			std::cerr << " exp_significant_weight[ipart]= " << op.significant_weight[ipart] << std::endl;
			std::cerr << " exp_max_weight[ipart]= " << op.max_weight[ipart] << std::endl;
			std::cerr << " ml_model.sigma2_noise[group_id]= " << baseMLO->mymodel.sigma2_noise[group_id] << std::endl;
			REPORT_ERROR("ERROR: op.sum_weight[ipart]==0");
		}
		double dLL;
		if ((baseMLO->iter==1 && baseMLO->do_firstiter_cc) || baseMLO->do_always_cc)
			dLL = -op.min_diff2[ipart];
		else
			dLL = log(op.sum_weight[ipart]) - op.min_diff2[ipart] - logsigma2;

		// Store dLL of each image in the output array, and keep track of total sum
		DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_DLL) = dLL;
		thr_sum_dLL += dLL;

		// Also store sum of Pmax
		thr_sum_Pmax += DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_PMAX);

	}

	// Now, inside a global_mutex, update the other weighted sums among all threads
	if (!baseMLO->do_skip_maximization)
	{
		pthread_mutex_lock(&global_mutex);
		for (int n = 0; n < baseMLO->mymodel.nr_groups; n++)
		{
			baseMLO->wsum_model.sigma2_noise[n] += thr_wsum_sigma2_noise[n];
			baseMLO->wsum_model.sumw_group[n] += thr_sumw_group[n];
			if (baseMLO->do_scale_correction)
			{
				baseMLO->wsum_model.wsum_signal_product_spectra[n] += thr_wsum_signal_product_spectra[n];
				baseMLO->wsum_model.wsum_reference_power_spectra[n] += thr_wsum_reference_power_spectra[n];
			}
		}
		for (int n = 0; n < baseMLO->mymodel.nr_classes; n++)
		{
			baseMLO->wsum_model.pdf_class[n] += thr_wsum_pdf_class[n];
			if (baseMLO->mymodel.ref_dim == 2)
			{
				XX(baseMLO->wsum_model.prior_offset_class[n]) += thr_wsum_prior_offsetx_class[n];
				YY(baseMLO->wsum_model.prior_offset_class[n]) += thr_wsum_prior_offsety_class[n];
			}

			if (!(baseMLO->do_skip_align || baseMLO->do_skip_rotate) )
				baseMLO->wsum_model.pdf_direction[n] += thr_wsum_pdf_direction[n];
		}
		baseMLO->wsum_model.sigma2_offset += thr_wsum_sigma2_offset;
		if (baseMLO->do_norm_correction)
			baseMLO->wsum_model.avg_norm_correction += thr_avg_norm_correction;
		baseMLO->wsum_model.LL += thr_sum_dLL;
		baseMLO->wsum_model.ave_Pmax += thr_sum_Pmax;
		pthread_mutex_unlock(&global_mutex);
	} // end if !do_skip_maximization

	CUDA_CPU_TOC("store_post_gpu");
}
