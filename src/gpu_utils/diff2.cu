#include "hip/hip_runtime.h"
#include "src/gpu_utils/diff2.h"
#include <vector>

#define cuda_block_size 32

__global__ void cuda_kernel_applyAB(double *img, double *myAB, double *shifted_img)
{
    int n = (blockIdx.x * blockDim.x + threadIdx.x)*2;
    *(shifted_img + n) = (*(myAB + n)) * (*(img + n))
    		- (*(myAB + n + 1)) * (*(img + n + 1));
    *(shifted_img + n + 1) = (*(myAB + n)) * (*(img + n + 1))
			+ (*(myAB + n + 1)) * (*(img + n));
}

//__global__ void cuda_kernel_diff2(Complex *ref, Complex* img, Complex* Minvsigma2, double* diff2)
//{
//    int n = threadIdx.x;
//    double diff_real = (*(ref + n)).real - (*(img + n)).real;
//	double diff_imag = (*(ref + n)).imag - (*(img + n)).imag;
//	// diff2 increment add needs to be atomic
//	diff2 += (diff_real * diff_real + diff_imag * diff_imag) * 0.5 * (*(Minvsigma2 + n));
//}

void cuda_applyAB(
		int img_num,
		double *h_exp_local_Fimgs_shifted,
		double *h_myAB,
		double *h_Fimg_otfshift)
{
	int num_blocks(ceil(img_num/cuda_block_size));

	// declare GPU memory pointers
	double * d_myAB;
	double * d_exp_local_Fimgs_shifted;
	double * d_Fimg_otfshift;

	int N = img_num  * sizeof(double) * 2; // x2 for real and imaginary part

	hipMalloc( (void**) &d_myAB, N);
	hipMalloc( (void**) &d_exp_local_Fimgs_shifted, N);
	hipMalloc( (void**) &d_Fimg_otfshift, N);

	hipMemcpy( d_myAB, h_myAB, N, hipMemcpyHostToDevice);
	hipMemcpy( d_exp_local_Fimgs_shifted, h_exp_local_Fimgs_shifted, N, hipMemcpyHostToDevice);

	//let's do a simple setup for now; each pixel is a thread, each row is a block
	cuda_kernel_applyAB<<<num_blocks, cuda_block_size>>>(d_exp_local_Fimgs_shifted, d_myAB, d_Fimg_otfshift);

	hipMemcpy( h_Fimg_otfshift, d_Fimg_otfshift, N, hipMemcpyDeviceToHost );
}
//
//void cuda_diff2(
//		long int ipart,
//		std::vector<MultidimArray<Complex > > &h_exp_local_Fimgs_shifted,
//		Complex *h_myAB,
//		MultidimArray<Complex > h_Fimg_otfshift)
//{
//	// Size of the image arrays
//	int img_num = NZYXSIZE(h_exp_local_Fimgs_shifted[ipart]);
//	int num_blocks(ceil(img_num/cuda_block_size));
//
//	// declare GPU memory pointers
//	Complex * d_myAB;
//	Complex * d_exp_local_Fimgs_shifted;
//	Complex * d_Fimg_otfshift;
//
//	int N = img_num  * sizeof(Complex);
//
//	hipMalloc( (void**) &d_myAB, N);
//	hipMalloc( (void**) &d_exp_local_Fimgs_shifted, N);
//	hipMalloc( (void**) &d_Fimg_otfshift, N);
//
//	hipMemcpy( d_myAB, h_myAB, N, hipMemcpyHostToDevice);
//	hipMemcpy( d_exp_local_Fimgs_shifted, h_exp_local_Fimgs_shifted[ipart], N, hipMemcpyHostToDevice);
//
//	//let's do a simple setup for now; each pixel is a thread, each row is a block
//	cuda_kernel_diff2<<<num_blocks, cuda_block_size>>>(d_exp_local_Fimgs_shifted, d_myAB, d_Fimg_otfshift);
//
//	hipMemcpy( h_Fimg_otfshift, d_Fimg_otfshift, N, hipMemcpyDeviceToHost );
//}
