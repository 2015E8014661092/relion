#include "hip/hip_runtime.h"
/*
 * cuda_ProjDiff_kenrels.cu

 *
 *  Created on: May 27, 2015
 *      Author: bjornf
 */
#include "src/gpu_utils/cuda_ProjDiff_kernels.cuh"
#include <vector>
#include <iostream>

#if !defined(CUDA_DOUBLE_PRECISION) && defined(USE_TEXINTERP)

__global__ void cuda_kernel_PAV_TTI_D2( FLOAT *g_eulers,
		                                FLOAT *g_imgs_real,
		                                FLOAT *g_imgs_imag,
										hipTextureObject_t texModel_real,
										hipTextureObject_t texModel_imag,
										FLOAT *g_Minvsigma2,
										FLOAT *g_diff2s,
										int image_size,
										FLOAT sum_init,
										int orientation_num,
										int translation_num,
										int significant_num,
										unsigned long *d_rotidx,
										unsigned long *d_transidx,
										unsigned long *d_trans_num,
										unsigned long *d_ihidden_overs,
										int my_r_max,
										int max_r2,
										int min_r2_nn,
										long int img_x,
										long int img_y,
										long int mdl_init_y,
										long int mdl_init_z
										)
{
	int bid = blockIdx.y * gridDim.x + blockIdx.x;
	int tid = threadIdx.x;

	FLOAT xp, yp, zp;
	long int r2;
	int pixel;
	bool is_neg_x;
	FLOAT ref_real;
	FLOAT ref_imag;

	// inside the padded 2D orientation grid
	if( bid < significant_num ) // we only need to make
	{
		__shared__ FLOAT s[BLOCK_SIZE];
		s[tid] = 0.0f;

		// index of comparison
		unsigned long int ix=d_rotidx[bid];
		unsigned long int iy=d_transidx[bid];

		unsigned pass_num(ceilf(   ((float)image_size) / (float)BLOCK_SIZE  ));
		unsigned long img_start(iy * image_size);
		unsigned long img_pixel_idx;

		for (unsigned pass = 0; pass < pass_num; pass++) // finish a reference proj in each block
		{
			pixel = (pass * BLOCK_SIZE) + tid;
			if(pixel<image_size)
			{
				int x = pixel % img_x;
				int y = (int)floorf( (float)pixel / (float)img_x);
				img_pixel_idx = img_start + pixel;

				// Dont search beyond square with side max_r
				if (y > my_r_max)
				{
					if (y >= img_y - my_r_max)
						y = y - img_y ;
					else
						x=r2;
				}

				r2 = x*x + y*y;
				if (r2 <= max_r2)
				{
					xp = __ldg(&g_eulers[ix*9])   * x + __ldg(&g_eulers[ix*9+1]) * y;  // FIXME: xp,yp,zp has has accuracy loss
					yp = __ldg(&g_eulers[ix*9+3]) * x + __ldg(&g_eulers[ix*9+4]) * y;  // compared to CPU-based projection. This
					zp = __ldg(&g_eulers[ix*9+6]) * x + __ldg(&g_eulers[ix*9+7]) * y;  // propagates to dx00, dx10, and so on.
					// Only asymmetric half is stored
					if (xp < 0)
					{
						// Get complex conjugated hermitian symmetry pair
						xp = -xp;
						yp = -yp;
						zp = -zp;
						is_neg_x = true;
					}
					else
					{
						is_neg_x = false;
					}
					yp -= mdl_init_y;
					zp -= mdl_init_z;

					ref_real=tex3D<FLOAT>(texModel_real,xp+0.5f,yp+0.5f,zp+0.5f);
					ref_imag=tex3D<FLOAT>(texModel_imag,xp+0.5f,yp+0.5f,zp+0.5f);

//					printf("%i, %i", x,y);
//					printf("%f, %f,%f", xp,yp,zp);
					if (is_neg_x)
					{
						ref_imag = -ref_imag;
					}
				}
				else
				{
					ref_real=0.0f;
					ref_imag=0.0f;
				}
				FLOAT diff_real =  ref_real - __ldg(&g_imgs_real[img_pixel_idx]); // TODO  Put g_img_* in texture (in such a way that fetching of next image might hit in cache)
				FLOAT diff_imag =  ref_imag - __ldg(&g_imgs_imag[img_pixel_idx]);

				s[tid] += (diff_real * diff_real + diff_imag * diff_imag) * 0.5f * __ldg(&g_Minvsigma2[pixel]);
//				printf(" diffs = %f, %f \n",ref_real,img_pixel_idx);
//				printf(" diffs = %i, %i ,%i \n",x,y);
			}
		}
		__syncthreads();

		for(int j=(BLOCK_SIZE/2); j>0; j>>=1)
		{
			if(tid<j)
			{
				s[tid] += s[tid+j];
			}
			__syncthreads();
		}
		if (tid == 0)
		{
			g_diff2s[ix * translation_num + iy] = s[0]+sum_init;
		}
	}
}
#elif !defined(CUDA_DOUBLE_PRECISION)
// __global__ void cuda_kernel_PAV_TTE_D2
#else
// __global__ void cuda_kernel_PAV_TGE_D2
#endif // !defined(CUDA_DOUBLE_PRECISION) && defined(USE_TEXINTERP)
