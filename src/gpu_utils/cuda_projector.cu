#include "src/gpu_utils/cuda_projector.h"
#include "src/gpu_utils/cuda_utils.cuh"
#include <signal.h>

#ifndef CUDA_DOUBLE_PRECISION

void CudaProjector::setMdlData(float *real, float *imag)
{
#ifdef CUDA_DEBUG
	if (mdlXYZ == 0)
	{
        printf("DEBUG_ERROR: Model dimensions must be set with setMdlDim before call to setMdlData.");
		raise(SIGSEGV);
	}
	if (mdlReal != 0)
	{
        printf("DEBUG_ERROR: Duplicated call to setMdlData.");
		raise(SIGSEGV);
	}
#endif
    mdlReal = new hipTextureObject_t();
    mdlImag = new hipTextureObject_t();
	texArrayReal = new hipArray_t();
	texArrayImag = new hipArray_t();

	// create channel to describe data type (bits,bits,bits,bits,type)
	hipChannelFormatDesc desc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipExtent volumeSize = make_hipExtent(mdlX, mdlY, mdlZ);

	struct hipResourceDesc resDesc_real, resDesc_imag;
	struct hipTextureDesc texDesc_real, texDesc_imag;

	hipMemcpy3DParms copyParams = {0};
	copyParams.extent = volumeSize;
	copyParams.kind   = hipMemcpyHostToDevice;


	HANDLE_ERROR(hipMalloc3DArray(texArrayReal, &desc, volumeSize));

	copyParams.dstArray = *texArrayReal;
	copyParams.srcPtr   = make_hipPitchedPtr(real,mdlX*sizeof(float), mdlY, mdlZ);
	HANDLE_ERROR(hipMemcpy3D(&copyParams));

	memset(&resDesc_real, 0, sizeof(hipResourceDesc));
    resDesc_real.resType = hipResourceTypeArray;
    resDesc_real.res.array.array = copyParams.dstArray;

    memset(&texDesc_real, 0, sizeof(hipTextureDesc));
    texDesc_real.filterMode       = hipFilterModeLinear;
    texDesc_real.readMode         = hipReadModeElementType;
    texDesc_real.normalizedCoords = false;
    for(int n=0; n<3; n++)
    	texDesc_real.addressMode[n]=hipAddressModeClamp;

	HANDLE_ERROR(hipCreateTextureObject(mdlReal, &resDesc_real, &texDesc_real, NULL));




	HANDLE_ERROR(hipMalloc3DArray(texArrayImag, &desc, volumeSize));

	copyParams.dstArray = *texArrayImag;
	copyParams.srcPtr   = make_hipPitchedPtr(imag,mdlX*sizeof(float), mdlY, mdlZ);
	HANDLE_ERROR(hipMemcpy3D(&copyParams));

	memset(&resDesc_imag, 0, sizeof(hipResourceDesc));
    resDesc_imag.resType = hipResourceTypeArray;
    resDesc_imag.res.array.array = copyParams.dstArray;

    memset(&texDesc_imag, 0, sizeof(hipTextureDesc));
    texDesc_imag.filterMode       = hipFilterModeLinear;
    texDesc_imag.readMode         = hipReadModeElementType;
    texDesc_imag.normalizedCoords = false;
    for(int n=0; n<3; n++)
    	texDesc_imag.addressMode[n]=hipAddressModeClamp;

	HANDLE_ERROR(hipCreateTextureObject(mdlImag, &resDesc_imag, &texDesc_imag, NULL));
}

#else

void CudaProjector::setMdlData(double *real, double *imag)
{
#ifdef CUDA_DEBUG
	if (mdlXYZ == 0)
	{
        printf("DEBUG_ERROR: Model dimensions must be set with setMdlDim before call to setMdlData.");
		raise(SIGSEGV);
	}
	if (mdlReal != 0)
	{
        printf("DEBUG_ERROR: Duplicated call to setMdlData.");
		raise(SIGSEGV);
	}
#endif

	HANDLE_ERROR(hipMalloc( (void**) &mdlReal, mdlXYZ * sizeof(double)));
	HANDLE_ERROR(hipMalloc( (void**) &mdlImag, mdlXYZ * sizeof(double)));

	HANDLE_ERROR(hipMemcpy( mdlReal, real, mdlXYZ * sizeof(FLOAT), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy( mdlImag, imag, mdlXYZ * sizeof(FLOAT), hipMemcpyHostToDevice));

}
#endif


void CudaProjector::setMdlData(Complex *data)
{
	XFLOAT *tmpReal = new XFLOAT[mdlXYZ];
	XFLOAT *tmpImag = new XFLOAT[mdlXYZ];

	for (unsigned long i = 0; i < mdlXYZ; i ++)
	{
		tmpReal[i] = (XFLOAT) data[i].real;
		tmpImag[i] = (XFLOAT) data[i].imag;
	}

	setMdlData(tmpReal, tmpImag);

	delete [] tmpReal;
	delete [] tmpImag;
}


CudaProjector::~CudaProjector()
{
	if (mdlReal != 0)
	{
#ifdef CUDA_DOUBLE_PRECISION
		hipFree(mdlReal);
		hipFree(mdlImag);
#else
		hipDestroyTextureObject(*mdlReal);
		hipDestroyTextureObject(*mdlImag);
		delete mdlReal;
		delete mdlImag;

		hipFreeArray(*texArrayReal);
		hipFreeArray(*texArrayImag);
		delete texArrayReal;
		delete texArrayImag;
		texArrayReal = 0;
		texArrayImag = 0;
#endif
		mdlReal = 0;
		mdlImag = 0;
	}
}

