#include "src/gpu_utils/cuda_projector.h"
#include "src/gpu_utils/cuda_utils.cuh"
#include <hip/hip_runtime.h>
#include <signal.h>

#ifndef CUDA_DOUBLE_PRECISION

void Cuda3DProjector::setMdlData(float *real, float *imag)
{
#ifdef CUDA_DEBUG
	if (mdlXYZ == 0)
	{
        printf("DEBUG_ERROR: Model dimensions must be set with setMdlDim before call to setMdlData.");
		raise(SIGSEGV);
	}
	if (mdlReal != 0)
	{
        printf("DEBUG_ERROR: Duplicated call to setMdlData.");
		raise(SIGSEGV);
	}
#endif
    mdlReal = (void*) new hipTextureObject_t();
    mdlImag = (void*) new hipTextureObject_t();
	texArrayReal = (void*) new hipArray_t();
	texArrayImag = (void*) new hipArray_t();

	// create channel to describe data type (bits,bits,bits,bits,type)
	hipChannelFormatDesc desc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipExtent volumeSize = make_hipExtent(mdlX, mdlY, mdlZ);

	struct hipResourceDesc resDesc_real, resDesc_imag;
	struct hipTextureDesc texDesc_real, texDesc_imag;

	hipMemcpy3DParms copyParams = {0};
	copyParams.extent   = volumeSize;
	copyParams.kind     = hipMemcpyHostToDevice;


	HANDLE_ERROR(hipMalloc3DArray((hipArray_t*)texArrayReal, &desc, volumeSize));

	copyParams.dstArray = *((hipArray_t*)texArrayReal);
	copyParams.srcPtr   = make_hipPitchedPtr((void*)real,mdlX*sizeof(float), mdlY, mdlZ);
	HANDLE_ERROR(hipMemcpy3D(&copyParams));

	memset(&resDesc_real, 0, sizeof(hipResourceDesc));
    resDesc_real.resType = hipResourceTypeArray;
    resDesc_real.res.array.array = copyParams.dstArray;

    memset(&texDesc_real, 0, sizeof(hipTextureDesc));
    texDesc_real.filterMode       = hipFilterModeLinear;
    texDesc_real.readMode         = hipReadModeElementType;
    texDesc_real.normalizedCoords = false;
    for(int n=0; n<3; n++)
    	texDesc_real.addressMode[n]=hipAddressModeClamp;

	HANDLE_ERROR(hipCreateTextureObject((hipTextureObject_t*)mdlReal, &resDesc_real, &texDesc_real, NULL));




	HANDLE_ERROR(hipMalloc3DArray((hipArray_t*)texArrayImag, &desc, volumeSize));

	copyParams.dstArray = *((hipArray_t*)texArrayImag);
	copyParams.srcPtr   = make_hipPitchedPtr((void*)imag,mdlX*sizeof(float), mdlY, mdlZ);
	HANDLE_ERROR(hipMemcpy3D(&copyParams));

	memset(&resDesc_imag, 0, sizeof(hipResourceDesc));
    resDesc_imag.resType = hipResourceTypeArray;
    resDesc_imag.res.array.array = copyParams.dstArray;

    memset(&texDesc_imag, 0, sizeof(hipTextureDesc));
    texDesc_imag.filterMode       = hipFilterModeLinear;
    texDesc_imag.readMode         = hipReadModeElementType;
    texDesc_imag.normalizedCoords = false;
    for(int n=0; n<3; n++)
    	texDesc_imag.addressMode[n]=hipAddressModeClamp;

	HANDLE_ERROR(hipCreateTextureObject((hipTextureObject_t*)mdlImag, &resDesc_imag, &texDesc_imag, NULL));
}

#else

void Cuda3DProjector::setMdlData(double *real, double *imag)
{
#ifdef CUDA_DEBUG
	if (mdlXYZ == 0)
	{
        printf("DEBUG_ERROR: Model dimensions must be set with setMdlDim before call to setMdlData.");
		raise(SIGSEGV);
	}
	if (mdlReal != 0)
	{
        printf("DEBUG_ERROR: Duplicated call to setMdlData.");
		raise(SIGSEGV);
	}
#endif
	CudaGlobalPtr<double> *r = new CudaGlobalPtr<double>();
	CudaGlobalPtr<double> *i = new CudaGlobalPtr<double>();

	r->h_ptr = real;
	i->h_ptr = imag;

	r->size = mdlXYZ;
	i->size = mdlXYZ;

	r->device_alloc();
	i->device_alloc();

	r->cp_to_device();
	i->cp_to_device();

	r->h_ptr = 0;
	i->h_ptr = 0;

	mdlReal = (void*) r;
	mdlImag = (void*) i;
}
:
#endif


void Cuda3DProjector::setMdlData(Complex *data)
{
	FLOAT *tmpReal = new FLOAT[mdlXYZ];
	FLOAT *tmpImag = new FLOAT[mdlXYZ];

	for (unsigned long i = 0; i < mdlXYZ; i ++)
	{
		tmpReal[i] = (FLOAT) data[i].real;
		tmpImag[i] = (FLOAT) data[i].imag;
	}

	setMdlData(tmpReal, tmpImag);

	delete [] tmpReal;
	delete [] tmpImag;
}


Cuda3DProjector::~Cuda3DProjector()
{
	if (mdlReal != 0)
	{
#ifdef CUDA_DOUBLE_PRECISION
		delete (CudaGlobalPtr<double>*) mdlReal;
		delete (CudaGlobalPtr<double>*) mdlImag;
#else
		hipDestroyTextureObject(*(hipTextureObject_t*) mdlReal);
		hipDestroyTextureObject(*(hipTextureObject_t*) mdlImag);
		delete (hipTextureObject_t*) mdlReal;
		delete (hipTextureObject_t*) mdlImag;

		hipFreeArray(*((hipArray_t*) texArrayReal));
		hipFreeArray(*((hipArray_t*) texArrayImag));
		delete (hipArray_t*) texArrayReal;
		delete (hipArray_t*) texArrayImag;
		texArrayReal = 0;
		texArrayImag = 0;
#endif
		mdlReal = 0;
		mdlImag = 0;
	}
}

