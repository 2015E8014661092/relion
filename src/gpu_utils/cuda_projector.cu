#include "src/gpu_utils/cuda_projector.h"
#include <signal.h>


void CudaProjector::setMdlDim(
		int xdim, int ydim, int zdim,
		int inity, int initz,
		int maxr, int paddingFactor)
{
	bool resizeTexure(true);

	if (xdim == mdlX &&
		ydim == mdlY &&
		zdim == mdlZ &&
		inity == mdlInitY &&
		initz == mdlInitZ &&
		maxr == mdlMaxR &&
		paddingFactor == padding_factor)
		resizeTexure = false;

	mdlX = xdim;
	mdlY = ydim;
	if(zdim==1)
		mdlZ=0;
	else
		mdlZ = zdim;
	mdlXYZ = xdim*ydim*zdim;
	mdlInitY = inity;
	mdlInitZ = initz;
	mdlMaxR = maxr;
	padding_factor = paddingFactor;

	if (! resizeTexure) return;

	clear();

#ifndef CUDA_DOUBLE_PRECISION

	mdlReal = new hipTextureObject_t();
	mdlImag = new hipTextureObject_t();

	// create channel to describe data type (bits,bits,bits,bits,type)
	hipChannelFormatDesc desc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

	struct hipResourceDesc resDesc_real, resDesc_imag;
	struct hipTextureDesc  texDesc;
	// -- Zero all data in objects handlers
	memset(&resDesc_real, 0, sizeof(hipResourceDesc));
	memset(&resDesc_imag, 0, sizeof(hipResourceDesc));
	memset(&texDesc, 0, sizeof(hipTextureDesc));

	if(mdlZ!=0)  // 3D model
	{
		texArrayReal = new hipArray_t();
		texArrayImag = new hipArray_t();

		// -- make extents for automatic pitch:ing (aligment) of allocated 3D arrays
		hipExtent volumeSize = make_hipExtent(mdlX, mdlY, mdlZ);

		// -- Allocate and copy data using very celver CUDA memcpy-functions
		HANDLE_ERROR(hipMalloc3DArray(texArrayReal, &desc, volumeSize));
		HANDLE_ERROR(hipMalloc3DArray(texArrayImag, &desc, volumeSize));

		// -- Descriptors of the channel(s) in the texture(s)
		resDesc_real.res.array.array = *texArrayReal;
		resDesc_imag.res.array.array = *texArrayImag;
		resDesc_real.resType = hipResourceTypeArray;
		resDesc_imag.resType = hipResourceTypeArray;
	}
	else // 2D model
	{
		HANDLE_ERROR(hipMallocPitch(&texArrayReal2D, &pitch2D, sizeof(float)*mdlX,mdlY));
		HANDLE_ERROR(hipMallocPitch(&texArrayImag2D, &pitch2D, sizeof(float)*mdlX,mdlY));

		// -- Descriptors of the channel(s) in the texture(s)
		resDesc_real.resType = hipResourceTypePitch2D;
		resDesc_real.res.pitch2D.devPtr = texArrayReal2D;
		resDesc_real.res.pitch2D.pitchInBytes =  pitch2D;
		resDesc_real.res.pitch2D.width = mdlX;
		resDesc_real.res.pitch2D.height = mdlY;
		resDesc_real.res.pitch2D.desc = desc;
		// -------------------------------------------------
		resDesc_imag.resType = hipResourceTypePitch2D;
		resDesc_imag.res.pitch2D.devPtr = texArrayImag2D;
		resDesc_imag.res.pitch2D.pitchInBytes =  pitch2D;
		resDesc_imag.res.pitch2D.width = mdlX;
		resDesc_imag.res.pitch2D.height = mdlY;
		resDesc_imag.res.pitch2D.desc = desc;
	}

	// -- Decriptors of the texture(s) and methods used for reading it(them) --
	texDesc.filterMode       = hipFilterModeLinear;
	texDesc.readMode         = hipReadModeElementType;
	texDesc.normalizedCoords = false;

	for(int n=0; n<3; n++)
		texDesc.addressMode[n]=hipAddressModeClamp;

	// -- Create texture object(s)
	HANDLE_ERROR(hipCreateTextureObject(mdlReal, &resDesc_real, &texDesc, NULL));
	HANDLE_ERROR(hipCreateTextureObject(mdlImag, &resDesc_imag, &texDesc, NULL));

#else

	DEBUG_HANDLE_ERROR(hipMalloc( (void**) &mdlReal, mdlXYZ * sizeof(double)));
	DEBUG_HANDLE_ERROR(hipMalloc( (void**) &mdlImag, mdlXYZ * sizeof(double)));

#endif
}

void CudaProjector::setMdlData(XFLOAT *real, XFLOAT *imag)
{
#ifdef CUDA_DEBUG
	if (mdlXYZ == 0)
	{
        printf("DEBUG_ERROR: Model dimensions must be set with setMdlDim before call to setMdlData.");
		raise(SIGSEGV);
	}
	if (mdlReal != 0)
	{
        printf("DEBUG_ERROR: Duplicated call to setMdlData.");
		raise(SIGSEGV);
	}
#endif

#ifndef CUDA_DOUBLE_PRECISION

	if(mdlZ!=0)  // 3D model
	{
		// -- make extents for automatic pitch:ing (aligment) of allocated 3D arrays
		hipExtent volumeSize = make_hipExtent(mdlX, mdlY, mdlZ);
		hipMemcpy3DParms copyParams = {0};
		copyParams.extent = volumeSize;
		copyParams.kind   = hipMemcpyHostToDevice;

		// -- Copy data
		copyParams.dstArray = *texArrayReal;
		copyParams.srcPtr   = make_hipPitchedPtr(real, mdlX * sizeof(float), mdlY, mdlZ);
		DEBUG_HANDLE_ERROR(hipMemcpy3D(&copyParams));
		copyParams.dstArray = *texArrayImag;
		copyParams.srcPtr   = make_hipPitchedPtr(imag, mdlX * sizeof(float), mdlY, mdlZ);
		DEBUG_HANDLE_ERROR(hipMemcpy3D(&copyParams));
	}
	else // 2D model
	{
		DEBUG_HANDLE_ERROR(hipMemcpy2D(texArrayReal2D, pitch2D, real, sizeof(float) * mdlX, sizeof(float) * mdlX, mdlY, hipMemcpyHostToDevice));
		DEBUG_HANDLE_ERROR(hipMemcpy2D(texArrayImag2D, pitch2D, imag, sizeof(float) * mdlX, sizeof(float) * mdlX, mdlY, hipMemcpyHostToDevice));
	}

#else

	DEBUG_HANDLE_ERROR(hipMemcpy( mdlReal, real, mdlXYZ * sizeof(XFLOAT), hipMemcpyHostToDevice));
	DEBUG_HANDLE_ERROR(hipMemcpy( mdlImag, imag, mdlXYZ * sizeof(XFLOAT), hipMemcpyHostToDevice));

#endif

}


void CudaProjector::setMdlData(Complex *data)
{
	XFLOAT *tmpReal = new XFLOAT[mdlXYZ];
	XFLOAT *tmpImag = new XFLOAT[mdlXYZ];

	for (unsigned long i = 0; i < mdlXYZ; i ++)
	{
		tmpReal[i] = (XFLOAT) data[i].real;
		tmpImag[i] = (XFLOAT) data[i].imag;
	}

	setMdlData(tmpReal, tmpImag);

	delete [] tmpReal;
	delete [] tmpImag;
}


void CudaProjector::clear()
{
	if (mdlReal != 0)
	{
#ifdef CUDA_DOUBLE_PRECISION
		hipFree(mdlReal);
		hipFree(mdlImag);
#else
		hipDestroyTextureObject(*mdlReal);
		hipDestroyTextureObject(*mdlImag);
		delete mdlReal;
		delete mdlImag;

		if(mdlZ!=0) //3D case
		{
			hipFreeArray(*texArrayReal);
			hipFreeArray(*texArrayImag);
			delete texArrayReal;
			delete texArrayImag;
		}
		else //2D case
		{
			hipFree(texArrayReal2D);
			hipFree(texArrayImag2D);
		}

		texArrayReal = 0;
		texArrayImag = 0;
#endif
		mdlReal = 0;
		mdlImag = 0;
	}
}

