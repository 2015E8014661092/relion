#include "hip/hip_runtime.h"
#include "src/gpu_utils/cuda_projector_plan.h"
#include "src/gpu_utils/cuda_utils_stl.cuh"
#include <hip/hip_runtime.h>

void CudaProjectorPlan::setup(
		HealpixSampling &sampling,
		std::vector<double> &directions_prior,
		std::vector<double> &psi_prior,
		std::vector<int> &pointer_dir_nonzeroprior,
		std::vector<int> &pointer_psi_nonzeroprior,
		MultidimArray<bool> *Mcoarse_significant,
		std::vector<double > &pdf_class,
		std::vector<MultidimArray<double> > &pdf_direction,
		unsigned long nr_dir,
		unsigned long nr_psi,
		unsigned long idir_min,
		unsigned long idir_max,
		unsigned long ipsi_min,
		unsigned long ipsi_max,
		unsigned long itrans_min,
		unsigned long itrans_max,
		unsigned long current_oversampling,
		unsigned long nr_oversampled_rot,
		unsigned iclass,
		bool coarse,
		bool inverseMatrix,
		bool do_skip_align,
		bool do_skip_rotate,
		int orientational_prior_mode)
{
	std::vector< double > rots, tilts, psis;
	std::vector< double > oversampled_rot, oversampled_tilt, oversampled_psi;

	rots.reserve(nr_dir * nr_psi * nr_oversampled_rot);
	tilts.reserve(nr_dir * nr_psi * nr_oversampled_rot);
	psis.reserve(nr_dir * nr_psi * nr_oversampled_rot);
	iorientclasses.reserve(nr_dir * nr_psi * nr_oversampled_rot);
	iover_rots.reserve(nr_dir * nr_psi * nr_oversampled_rot);

	for (long int idir = idir_min, iorient = 0; idir <= idir_max; idir++)
	{
		for (long int ipsi = ipsi_min, ipart = 0; ipsi <= ipsi_max; ipsi++, iorient++)
		{
			long int iorientclass = iclass * nr_dir * nr_psi + iorient;

			// Get prior for this direction and skip calculation if prior==0
			double pdf_orientation;
			if (do_skip_align || do_skip_rotate)
			{
				pdf_orientation = pdf_class[iclass];
			}
			else if (orientational_prior_mode == NOPRIOR)
			{
				pdf_orientation = DIRECT_MULTIDIM_ELEM(pdf_direction[iclass], idir);
			}
			else
			{
				pdf_orientation = directions_prior[idir] * psi_prior[ipsi];
			}

			// In the first pass, always proceed
			// In the second pass, check whether one of the translations for this orientation of any of the particles had a significant weight in the first pass
			// if so, proceed with projecting the reference in that direction

			bool do_proceed(false);

			if (coarse && pdf_orientation > 0.)
				do_proceed = true;
			else if (pdf_orientation > 0.)
			{
				long int nr_trans = itrans_max - itrans_min + 1;
				for (long int ipart = 0; ipart < YSIZE(*Mcoarse_significant); ipart++)
				{
					long int ihidden = iorient * nr_trans;
					for (long int itrans = itrans_min; itrans <= itrans_max; itrans++, ihidden++)
					{
						if (DIRECT_A2D_ELEM(*Mcoarse_significant, ipart, ihidden))
						{
							do_proceed = true;
							break;
						}
					}
				}
			}

			if (do_proceed)
			{
				// Now get the oversampled (rot, tilt, psi) triplets
				// This will be only the original (rot,tilt,psi) triplet in the first pass (sp.current_oversampling==0)
				sampling.getOrientations(idir, ipsi, current_oversampling, oversampled_rot, oversampled_tilt, oversampled_psi,
						pointer_dir_nonzeroprior, directions_prior, pointer_psi_nonzeroprior, psi_prior);

				// Loop over all oversampled orientations (only a single one in the first pass)
				for (long int iover_rot = 0; iover_rot < nr_oversampled_rot; iover_rot++, ipart++)
				{
					rots.push_back(oversampled_rot[iover_rot]);
					tilts.push_back(oversampled_tilt[iover_rot]);
					psis.push_back(oversampled_psi[iover_rot]);
					iorientclasses.push_back(iorientclass);
					iover_rots.push_back(iover_rot);

					orientation_num ++;
				}
			}
		}
	}


	double alpha(.0), beta(.0), gamma(.0);
	double ca(.0), sa(.0), cb(.0), sb(.0), cg(.0), sg(.0);
	double cc(.0), cs(.0), sc(.0), ss(.0);

	if (eulers == NULL)
	{
		eulers = new CudaGlobalPtr<XFLOAT,false>(9*orientation_num);
		eulers->device_alloc();
		free_device = true;
	}

	for (long int i = 0; i < rots.size(); i++)
	{
		alpha = DEG2RAD(rots[i]);
		beta  = DEG2RAD(tilts[i]);
		gamma = DEG2RAD(psis[i]);

		sincos(alpha, &sa, &ca);
		sincos(beta,  &sb, &cb);
		sincos(gamma, &sg, &cg);

		cc = cb * ca;
		cs = cb * sa;
		sc = sb * ca;
		ss = sb * sa;

		if(inverseMatrix)
		{
			(*eulers)[9 * i + 0] = ( cg * cc - sg * sa) ;// * padding_factor; //00
			(*eulers)[9 * i + 1] = (-sg * cc - cg * sa) ;// * padding_factor; //10
			(*eulers)[9 * i + 2] = ( sc )               ;// * padding_factor; //20
			(*eulers)[9 * i + 3] = ( cg * cs + sg * ca) ;// * padding_factor; //01
			(*eulers)[9 * i + 4] = (-sg * cs + cg * ca) ;// * padding_factor; //11
			(*eulers)[9 * i + 5] = ( ss )               ;// * padding_factor; //21
			(*eulers)[9 * i + 6] = (-cg * sb )          ;// * padding_factor; //02
			(*eulers)[9 * i + 7] = ( sg * sb )          ;// * padding_factor; //12
			(*eulers)[9 * i + 8] = ( cb )               ;// * padding_factor; //22
		}
		else
		{
			(*eulers)[9 * i + 0] = ( cg * cc - sg * sa) ;// * padding_factor; //00
			(*eulers)[9 * i + 1] = ( cg * cs + sg * ca) ;// * padding_factor; //01
			(*eulers)[9 * i + 2] = (-cg * sb )          ;// * padding_factor; //02
			(*eulers)[9 * i + 3] = (-sg * cc - cg * sa) ;// * padding_factor; //10
			(*eulers)[9 * i + 4] = (-sg * cs + cg * ca) ;// * padding_factor; //11
			(*eulers)[9 * i + 5] = ( sg * sb )          ;// * padding_factor; //12
			(*eulers)[9 * i + 6] = ( sc )               ;// * padding_factor; //20
			(*eulers)[9 * i + 7] = ( ss )               ;// * padding_factor; //21
			(*eulers)[9 * i + 8] = ( cb )               ;// * padding_factor; //22
		}
	}

	eulers->cp_to_device();
}

void CudaProjectorPlan::printTo(std::ostream &os) // print
{
	os << "orientation_num = " << orientation_num << std::endl;
	os << "free_device = " << free_device << std::endl;
	os << "iorientclasses.size = " << iorientclasses.size() << std::endl;
	os << "iover_rots.size = " << iover_rots.size() << std::endl;
	os << std::endl << "iorientclasses\tiover_rots\teulers" << std::endl;

	for (int i = 0; i < iorientclasses.size(); i ++)
	{
		os << iorientclasses[i] << "\t\t" << iover_rots[i] << "\t";
		for (int j = 0; j < 9; j++)
			os << (*eulers)[i * 9 + j] << "\t";
		os << std::endl;
	}
}

CudaProjectorPlan::~CudaProjectorPlan()
{
	if(free_device)
		delete eulers;
}
