#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <ctime>
#include <iostream>
#include "src/gpu_utils/cuda_mem_utils.h"
#include "src/complex.h"
#include <fstream>
#include <hip/hip_runtime.h>
#include <signal.h>

#include "src/image.h"
#include "src/autopicker.h"
#include "src/gpu_utils/cuda_autopicker.h"

#ifdef CUDA_FORCESTL
#include "src/gpu_utils/cuda_utils_stl.cuh"
#else
#include "src/gpu_utils/cuda_utils_cub.cuh"
#endif

static pthread_mutex_t global_mutex = PTHREAD_MUTEX_INITIALIZER;

AutoPickerCuda::AutoPickerCuda(AutoPicker *basePicker, int dev_id) :
	basePckr(basePicker)
{

	/*======================================================
	                    DEVICE SETTINGS
	======================================================*/
	device_id = dev_id;
	int devCount;
	HANDLE_ERROR(hipGetDeviceCount(&devCount));

	if(dev_id >= devCount)
	{
		std::cerr << " using device_id=" << dev_id << " (device no. " << dev_id+1 << ") which is higher than the available number of devices=" << devCount << std::endl;
		REPORT_ERROR("ERROR: Assigning a thread to a non-existent device (index likely too high)");
	}
	else
		HANDLE_ERROR(hipSetDevice(dev_id));


	/*======================================================
	                    CUSTOM ALLOCATOR
	======================================================*/

#ifdef CUDA_NO_CUSTOM_ALLOCATION
	printf(" DEBUG: Custom allocator is disabled.\n");
	allocator = new CudaCustomAllocator(0, 1);
#else
	size_t allocationSize(0);

	size_t free, total;
	HANDLE_ERROR(hipMemGetInfo( &free, &total ));

	if (basePckr->available_gpu_memory > 0)
		allocationSize = basePckr->available_gpu_memory * (1000*1000*1000);
	else
		allocationSize = (float)free * .7;

	if (allocationSize > free)
	{
		printf(" WARNING: Required memory per thread, via \"--gpu_memory_per_thread\", not available on device. (Defaulting to less)\n");
		allocationSize = (float)free * .7; //Lets leave some for other processes for now
	}

	int memAlignmentSize;
	hipDeviceGetAttribute ( &memAlignmentSize, hipDeviceAttributeTextureAlignment, dev_id );

	allocator = new CudaCustomAllocator(allocationSize, memAlignmentSize);
#endif
};


void AutoPickerCuda::run()
{

	int barstep;
	if (basePckr->verb > 0)
	{
		std::cout << " Autopicking ..." << std::endl;
		init_progress_bar(basePckr->fn_micrographs.size());
		barstep = XMIPP_MAX(1,basePckr->fn_micrographs.size() / 60);
	}


	for (long int imic = 0; imic < basePckr->fn_micrographs.size(); imic++)
	{
		if (basePckr->verb > 0 && imic % barstep == 0)
			progress_bar(imic);

		autoPickOneMicrograph(basePckr->fn_micrographs[imic]);
	}

	if (basePckr->verb > 0)
		progress_bar(basePckr->fn_micrographs.size());

}


void AutoPickerCuda::autoPickOneMicrograph(FileName &fn_mic)
{
	Image<RFLOAT> Imic;
	MultidimArray<Complex > Faux, Faux2, Fmic;
	MultidimArray<RFLOAT> Maux, Mstddev, Mmean, Mdiff2, MsumX2, Mccf_best, Mpsi_best, Fctf;
	FourierTransformer transformer;
	RFLOAT sum_ref_under_circ_mask, sum_ref2_under_circ_mask;
	int my_skip_side = basePckr->autopick_skip_side + basePckr->particle_size/2;
	CTF ctf;

	int min_distance_pix = ROUND(basePckr->min_particle_distance / basePckr->angpix);

#ifdef DEBUG
	Image<RFLOAT> tt;
	tt().resize(micrograph_size, micrograph_size);
	std::cerr << " fn_mic= " << fn_mic << std::endl;
#endif
	// Read in the micrograph
	Imic.read(fn_mic);
	Imic().setXmippOrigin();

	// Let's just check the square size again....
	RFLOAT my_size, my_xsize, my_ysize;
	my_xsize = XSIZE(Imic());
	my_ysize = YSIZE(Imic());
	my_size = (my_xsize != my_ysize) ? XMIPP_MAX(my_xsize, my_ysize) : my_xsize;

	if (my_size != basePckr->micrograph_size || my_xsize != basePckr->micrograph_xsize || my_ysize != basePckr->micrograph_ysize)
	{
		Imic().printShape();
		std::cerr << " micrograph_size= " << basePckr->micrograph_size << " micrograph_xsize= " << basePckr->micrograph_xsize << " micrograph_ysize= " << basePckr->micrograph_ysize << std::endl;
		REPORT_ERROR("AutoPicker::autoPickOneMicrograph ERROR: No differently sized micrographs are allowed in one run, sorry you will have to run separately for each size...");
	}

	// Set mean to zero and stddev to 1 to prevent numerical problems with one-sweep stddev calculations....
    RFLOAT avg0, stddev0, minval0, maxval0;
	Imic().computeStats(avg0, stddev0, minval0, maxval0);
	FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(Imic())
	{
		// Remove pixel values that are too far away from the mean
		if ( ABS(DIRECT_MULTIDIM_ELEM(Imic(), n) - avg0) / stddev0 > basePckr->outlier_removal_zscore)
			DIRECT_MULTIDIM_ELEM(Imic(), n) = avg0;

		DIRECT_MULTIDIM_ELEM(Imic(), n) = (DIRECT_MULTIDIM_ELEM(Imic(), n) - avg0) / stddev0;
	}

	if (basePckr->micrograph_xsize !=basePckr->micrograph_ysize)
	{
		// Window non-square micrographs to be a square with the largest side
		rewindow(Imic, basePckr->micrograph_size);

		// Fill region outside the original window with white Gaussian noise to prevent all-zeros in Mstddev
		FOR_ALL_ELEMENTS_IN_ARRAY2D(Imic())
		{
			if (i < FIRST_XMIPP_INDEX(basePckr->micrograph_ysize)
					|| i > LAST_XMIPP_INDEX(basePckr->micrograph_ysize)
					|| j < FIRST_XMIPP_INDEX(basePckr->micrograph_xsize)
					|| j > LAST_XMIPP_INDEX(basePckr->micrograph_xsize) )
				A2D_ELEM(Imic(), i, j) = rnd_gaus(0.,1.);
		}
	}

	// Read in the CTF information if needed
	if (basePckr->do_ctf)
	{
		// Search for this micrograph in the metadata table
		FOR_ALL_OBJECTS_IN_METADATA_TABLE(basePckr->MDmic)
		{
			FileName fn_tmp;
			basePckr->MDmic.getValue(EMDL_MICROGRAPH_NAME, fn_tmp);
			if (fn_tmp==fn_mic)
			{
				ctf.read(basePckr->MDmic, basePckr->MDmic);
				Fctf.resize(basePckr->downsize_mic, basePckr->downsize_mic/2 + 1);
				ctf.getFftwImage(Fctf, basePckr->micrograph_size, basePckr->micrograph_size, basePckr->angpix, false, false, basePckr->intact_ctf_first_peak, true);
				break;
			}
		}
#ifdef DEBUG
		std::cerr << " Read CTF info from" << fn_mic.withoutExtension()<<"_ctf.star" << std::endl;
		Image<RFLOAT> Ictf;
		Ictf()=Fctf;
		Ictf.write("Mmic_ctf.spi");
#endif
	}

	Mccf_best.resize(basePckr->micrograph_size, basePckr->micrograph_size);
	Mpsi_best.resize(basePckr->micrograph_size, basePckr->micrograph_size);

	RFLOAT normfft = (RFLOAT)(basePckr->micrograph_size * basePckr->micrograph_size) / (RFLOAT)basePckr->nr_pixels_circular_mask;;
	if (basePckr->do_read_fom_maps)
	{
		FileName fn_tmp=fn_mic.withoutExtension()+"_"+basePckr->fn_out+"_stddevNoise.spi";
		Image<RFLOAT> It;
		It.read(fn_tmp);
		Mstddev = It();
	}
	else
	{
		/*
		 * Squared difference FOM:
		 * Sum ( (X-mu)/sig  - A )^2 =
		 *  = Sum((X-mu)/sig)^2 - 2 Sum (A*(X-mu)/sig) + Sum(A)^2
		 *  = (1/sig^2)*Sum(X^2) - (2*mu/sig^2)*Sum(X) + (mu^2/sig^2)*Sum(1) - (2/sig)*Sum(AX) + (2*mu/sig)*Sum(A) + Sum(A^2)
		 *
		 * However, the squared difference with an "empty" ie all-zero reference is:
		 * Sum ( (X-mu)/sig)^2
		 *
		 * The ratio of the probabilities thereby becomes:
		 * P(ref) = 1/sqrt(2pi) * exp (( (X-mu)/sig  - A )^2 / -2 )   // assuming sigma = 1!
		 * P(zero) = 1/sqrt(2pi) * exp (( (X-mu)/sig )^2 / -2 )
		 *
		 * P(ref)/P(zero) = exp(( (X-mu)/sig  - A )^2 / -2) / exp ( ( (X-mu)/sig )^2 / -2)
		 *                = exp( (- (2/sig)*Sum(AX) + (2*mu/sig)*Sum(A) + Sum(A^2)) / - 2 )
		 *
		 *                Therefore, I do not need to calculate (X-mu)/sig beforehand!!!
		 *
		 */

		// Fourier Transform (and downscale) Imic()
		CenterFFT(Imic(), true);
		transformer.FourierTransform(Imic(), Fmic);

		// Also calculate the FFT of the squared micrograph
		Maux.resize(Imic());
		FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(Maux)
		{
			DIRECT_MULTIDIM_ELEM(Maux, n) = DIRECT_MULTIDIM_ELEM(Imic(), n) * DIRECT_MULTIDIM_ELEM(Imic(), n);
		}
		MultidimArray<Complex > Fmic2;
		transformer.FourierTransform(Maux, Fmic2);

#ifdef DEBUG
		std::cerr << " nr_pixels_circular_invmask= " << nr_pixels_circular_invmask << std::endl;
		std::cerr << " nr_pixels_circular_mask= " << nr_pixels_circular_mask << std::endl;
		windowFourierTransform(Finvmsk, Faux2, micrograph_size);
		transformer.inverseFourierTransform(Faux2, tt());
		CenterFFT(tt(), false);
		tt.write("Minvmask.spi");
		windowFourierTransform(Fmsk, Faux2, micrograph_size);
		transformer.inverseFourierTransform(Faux2, tt());
		CenterFFT(tt(), false);
		tt.write("Mmask.spi");
#endif

		// The following calculate mu and sig under the solvent area at every position in the micrograph
		basePckr->calculateStddevAndMeanUnderMask(Fmic, Fmic2, basePckr->Finvmsk,basePckr->nr_pixels_circular_invmask, Mstddev, Mmean);

		if (basePckr->do_write_fom_maps)
		{
			// TMP output
			FileName fn_tmp=fn_mic.withoutExtension()+"_"+basePckr->fn_out+"_stddevNoise.spi";
			Image<RFLOAT> It;
			It() = Mstddev;
			It.write(fn_tmp);
		}

		// From now on use downsized Fmic, as the cross-correlation with the references can be done at lower resolution
		windowFourierTransform(Fmic, Faux, basePckr->downsize_mic);
		Fmic = Faux;

	}// end if do_read_fom_maps

	// Now start looking for the peaks of all references
	// Clear the output vector with all peaks
	std::vector<Peak> peaks;
	peaks.clear();
	for (int iref = 0; iref < basePckr->Mrefs.size(); iref++)
	{
		RFLOAT expected_Pratio; // the expectedFOM for this (ctf-corrected) reference
		if (basePckr->do_read_fom_maps)
		{
			FileName fn_tmp;
			Image<RFLOAT> It;
			fn_tmp.compose(fn_mic.withoutExtension()+"_"+basePckr->fn_out+"_ref", iref,"_bestCCF.spi");
			It.read(fn_tmp);
			Mccf_best = It();
			// Retrieve expected_Pratio from the header of the image..
			It.MDMainHeader.getValue(EMDL_IMAGE_STATS_MAX, expected_Pratio);
			fn_tmp.compose(fn_mic.withoutExtension()+"_"+basePckr->fn_out+"_ref", iref,"_bestPSI.spi");
			It.read(fn_tmp);
			Mpsi_best = It();

		} //end else if do_read_fom_maps
		else
		{
			Mccf_best.initConstant(-LARGE_NUMBER);
			bool is_first_psi = true;
			for (RFLOAT psi = 0. ; psi < 360.; psi+=basePckr->psi_sampling)
			{

				// Get the Euler matrix
				Matrix2D<RFLOAT> A(3,3);
				Euler_angles2matrix(0., 0., psi, A);

				// Now get the FT of the rotated (non-ctf-corrected) template
				Faux.initZeros(basePckr->downsize_mic, basePckr->downsize_mic/2 + 1);
				basePckr->PPref[iref].get2DFourierTransform(Faux, A, IS_NOT_INV);

#ifdef DEBUG
				std::cerr << " psi= " << psi << std::endl;
				windowFourierTransform(Faux, Faux2, micrograph_size);
				transformer.inverseFourierTransform(Faux2, tt());
				CenterFFT(tt(), false);
				tt.write("Mref_rot.spi");

				windowFourierTransform(Fmic, Faux2, micrograph_size);
				transformer.inverseFourierTransform(Faux2, tt());
				CenterFFT(tt(), false);
				tt.write("Mmic.spi");

#endif

				// Apply the CTF on-the-fly (so same PPref can be used for many different micrographs)
				if (basePckr->do_ctf)
				{
					FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(Faux)
					{
						DIRECT_MULTIDIM_ELEM(Faux, n) *= DIRECT_MULTIDIM_ELEM(Fctf, n);
					}
#ifdef DEBUG
				windowFourierTransform(Faux, Faux2, micrograph_size);
				transformer.inverseFourierTransform(Faux2, Maux);
				CenterFFT(Maux, false);
				Maux.setXmippOrigin();
				tt().resize(particle_size, particle_size);
				tt().setXmippOrigin();
				FOR_ALL_ELEMENTS_IN_ARRAY2D(tt())
				{
					A2D_ELEM(tt(), i, j) = A2D_ELEM(Maux, i, j);
				}
				tt.write("Mref_rot_ctf.spi");
#endif
				}

				if (is_first_psi)
				{
					// Calculate the expected ratio of probabilities for this CTF-corrected reference
					// and the sum_ref_under_circ_mask and sum_ref_under_circ_mask2
					// Do this also if we're not recalculating the fom maps...

					windowFourierTransform(Faux, Faux2, basePckr->micrograph_size);
					transformer.inverseFourierTransform(Faux2, Maux);
					CenterFFT(Maux, false);
					Maux.setXmippOrigin();
					// TODO: check whether I need CenterFFT(Maux, false)

					sum_ref_under_circ_mask = 0.;
					sum_ref2_under_circ_mask = 0.;
					RFLOAT suma2 = 0.;
					RFLOAT sumn = 1.;
					MultidimArray<RFLOAT> Mctfref(basePckr->particle_size, basePckr->particle_size);
					Mctfref.setXmippOrigin();
					FOR_ALL_ELEMENTS_IN_ARRAY2D(Mctfref) // only loop over smaller Mctfref, but take values from large Maux!
					{
						if (i*i + j*j < basePckr->particle_radius2)
						{
							suma2 += A2D_ELEM(Maux, i, j) * A2D_ELEM(Maux, i, j);
							suma2 += 2. * A2D_ELEM(Maux, i, j) * rnd_gaus(0., 1.);
							sum_ref_under_circ_mask += A2D_ELEM(Maux, i, j);
							sum_ref2_under_circ_mask += A2D_ELEM(Maux, i, j) * A2D_ELEM(Maux, i, j);
							sumn += 1.;
						}
#ifdef DEBUG
						A2D_ELEM(Mctfref, i, j) = A2D_ELEM(Maux, i, j);
#endif
					}
					sum_ref_under_circ_mask /= sumn;
					sum_ref2_under_circ_mask /= sumn;
					expected_Pratio = exp(suma2 / (2. * sumn));
#ifdef DEBUG
					std::cerr << " expected_Pratio["<<iref<<"]= " << expected_Pratio << std::endl;
					tt()=Mctfref;
					tt.write("Mctfref.spi");
#endif
				}

				// Now multiply template and micrograph to calculate the cross-correlation
				FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(Faux)
				{
					DIRECT_MULTIDIM_ELEM(Faux, n) = conj(DIRECT_MULTIDIM_ELEM(Faux, n)) * DIRECT_MULTIDIM_ELEM(Fmic, n);
				}
				windowFourierTransform(Faux, Faux2, basePckr->micrograph_size);
				transformer.inverseFourierTransform(Faux2, Maux);
				CenterFFT(Maux, false);
#ifdef DEBUG
				tt()=Maux*normfft;
				tt.write("Mcc.spi");
#endif

				// Calculate ratio of prabilities P(ref)/P(zero)
				// Keep track of the best values and their corresponding iref and psi

				// So now we already had precalculated: Mdiff2 = 1/sig*Sum(X^2) - 2/sig*Sum(X) + mu^2/sig*Sum(1)
				// Still to do (per reference): - 2/sig*Sum(AX) + 2*mu/sig*Sum(A) + Sum(A^2)
				FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(Maux)
				{
					RFLOAT diff2 = - 2. * normfft * DIRECT_MULTIDIM_ELEM(Maux, n);
					diff2 += 2. * DIRECT_MULTIDIM_ELEM(Mmean, n) * sum_ref_under_circ_mask;
					if (DIRECT_MULTIDIM_ELEM(Mstddev, n) > 1E-10)
						diff2 /= DIRECT_MULTIDIM_ELEM(Mstddev, n);
					diff2 += sum_ref2_under_circ_mask;
#ifdef DEBUG
						/*
						if (diff2 < 0. || n==28800 || n==0)
						{
							std::cerr << " n= "<<n<< "diff2= " << diff2 << " old Mdiff2=" <<DIRECT_MULTIDIM_ELEM(Mdiff2, n)
									<< " -2AX/sig " << - 2. * normfft * DIRECT_MULTIDIM_ELEM(Maux, n) / DIRECT_MULTIDIM_ELEM(Mstddev, n)
									<< " 2Amu/sig= " << 2. * DIRECT_MULTIDIM_ELEM(Mmean, n) * sum_ref_under_circ_mask[iref] / DIRECT_MULTIDIM_ELEM(Mstddev, n)
									<< " A2=" <<  sum_ref2_under_circ_mask[iref]
									<< " stddev= " <<  DIRECT_MULTIDIM_ELEM(Mstddev, n) << " avg= "<< DIRECT_MULTIDIM_ELEM(Mmean, n)
									<< std::endl;
						}
						*/
#endif
					diff2 = exp(- diff2 / 2.); // exponentiate to reflect the Gaussian error model. sigma=1 after normalization, 0.4=1/sqrt(2pi)

					// Store fraction of (1 - probability-ratio) wrt  (1 - expected Pratio)
					diff2 = (diff2 - 1.) / (expected_Pratio - 1.);
#ifdef DEBUG
					DIRECT_MULTIDIM_ELEM(Maux, n) = diff2;
#endif
					if (diff2 > DIRECT_MULTIDIM_ELEM(Mccf_best, n))
					{
						DIRECT_MULTIDIM_ELEM(Mccf_best, n) = diff2;
						DIRECT_MULTIDIM_ELEM(Mpsi_best, n) = psi;
					}
				}
#ifdef DEBUG
				std::cerr << " Maux.computeMax()= " << Maux.computeMax() << std::endl;
				tt()=Maux;
				tt.write("Mccf.spi");
			    std::cerr << " Press any key to continue... "  << std::endl;
			    char c;
			    std::cin >> c;

#endif
			    is_first_psi = false;
			} // end for psi


			if (basePckr->do_write_fom_maps)
			{
				// TMP output
				FileName fn_tmp;
				Image<RFLOAT> It;
				It() = Mccf_best;
				// Store expected_Pratio in the header of the image..
				It.MDMainHeader.setValue(EMDL_IMAGE_STATS_MAX, expected_Pratio);;
				fn_tmp.compose(fn_mic.withoutExtension()+"_"+basePckr->fn_out+"_ref", iref,"_bestCCF.spi");
				It.write(fn_tmp);

				It() = Mpsi_best;
				fn_tmp.compose(fn_mic.withoutExtension()+"_"+basePckr->fn_out+"_ref", iref,"_bestPSI.spi");
				It.write(fn_tmp);
			} // end if do_write_fom_maps

		} // end if do_read_fom_maps

		// Now that we have Mccf_best and Mpsi_best, get the peaks
		std::vector<Peak> my_ref_peaks;
		Mstddev.setXmippOrigin();
		Mccf_best.setXmippOrigin();
		Mpsi_best.setXmippOrigin();
		basePckr->peakSearch(Mccf_best, Mpsi_best, Mstddev, iref, my_skip_side, my_ref_peaks);

		basePckr->prunePeakClusters(my_ref_peaks, min_distance_pix);

		// append the peaks of this reference to all the other peaks
		peaks.insert(peaks.end(), my_ref_peaks.begin(), my_ref_peaks.end());

	} // end for iref


	//Now that we have done all references, prune the list again...
	basePckr->prunePeakClusters(peaks, min_distance_pix);

	// And remove all too close neighbours
	basePckr->removeTooCloselyNeighbouringPeaks(peaks, min_distance_pix);

	// Write out a STAR file with the coordinates
	MetaDataTable MDout;
	for (int ipeak =0; ipeak < peaks.size(); ipeak++)
	{
		MDout.addObject();
		MDout.setValue(EMDL_IMAGE_COORD_X, (RFLOAT)(peaks[ipeak].x));
		MDout.setValue(EMDL_IMAGE_COORD_Y, (RFLOAT)(peaks[ipeak].y));
		MDout.setValue(EMDL_ORIENT_PSI, peaks[ipeak].psi);
		MDout.setValue(EMDL_PARTICLE_CLASS, peaks[ipeak].ref + 1); // start counting at 1
		MDout.setValue(EMDL_PARTICLE_AUTOPICK_FOM, peaks[ipeak].fom);
	}
	FileName fn_tmp = fn_mic.withoutExtension() + "_" + basePckr->fn_out + ".star";
	MDout.write(fn_tmp);

}
