#include "hip/hip_runtime.h"
#include "src/gpu_utils/cuda_backprojector.h"
#include "src/gpu_utils/cuda_device_utils.cuh"
#include <signal.h>

#define BACKPROJECTION4_BLOCK_SIZE 64
#define BACKPROJECTION4_GROUP_SIZE 16
#define BACKPROJECTION4_PREFETCH_COUNT 3
#define BP_2D_BLOCK_SIZE 128

void CudaBackprojector::setMdlDim(
			int xdim, int ydim, int zdim,
			int inity, int initz,
			int max_r, int paddingFactor)
{
	if (xdim != mdlX ||
		ydim != mdlY ||
		zdim != mdlZ ||
		inity != mdlInitY ||
		initz != mdlInitZ ||
		max_r != maxR ||
		paddingFactor != padding_factor)
	{
		mdlX = xdim;
		mdlY = ydim;
		mdlZ = zdim;
		if (mdlZ < 1) mdlZ = 1;
		mdlXYZ = xdim*ydim*zdim;
		mdlInitY = inity;
		mdlInitZ = initz;
		maxR = max_r;
		maxR2 = max_r*max_r;
		padding_factor = paddingFactor;

		clear();

		//Allocate space for model
		HANDLE_ERROR(hipMalloc( (void**) &d_mdlReal,   mdlXYZ * sizeof(XFLOAT)));
		HANDLE_ERROR(hipMalloc( (void**) &d_mdlImag,   mdlXYZ * sizeof(XFLOAT)));
		HANDLE_ERROR(hipMalloc( (void**) &d_mdlWeight, mdlXYZ * sizeof(XFLOAT)));
	}
}

void CudaBackprojector::initMdl()
{
#ifdef CUDA_DEBUG
	if (mdlXYZ == 0)
	{
        printf("Model dimensions must be set with setMdlDim before call to setupMdl.");
		raise(SIGSEGV);
	}
	if (voxelCount != 0)
	{
        printf("DEBUG_ERROR: Duplicated call to model setup");
		raise(SIGSEGV);
	}
#endif

	//Initiate model with zeros
	DEBUG_HANDLE_ERROR(hipMemset( d_mdlReal,   0, mdlXYZ * sizeof(XFLOAT)));
	DEBUG_HANDLE_ERROR(hipMemset( d_mdlImag,   0, mdlXYZ * sizeof(XFLOAT)));
	DEBUG_HANDLE_ERROR(hipMemset( d_mdlWeight, 0, mdlXYZ * sizeof(XFLOAT)));
}

__global__ void cuda_kernel_backproject2D(
		XFLOAT *g_model_real,
		XFLOAT *g_model_imag,
		XFLOAT *g_model_weight,
		XFLOAT *g_wavgs_real,
		XFLOAT *g_wavgs_imag,
		XFLOAT *g_Fweights,
		XFLOAT *g_eulers,
		int max_r,
		int max_r2,
		XFLOAT padding_factor,
		unsigned img_x,
		unsigned img_y,
		unsigned img_xy,
		unsigned mdl_x,
		int mdl_inity)
{
	unsigned tid = threadIdx.x;
	unsigned img = blockIdx.x;

	__shared__ XFLOAT s_eulers[4];

	if (tid == 0)
		s_eulers[0] = g_eulers[img*9+0] * padding_factor;
	else if (tid == 1)
		s_eulers[1] = g_eulers[img*9+1] * padding_factor;
	else if (tid == 2)
		s_eulers[2] = g_eulers[img*9+3] * padding_factor;
	else if (tid == 3)
		s_eulers[3] = g_eulers[img*9+4] * padding_factor;

	__syncthreads();

	int pixel_pass_num(ceilf((float)img_xy/(float)BP_2D_BLOCK_SIZE));

	for (unsigned pass = 0; pass < pixel_pass_num; pass++)
    {
		unsigned pixel = (pass * BP_2D_BLOCK_SIZE) + tid;

		if (pixel >= img_xy)
			continue;

		int x = pixel % img_x;
		int y = (int)floorf( (float)pixel / (float)img_x);

		pixel += img * img_xy;

		// Don't search beyond square with side max_r
		if (y > max_r)
		{
			if (y >= img_y - max_r)
				y -= img_y;
			else
				continue;
		}

		if (x * x + y * y > max_r2)
			continue;

		// Get the weight
		XFLOAT weight = g_Fweights[pixel];

		if (weight > 0.f)
		{
			// Get the relevant value in the input image
			XFLOAT real = g_wavgs_real[pixel];
			XFLOAT imag = g_wavgs_imag[pixel];

			// Get logical coordinates in the 3D map
			XFLOAT xp = (s_eulers[0] * x + s_eulers[1] * y );
			XFLOAT yp = (s_eulers[2] * x + s_eulers[3] * y );

			// Only asymmetric half is stored
			if (xp < 0)
			{
				// Get complex conjugated hermitian symmetry pair
				xp = -xp;
				yp = -yp;
				imag = -imag;
			}

			int x0 = floorf(xp);
			XFLOAT fx = xp - x0;
			int x1 = x0 + 1;

			int y0 = floorf(yp);
			XFLOAT fy = yp - y0;
			y0 -= mdl_inity;
			int y1 = y0 + 1;

			XFLOAT mfx = 1.f - fx;
			XFLOAT mfy = 1.f - fy;

			XFLOAT dd00 = mfy * mfx;
			XFLOAT dd01 = mfy *  fx;
			XFLOAT dd10 =  fy * mfx;
			XFLOAT dd11 =  fy *  fx;

			cuda_atomic_add(&g_model_real  [y0 * mdl_x + x0], dd00 * real);
			cuda_atomic_add(&g_model_imag  [y0 * mdl_x + x0], dd00 * imag);
			cuda_atomic_add(&g_model_weight[y0 * mdl_x + x0], dd00 * weight);

			cuda_atomic_add(&g_model_real  [y0 * mdl_x + x1], dd01 * real);
			cuda_atomic_add(&g_model_imag  [y0 * mdl_x + x1], dd01 * imag);
			cuda_atomic_add(&g_model_weight[y0 * mdl_x + x1], dd01 * weight);

			cuda_atomic_add(&g_model_real  [y1 * mdl_x + x0], dd10 * real);
			cuda_atomic_add(&g_model_imag  [y1 * mdl_x + x0], dd10 * imag);
			cuda_atomic_add(&g_model_weight[y1 * mdl_x + x0], dd10 * weight);

			cuda_atomic_add(&g_model_real  [y1 * mdl_x + x1], dd11 * real);
			cuda_atomic_add(&g_model_imag  [y1 * mdl_x + x1], dd11 * imag);
			cuda_atomic_add(&g_model_weight[y1 * mdl_x + x1], dd11 * weight);
		}
	}
}

__global__ void cuda_kernel_backproject3D_scatter(
		XFLOAT *g_model_real,
		XFLOAT *g_model_imag,
		XFLOAT *g_model_weight,
		XFLOAT *g_wavgs_real,
		XFLOAT *g_wavgs_imag,
		XFLOAT *g_Fweights,
		XFLOAT *g_eulers,
		int max_r,
		int max_r2,
		XFLOAT padding_factor,
		unsigned img_x,
		unsigned img_y,
		unsigned img_xy,
		unsigned mdl_x,
		unsigned mdl_y,
		int mdl_inity,
		int mdl_initz)
{
	unsigned tid = threadIdx.x;
	unsigned img = blockIdx.x;

	__shared__ XFLOAT s_eulers[9];

	if (tid < 9)
		s_eulers[tid] = g_eulers[img*9+tid];

	__syncthreads();

	int pixel_pass_num(ceilf((float)img_xy/(float)BP_2D_BLOCK_SIZE));
	for (unsigned pass = 0; pass < pixel_pass_num; pass++)
    {
		unsigned pixel = (pass * BP_2D_BLOCK_SIZE) + tid;

		if (pixel >= img_xy)
			continue;

		int x = pixel % img_x;
		int y = (int)floorf( (float)pixel / (float)img_x);

		pixel += img * img_xy;

		// Don't search beyond square with side max_r
		if (y > max_r)
		{
			if (y >= img_y - max_r)
				y -= img_y;
		}

		if (x * x + y * y > max_r2)
			continue;

		// Get the weight
		XFLOAT Fweights = g_Fweights[pixel];

		if (Fweights > 0.f)
		{
			// Get the relevant value in the input image
			XFLOAT real = g_wavgs_real[pixel];
			XFLOAT imag = g_wavgs_imag[pixel];

			// Get logical coordinates in the 3D map
			XFLOAT xp = (s_eulers[0] * x + s_eulers[1] * y ) * padding_factor;
			XFLOAT yp = (s_eulers[3] * x + s_eulers[4] * y ) * padding_factor;
			XFLOAT zp = (s_eulers[6] * x + s_eulers[7] * y ) * padding_factor;

			// Only asymmetric half is stored
			if (xp < 0.f)
			{
				// Get complex conjugated hermitian symmetry pair
				xp = -xp;
				yp = -yp;
				zp = -zp;
				imag = -imag;
			}

			int x0 = floorf(xp);
			XFLOAT fx = xp - x0;
			int x1 = x0 + 1;

			int y0 = floorf(yp);
			XFLOAT fy = yp - y0;
			y0 -= mdl_inity;
			int y1 = y0 + 1;

			int z0 = floorf(zp);
			XFLOAT fz = zp - z0;
			z0 -= mdl_initz;
			int z1 = z0 + 1;

			XFLOAT mfx = 1.f - fx;
			XFLOAT mfy = 1.f - fy;
			XFLOAT mfz = 1.f - fz;

			XFLOAT dd000 = mfz * mfy * mfx;

			cuda_atomic_add(&g_model_real  [z0 * mdl_x * mdl_y + y0 * mdl_x + x0], dd000 * real);
			cuda_atomic_add(&g_model_imag  [z0 * mdl_x * mdl_y + y0 * mdl_x + x0], dd000 * imag);
			cuda_atomic_add(&g_model_weight[z0 * mdl_x * mdl_y + y0 * mdl_x + x0], dd000 * Fweights);

			XFLOAT dd001 = mfz * mfy *  fx;

			cuda_atomic_add(&g_model_real  [z0 * mdl_x * mdl_y + y0 * mdl_x + x1], dd001 * real);
			cuda_atomic_add(&g_model_imag  [z0 * mdl_x * mdl_y + y0 * mdl_x + x1], dd001 * imag);
			cuda_atomic_add(&g_model_weight[z0 * mdl_x * mdl_y + y0 * mdl_x + x1], dd001 * Fweights);

			XFLOAT dd010 = mfz *  fy * mfx;

			cuda_atomic_add(&g_model_real  [z0 * mdl_x * mdl_y + y1 * mdl_x + x0], dd010 * real);
			cuda_atomic_add(&g_model_imag  [z0 * mdl_x * mdl_y + y1 * mdl_x + x0], dd010 * imag);
			cuda_atomic_add(&g_model_weight[z0 * mdl_x * mdl_y + y1 * mdl_x + x0], dd010 * Fweights);

			XFLOAT dd011 = mfz *  fy *  fx;

			cuda_atomic_add(&g_model_real  [z0 * mdl_x * mdl_y + y1 * mdl_x + x1], dd011 * real);
			cuda_atomic_add(&g_model_imag  [z0 * mdl_x * mdl_y + y1 * mdl_x + x1], dd011 * imag);
			cuda_atomic_add(&g_model_weight[z0 * mdl_x * mdl_y + y1 * mdl_x + x1], dd011 * Fweights);

			XFLOAT dd100 =  fz * mfy * mfx;

			cuda_atomic_add(&g_model_real  [z1 * mdl_x * mdl_y + y0 * mdl_x + x0], dd100 * real);
			cuda_atomic_add(&g_model_imag  [z1 * mdl_x * mdl_y + y0 * mdl_x + x0], dd100 * imag);
			cuda_atomic_add(&g_model_weight[z1 * mdl_x * mdl_y + y0 * mdl_x + x0], dd100 * Fweights);

			XFLOAT dd101 =  fz * mfy *  fx;

			cuda_atomic_add(&g_model_real  [z1 * mdl_x * mdl_y + y0 * mdl_x + x1], dd101 * real);
			cuda_atomic_add(&g_model_imag  [z1 * mdl_x * mdl_y + y0 * mdl_x + x1], dd101 * imag);
			cuda_atomic_add(&g_model_weight[z1 * mdl_x * mdl_y + y0 * mdl_x + x1], dd101 * Fweights);

			XFLOAT dd110 =  fz *  fy * mfx;

			cuda_atomic_add(&g_model_real  [z1 * mdl_x * mdl_y + y1 * mdl_x + x0], dd110 * real);
			cuda_atomic_add(&g_model_imag  [z1 * mdl_x * mdl_y + y1 * mdl_x + x0], dd110 * imag);
			cuda_atomic_add(&g_model_weight[z1 * mdl_x * mdl_y + y1 * mdl_x + x0], dd110 * Fweights);

			XFLOAT dd111 =  fz *  fy *  fx;

			cuda_atomic_add(&g_model_real  [z1 * mdl_x * mdl_y + y1 * mdl_x + x1], dd111 * real);
			cuda_atomic_add(&g_model_imag  [z1 * mdl_x * mdl_y + y1 * mdl_x + x1], dd111 * imag);
			cuda_atomic_add(&g_model_weight[z1 * mdl_x * mdl_y + y1 * mdl_x + x1], dd111 * Fweights);

		}
	}
}


void CudaBackprojector::backproject(
		XFLOAT *d_real,
		XFLOAT *d_imag,
		XFLOAT *d_weight,
		XFLOAT *d_eulers,
		int imgX,
		int imgY,
		unsigned long imageCount)
{

	if(mdlZ==1)
	{
		cuda_kernel_backproject2D<<<imageCount,BP_2D_BLOCK_SIZE,0,stream>>>(
			d_mdlReal,
			d_mdlImag,
			d_mdlWeight,
			d_real,
			d_imag,
			d_weight,
			d_eulers,
			maxR,
			maxR2,
			padding_factor,
			imgX,
			imgY,
			imgX*imgY,
			mdlX,
			mdlInitY);
	}
	else
	{
		cuda_kernel_backproject3D_scatter<<<imageCount,BP_2D_BLOCK_SIZE,0,stream>>>(
				d_mdlReal,
				d_mdlImag,
				d_mdlWeight,
				d_real,
				d_imag,
				d_weight,
				d_eulers,
				maxR,
				maxR2,
				padding_factor,
				imgX,
				imgY,
				imgX*imgY,
				mdlX,
				mdlY,
				mdlInitY,
				mdlInitZ);
	}
}


void CudaBackprojector::getMdlData(XFLOAT *r, XFLOAT *i, XFLOAT * w)
{
	DEBUG_HANDLE_ERROR(hipStreamSynchronize(stream)); //Make sure to wait for remaining kernel executions

	DEBUG_HANDLE_ERROR(hipMemcpyAsync( r, d_mdlReal,   mdlXYZ * sizeof(XFLOAT), hipMemcpyDeviceToHost, stream));
	DEBUG_HANDLE_ERROR(hipMemcpyAsync( i, d_mdlImag,   mdlXYZ * sizeof(XFLOAT), hipMemcpyDeviceToHost, stream));
	DEBUG_HANDLE_ERROR(hipMemcpyAsync( w, d_mdlWeight, mdlXYZ * sizeof(XFLOAT), hipMemcpyDeviceToHost, stream));

	DEBUG_HANDLE_ERROR(hipStreamSynchronize(stream)); //Wait for copy
}

void CudaBackprojector::clear()
{
	if (d_mdlReal != NULL)
	{
		DEBUG_HANDLE_ERROR(hipFree(d_mdlReal));
		DEBUG_HANDLE_ERROR(hipFree(d_mdlImag));
		DEBUG_HANDLE_ERROR(hipFree(d_mdlWeight));

		d_mdlReal = d_mdlImag = d_mdlWeight = NULL;
	}
}

CudaBackprojector::~CudaBackprojector()
{
	clear();
}
